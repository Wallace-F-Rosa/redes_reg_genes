
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <string>
#include <limits>
#include <stdio.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;

__global__ void passo_bool_Apoptosis_Network(unsigned long long * d_v)
{
    __shared__ bool etc;
    unsigned long long v=0,aux=0;
    //bool v[4] = {true};
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    
    v = tid;
    
    aux |= (( v>>3 )%2)<<3;
    aux |= (( v>>2 )%2 <<2);
    aux |= ( ( (v>>1)%2 & ( ( ( (v>>3)%2 & (v>>2)%2 ) ) ) ) & ! ( (v>>1)%2 ) ) << 2;
    aux |= ( (v>>3)%2 ) << 3;
    /*v[4] = ( ( v[38] ) & ! ( v[1] ) );
    v[5] = ( ( v[20] & ( ( ( v[38] ) ) ) ) & ! ( v[6] ) ) | ( ( v[12] & ( ( ( v[38] ) ) ) ) & ! ( v[6] ) );
    v[6] = ( ( ( v[25] ) & ! ( v[4] ) ) & ! ( v[38] ) );
    v[7] = ( v[11] );
    v[8] = ( ( v[2] ) & ! ( v[17] ) ) | ( ( v[12] ) & ! ( v[17] ) ) | ( v[10] );
    v[9] = ( v[8] );
    v[10] = ( ( v[8] ) & ! ( v[17] ) );
    v[11] = ( ( v[2] ) & ! ( v[17] ) ) | ( ( v[12] ) & ! ( v[17] ) );
    v[12] = ( ( v[10] ) & ! ( v[37] ) ) | ( ( v[15] ) & ! ( v[37] ) );
    v[13] = ( ( ( v[8] ) & ! ( v[1] ) ) & ! ( v[17] ) ) | ( ( ( v[7] ) & ! ( v[1] ) ) & ! ( v[17] ) );
    v[14] = ( v[9] & ( ( ( v[8] ) ) ) );
    v[15] = ( v[34] );
    v[16] = ( v[39] );
    v[17] = ( ( ( v[25] ) & ! ( v[24] ) ) & ! ( v[8] & ( ( ( v[10] ) ) ) ) );
    v[18] = ( ( v[1] ) & ! ( v[0] ) ) | ( ( v[26] ) & ! ( v[0] ) );
    v[19] = ( ( v[25] ) & ! ( v[18] ) );
    v[20] = ( v[21] );
    v[21] = ( ( v[22] ) & ! ( v[1] ) );
    v[22] = ( v[35] );
    v[23] = ( v[1] ) | ( v[38] );
    v[24] = ( ( v[5] ) & ! ( v[6] ) );
    v[25] = ! ( ( v[19] ) );
    v[26] = ( v[36] );
    v[27] = ( v[16] );
    v[28] = ( v[16] );
    v[29] = ( ( v[27] & ( ( ( v[28] ) ) ) ) & ! ( v[30] ) );
    v[30] = ( v[38] );
    v[31] = ( v[34] );
    v[32] = ( v[40] );
    v[33] = ( v[40] );
    v[34] = ( v[32] );
    v[35] = ( v[34] );
    v[36] = ( v[31] );
    v[37] = ( v[35] );
    v[38] = ( ( v[20] ) & ! ( v[23] ) ) | ( ( v[14] ) & ! ( v[23] ) ); */
    
    d_v[tid] = aux;
}


/* __global__ void passo_bool_IL_6_Signaling(bool * d_v)
{
    __shared__ bool etc;
    bool v[87] = {true};

    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    
    for(int i = 0; i < 87; i++)
        v[i] = (tid>>i)%2;

    v[0] = ( ( v[63] ) & !( v[71] ) );
    v[1] = ( ( ( v[72] & ( ( ( v[42] & v[46] ) ) ) ) & !( v[73] ) ) & !( v[74] ) );
    v[2] = ( ( ( ( v[63] ) & !( v[27] ) ) & !( v[7] ) ) & !( v[3] ) );
    v[3] = ( !( ( v[1] ) ) ) | !( v[1] );
    v[4] = ( v[30] );
    v[5] = ( v[4] );
    v[6] = ( v[5] );
    v[7] = ( !( ( v[1] ) ) ) | !( v[1] );
    v[8] = ( v[63] ) | ( v[43] );
    v[9] = ( v[63] );
    v[10] = ( v[63] );
    v[11] = ( v[63] );
    v[12] = ( ( v[10] & ( ( ( v[63] ) ) ) ) & !( v[71] ) );
    v[13] = ( v[8] ) | ( v[9] );
    v[14] = ( v[34] ) | ( v[75] );
    v[15] = ( v[24] ) | ( v[25] );
    v[16] = ( v[29] ) | ( v[26] );
    v[17] = ( v[47] ) | ( v[41] );
    v[18] = ( v[47] ) | ( v[6] );
    v[19] = ( v[36] );
    v[20] = ( !( ( v[1] ) ) ) | !( v[1] );
    v[21] = ( v[26] ) | ( v[19] & ( ( ( v[46] ) ) ) );
    v[22] = ( v[21] & ( ( ( v[14] ) ) ) );
    v[23] = ( !( ( v[6] ) | ( v[40] ) ) ) | !( v[40] | v[6] );
    v[24] = ( v[76] );
    v[25] = ( ( v[76] ) & !( v[23] ) );
    v[26] = ( v[29] ) | ( v[56] );
    v[27] = ( !( ( v[1] ) ) ) | !( v[1] );
    v[28] = ( v[77] & ( ( ( v[15] ) ) ) );
    v[29] = ( ( ( v[28] & ( ( ( v[34] ) ) ) ) & !( v[57] ) ) & !( v[56] ) );
    v[30] = ( v[48] );
    v[31] = ( !( ( v[59] ) | ( v[58] ) ) ) | !( v[58] | v[59] );
    v[32] = ( v[47] ) | ( v[19] );
    v[33] = ( !( ( v[32] ) | ( v[56] ) | ( v[57] ) ) ) | !( v[32] | v[56] | v[57] );
    v[34] = ( !( ( v[58] ) | ( v[56] ) | ( v[57] ) | ( v[59] ) ) ) | !( v[56] | v[58] | v[57] | v[59] );
    v[35] = ( v[63] );
    v[36] = ( v[52] );
    v[37] = ( v[39] );
    v[38] = ( v[29] );
    v[39] = ( v[51] );
    v[40] = ( v[43] );
    v[41] = ( v[78] );
    v[42] = ( v[78] );
    v[43] = ( v[38] );
    v[44] = ( v[41] ) | ( v[1] ) | ( v[19] );
    v[45] = ( v[22] & ( ( ( v[79] ) ) ) ) | ( v[53] );
    v[46] = ( ( v[45] ) & !( v[80] ) );
    v[47] = ( v[37] );
    v[48] = ( v[22] );
    v[49] = ( v[9] ) | ( v[8] );
    v[50] = ( ( v[63] ) | ( v[44] ) | ( v[6] ) | ( v[19] ) ) | !( v[8] | v[63] | v[44] | v[20] | v[19] | v[6] );
    v[51] = ( v[70] );
    v[52] = ( v[53] );
    v[53] = ( ( v[26] ) & !( v[54] ) );
    v[54] = ( ( v[22] ) & !( v[57] ) );
    v[55] = ( v[13] & ( ( ( v[71] & v[63] ) ) ) );
    v[56] = ( ( ( ( v[29] & ( ( ( v[34] ) ) ) ) & !( v[81] ) ) & !( v[59] ) ) & !( v[82] ) );
    v[57] = ( ( ( v[22] & ( ( ( v[56] ) ) ) ) & !( v[82] ) ) & !( v[81] ) );
    v[58] = ( v[63] );
    v[59] = ( v[43] ) | ( v[71] ) | ( v[63] ) | ( v[19] );
    v[60] = ( ( ( ( v[29] & ( ( ( v[34] ) ) ) ) & !( v[56] ) ) & !( v[83] ) ) & !( v[57] ) );
    v[61] = ( ( ( v[18] & ( ( ( v[60] ) ) ) ) & !( v[84] ) ) & !( v[85] ) );
    v[62] = ( ( ( ( v[29] & ( ( ( v[34] ) ) ) ) & !( v[56] ) ) & !( v[83] ) ) & !( v[57] ) );
    v[63] = ( ( ( v[17] & ( ( ( v[62] ) ) ) ) & !( v[84] ) ) & !( v[86] ) );
    v[64] = ( v[56] ) | ( v[57] );
    v[65] = ( v[56] ) | ( v[57] );
    v[66] = ( v[56] ) | ( v[57] );
    v[67] = ( v[58] );
    v[68] = ( v[59] );
    v[69] = ( v[8] ) | ( v[9] );
    v[70] = ( ( v[16] ) & !( v[58] ) );
    for(int i = 0; i < 87; i++)
        etc ^= v[i];
}



__global__ void passo_bool_CD4_T_cell_signaling(bool * d_v)
{
    
    bool v[188]={true};

    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;

    for(int i = 0; i < 188; i++)
        v[i] = (tid>>i)%2;

    v[0] = ( v[105] );
    v[1] = ( v[131] ) | ( v[86] ) | ( v[24] );
    v[2] = ( v[103] );
    v[3] = ( v[101] ) | ( v[147] );
    v[4] = ( v[9] );
    v[5] = ( v[120] );
    v[6] = ( v[4] & ( ( ( v[9] ) ) ) );
    v[7] = ( v[154] ) | ( v[141] );
    v[8] = ( v[20] );
    v[9] = ( v[11] ) | ( v[109] );
    v[10] = ( v[138] ) | ( v[155] ) | ( v[7] );
    v[11] = ( v[10] );
    v[12] = ( v[156] ) | ( v[157] );
    v[13] = ( v[141] );
    v[14] = ( v[141] & ( ( ( v[158] & v[13] ) ) ) );
    v[15] = ( v[76] );
    v[16] = ( v[15] );
    v[17] = ( v[26] & ( ( ( v[7] ) ) ) );
    v[18] = ( v[8] ) | ( v[123] );
    v[19] = ( ! ( ( v[88] ) ) ) | ! ( v[88] );
    v[20] = ( v[17] ) | ( v[112] );
    v[21] = ( v[110] ) | ( v[111] );
    v[22] = ( v[34] );
    v[23] = ( v[150] );
    v[24] = ( v[90] );
    v[25] = ( v[131] );
    v[26] = ( v[27] & ( ( ( v[138] ) ) ) );
    v[27] = ( v[7] );
    v[28] = ( ( ( v[126] & ( ( ( v[132] & v[97] ) ) ) ) & ! ( v[130] & ( ( ( v[118] ) ) ) ) ) & ! ( v[129] ) ) | ( v[97] & ( ( ( v[28] & v[132] ) ) ) );
    v[29] = ( v[13] & ( ( ( v[141] ) ) ) ) | ( v[10] );
    v[30] = ( v[3] & ( ( ( v[37] ) ) ) );
    v[31] = ( v[136] & ( ( ( v[43] ) ) ) );
    v[32] = ( v[159] & ( ( ( v[141] ) ) ) );
    v[33] = ( v[13] ) | ( v[159] );
    v[34] = ( v[22] ) | ( ( v[134] ) & ! ( v[140] ) );
    v[35] = ( v[141] ) | ( v[134] );
    v[36] = ( ! ( ( v[0] ) ) ) | ! ( v[0] );
    v[37] = ( v[113] );
    v[38] = ( v[87] );
    v[39] = ( v[160] );
    v[40] = ( v[161] );
    v[41] = ( v[162] );
    v[42] = ( v[163] );
    v[43] = ( v[87] ) | ( v[136] );
    v[44] = ( v[140] );
    v[45] = ( v[156] );
    v[46] = ( v[164] );
    v[47] = ( ( ( v[131] & ( ( ( v[151] & v[97] ) ) ) ) & ! ( v[28] ) ) & ! ( v[130] ) ) | ( ( ( v[2] ) & ! ( v[28] ) ) & ! ( v[130] ) ) | ( ( ( v[1] & ( ( ( v[131] ) ) ) ) & ! ( v[28] ) ) & ! ( v[130] ) ) | ( ( ( v[119] & ( ( ( v[140] & v[151] & v[97] ) ) ) ) & ! ( v[28] ) ) & ! ( v[130] ) ) | ( ( ( v[44] ) & ! ( v[28] ) ) & ! ( v[130] ) );
    v[48] = ( v[47] & ( ( ( v[165] & v[166] ) ) ) ) | ( v[167] & ( ( ( v[165] & v[166] ) ) ) );
    v[49] = ( ! ( ( v[50] ) ) ) | ! ( v[50] );
    v[50] = ( v[6] ) | ( v[99] ) | ( v[141] );
    v[51] = ( v[97] & ( ( ( v[34] | v[130] ) & ( ( ( v[151] ) ) ) ) ) );
    v[52] = ( v[51] & ( ( ( v[168] & v[169] ) ) ) ) | ( v[170] & ( ( ( v[168] & v[169] ) ) ) );
    v[53] = ( v[78] ) | ( v[159] );
    v[54] = ( v[159] );
    v[55] = ( v[171] & ( ( ( v[172] & v[173] & v[174] ) ) ) );
    v[56] = ( ( ( ( v[97] & ( ( ( v[130] & v[98] & v[151] & v[118] ) ) ) ) & ! ( v[129] & ( ( ( v[28] ) ) ) ) ) & ! ( v[134] & ( ( ( v[28] ) ) ) ) ) & ! ( v[132] & ( ( ( v[28] ) ) ) ) );
    v[57] = ( v[175] );
    v[58] = ( ( ( v[97] & ( ( ( ! v[28] ) ) ) ) & ! ( v[140] & ( ( ( v[98] ) ) ) ) ) & ! ( v[132] & ( ( ( v[134] ) ) ) ) ) | ( ( ( v[98] ) & ! ( v[140] & ( ( ( v[98] ) ) ) ) ) & ! ( v[132] & ( ( ( v[134] ) ) ) ) );
    v[59] = ( v[97] & ( ( ( v[130] & v[151] ) ) ) );
    v[60] = ( v[176] & ( ( ( v[177] & v[171] ) ) ) ) | ( v[59] & ( ( ( v[177] & v[171] ) ) ) );
    v[61] = ( v[131] ) | ( v[129] ) | ( v[132] ) | ( v[130] );
    v[62] = ( v[178] );
    v[63] = ( v[97] & ( ( ( v[130] & v[151] ) ) ) );
    v[64] = ( v[63] & ( ( ( v[53] & v[177] & v[130] & v[118] ) ) ) ) | ( v[179] & ( ( ( v[53] & v[177] & v[130] & v[118] ) ) ) );
    v[65] = ( v[177] & ( ( ( v[180] & v[181] ) ) ) );
    v[66] = ( v[58] & ( ( ( v[171] & v[173] ) & ( ( ( ! v[67] ) ) ) ) ) ) | ( v[182] & ( ( ( v[171] & v[173] ) & ( ( ( ! v[67] ) ) ) ) ) );
    v[67] = ( v[28] & ( ( ( v[97] ) ) ) ) | ( v[132] & ( ( ( v[97] ) ) ) ) | ( v[126] & ( ( ( v[97] ) ) ) ) | ( v[98] & ( ( ( v[97] ) ) ) );
    v[68] = ( v[58] & ( ( ( v[67] & v[171] & v[173] ) ) ) ) | ( v[182] & ( ( ( v[67] & v[171] & v[173] ) ) ) );
    v[69] = ( v[79] ) | ( ( ( ( v[34] & ( ( ( v[151] & v[97] ) ) ) ) & ! ( v[28] ) ) & ! ( v[140] & ( ( ( v[119] ) ) ) ) ) & ! ( v[78] ) );
    v[70] = ( v[69] & ( ( ( v[71] & v[171] ) ) ) ) | ( v[183] & ( ( ( v[71] & v[171] ) ) ) );
    v[71] = ( ! ( ( v[133] ) ) ) | ! ( v[133] );
    v[72] = ( v[133] );
    v[73] = ( v[69] & ( ( ( v[171] & v[72] ) ) ) ) | ( v[183] & ( ( ( v[171] & v[72] ) ) ) );
    v[74] = ( v[177] & ( ( ( v[184] & v[185] ) ) ) );
    v[75] = ( v[186] ) | ( v[85] );
    v[76] = ( v[111] );
    v[77] = ( v[57] );
    v[78] = ( v[129] );
    v[79] = ( v[34] );
    v[80] = ( v[152] );
    v[81] = ( v[89] );
    v[82] = ( v[125] );
    v[83] = ( ( v[85] ) & ! ( v[128] ) ) | ( ( v[66] ) & ! ( v[128] ) ) | ( ( v[75] ) & ! ( v[128] ) ) | ( ( v[62] ) & ! ( v[128] ) );
    v[84] = ( v[53] & ( ( ( v[54] ) ) ) );
    v[85] = ( v[66] );
    v[86] = ( v[152] & ( ( ( v[20] ) ) ) ) | ( v[92] ) | ( v[95] );
    v[87] = ( v[148] );
    v[88] = ( v[104] ) | ( v[117] );
    v[89] = ( v[12] ) | ( v[85] & ( ( ( v[173] ) ) ) ) | ( v[14] );
    v[90] = ( v[114] ) | ( v[5] ) | ( v[104] );
    v[91] = ( v[94] );
    v[92] = ( v[94] );
    v[93] = ( v[94] );
    v[94] = ( v[32] & ( ( ( v[33] ) ) ) );
    v[95] = ( v[139] );
    v[96] = ( v[117] );
    v[97] = ( v[12] & ( ( ( v[141] ) ) ) ) | ( v[141] & ( ( ( v[12] ) ) ) ) | ( ( v[16] & ( ( ( v[103] ) ) ) ) & ! ( v[36] ) );
    v[98] = ( ! ( ( v[49] ) | ( v[28] ) ) ) | ! ( v[28] | v[49] );
    v[99] = ( v[144] );
    v[100] = ( v[10] );
    v[101] = ( v[18] ) | ( v[102] & ( ( ( v[146] ) ) ) );
    v[102] = ( v[125] );
    v[103] = ( v[91] ) | ( v[93] );
    v[104] = ( v[152] ) | ( v[18] ) | ( v[102] );
    v[105] = ( v[107] );
    v[106] = ( v[12] & ( ( ( v[45] ) ) ) ) | ( v[66] ) | ( v[124] ) | ( v[31] ) | ( v[121] ) | ( v[26] );
    v[107] = ( v[106] );
    v[108] = ( v[150] );
    v[109] = ( v[21] );
    v[110] = ( v[41] );
    v[111] = ( v[148] ) | ( v[82] ) | ( v[87] );
    v[112] = ( v[26] );
    v[113] = ( v[116] );
    v[114] = ( v[121] );
    v[115] = ( v[21] );
    v[116] = ( v[120] );
    v[117] = ( v[122] );
    v[118] = ( v[143] & ( ( ( v[130] ) ) ) ) | ( v[118] & ( ( ( v[143] | v[130] ) ) ) );
    v[119] = ( ( v[140] ) & ! ( v[34] ) );
    v[120] = ( v[23] ) | ( v[8] & ( ( ( v[20] ) ) ) ) | ( v[108] );
    v[121] = ( v[137] ) | ( v[115] );
    v[122] = ( v[123] ) | ( v[10] );
    v[123] = ( v[39] ) | ( v[26] );
    v[124] = ( v[31] ) | ( v[173] );
    v[125] = ( v[148] ) | ( v[38] );
    v[126] = ( v[143] );
    v[127] = ( v[134] ) | ( v[130] );
    v[128] = ( v[130] );
    v[129] = ( ( v[46] ) & ! ( v[127] ) ) | ( ( v[48] ) & ! ( v[127] ) ) | ( ( v[65] ) & ! ( v[127] ) );
    v[130] = ( v[60] ) | ( v[65] ) | ( v[74] ) | ( v[64] ) | ( v[52] );
    v[131] = ( ( v[103] & ( ( ( v[145] ) ) ) ) & ! ( v[34] ) ) | ( ( v[84] ) & ! ( v[34] ) );
    v[132] = ( v[66] ) | ( v[70] ) | ( v[55] ) | ( v[135] ) | ( v[89] ) | ( v[83] );
    v[133] = ( v[73] ) | ( v[68] );
    v[134] = ( v[70] );
    v[135] = ( v[66] );
    v[136] = ( v[29] ) | ( v[173] & ( ( ( v[66] ) ) ) );
    v[137] = ( v[43] );
    v[138] = ( v[7] ) | ( v[27] );
    v[139] = ( v[144] );
    v[140] = ( ( v[140] ) & ! ( v[34] ) ) | ( ( v[129] ) & ! ( v[34] ) );
    v[141] = ( v[156] & ( ( ( v[12] ) ) ) );
    v[142] = ( v[28] & ( ( ( v[151] & v[97] ) ) ) );
    v[143] = ( v[142] ) | ( v[187] );
    v[144] = ( v[77] );
    v[145] = ( v[53] & ( ( ( v[54] ) ) ) );
    v[146] = ( v[125] );
    v[147] = ( v[80] & ( ( ( v[152] ) ) ) );
    v[148] = ( v[81] & ( ( ( v[13] ) ) ) );
    v[149] = ( v[40] );
    v[150] = ( v[149] );
    v[151] = ( v[151] ) | ( v[133] );
    v[152] = ( v[153] ) | ( v[100] ) | ( v[20] & ( ( ( v[112] ) ) ) ) | ( v[146] );
    v[153] = ( v[138] );
    for(int i = 0; i < 188; i++)
        d_v[i] = v[i];
}
 */

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[1];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    unsigned long long *d_v;
    hipMalloc(&d_v,sizeof(unsigned long long)*MAX_ESTADO);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime=0;

    hipEventRecord(start);

    passo_bool_Apoptosis_Network<<<grid,block>>>(d_v);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout.precision(20);
    cout <<"\n" << (elapsedTime) << " ms\n";
    
    hipFree(d_v);
    return 0;
}