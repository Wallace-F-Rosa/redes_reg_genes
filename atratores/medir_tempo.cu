
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <string>
#include <limits>
#include <stdio.h>
#include <stdlib.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;

//REDE 1
__global__ void passo_bool_1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>1)%2 & ( ( ( (v>>2)%2 ) ) ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 1 ;
        aux |=(unsigned long long) ( (v>>3)%2 )<< 2 ;
        aux |=(unsigned long long) ( (v>>4)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 4 ;
        aux |=(unsigned long long) ( (v>>9)%2 ) | ( (v>>29)%2 )<< 5 ;
        aux |=(unsigned long long) ( (v>>7)%2 )<< 6 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 7 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>25)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>24)%2 )<< 10 ;
        aux |=(unsigned long long) ( (v>>12)%2 )<< 11 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 12 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 13 ;
        aux |=(unsigned long long) ( ( (v>>15)%2 ) & ! ( (v>>11)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>10)%2 ) | ( (v>>24)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>0)%2 & ( ( ( (v>>1)%2 & (v>>20)%2 ) ) ) )<< 16 ;
        aux |=(unsigned long long) ( ( (v>>18)%2 ) & ! ( (v>>31)%2 ) )<< 17 ;
        aux |=(unsigned long long) ( (v>>19)%2 )<< 18 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>21)%2 )<< 20 ;
        aux |=(unsigned long long) ( (v>>22)%2 )<< 21 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 22 ;
        aux |=(unsigned long long) ! ( ( (v>>5)%2 ) )<< 23 ;
        aux |=(unsigned long long) ( (v>>27)%2 )<< 24 ;
        aux |=(unsigned long long) ( (v>>30)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>16)%2 & ( ( ( (v>>17)%2 ) ) ) )<< 26 ;
        aux |=(unsigned long long) ( (v>>26)%2 )<< 27 ;
        aux |=(unsigned long long) ( ( (v>>23)%2 ) & ! ( (v>>31)%2 ) )<< 28 ;
        aux |=(unsigned long long) ( (v>>25)%2 )<< 29 ;
        aux |=(unsigned long long) ( (v>>14)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>33)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 33 ;
    
        estado[tid] = aux;
    }
}

unsigned long long confere_bool_1(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( (v>>1)%2 & ( ( ( (v>>2)%2 ) ) ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 1 ;
        aux |=(unsigned long long) ( (v>>3)%2 )<< 2 ;
        aux |=(unsigned long long) ( (v>>4)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 4 ;
        aux |=(unsigned long long) ( (v>>9)%2 ) | ( (v>>29)%2 )<< 5 ;
        aux |=(unsigned long long) ( (v>>7)%2 )<< 6 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 7 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>25)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>24)%2 )<< 10 ;
        aux |=(unsigned long long) ( (v>>12)%2 )<< 11 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 12 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 13 ;
        aux |=(unsigned long long) ( ( (v>>15)%2 ) & ! ( (v>>11)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>10)%2 ) | ( (v>>24)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>0)%2 & ( ( ( (v>>1)%2 & (v>>20)%2 ) ) ) )<< 16 ;
        aux |=(unsigned long long) ( ( (v>>18)%2 ) & ! ( (v>>31)%2 ) )<< 17 ;
        aux |=(unsigned long long) ( (v>>19)%2 )<< 18 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>21)%2 )<< 20 ;
        aux |=(unsigned long long) ( (v>>22)%2 )<< 21 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 22 ;
        aux |=(unsigned long long) ! ( ( (v>>5)%2 ) )<< 23 ;
        aux |=(unsigned long long) ( (v>>27)%2 )<< 24 ;
        aux |=(unsigned long long) ( (v>>30)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>16)%2 & ( ( ( (v>>17)%2 ) ) ) )<< 26 ;
        aux |=(unsigned long long) ( (v>>26)%2 )<< 27 ;
        aux |=(unsigned long long) ( ( (v>>23)%2 ) & ! ( (v>>31)%2 ) )<< 28 ;
        aux |=(unsigned long long) ( (v>>25)%2 )<< 29 ;
        aux |=(unsigned long long) ( (v>>14)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>33)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 33 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>2)%2 ) * 2) >= 4 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2 + ( (v>>29)%2 ) * 2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>11)%2 ) * -2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>24)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>1)%2 ) * 2 + ( (v>>20)%2 ) * 2) >= 6 ) << 16 ;

    
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_1_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>31)%2 ) * -2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * -2) >= 0 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2 + ( (v>>17)%2 ) * 2) >= 4 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2 + ( (v>>31)%2 ) * -2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>14)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 33 ;

    
        estado[tid] |= aux;
    }
}

unsigned long long confere_tlf_1(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>2)%2 ) * 2) >= 4 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2 + ( (v>>29)%2 ) * 2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>11)%2 ) * -2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>24)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>1)%2 ) * 2 + ( (v>>20)%2 ) * 2) >= 6 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>31)%2 ) * -2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * -2) >= 0 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2 + ( (v>>17)%2 ) * 2) >= 4 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2 + ( (v>>31)%2 ) * -2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>14)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 33 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 2

__global__ void passo_bool_2_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>25)%2 )<< 0 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 1 ;
        aux |=(unsigned long long) ( ( (v>>13)%2 & ( ( ( (v>>3)%2 & (v>>24)%2 ) ) ) ) & ! ( (v>>17)%2 ) )<< 2 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 3 ;
        aux |=(unsigned long long) ( ( (v>>38)%2 ) & ! ( (v>>1)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) ) | ( ( (v>>12)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>38)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>11)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) ) | ( (v>>10)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 ) & ! ( (v>>17)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 ) & ! ( (v>>37)%2 ) ) | ( ( (v>>15)%2 ) & ! ( (v>>37)%2 ) )<< 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) ) | ( ( ( (v>>7)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( (v>>9)%2 & ( ( ( (v>>8)%2 ) ) ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>39)%2 )<< 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>24)%2 ) ) & ! ( (v>>8)%2 & ( ( ( (v>>10)%2 ) ) ) ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>1)%2 ) & ! ( (v>>0)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>0)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( ( (v>>25)%2 ) & ! ( (v>>18)%2 ) )<< 19 ;
        aux |=(unsigned long long) ( (v>>21)%2 )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>22)%2 ) & ! ( (v>>1)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>1)%2 ) | ( (v>>38)%2 )<< 23 ;
        aux |=(unsigned long long) ( ( (v>>5)%2 ) & ! ( (v>>6)%2 ) )<< 24 ;
        aux |=(unsigned long long) ! ( ( (v>>19)%2 ) )<< 25 ;
        
        
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_2_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
        aux |=(unsigned long long) ( (v>>36)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 27 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 28 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) & ( ( ( (v>>28)%2 ) ) ) ) & (! ( (v>>30)%2 )) )<< 29 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 37 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 ) & ! ( (v>>23)%2 ) ) | ( ( (v>>14)%2 ) & ! ( (v>>23)%2 ) )<< 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
        
        estado[tid] |= aux;
    }
}


unsigned long long confere_bool_2(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( (v>>25)%2 )<< 0 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 1 ;
        aux |=(unsigned long long) ( ( (v>>13)%2 & ( ( ( (v>>3)%2 & (v>>24)%2 ) ) ) ) & ! ( (v>>17)%2 ) )<< 2 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 3 ;
        aux |=(unsigned long long) ( ( (v>>38)%2 ) & ! ( (v>>1)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) ) | ( ( (v>>12)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>38)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>11)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) ) | ( (v>>10)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 ) & ! ( (v>>17)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 ) & ! ( (v>>37)%2 ) ) | ( ( (v>>15)%2 ) & ! ( (v>>37)%2 ) )<< 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) ) | ( ( ( (v>>7)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( (v>>9)%2 & ( ( ( (v>>8)%2 ) ) ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>39)%2 )<< 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>24)%2 ) ) & ! ( (v>>8)%2 & ( ( ( (v>>10)%2 ) ) ) ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>1)%2 ) & ! ( (v>>0)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>0)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( ( (v>>25)%2 ) & ! ( (v>>18)%2 ) )<< 19 ;
        aux |=(unsigned long long) ( (v>>21)%2 )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>22)%2 ) & ! ( (v>>1)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>1)%2 ) | ( (v>>38)%2 )<< 23 ;
        aux |=(unsigned long long) ( ( (v>>5)%2 ) & ! ( (v>>6)%2 ) )<< 24 ;
        aux |=(unsigned long long) ! ( ( (v>>19)%2 ) )<< 25 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 27 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 28 ;
        aux |=(unsigned long long) ( ( (v>>27)%2 & ( ( ( (v>>28)%2 ) ) ) ) & ! ( (v>>30)%2 ) )<< 29 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 37 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 ) & ! ( (v>>23)%2 ) ) | ( ( (v>>14)%2 ) & ! ( (v>>23)%2 ) )<< 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
    
        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_2_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>3)%2 ) * 2 + ( (v>>24)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 6 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>38)%2 ) * 6 + ( (v>>6)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 8 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>4)%2 ) * -2 + ( (v>>38)%2 ) * -2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2 + ( (v>>10)%2 ) * 10) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>37)%2 ) * -6 + ( (v>>15)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>17)%2 ) * -6 + ( (v>>7)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 4 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 6 + ( (v>>24)%2 ) * -6 + ( (v>>8)%2 ) * -2 + ( (v>>10)%2 ) * -2) >= 4 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>0)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>18)%2 ) * -2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>38)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>6)%2 ) * -2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * -2) >= 0 ) << 25 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_2_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2 + ( (v>>28)%2 ) * 2 + ( (v>>30)%2 ) * -2) >= 4 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>23)%2 ) * -6 + ( (v>>14)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
        
        estado[tid] |= aux;
    }
}


unsigned long long confere_tlf_2(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>3)%2 ) * 2 + ( (v>>24)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 6 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>38)%2 ) * 6 + ( (v>>6)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 8 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>4)%2 ) * -2 + ( (v>>38)%2 ) * -2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2 + ( (v>>10)%2 ) * 10) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>37)%2 ) * -6 + ( (v>>15)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>17)%2 ) * -6 + ( (v>>7)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 4 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 6 + ( (v>>24)%2 ) * -6 + ( (v>>8)%2 ) * -2 + ( (v>>10)%2 ) * -2) >= 4 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>0)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>18)%2 ) * -2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>38)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>6)%2 ) * -2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * -2) >= 0 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2 + ( (v>>28)%2 ) * 2 + ( (v>>30)%2 ) * -2) >= 4 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>23)%2 ) * -6 + ( (v>>14)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
        
        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 3

__global__ void passo_bool_3_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) & ! ( (v>>34)%2 ) ) & ! ( (v>>26)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>23)%2 )<< 1 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 2 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 ) ) | ! ( (v>>8)%2 | (v>>30)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>39)%2 & ( ( ( ! (v>>0)%2 ) ) | ( ( (v>>8)%2 ) ) ) ) | ( (v>>8)%2 & ( ( ( (v>>39)%2 ) ) | ( ( ! (v>>0)%2 ) ) ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>25)%2 & ( ( ( (v>>39)%2 & (v>>33)%2 ) ) ) ) & ! ( (v>>0)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( (v>>15)%2 & ( ( ( (v>>16)%2 ) ) ) ) | ( (v>>38)%2 & ( ( ( (v>>37)%2 ) ) ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>6)%2 ) & ! ( (v>>7)%2 ) ) | ( ( (v>>9)%2 ) & ! ( (v>>7)%2 ) )<< 8 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 & ( ( ( ! (v>>41)%2 ) ) ) ) & ! ( (v>>11)%2 ) ) | ( ( (v>>41)%2 & ( ( ( ! (v>>40)%2 ) ) ) ) & ! ( (v>>11)%2 ) ) | ( ( (v>>34)%2 ) & ! ( (v>>11)%2 ) ) ) | ! ( (v>>41)%2 | (v>>34)%2 | (v>>11)%2 | (v>>40)%2 )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( (v>>3)%2 ) ) & ( ( (v>>4)%2 ) ) ) ) & ! ( (v>>21)%2 ) ) | ( ( (v>>18)%2 & ( ( ( (v>>3)%2 ) ) & ( ( (v>>4)%2 ) ) ) ) & ! ( (v>>21)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( ! (v>>4)%2 & ! (v>>8)%2 & ! (v>>19)%2 & ! (v>>14)%2 ) ) ) ) | ( (v>>8)%2 ) | ( (v>>14)%2 & ( ( ( ! (v>>4)%2 & ! (v>>8)%2 & ! (v>>20)%2 & ! (v>>19)%2 ) ) ) ) | ( (v>>19)%2 ) | ( (v>>4)%2 ) ) | ! ( (v>>4)%2 | (v>>8)%2 | (v>>20)%2 | (v>>19)%2 | (v>>14)%2 )<< 11 ;
        aux |=(unsigned long long) ( (v>>23)%2 )<< 12 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>42)%2 ) ) | ( (v>>35)%2 & ( ( ( (v>>2)%2 ) ) ) )<< 13 ;
        aux |=(unsigned long long) ! ( ( (v>>39)%2 ) | ( (v>>8)%2 ) | ( (v>>34)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>17)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>11)%2 ) & ! ( (v>>8)%2 & ( ( ( (v>>39)%2 ) ) ) ) )<< 18 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 19 ;
    
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_3_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];

        aux |=(unsigned long long) ( (v>>39)%2 & ( ( ( (v>>11)%2 ) ) ) ) | ( ( (v>>11)%2 ) & ! ( (v>>34)%2 & ( ( ( (v>>23)%2 ) ) ) ) )<< 20 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) & ! ( (v>>43)%2 ) ) & ! ( (v>>4)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>24)%2 & ( ( ( (v>>22)%2 ) ) ) )<< 23 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 24 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 26 ;
        aux |=(unsigned long long) ! ( ( (v>>43)%2 ) )<< 27 ;
        aux |=(unsigned long long) ( (v>>43)%2 & ( ( ( (v>>8)%2 ) ) ) )<< 28 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 29 ;
        aux |=(unsigned long long) ! ( ( (v>>43)%2 ) | ( (v>>0)%2 ) )<< 30 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>26)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>12)%2 )<< 38 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 39 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;
    
        estado[tid] |= aux;
    }
}

unsigned long long confere_bool_3(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) & ! ( (v>>34)%2 ) ) & ! ( (v>>26)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>23)%2 )<< 1 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 2 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 ) ) | ! ( (v>>8)%2 | (v>>30)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>39)%2 & ( ( ( ! (v>>0)%2 ) ) | ( ( (v>>8)%2 ) ) ) ) | ( (v>>8)%2 & ( ( ( (v>>39)%2 ) ) | ( ( ! (v>>0)%2 ) ) ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>25)%2 & ( ( ( (v>>39)%2 & (v>>33)%2 ) ) ) ) & ! ( (v>>0)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( (v>>15)%2 & ( ( ( (v>>16)%2 ) ) ) ) | ( (v>>38)%2 & ( ( ( (v>>37)%2 ) ) ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>6)%2 ) & ! ( (v>>7)%2 ) ) | ( ( (v>>9)%2 ) & ! ( (v>>7)%2 ) )<< 8 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 & ( ( ( ! (v>>41)%2 ) ) ) ) & ! ( (v>>11)%2 ) ) | ( ( (v>>41)%2 & ( ( ( ! (v>>40)%2 ) ) ) ) & ! ( (v>>11)%2 ) ) | ( ( (v>>34)%2 ) & ! ( (v>>11)%2 ) ) ) | ! ( (v>>41)%2 | (v>>34)%2 | (v>>11)%2 | (v>>40)%2 )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( (v>>3)%2 ) ) & ( ( (v>>4)%2 ) ) ) ) & ! ( (v>>21)%2 ) ) | ( ( (v>>18)%2 & ( ( ( (v>>3)%2 ) ) & ( ( (v>>4)%2 ) ) ) ) & ! ( (v>>21)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( ! (v>>4)%2 & ! (v>>8)%2 & ! (v>>19)%2 & ! (v>>14)%2 ) ) ) ) | ( (v>>8)%2 ) | ( (v>>14)%2 & ( ( ( ! (v>>4)%2 & ! (v>>8)%2 & ! (v>>20)%2 & ! (v>>19)%2 ) ) ) ) | ( (v>>19)%2 ) | ( (v>>4)%2 ) ) | ! ( (v>>4)%2 | (v>>8)%2 | (v>>20)%2 | (v>>19)%2 | (v>>14)%2 )<< 11 ;
        aux |=(unsigned long long) ( (v>>23)%2 )<< 12 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>42)%2 ) ) | ( (v>>35)%2 & ( ( ( (v>>2)%2 ) ) ) )<< 13 ;
        aux |=(unsigned long long) ! ( ( (v>>39)%2 ) | ( (v>>8)%2 ) | ( (v>>34)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>17)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>11)%2 ) & ! ( (v>>8)%2 & ( ( ( (v>>39)%2 ) ) ) ) )<< 18 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>39)%2 & ( ( ( (v>>11)%2 ) ) ) ) | ( ( (v>>11)%2 ) & ! ( (v>>34)%2 & ( ( ( (v>>23)%2 ) ) ) ) )<< 20 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) & ! ( (v>>43)%2 ) ) & ! ( (v>>4)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>24)%2 & ( ( ( (v>>22)%2 ) ) ) )<< 23 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 24 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 26 ;
        aux |=(unsigned long long) ! ( ( (v>>43)%2 ) )<< 27 ;
        aux |=(unsigned long long) ( (v>>43)%2 & ( ( ( (v>>8)%2 ) ) ) )<< 28 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 29 ;
        aux |=(unsigned long long) ! ( ( (v>>43)%2 ) | ( (v>>0)%2 ) )<< 30 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>26)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>12)%2 )<< 38 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 39 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_3_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>34)%2 ) * -2 + ( (v>>26)%2 ) * -2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>30)%2 ) * -2) >= 0 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 4 + ( (v>>0)%2 ) * -4 + ( (v>>8)%2 ) * 4) >= 4 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>39)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>0)%2 ) * -2) >= 6 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 6 + ( (v>>16)%2 ) * 6 + ( (v>>38)%2 ) * 6 + ( (v>>37)%2 ) * 6) >= 12 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>7)%2 ) * -6 + ( (v>>9)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( 0 * -2 + 0 * -2 + ( (v>>11)%2 ) * -14 + ( (v>>34)%2 ) * 2) >= -2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>3)%2 ) * 6 + ( (v>>4)%2 ) * 6 + ( (v>>21)%2 ) * -6 + ( (v>>18)%2 ) * 2) >= 14 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * -2 + ( (v>>4)%2 ) * 2 + ( (v>>8)%2 ) * 2 + ( (v>>19)%2 ) * 2 + ( (v>>14)%2 ) * -2) >= -2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 6 + 0 * -2 + ( (v>>35)%2 ) * 2) >= 6 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * -2 + ( (v>>8)%2 ) * -2 + ( (v>>34)%2 ) * -2) >= 0 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 6 + ( (v>>8)%2 ) * -2 + ( (v>>39)%2 ) * -2) >= 4 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>11)%2 ) * 14 + ( (v>>34)%2 ) * -2 + ( (v>>23)%2 ) * -2) >= 12 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2 + 0 * -2 + ( (v>>4)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 22 ;
    
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_3_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2 + ( (v>>22)%2 ) * 2) >= 4 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * -2) >= 0 ) << 27 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>8)%2 ) * 2) >= 4 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( 0 * -2 + ( (v>>0)%2 ) * -2) >= 0 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;
    
        estado[tid] |= aux;
    }
}

unsigned long long confere_tlf_3(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>34)%2 ) * -2 + ( (v>>26)%2 ) * -2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>30)%2 ) * -2) >= 0 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 4 + ( (v>>0)%2 ) * -4 + ( (v>>8)%2 ) * 4) >= 4 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>39)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>0)%2 ) * -2) >= 6 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 6 + ( (v>>16)%2 ) * 6 + ( (v>>38)%2 ) * 6 + ( (v>>37)%2 ) * 6) >= 12 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>7)%2 ) * -6 + ( (v>>9)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( 0 * -2 + 0 * -2 + ( (v>>11)%2 ) * -14 + ( (v>>34)%2 ) * 2) >= -2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>3)%2 ) * 6 + ( (v>>4)%2 ) * 6 + ( (v>>21)%2 ) * -6 + ( (v>>18)%2 ) * 2) >= 14 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * -2 + ( (v>>4)%2 ) * 2 + ( (v>>8)%2 ) * 2 + ( (v>>19)%2 ) * 2 + ( (v>>14)%2 ) * -2) >= -2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 6 + 0 * -2 + ( (v>>35)%2 ) * 2) >= 6 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * -2 + ( (v>>8)%2 ) * -2 + ( (v>>34)%2 ) * -2) >= 0 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 6 + ( (v>>8)%2 ) * -2 + ( (v>>39)%2 ) * -2) >= 4 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>11)%2 ) * 14 + ( (v>>34)%2 ) * -2 + ( (v>>23)%2 ) * -2) >= 12 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2 + 0 * -2 + ( (v>>4)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2 + ( (v>>22)%2 ) * 2) >= 4 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * -2) >= 0 ) << 27 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>8)%2 ) * 2) >= 4 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( 0 * -2 + ( (v>>0)%2 ) * -2) >= 0 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}


//REDE 5
__global__ void passo_bool_5_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( (v>>40)%2 ) & ! ( (v>>29)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>29)%2 ) ) | ( ( (v>>0)%2 ) & ! ( (v>>29)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>41)%2 ) ) | ( ( ( (v>>26)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>41)%2 ) ) | ( ( ( (v>>1)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>41)%2 ) )<< 1 ;
        aux |=(unsigned long long) ( (v>>29)%2 ) | ( (v>>39)%2 ) | ( (v>>15)%2 )<< 2 ;
        aux |=(unsigned long long) ( ( ( ( ( (v>>30)%2 ) & ! ( (v>>33)%2 ) ) & ! ( (v>>39)%2 ) ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) ) | ( ( ( ( ( (v>>3)%2 ) & ! ( (v>>33)%2 ) ) & ! ( (v>>39)%2 ) ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) )<< 3 ;
        aux |=(unsigned long long) ( ( ( ( (v>>36)%2 ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>39)%2 ) ) | ( ( ( ( (v>>4)%2 ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>39)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( (v>>2)%2 )<< 5 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 6 ;
        aux |=(unsigned long long) ( ( (v>>34)%2 ) & ! ( (v>>33)%2 ) ) | ( ( (v>>22)%2 ) & ! ( (v>>33)%2 ) ) | ( ( (v>>2)%2 ) & ! ( (v>>33)%2 ) ) | ( ( (v>>39)%2 ) & ! ( (v>>33)%2 ) )<< 7 ;
        aux |=(unsigned long long) ( (v>>7)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>4)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>9)%2 )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>43)%2 ) & ! ( (v>>36)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 12 ;
        aux |=(unsigned long long) ( ( (v>>44)%2 ) & ! ( (v>>36)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 14 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 16 ;
        aux |=(unsigned long long) ( ( (v>>4)%2 ) & ! ( (v>>32)%2 ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>17)%2 ) & ! ( (v>>31)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>19)%2 )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 ) & ! ( (v>>41)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 23 ;
        aux |=(unsigned long long) ( (v>>20)%2 )<< 24 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_5_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>41)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>2)%2 )<< 27 ;
        aux |=(unsigned long long) ( ( ( ( (v>>30)%2 ) & ! ( (v>>39)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>4)%2 ) ) | ( ( ( ( (v>>28)%2 ) & ! ( (v>>39)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>4)%2 ) ) | ( ( ( ( (v>>33)%2 ) & ! ( (v>>39)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>4)%2 ) )<< 28 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>0)%2 ) ) | ( ( ( (v>>35)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>0)%2 ) ) | ( ( ( (v>>1)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>0)%2 ) )<< 29 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>39)%2 ) | ( (v>>32)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>6)%2 ) | ( (v>>15)%2 ) | ( (v>>23)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>24)%2 ) | ( (v>>14)%2 ) | ( (v>>10)%2 )<< 33 ;
        aux |=(unsigned long long) ( ( (v>>11)%2 ) & ! ( (v>>4)%2 ) )<< 34 ;
        aux |=(unsigned long long) ( ( (v>>21)%2 ) & ! ( (v>>31)%2 ) ) | ( ( (v>>16)%2 ) & ! ( (v>>31)%2 ) )<< 35 ;
        aux |=(unsigned long long) ( (v>>18)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>3)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>37)%2 )<< 38 ;
        aux |=(unsigned long long) ( ( ( ( (v>>39)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) ) | ( ( ( ( (v>>32)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) )<< 39 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>0)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>28)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>39)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>4)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>3)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) )<< 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>44)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 45 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 46 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 47 ;
        aux |=(unsigned long long) ( (v>>48)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;
        
        estado[tid] |= aux;
    }
}

unsigned long long confere_bool_5(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( (v>>40)%2 ) & ! ( (v>>29)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>29)%2 ) ) | ( ( (v>>0)%2 ) & ! ( (v>>29)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>41)%2 ) ) | ( ( ( (v>>26)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>41)%2 ) ) | ( ( ( (v>>1)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>41)%2 ) )<< 1 ;
        aux |=(unsigned long long) ( (v>>29)%2 ) | ( (v>>39)%2 ) | ( (v>>15)%2 )<< 2 ;
        aux |=(unsigned long long) ( ( ( ( ( (v>>30)%2 ) & ! ( (v>>33)%2 ) ) & ! ( (v>>39)%2 ) ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) ) | ( ( ( ( ( (v>>3)%2 ) & ! ( (v>>33)%2 ) ) & ! ( (v>>39)%2 ) ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) )<< 3 ;
        aux |=(unsigned long long) ( ( ( ( (v>>36)%2 ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>39)%2 ) ) | ( ( ( ( (v>>4)%2 ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>39)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( (v>>2)%2 )<< 5 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 6 ;
        aux |=(unsigned long long) ( ( (v>>34)%2 ) & ! ( (v>>33)%2 ) ) | ( ( (v>>22)%2 ) & ! ( (v>>33)%2 ) ) | ( ( (v>>2)%2 ) & ! ( (v>>33)%2 ) ) | ( ( (v>>39)%2 ) & ! ( (v>>33)%2 ) )<< 7 ;
        aux |=(unsigned long long) ( (v>>7)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>4)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>9)%2 )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>43)%2 ) & ! ( (v>>36)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 12 ;
        aux |=(unsigned long long) ( ( (v>>44)%2 ) & ! ( (v>>36)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 14 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 16 ;
        aux |=(unsigned long long) ( ( (v>>4)%2 ) & ! ( (v>>32)%2 ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>17)%2 ) & ! ( (v>>31)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( (v>>28)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>19)%2 )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 ) & ! ( (v>>41)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 23 ;
        aux |=(unsigned long long) ( (v>>20)%2 )<< 24 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>2)%2 )<< 27 ;
        aux |=(unsigned long long) ( ( ( ( (v>>30)%2 ) & ! ( (v>>39)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>4)%2 ) ) | ( ( ( ( (v>>28)%2 ) & ! ( (v>>39)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>4)%2 ) ) | ( ( ( ( (v>>33)%2 ) & ! ( (v>>39)%2 ) ) & ! ( (v>>3)%2 ) ) & ! ( (v>>4)%2 ) )<< 28 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>0)%2 ) ) | ( ( ( (v>>35)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>0)%2 ) ) | ( ( ( (v>>1)%2 ) & ! ( (v>>40)%2 ) ) & ! ( (v>>0)%2 ) )<< 29 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>39)%2 ) | ( (v>>32)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>6)%2 ) | ( (v>>15)%2 ) | ( (v>>23)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>24)%2 ) | ( (v>>14)%2 ) | ( (v>>10)%2 )<< 33 ;
        aux |=(unsigned long long) ( ( (v>>11)%2 ) & ! ( (v>>4)%2 ) )<< 34 ;
        aux |=(unsigned long long) ( ( (v>>21)%2 ) & ! ( (v>>31)%2 ) ) | ( ( (v>>16)%2 ) & ! ( (v>>31)%2 ) )<< 35 ;
        aux |=(unsigned long long) ( (v>>18)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>3)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>37)%2 )<< 38 ;
        aux |=(unsigned long long) ( ( ( ( (v>>39)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) ) | ( ( ( ( (v>>32)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>28)%2 ) ) & ! ( (v>>3)%2 ) )<< 39 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>0)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>28)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>39)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>4)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) ) | ( ( ( (v>>3)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>29)%2 ) )<< 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>44)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 45 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 46 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 47 ;
        aux |=(unsigned long long) ( (v>>48)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_5_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2 + ( (v>>29)%2 ) * -14 + ( (v>>26)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>40)%2 ) * -14 + 0 * -14 + ( (v>>26)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>39)%2 ) * 2 + ( (v>>15)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2 + ( (v>>33)%2 ) * -6 + ( (v>>39)%2 ) * -6 + ( (v>>4)%2 ) * -6 + ( (v>>28)%2 ) * -6 + ( (v>>3)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>28)%2 ) * -6 + ( (v>>3)%2 ) * -6 + ( (v>>39)%2 ) * -6 + ( (v>>4)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2 + ( (v>>33)%2 ) * -30 + ( (v>>22)%2 ) * 2 + ( (v>>2)%2 ) * 2 + ( (v>>39)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>36)%2 ) * -2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>36)%2 ) * -2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2 + ( (v>>32)%2 ) * -2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2 + ( (v>>31)%2 ) * -2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>41)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2) >= 2 ) << 24 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_5_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>49)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2 + ( (v>>39)%2 ) * -14 + ( (v>>3)%2 ) * -14 + ( (v>>4)%2 ) * -14 + ( (v>>28)%2 ) * 2 + ( (v>>33)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>40)%2 ) * -14 + ( (v>>0)%2 ) * -14 + ( (v>>35)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>32)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>15)%2 ) * 2 + ( (v>>23)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2 + ( (v>>14)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2 + ( (v>>4)%2 ) * -2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + ( (v>>31)%2 ) * -6 + ( (v>>16)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>4)%2 ) * -6 + ( (v>>28)%2 ) * -6 + ( (v>>3)%2 ) * -6 + ( (v>>32)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2 + ( (v>>1)%2 ) * -254 + ( (v>>29)%2 ) * -254 + ( (v>>0)%2 ) * 2 + 0 * 2 + ( (v>>28)%2 ) * 2 + ( (v>>39)%2 ) * 2 + ( (v>>4)%2 ) * 2 + ( (v>>3)%2 ) * 2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>44)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 45 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 46 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 47 ;
        aux |=(unsigned long long) ( (v>>48)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;
        
        estado[tid] |= aux;
    }
}

unsigned long long confere_tlf_5(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2 + ( (v>>29)%2 ) * -14 + ( (v>>26)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>40)%2 ) * -14 + 0 * -14 + ( (v>>26)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>39)%2 ) * 2 + ( (v>>15)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2 + ( (v>>33)%2 ) * -6 + ( (v>>39)%2 ) * -6 + ( (v>>4)%2 ) * -6 + ( (v>>28)%2 ) * -6 + ( (v>>3)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>28)%2 ) * -6 + ( (v>>3)%2 ) * -6 + ( (v>>39)%2 ) * -6 + ( (v>>4)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2 + ( (v>>33)%2 ) * -30 + ( (v>>22)%2 ) * 2 + ( (v>>2)%2 ) * 2 + ( (v>>39)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>36)%2 ) * -2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>36)%2 ) * -2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2 + ( (v>>32)%2 ) * -2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2 + ( (v>>31)%2 ) * -2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>41)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>49)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2 + ( (v>>39)%2 ) * -14 + ( (v>>3)%2 ) * -14 + ( (v>>4)%2 ) * -14 + ( (v>>28)%2 ) * 2 + ( (v>>33)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>40)%2 ) * -14 + ( (v>>0)%2 ) * -14 + ( (v>>35)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>32)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>15)%2 ) * 2 + ( (v>>23)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2 + ( (v>>14)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2 + ( (v>>4)%2 ) * -2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + ( (v>>31)%2 ) * -6 + ( (v>>16)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>4)%2 ) * -6 + ( (v>>28)%2 ) * -6 + ( (v>>3)%2 ) * -6 + ( (v>>32)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2 + ( (v>>1)%2 ) * -254 + ( (v>>29)%2 ) * -254 + ( (v>>0)%2 ) * 2 + 0 * 2 + ( (v>>28)%2 ) * 2 + ( (v>>39)%2 ) * 2 + ( (v>>4)%2 ) * 2 + ( (v>>3)%2 ) * 2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>44)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 45 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 46 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 47 ;
        aux |=(unsigned long long) ( (v>>48)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}


//REDE 6
__global__ void passo_bool_6_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>29)%2 & ( ( ( (v>>0)%2 | (v>>52)%2 ) ) ) ) & ! ( (v>>31)%2 ) ) & ! ( (v>>33)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>2)%2 & ( ( ( (v>>44)%2 & (v>>31)%2 ) ) & ( ( (v>>5)%2 ) ) ) ) | ( (v>>30)%2 & ( ( ( (v>>5)%2 ) ) & ( ( (v>>6)%2 ) ) & ( ( (v>>44)%2 & (v>>31)%2 ) ) ) )<< 1 ;
        aux |=(unsigned long long) ( (v>>26)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>30)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 2 ;
        aux |=(unsigned long long) ( (v>>37)%2 ) | ( (v>>3)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>4)%2 ) | ( (v>>37)%2 )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>5)%2 ) & ! ( (v>>35)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( (v>>2)%2 & ( ( ( (v>>30)%2 ) ) ) ) | ( ( (v>>5)%2 ) & ! ( (v>>34)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>9)%2 )<< 7 ;
        aux |=(unsigned long long) ( (v>>10)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>18)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>36)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 9 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 10 ;
        aux |=(unsigned long long) ( (v>>32)%2 & ( ( ( (v>>40)%2 ) ) ) )<< 11 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 12 ;
        aux |=(unsigned long long) ( (v>>52)%2 ) | ( (v>>0)%2 )<< 13 ;
    
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_6_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>52)%2 ) | ( (v>>0)%2 )<< 13 ;
        aux |=(unsigned long long) ( ( (v>>52)%2 ) & ! ( (v>>15)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>28)%2 & ( ( ( (v>>25)%2 ) ) ) ) | ( (v>>22)%2 & ( ( ( (v>>25)%2 ) ) ) )<< 15 ;
        aux |=(unsigned long long) ( (v>>18)%2 ) | ( (v>>45)%2 ) | ( (v>>10)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>18)%2 ) | ( (v>>16)%2 )<< 17 ;
        aux |=(unsigned long long) ( (v>>9)%2 ) | ( (v>>31)%2 ) | ( ( ( (v>>44)%2 ) & ! ( (v>>20)%2 ) ) & ! ( (v>>23)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( (v>>20)%2 ) | ( (v>>49)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>51)%2 ) | ( (v>>31)%2 ) | ( (v>>48)%2 & ( ( ( (v>>40)%2 ) ) ) )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>7)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>24)%2 ) ) | ( ( (v>>8)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>24)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>48)%2 ) | ( (v>>49)%2 ) | ( (v>>15)%2 ) | ( (v>>14)%2 & ( ( ( (v>>52)%2 ) ) ) )<< 22 ;
        aux |=(unsigned long long) ( (v>>24)%2 )<< 23 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( ( ( (v>>46)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) ) | ( ( ( (v>>47)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) ) | ( ( ( (v>>7)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) ) | ( ( ( (v>>8)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) )<< 24 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( (v>>47)%2 )<< 25 ;
    
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_6_parte3(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>3)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>26)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 26 ;
        aux |=(unsigned long long) ( (v>>52)%2 & ( ( ( (v>>4)%2 ) ) ) )<< 27 ;
        aux |=(unsigned long long) ( (v>>22)%2 & ( ( ( (v>>4)%2 ) ) ) ) | ( (v>>24)%2 & ( ( ( (v>>4)%2 ) ) ) )<< 28 ;
        aux |=(unsigned long long) ( (v>>4)%2 )<< 29 ;
        aux |=(unsigned long long) ( (v>>3)%2 ) | ( (v>>30)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>18)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>36)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 31 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 32 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) & ! ( (v>>23)%2 ) ) & ! ( (v>>19)%2 ) ) | ( (v>>36)%2 & ( ( ( (v>>0)%2 ) ) ) )<< 33 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>1)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 35 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) ) | ( ( ( (v>>12)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) ) | ( ( ( (v>>0)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) ) | ( ( ( (v>>1)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) )<< 36 ;
        aux |=(unsigned long long) ( (v>>7)%2 ) | ( (v>>8)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( (v>>14)%2 )<< 38 ;
        aux |=(unsigned long long) ( (v>>33)%2 ) | ( (v>>32)%2 )<< 39 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) & ! ( (v>>30)%2 ) ) & ! ( (v>>26)%2 ) )<< 40 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>7)%2 & ( ( ( (v>>21)%2 & (v>>37)%2 ) ) ) )<< 42 ;
        aux |=(unsigned long long) ( (v>>8)%2 & ( ( ( (v>>21)%2 ) ) & ( ( (v>>8)%2 ) ) & ( ( (v>>37)%2 ) ) ) )<< 43 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 45 ;
        aux |=(unsigned long long) ( ( (v>>7)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>21)%2 ) )<< 46 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>21)%2 ) )<< 47 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>7)%2 & ( ( ( (v>>41)%2 ) ) & ( ( (v>>37)%2 ) ) ) )<< 50 ;
        aux |=(unsigned long long) ( (v>>50)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;
    
        estado[tid] |= aux;
    }
}

unsigned long long confere_bool_6(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>29)%2 & ( ( ( (v>>0)%2 | (v>>52)%2 ) ) ) ) & ! ( (v>>31)%2 ) ) & ! ( (v>>33)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>2)%2 & ( ( ( (v>>44)%2 & (v>>31)%2 ) ) & ( ( (v>>5)%2 ) ) ) ) | ( (v>>30)%2 & ( ( ( (v>>5)%2 ) ) & ( ( (v>>6)%2 ) ) & ( ( (v>>44)%2 & (v>>31)%2 ) ) ) )<< 1 ;
        aux |=(unsigned long long) ( (v>>26)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>30)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 2 ;
        aux |=(unsigned long long) ( (v>>37)%2 ) | ( (v>>3)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>4)%2 ) | ( (v>>37)%2 )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>5)%2 ) & ! ( (v>>35)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( (v>>2)%2 & ( ( ( (v>>30)%2 ) ) ) ) | ( ( (v>>5)%2 ) & ! ( (v>>34)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>9)%2 )<< 7 ;
        aux |=(unsigned long long) ( (v>>10)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>18)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>36)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 9 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 10 ;
        aux |=(unsigned long long) ( (v>>32)%2 & ( ( ( (v>>40)%2 ) ) ) )<< 11 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 12 ;
        aux |=(unsigned long long) ( (v>>52)%2 ) | ( (v>>0)%2 )<< 13 ;
        aux |=(unsigned long long) ( ( (v>>52)%2 ) & ! ( (v>>15)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>28)%2 & ( ( ( (v>>25)%2 ) ) ) ) | ( (v>>22)%2 & ( ( ( (v>>25)%2 ) ) ) )<< 15 ;
        aux |=(unsigned long long) ( (v>>18)%2 ) | ( (v>>45)%2 ) | ( (v>>10)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>18)%2 ) | ( (v>>16)%2 )<< 17 ;
        aux |=(unsigned long long) ( (v>>9)%2 ) | ( (v>>31)%2 ) | ( ( ( (v>>44)%2 ) & ! ( (v>>20)%2 ) ) & ! ( (v>>23)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( (v>>20)%2 ) | ( (v>>49)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>51)%2 ) | ( (v>>31)%2 ) | ( (v>>48)%2 & ( ( ( (v>>40)%2 ) ) ) )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>7)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>24)%2 ) ) | ( ( (v>>8)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>24)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>48)%2 ) | ( (v>>49)%2 ) | ( (v>>15)%2 ) | ( (v>>14)%2 & ( ( ( (v>>52)%2 ) ) ) )<< 22 ;
        aux |=(unsigned long long) ( (v>>24)%2 )<< 23 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( ( ( (v>>46)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) ) | ( ( ( (v>>47)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) ) | ( ( ( (v>>7)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) ) | ( ( ( (v>>8)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>16)%2 ) ) & ! ( (v>>21)%2 ) )<< 24 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( (v>>47)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>3)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>26)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 26 ;
        aux |=(unsigned long long) ( (v>>52)%2 & ( ( ( (v>>4)%2 ) ) ) )<< 27 ;
        aux |=(unsigned long long) ( (v>>22)%2 & ( ( ( (v>>4)%2 ) ) ) ) | ( (v>>24)%2 & ( ( ( (v>>4)%2 ) ) ) )<< 28 ;
        aux |=(unsigned long long) ( (v>>4)%2 )<< 29 ;
        aux |=(unsigned long long) ( (v>>3)%2 ) | ( (v>>30)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>18)%2 & ( ( ( (v>>5)%2 ) ) ) ) | ( (v>>36)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 31 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 32 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) & ! ( (v>>23)%2 ) ) & ! ( (v>>19)%2 ) ) | ( (v>>36)%2 & ( ( ( (v>>0)%2 ) ) ) )<< 33 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>1)%2 & ( ( ( (v>>5)%2 ) ) ) )<< 35 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) ) | ( ( ( (v>>12)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) ) | ( ( ( (v>>0)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) ) | ( ( ( (v>>1)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>27)%2 ) )<< 36 ;
        aux |=(unsigned long long) ( (v>>7)%2 ) | ( (v>>8)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( (v>>14)%2 )<< 38 ;
        aux |=(unsigned long long) ( (v>>33)%2 ) | ( (v>>32)%2 )<< 39 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) & ! ( (v>>30)%2 ) ) & ! ( (v>>26)%2 ) )<< 40 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>7)%2 & ( ( ( (v>>21)%2 & (v>>37)%2 ) ) ) )<< 42 ;
        aux |=(unsigned long long) ( (v>>8)%2 & ( ( ( (v>>21)%2 ) ) & ( ( (v>>8)%2 ) ) & ( ( (v>>37)%2 ) ) ) )<< 43 ;
        aux |=(unsigned long long) ( (v>>42)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 45 ;
        aux |=(unsigned long long) ( ( (v>>7)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>21)%2 ) )<< 46 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>21)%2 ) )<< 47 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>7)%2 & ( ( ( (v>>41)%2 ) ) & ( ( (v>>37)%2 ) ) ) )<< 50 ;
        aux |=(unsigned long long) ( (v>>50)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_6_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 6 + ( (v>>0)%2 ) * 2 + 0 * 2 + ( (v>>31)%2 ) * -6 + ( (v>>33)%2 ) * -6) >= 8 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 6 + ( (v>>44)%2 ) * 10 + ( (v>>31)%2 ) * 10 + ( (v>>5)%2 ) * 10 + ( (v>>30)%2 ) * 2 + ( (v>>6)%2 ) * 2) >= 34 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>30)%2 ) * 2) >= 8 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 2 + ( (v>>3)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>35)%2 ) * -2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 6 + ( (v>>30)%2 ) * 6 + ( (v>>5)%2 ) * 6 + ( (v>>34)%2 ) * -6) >= 6 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>36)%2 ) * 2) >= 8 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2 + ( (v>>40)%2 ) * 2) >= 4 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>0)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>15)%2 ) * -2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>25)%2 ) * 6 + ( (v>>22)%2 ) * 2) >= 8 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>45)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 16 ;
        
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_6_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>16)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 14 + ( (v>>31)%2 ) * 14 + ( (v>>44)%2 ) * 2 + ( (v>>20)%2 ) * -2 + ( (v>>23)%2 ) * -2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>49)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>51)%2 ) * 6 + ( (v>>31)%2 ) * 6 + ( (v>>48)%2 ) * 2 + ( (v>>40)%2 ) * 2) >= 4 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>37)%2 ) * 6 + ( (v>>24)%2 ) * -6 + ( (v>>8)%2 ) * 2) >= 8 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 6 + ( (v>>49)%2 ) * 6 + ( (v>>15)%2 ) * 6 + ( (v>>14)%2 ) * 2 + ( (v>>52)%2 ) * 2) >= 4 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 202 + ( (v>>46)%2 ) * 10 + ( (v>>16)%2 ) * -54 + ( (v>>21)%2 ) * -54 + ( (v>>47)%2 ) * 10 + ( (v>>7)%2 ) * 2 + ( (v>>37)%2 ) * 6 + ( (v>>8)%2 ) * 2) >= 8 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>47)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>26)%2 ) * 2) >= 8 ) << 26 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>4)%2 ) * 2) >= 4 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>4)%2 ) * 6 + ( (v>>24)%2 ) * 2) >= 8 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2 + ( (v>>30)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>36)%2 ) * 2) >= 8 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 6 + ( (v>>23)%2 ) * -6 + ( (v>>19)%2 ) * -6 + ( (v>>36)%2 ) * 14 + ( (v>>0)%2 ) * 14) >= 6 ) << 33 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_6_parte3(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>5)%2 ) * 2) >= 4 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>19)%2 ) * -30 + ( (v>>27)%2 ) * -30 + ( (v>>12)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>14)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>32)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>30)%2 ) * -2 + ( (v>>26)%2 ) * -2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>21)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 6 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>21)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 6 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>21)%2 ) * -2) >= 4 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>21)%2 ) * -2) >= 4 ) << 47 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 48 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 49 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>41)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 6 ) << 50 ;
        aux |=(unsigned long long) ( ( ( (v>>50)%2 ) * 2) >= 2 ) << 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;
        
        estado[tid] |= aux;
    }
}

unsigned long long confere_tlf_6(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 6 + ( (v>>0)%2 ) * 2 + 0 * 2 + ( (v>>31)%2 ) * -6 + ( (v>>33)%2 ) * -6) >= 8 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 6 + ( (v>>44)%2 ) * 10 + ( (v>>31)%2 ) * 10 + ( (v>>5)%2 ) * 10 + ( (v>>30)%2 ) * 2 + ( (v>>6)%2 ) * 2) >= 34 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>30)%2 ) * 2) >= 8 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 2 + ( (v>>3)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>35)%2 ) * -2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 6 + ( (v>>30)%2 ) * 6 + ( (v>>5)%2 ) * 6 + ( (v>>34)%2 ) * -6) >= 6 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>36)%2 ) * 2) >= 8 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2 + ( (v>>40)%2 ) * 2) >= 4 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>0)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>15)%2 ) * -2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>25)%2 ) * 6 + ( (v>>22)%2 ) * 2) >= 8 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>45)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>16)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 14 + ( (v>>31)%2 ) * 14 + ( (v>>44)%2 ) * 2 + ( (v>>20)%2 ) * -2 + ( (v>>23)%2 ) * -2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>49)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>51)%2 ) * 6 + ( (v>>31)%2 ) * 6 + ( (v>>48)%2 ) * 2 + ( (v>>40)%2 ) * 2) >= 4 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>37)%2 ) * 6 + ( (v>>24)%2 ) * -6 + ( (v>>8)%2 ) * 2) >= 8 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 6 + ( (v>>49)%2 ) * 6 + ( (v>>15)%2 ) * 6 + ( (v>>14)%2 ) * 2 + ( (v>>52)%2 ) * 2) >= 4 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 202 + ( (v>>46)%2 ) * 10 + ( (v>>16)%2 ) * -54 + ( (v>>21)%2 ) * -54 + ( (v>>47)%2 ) * 10 + ( (v>>7)%2 ) * 2 + ( (v>>37)%2 ) * 6 + ( (v>>8)%2 ) * 2) >= 8 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>47)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>26)%2 ) * 2) >= 8 ) << 26 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>4)%2 ) * 2) >= 4 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>4)%2 ) * 6 + ( (v>>24)%2 ) * 2) >= 8 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>4)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2 + ( (v>>30)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>5)%2 ) * 6 + ( (v>>36)%2 ) * 2) >= 8 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 6 + ( (v>>23)%2 ) * -6 + ( (v>>19)%2 ) * -6 + ( (v>>36)%2 ) * 14 + ( (v>>0)%2 ) * 14) >= 6 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>5)%2 ) * 2) >= 4 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>19)%2 ) * -30 + ( (v>>27)%2 ) * -30 + ( (v>>12)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>14)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>32)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>30)%2 ) * -2 + ( (v>>26)%2 ) * -2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>21)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 6 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>21)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 6 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>21)%2 ) * -2) >= 4 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>21)%2 ) * -2) >= 4 ) << 47 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 48 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 49 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>41)%2 ) * 2 + ( (v>>37)%2 ) * 2) >= 6 ) << 50 ;
        aux |=(unsigned long long) ( ( ( (v>>50)%2 ) * 2) >= 2 ) << 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}


//REDE 7
__global__ void passo_bool_7_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( (v>>28)%2 ) & !( (v>>33)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>20)%2 & ( ( ( (v>>12)%2 | (v>>2)%2 ) ) ) )<< 1 ;
        aux |=(unsigned long long) ( (v>>19)%2 ) | ( (v>>46)%2 )<< 2 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 3 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 & ( ( ( (v>>47)%2 ) ) ) ) & !( (v>>5)%2 ) ) & !( (v>>10)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( (v>>6)%2 & ( ( ( (v>>0)%2 ) ) ) )<< 5 ;
        aux |=(unsigned long long) ( (v>>25)%2 )<< 6 ;
        aux |=(unsigned long long) ( (v>>6)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( ( (v>>50)%2 ) & !( (v>>30)%2 ) ) & !( (v>>17)%2 ) ) | ( ( ( (v>>40)%2 ) & !( (v>>30)%2 ) ) & !( (v>>17)%2 ) )<< 8 ;
        aux |=(unsigned long long) ( (v>>46)%2 ) | ( (v>>19)%2 ) | ( (v>>10)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>24)%2 )<< 10 ;
        aux |=(unsigned long long) ( ( ( (v>>51)%2 ) & !( (v>>17)%2 ) ) & !( (v>>30)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( (v>>10)%2 & ( ( ( (v>>37)%2 ) & ( ( ( (v>>9)%2 | (v>>6)%2 ) ) ) ) ) )<< 12 ;
        aux |=(unsigned long long) ( ( (v>>19)%2 ) & !( (v>>0)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) & !( (v>>40)%2 ) ) & !( (v>>17)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>29)%2 ) | ( (v>>17)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>47)%2 ) | ( (v>>38)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>14)%2 ) | ( (v>>43)%2 ) | ( (v>>8)%2 )<< 17 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 18 ;
        aux |=(unsigned long long) ( (v>>41)%2 & ( ( ( (v>>42)%2 ) ) ) ) | ( (v>>21)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>26)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>42)%2 & ( ( ( (v>>26)%2 ) ) ) )<< 19 ;
        aux |=(unsigned long long) ( (v>>19)%2 )<< 20 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 21 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 22 ;
    
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_7_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( (v>>0)%2 ) & !( (v>>44)%2 ) ) | ( ( (v>>47)%2 ) & !( (v>>44)%2 ) )<< 23 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) & !( (v>>1)%2 ) ) & !( (v>>32)%2 ) ) | ( ( ( (v>>21)%2 ) & !( (v>>1)%2 ) ) & !( (v>>32)%2 ) )<< 24 ;
        aux |=(unsigned long long) ( (v>>46)%2 ) | ( (v>>10)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>25)%2 & ( ( ( (v>>0)%2 ) ) ) )<< 27 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 28 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( (v>>36)%2 & ( ( ( (v>>39)%2 ) ) ) )<< 29 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>8)%2 ) | ( (v>>11)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 33 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 & ( ( ( (v>>27)%2 ) ) ) ) & !( (v>>45)%2 ) )<< 34 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) & !( (v>>0)%2 ) ) & !( (v>>10)%2 ) ) | ( ( ( (v>>30)%2 ) & !( (v>>0)%2 ) ) & !( (v>>10)%2 ) )<< 35 ;
        aux |=(unsigned long long) ( (v>>39)%2 ) | ( (v>>31)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>10)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 38 ;
        aux |=(unsigned long long) ( ( (v>>17)%2 ) & !( (v>>37)%2 ) )<< 39 ;
        aux |=(unsigned long long) ( (v>>10)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>3)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>27)%2 )<< 44 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & !( (v>>0)%2 ) )<< 45 ;
        aux |=(unsigned long long) ( (v>>41)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>21)%2 & ( ( ( (v>>26)%2 ) ) ) ) | ( (v>>26)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>42)%2 & ( ( ( (v>>21)%2 ) ) ) )<< 46 ;
        aux |=(unsigned long long) ( (v>>46)%2 & ( ( ( (v>>54)%2 ) ) ) ) | ( (v>>3)%2 & ( ( ( (v>>54)%2 ) ) ) )<< 47 ;
        aux |=(unsigned long long) ( (v>>28)%2 & ( ( ( (v>>10)%2 ) ) ) )<< 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>50)%2 )<< 50 ;
        aux |=(unsigned long long) ( (v>>51)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;
    
        estado[tid] |= aux;
    }
}

unsigned long long confere_bool_7(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( (v>>28)%2 ) & !( (v>>33)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>20)%2 & ( ( ( (v>>12)%2 | (v>>2)%2 ) ) ) )<< 1 ;
        aux |=(unsigned long long) ( (v>>19)%2 ) | ( (v>>46)%2 )<< 2 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 3 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 & ( ( ( (v>>47)%2 ) ) ) ) & !( (v>>5)%2 ) ) & !( (v>>10)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( (v>>6)%2 & ( ( ( (v>>0)%2 ) ) ) )<< 5 ;
        aux |=(unsigned long long) ( (v>>25)%2 )<< 6 ;
        aux |=(unsigned long long) ( (v>>6)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( ( (v>>50)%2 ) & !( (v>>30)%2 ) ) & !( (v>>17)%2 ) ) | ( ( ( (v>>40)%2 ) & !( (v>>30)%2 ) ) & !( (v>>17)%2 ) )<< 8 ;
        aux |=(unsigned long long) ( (v>>46)%2 ) | ( (v>>19)%2 ) | ( (v>>10)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>24)%2 )<< 10 ;
        aux |=(unsigned long long) ( ( ( (v>>51)%2 ) & !( (v>>17)%2 ) ) & !( (v>>30)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( (v>>10)%2 & ( ( ( (v>>37)%2 ) & ( ( ( (v>>9)%2 | (v>>6)%2 ) ) ) ) ) )<< 12 ;
        aux |=(unsigned long long) ( ( (v>>19)%2 ) & !( (v>>0)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) & !( (v>>40)%2 ) ) & !( (v>>17)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>29)%2 ) | ( (v>>17)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>47)%2 ) | ( (v>>38)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>14)%2 ) | ( (v>>43)%2 ) | ( (v>>8)%2 )<< 17 ;
        aux |=(unsigned long long) ( (v>>45)%2 )<< 18 ;
        aux |=(unsigned long long) ( (v>>41)%2 & ( ( ( (v>>42)%2 ) ) ) ) | ( (v>>21)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>26)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>42)%2 & ( ( ( (v>>26)%2 ) ) ) )<< 19 ;
        aux |=(unsigned long long) ( (v>>19)%2 )<< 20 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 21 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 22 ;
        aux |=(unsigned long long) ( ( (v>>0)%2 ) & !( (v>>44)%2 ) ) | ( ( (v>>47)%2 ) & !( (v>>44)%2 ) )<< 23 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) & !( (v>>1)%2 ) ) & !( (v>>32)%2 ) ) | ( ( ( (v>>21)%2 ) & !( (v>>1)%2 ) ) & !( (v>>32)%2 ) )<< 24 ;
        aux |=(unsigned long long) ( (v>>46)%2 ) | ( (v>>10)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>25)%2 & ( ( ( (v>>0)%2 ) ) ) )<< 27 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 28 ;
        aux |=(unsigned long long) ( (v>>15)%2 ) | ( (v>>36)%2 & ( ( ( (v>>39)%2 ) ) ) )<< 29 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>8)%2 ) | ( (v>>11)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 33 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 & ( ( ( (v>>27)%2 ) ) ) ) & !( (v>>45)%2 ) )<< 34 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) & !( (v>>0)%2 ) ) & !( (v>>10)%2 ) ) | ( ( ( (v>>30)%2 ) & !( (v>>0)%2 ) ) & !( (v>>10)%2 ) )<< 35 ;
        aux |=(unsigned long long) ( (v>>39)%2 ) | ( (v>>31)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>10)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 38 ;
        aux |=(unsigned long long) ( ( (v>>17)%2 ) & !( (v>>37)%2 ) )<< 39 ;
        aux |=(unsigned long long) ( (v>>10)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>43)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>3)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>27)%2 )<< 44 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & !( (v>>0)%2 ) )<< 45 ;
        aux |=(unsigned long long) ( (v>>41)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>21)%2 & ( ( ( (v>>26)%2 ) ) ) ) | ( (v>>26)%2 & ( ( ( (v>>52)%2 ) ) ) ) | ( (v>>42)%2 & ( ( ( (v>>21)%2 ) ) ) )<< 46 ;
        aux |=(unsigned long long) ( (v>>46)%2 & ( ( ( (v>>54)%2 ) ) ) ) | ( (v>>3)%2 & ( ( ( (v>>54)%2 ) ) ) )<< 47 ;
        aux |=(unsigned long long) ( (v>>28)%2 & ( ( ( (v>>10)%2 ) ) ) )<< 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>50)%2 )<< 50 ;
        aux |=(unsigned long long) ( (v>>51)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_7_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>33)%2 ) * -2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 6 + ( (v>>12)%2 ) * 2 + ( (v>>2)%2 ) * 2) >= 8 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2 + ( (v>>46)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>49)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>47)%2 ) * 2 + ( (v>>5)%2 ) * -2 + ( (v>>10)%2 ) * -2) >= 4 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 4 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>30)%2 ) * -6 + ( (v>>17)%2 ) * -6 + ( (v>>40)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>19)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>17)%2 ) * -2 + ( (v>>30)%2 ) * -2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 6 + ( (v>>37)%2 ) * 6 + ( (v>>9)%2 ) * 2 + ( (v>>6)%2 ) * 2) >= 14 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2 + ( (v>>0)%2 ) * -2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2 + ( (v>>40)%2 ) * -2 + ( (v>>17)%2 ) * -2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>17)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>38)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>14)%2 ) * 2 + ( (v>>43)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 6 + ( (v>>42)%2 ) * 14 + ( (v>>21)%2 ) * 6 + 0 * 14 + ( (v>>26)%2 ) * 10) >= 20 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 22 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_7_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>44)%2 ) * -6 + ( (v>>47)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>32)%2 ) * -6 + ( (v>>21)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 4 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 6 + ( (v>>36)%2 ) * 2 + ( (v>>39)%2 ) * 2) >= 4 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>11)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2 + ( (v>>27)%2 ) * 2 + ( (v>>45)%2 ) * -2) >= 4 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>0)%2 ) * -6 + ( (v>>10)%2 ) * -6 + ( (v>>30)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>31)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2 + ( (v>>37)%2 ) * -2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2) >= 2 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>53)%2 ) * 2) >= 2 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>0)%2 ) * -2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 6 + 0 * 14 + ( (v>>21)%2 ) * 14 + ( (v>>26)%2 ) * 10 + ( (v>>42)%2 ) * 6) >= 20 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + 0 * 6 + ( (v>>3)%2 ) * 2) >= 8 ) << 47 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 4 ) << 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>50)%2 )<< 50 ;
        aux |=(unsigned long long) ( (v>>51)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;
        
        estado[tid] |= aux;
    }
}


unsigned long long confere_tlf_7(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>33)%2 ) * -2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 6 + ( (v>>12)%2 ) * 2 + ( (v>>2)%2 ) * 2) >= 8 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2 + ( (v>>46)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>49)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>47)%2 ) * 2 + ( (v>>5)%2 ) * -2 + ( (v>>10)%2 ) * -2) >= 4 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 4 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>30)%2 ) * -6 + ( (v>>17)%2 ) * -6 + ( (v>>40)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>19)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>24)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>17)%2 ) * -2 + ( (v>>30)%2 ) * -2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 6 + ( (v>>37)%2 ) * 6 + ( (v>>9)%2 ) * 2 + ( (v>>6)%2 ) * 2) >= 14 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2 + ( (v>>0)%2 ) * -2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2 + ( (v>>40)%2 ) * -2 + ( (v>>17)%2 ) * -2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>17)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>38)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>14)%2 ) * 2 + ( (v>>43)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 6 + ( (v>>42)%2 ) * 14 + ( (v>>21)%2 ) * 6 + 0 * 14 + ( (v>>26)%2 ) * 10) >= 20 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>44)%2 ) * -6 + ( (v>>47)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>32)%2 ) * -6 + ( (v>>21)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 4 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 6 + ( (v>>36)%2 ) * 2 + ( (v>>39)%2 ) * 2) >= 4 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>11)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2 + ( (v>>27)%2 ) * 2 + ( (v>>45)%2 ) * -2) >= 4 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>0)%2 ) * -6 + ( (v>>10)%2 ) * -6 + ( (v>>30)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2 + ( (v>>31)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2 + ( (v>>37)%2 ) * -2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2) >= 2 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>53)%2 ) * 2) >= 2 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>0)%2 ) * -2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 6 + 0 * 14 + ( (v>>21)%2 ) * 14 + ( (v>>26)%2 ) * 10 + ( (v>>42)%2 ) * 6) >= 20 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + 0 * 6 + ( (v>>3)%2 ) * 2) >= 8 ) << 47 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 4 ) << 48 ;
        aux |=(unsigned long long) ( (v>>49)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>50)%2 )<< 50 ;
        aux |=(unsigned long long) ( (v>>51)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 52 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 8
__global__ void passo_bool_8_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>6)%2 ) | ( (v>>1)%2 )<< 1 ;
        aux |=(unsigned long long) ( ( ( ( (v>>6)%2 ) & ! ( (v>>30)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>3)%2 ) ) | ( ( ( ( (v>>19)%2 ) & ! ( (v>>30)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>3)%2 ) )<< 2 ;
        aux |=(unsigned long long) ( ( ( ( ( (v>>46)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>2)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>9)%2 ) ) | ( ( ( ( ( (v>>33)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>2)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>9)%2 ) )<< 3 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 & ( ( ( (v>>21)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 ) & ! ( (v>>1)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 & ( ( ( (v>>2)%2 ) ) ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>20)%2 ) ) | ( ( (v>>9)%2 ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>51)%2 & ( ( ( (v>>2)%2 ) ) ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>20)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>42)%2 ) )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>12)%2 ) & ! ( (v>>1)%2 ) )<< 8 ;
        aux |=(unsigned long long) ( ( (v>>15)%2 & ( ( ( (v>>7)%2 ) ) | ( ( (v>>13)%2 ) & ( ( ( ! (v>>11)%2 ) ) ) ) | ( ( (v>>13)%2 & (v>>23)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>31)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) & ! ( (v>>9)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>4)%2 & ( ( ( (v>>21)%2 ) ) ) ) & ! ( (v>>9)%2 ) ) & ! ( (v>>1)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>1)%2 ) )<< 12 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 & ( ( ( (v>>14)%2 ) ) ) ) & ! ( (v>>53)%2 ) ) & ! ( (v>>1)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>32)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>10)%2 ) & ! ( (v>>1)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 15 ;
        aux |=(unsigned long long) ( ( ( ( (v>>36)%2 & ( ( ( (v>>16)%2 ) ) ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>23)%2 ) ) & ! ( (v>>54)%2 ) ) | ( ( ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>23)%2 ) ) & ! ( (v>>54)%2 ) )<< 16 ;
        aux |=(unsigned long long) ( ( (v>>42)%2 ) & ! ( (v>>1)%2 ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>26)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>52)%2 ) & ! ( (v>>1)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>4)%2 & ( ( ( (v>>21)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 19 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) & ! ( (v>>2)%2 ) ) & ! ( (v>>1)%2 ) )<< 20 ;
        aux |=(unsigned long long) ( ( ( ( (v>>54)%2 & ( ( ( (v>>22)%2 ) ) ) ) & ! ( (v>>34)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>43)%2 ) ) | ( ( ( ( (v>>23)%2 & ( ( ( (v>>22)%2 ) ) ) ) & ! ( (v>>34)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>43)%2 ) ) | ( ( ( ( (v>>55)%2 & ( ( ( (v>>22)%2 ) ) ) ) & ! ( (v>>34)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>43)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>32)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) )<< 22 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>47)%2 ) ) | ( ( ( (v>>32)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>47)%2 ) ) | ( ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>47)%2 ) )<< 23 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 & ( ( ( (v>>25)%2 ) ) ) ) & ! ( (v>>24)%2 ) ) & ! ( (v>>1)%2 ) )<< 24 ;
        aux |=(unsigned long long) ( ( (v>>23)%2 & ( ( ( (v>>46)%2 | (v>>33)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 25 ;
        aux |=(unsigned long long) ( ( (v>>27)%2 & ( ( ( (v>>23)%2 | (v>>54)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 26 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 & ( ( ( (v>>47)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 27 ;
        aux |=(unsigned long long) ( ( ( ( (v>>21)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) ) | ( ( ( ( (v>>40)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) ) | ( ( ( ( (v>>24)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) ) | ( ( ( ( (v>>26)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) )<< 28 ;
        aux |=(unsigned long long) ( ( (v>>56)%2 ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>48)%2 ) & ! ( (v>>52)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>26)%2 ) & ! ( (v>>52)%2 ) ) & ! ( (v>>1)%2 ) )<< 29 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 & ( ( ( (v>>46)%2 & (v>>37)%2 & (v>>33)%2 ) ) ) ) & ! ( (v>>9)%2 ) ) & ! ( (v>>1)%2 ) )<< 30 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_8_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) )<< 31 ;
        aux |=(unsigned long long) ( ( (v>>37)%2 ) & ! ( (v>>1)%2 ) )<< 32 ;
        aux |=(unsigned long long) ( ( (v>>37)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>50)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>11)%2 & ( ( ( (v>>51)%2 & (v>>20)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 33 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) & ! ( (v>>57)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>34)%2 ) & ! ( (v>>57)%2 ) ) & ! ( (v>>1)%2 ) )<< 34 ;
        aux |=(unsigned long long) ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) )<< 35 ;
        aux |=(unsigned long long) ( ( (v>>58)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>42)%2 ) & ! ( (v>>1)%2 ) )<< 36 ;
        aux |=(unsigned long long) ( ( (v>>36)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) )<< 37 ;
        aux |=(unsigned long long) ( ( (v>>36)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>18)%2 ) & ! ( (v>>1)%2 ) )<< 38 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>35)%2 ) )<< 39 ;
        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 40 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>38)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>1)%2 ) )<< 41 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) & ! ( (v>>7)%2 ) ) & ! ( (v>>1)%2 ) )<< 42 ;
        aux |=(unsigned long long) ( ( (v>>17)%2 ) & ! ( (v>>1)%2 ) )<< 43 ;
        aux |=(unsigned long long) ( ( ( ( (v>>28)%2 ) & ! ( (v>>54)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>23)%2 ) )<< 44 ;
        aux |=(unsigned long long) ( ( (v>>36)%2 ) & ! ( (v>>1)%2 ) )<< 45 ;
        aux |=(unsigned long long) ( ( (v>>28)%2 ) & ! ( (v>>1)%2 ) )<< 46 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>28)%2 ) & ! ( (v>>1)%2 ) )<< 47 ;
        aux |=(unsigned long long) ( ( ( (v>>55)%2 ) & ! ( (v>>5)%2 ) ) & ! ( (v>>1)%2 ) )<< 48 ;
        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 49 ;
        aux |=(unsigned long long) ( ( (v>>59)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>37)%2 & ( ( ( (v>>49)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 50 ;
        aux |=(unsigned long long) ( ( ( ( (v>>49)%2 ) & ! ( (v>>20)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>0)%2 ) )<< 51 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>12)%2 ) )<< 52 ;
        aux |=(unsigned long long) ( ( (v>>15)%2 & ( ( ( (v>>42)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;
        aux |=(unsigned long long) ( (v>>55)%2 )<< 55 ;
        aux |=(unsigned long long) ( (v>>56)%2 )<< 56 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 57 ;
        aux |=(unsigned long long) ( (v>>58)%2 )<< 58 ;
        aux |=(unsigned long long) ( (v>>59)%2 )<< 59 ;
    
        estado[tid] |= aux;
    }
}



unsigned long long confere_bool_8(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 0 ;
        aux |=(unsigned long long) ( (v>>6)%2 ) | ( (v>>1)%2 )<< 1 ;
        aux |=(unsigned long long) ( ( ( ( (v>>6)%2 ) & ! ( (v>>30)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>3)%2 ) ) | ( ( ( ( (v>>19)%2 ) & ! ( (v>>30)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>3)%2 ) )<< 2 ;
        aux |=(unsigned long long) ( ( ( ( ( (v>>46)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>2)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>9)%2 ) ) | ( ( ( ( ( (v>>33)%2 ) & ! ( (v>>19)%2 ) ) & ! ( (v>>2)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>9)%2 ) )<< 3 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 & ( ( ( (v>>21)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 ) & ! ( (v>>1)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 & ( ( ( (v>>2)%2 ) ) ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>20)%2 ) ) | ( ( (v>>9)%2 ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>51)%2 & ( ( ( (v>>2)%2 ) ) ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>20)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>42)%2 ) )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>12)%2 ) & ! ( (v>>1)%2 ) )<< 8 ;
        aux |=(unsigned long long) ( ( (v>>15)%2 & ( ( ( (v>>7)%2 ) ) | ( ( (v>>13)%2 ) & ( ( ( ! (v>>11)%2 ) ) ) ) | ( ( (v>>13)%2 & (v>>23)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>31)%2 & ( ( ( (v>>37)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) & ! ( (v>>9)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>4)%2 & ( ( ( (v>>21)%2 ) ) ) ) & ! ( (v>>9)%2 ) ) & ! ( (v>>1)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( ( (v>>48)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>1)%2 ) )<< 12 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 & ( ( ( (v>>14)%2 ) ) ) ) & ! ( (v>>53)%2 ) ) & ! ( (v>>1)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>32)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>10)%2 ) & ! ( (v>>1)%2 ) )<< 14 ;
        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 15 ;
        aux |=(unsigned long long) ( ( ( ( (v>>36)%2 & ( ( ( (v>>16)%2 ) ) ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>23)%2 ) ) & ! ( (v>>54)%2 ) ) | ( ( ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>23)%2 ) ) & ! ( (v>>54)%2 ) )<< 16 ;
        aux |=(unsigned long long) ( ( (v>>42)%2 ) & ! ( (v>>1)%2 ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>26)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>52)%2 ) & ! ( (v>>1)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>4)%2 & ( ( ( (v>>21)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 19 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) & ! ( (v>>2)%2 ) ) & ! ( (v>>1)%2 ) )<< 20 ;
        aux |=(unsigned long long) ( ( ( ( (v>>54)%2 & ( ( ( (v>>22)%2 ) ) ) ) & ! ( (v>>34)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>43)%2 ) ) | ( ( ( ( (v>>23)%2 & ( ( ( (v>>22)%2 ) ) ) ) & ! ( (v>>34)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>43)%2 ) ) | ( ( ( ( (v>>55)%2 & ( ( ( (v>>22)%2 ) ) ) ) & ! ( (v>>34)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>43)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>32)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) )<< 22 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>47)%2 ) ) | ( ( ( (v>>32)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>47)%2 ) ) | ( ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>47)%2 ) )<< 23 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 & ( ( ( (v>>25)%2 ) ) ) ) & ! ( (v>>24)%2 ) ) & ! ( (v>>1)%2 ) )<< 24 ;
        aux |=(unsigned long long) ( ( (v>>23)%2 & ( ( ( (v>>46)%2 | (v>>33)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 25 ;
        aux |=(unsigned long long) ( ( (v>>27)%2 & ( ( ( (v>>23)%2 | (v>>54)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 26 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 & ( ( ( (v>>47)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 27 ;
        aux |=(unsigned long long) ( ( ( ( (v>>21)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) ) | ( ( ( ( (v>>40)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) ) | ( ( ( ( (v>>24)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) ) | ( ( ( ( (v>>26)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>56)%2 ) ) & ! ( (v>>44)%2 ) )<< 28 ;
        aux |=(unsigned long long) ( ( (v>>56)%2 ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>48)%2 ) & ! ( (v>>52)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>26)%2 ) & ! ( (v>>52)%2 ) ) & ! ( (v>>1)%2 ) )<< 29 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 & ( ( ( (v>>46)%2 & (v>>37)%2 & (v>>33)%2 ) ) ) ) & ! ( (v>>9)%2 ) ) & ! ( (v>>1)%2 ) )<< 30 ;
        aux |=(unsigned long long) ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) )<< 31 ;
        aux |=(unsigned long long) ( ( (v>>37)%2 ) & ! ( (v>>1)%2 ) )<< 32 ;
        aux |=(unsigned long long) ( ( (v>>37)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>50)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>11)%2 & ( ( ( (v>>51)%2 & (v>>20)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 33 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) & ! ( (v>>57)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>34)%2 ) & ! ( (v>>57)%2 ) ) & ! ( (v>>1)%2 ) )<< 34 ;
        aux |=(unsigned long long) ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) )<< 35 ;
        aux |=(unsigned long long) ( ( (v>>58)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>42)%2 ) & ! ( (v>>1)%2 ) )<< 36 ;
        aux |=(unsigned long long) ( ( (v>>36)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>41)%2 ) & ! ( (v>>1)%2 ) )<< 37 ;
        aux |=(unsigned long long) ( ( (v>>36)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>18)%2 ) & ! ( (v>>1)%2 ) )<< 38 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>35)%2 ) )<< 39 ;
        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 40 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>1)%2 ) ) | ( ( ( (v>>38)%2 ) & ! ( (v>>16)%2 ) ) & ! ( (v>>1)%2 ) )<< 41 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) & ! ( (v>>7)%2 ) ) & ! ( (v>>1)%2 ) )<< 42 ;
        aux |=(unsigned long long) ( ( (v>>17)%2 ) & ! ( (v>>1)%2 ) )<< 43 ;
        aux |=(unsigned long long) ( ( ( ( (v>>28)%2 ) & ! ( (v>>54)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>23)%2 ) )<< 44 ;
        aux |=(unsigned long long) ( ( (v>>36)%2 ) & ! ( (v>>1)%2 ) )<< 45 ;
        aux |=(unsigned long long) ( ( (v>>28)%2 ) & ! ( (v>>1)%2 ) )<< 46 ;
        aux |=(unsigned long long) ( ( (v>>47)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>28)%2 ) & ! ( (v>>1)%2 ) )<< 47 ;
        aux |=(unsigned long long) ( ( ( (v>>55)%2 ) & ! ( (v>>5)%2 ) ) & ! ( (v>>1)%2 ) )<< 48 ;
        aux |=(unsigned long long) ( ( (v>>33)%2 ) & ! ( (v>>1)%2 ) )<< 49 ;
        aux |=(unsigned long long) ( ( (v>>59)%2 ) & ! ( (v>>1)%2 ) ) | ( ( (v>>37)%2 & ( ( ( (v>>49)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 50 ;
        aux |=(unsigned long long) ( ( ( ( (v>>49)%2 ) & ! ( (v>>20)%2 ) ) & ! ( (v>>1)%2 ) ) & ! ( (v>>0)%2 ) )<< 51 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>12)%2 ) )<< 52 ;
        aux |=(unsigned long long) ( ( (v>>15)%2 & ( ( ( (v>>42)%2 ) ) ) ) & ! ( (v>>1)%2 ) )<< 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;
        aux |=(unsigned long long) ( (v>>55)%2 )<< 55 ;
        aux |=(unsigned long long) ( (v>>56)%2 )<< 56 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 57 ;
        aux |=(unsigned long long) ( (v>>58)%2 )<< 58 ;
        aux |=(unsigned long long) ( (v>>59)%2 )<< 59 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_8_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>30)%2 ) * -6 + ( (v>>1)%2 ) * -6 + ( (v>>3)%2 ) * -6 + ( (v>>19)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>19)%2 ) * -6 + ( (v>>2)%2 ) * -6 + ( (v>>1)%2 ) * -6 + ( (v>>9)%2 ) * -6 + ( (v>>33)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>21)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2 + ( (v>>2)%2 ) * 6 + ( (v>>1)%2 ) * -38 + ( (v>>20)%2 ) * -6 + ( (v>>9)%2 ) * 26 + ( (v>>51)%2 ) * 2) >= 8 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>1)%2 ) * -2 + ( (v>>42)%2 ) * -2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 22 + ( (v>>7)%2 ) * 10 + ( (v>>13)%2 ) * 6 + ( (v>>11)%2 ) * -2 + ( (v>>23)%2 ) * 2 + ( (v>>1)%2 ) * -22) >= 28 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 6 + ( (v>>9)%2 ) * -10 + ( (v>>1)%2 ) * -10 + ( (v>>4)%2 ) * 2 + ( (v>>21)%2 ) * 2) >= 4 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>14)%2 ) * 2 + ( (v>>53)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 4 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -30 + ( (v>>32)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 14 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_8_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>16)%2 ) * 2 + ( (v>>1)%2 ) * -10 + ( (v>>23)%2 ) * -10 + 0 * -10 + ( (v>>41)%2 ) * 6) >= 4 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>52)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 6 + ( (v>>1)%2 ) * -10 + ( (v>>4)%2 ) * 2 + ( (v>>21)%2 ) * 2) >= 4 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>2)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>22)%2 ) * 14 + ( (v>>34)%2 ) * -14 + ( (v>>1)%2 ) * -14 + ( (v>>43)%2 ) * -14 + ( (v>>23)%2 ) * 2 + ( (v>>55)%2 ) * 2) >= 16 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>1)%2 ) * -14 + ( (v>>32)%2 ) * 2 + ( (v>>46)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -14 + ( (v>>47)%2 ) * -14 + ( (v>>32)%2 ) * 2 + ( (v>>33)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2 + ( (v>>25)%2 ) * 2 + ( (v>>24)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 4 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 6 + ( (v>>46)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -6) >= 8 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 6 + ( (v>>23)%2 ) * 2 + 0 * 2 + ( (v>>1)%2 ) * -6) >= 8 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>47)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + ( (v>>1)%2 ) * -30 + 0 * -30 + ( (v>>44)%2 ) * -30 + ( (v>>40)%2 ) * 2 + ( (v>>24)%2 ) * 2 + ( (v>>26)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( 0 * 10 + ( (v>>1)%2 ) * -22 + ( (v>>48)%2 ) * 2 + ( (v>>52)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2 + ( (v>>46)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>9)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 8 ) << 30 ;
        
        estado[tid] |= aux;
    }
}


__global__ void passo_tlf_8_parte3(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 14 + ( (v>>1)%2 ) * -50 + ( (v>>50)%2 ) * 14 + ( (v>>11)%2 ) * 2 + ( (v>>51)%2 ) * 2 + ( (v>>20)%2 ) * 2) >= 6 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + 0 * -6 + ( (v>>1)%2 ) * -6 + ( (v>>34)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>1)%2 ) * -6 + ( (v>>42)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>41)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>18)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -2 + ( (v>>35)%2 ) * -2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>16)%2 ) * -6 + ( (v>>1)%2 ) * -6 + ( (v>>38)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2 + ( (v>>7)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 2 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + 0 * -2 + ( (v>>1)%2 ) * -2 + ( (v>>23)%2 ) * -2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>28)%2 ) * 2) >= 2 ) << 47 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>5)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 2 ) << 48 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 49 ;
        aux |=(unsigned long long) ( ( 0 * 6 + ( (v>>1)%2 ) * -10 + ( (v>>37)%2 ) * 2 + ( (v>>49)%2 ) * 2) >= 4 ) << 50 ;
        aux |=(unsigned long long) ( ( ( (v>>49)%2 ) * 2 + ( (v>>20)%2 ) * -2 + ( (v>>1)%2 ) * -2 + ( (v>>0)%2 ) * -2) >= 2 ) << 51 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>1)%2 ) * -2 + ( (v>>12)%2 ) * -2) >= 2 ) << 52 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>42)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;
        aux |=(unsigned long long) ( (v>>55)%2 )<< 55 ;
        aux |=(unsigned long long) ( (v>>56)%2 )<< 56 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 57 ;
        aux |=(unsigned long long) ( (v>>58)%2 )<< 58 ;
        aux |=(unsigned long long) ( (v>>59)%2 )<< 59 ;
        
        estado[tid] |= aux;
    }
}

unsigned long long confere_tlf_8(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>1)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>6)%2 ) * 2 + ( (v>>30)%2 ) * -6 + ( (v>>1)%2 ) * -6 + ( (v>>3)%2 ) * -6 + ( (v>>19)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>19)%2 ) * -6 + ( (v>>2)%2 ) * -6 + ( (v>>1)%2 ) * -6 + ( (v>>9)%2 ) * -6 + ( (v>>33)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>21)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * 2 + ( (v>>2)%2 ) * 6 + ( (v>>1)%2 ) * -38 + ( (v>>20)%2 ) * -6 + ( (v>>9)%2 ) * 26 + ( (v>>51)%2 ) * 2) >= 8 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>1)%2 ) * -2 + ( (v>>42)%2 ) * -2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 22 + ( (v>>7)%2 ) * 10 + ( (v>>13)%2 ) * 6 + ( (v>>11)%2 ) * -2 + ( (v>>23)%2 ) * 2 + ( (v>>1)%2 ) * -22) >= 28 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 6 + ( (v>>9)%2 ) * -10 + ( (v>>1)%2 ) * -10 + ( (v>>4)%2 ) * 2 + ( (v>>21)%2 ) * 2) >= 4 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>14)%2 ) * 2 + ( (v>>53)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 4 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -30 + ( (v>>32)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>10)%2 ) * 2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>16)%2 ) * 2 + ( (v>>1)%2 ) * -10 + ( (v>>23)%2 ) * -10 + 0 * -10 + ( (v>>41)%2 ) * 6) >= 4 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>52)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 6 + ( (v>>1)%2 ) * -10 + ( (v>>4)%2 ) * 2 + ( (v>>21)%2 ) * 2) >= 4 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>2)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>22)%2 ) * 14 + ( (v>>34)%2 ) * -14 + ( (v>>1)%2 ) * -14 + ( (v>>43)%2 ) * -14 + ( (v>>23)%2 ) * 2 + ( (v>>55)%2 ) * 2) >= 16 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>1)%2 ) * -14 + ( (v>>32)%2 ) * 2 + ( (v>>46)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -14 + ( (v>>47)%2 ) * -14 + ( (v>>32)%2 ) * 2 + ( (v>>33)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 2 + ( (v>>25)%2 ) * 2 + ( (v>>24)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 4 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>23)%2 ) * 6 + ( (v>>46)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -6) >= 8 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 6 + ( (v>>23)%2 ) * 2 + 0 * 2 + ( (v>>1)%2 ) * -6) >= 8 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>47)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + ( (v>>1)%2 ) * -30 + 0 * -30 + ( (v>>44)%2 ) * -30 + ( (v>>40)%2 ) * 2 + ( (v>>24)%2 ) * 2 + ( (v>>26)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( 0 * 10 + ( (v>>1)%2 ) * -22 + ( (v>>48)%2 ) * 2 + ( (v>>52)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>26)%2 ) * 2 + ( (v>>46)%2 ) * 2 + ( (v>>37)%2 ) * 2 + ( (v>>33)%2 ) * 2 + ( (v>>9)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 8 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>37)%2 ) * 14 + ( (v>>1)%2 ) * -50 + ( (v>>50)%2 ) * 14 + ( (v>>11)%2 ) * 2 + ( (v>>51)%2 ) * 2 + ( (v>>20)%2 ) * 2) >= 6 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + 0 * -6 + ( (v>>1)%2 ) * -6 + ( (v>>34)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>1)%2 ) * -6 + ( (v>>42)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>41)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>18)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2 + ( (v>>1)%2 ) * -2 + ( (v>>35)%2 ) * -2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>16)%2 ) * -6 + ( (v>>1)%2 ) * -6 + ( (v>>38)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2 + ( (v>>7)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 2 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + 0 * -2 + ( (v>>1)%2 ) * -2 + ( (v>>23)%2 ) * -2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>28)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>28)%2 ) * 2) >= 2 ) << 47 ;
        aux |=(unsigned long long) ( ( 0 * 2 + ( (v>>5)%2 ) * -2 + ( (v>>1)%2 ) * -2) >= 2 ) << 48 ;
        aux |=(unsigned long long) ( ( ( (v>>33)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 49 ;
        aux |=(unsigned long long) ( ( 0 * 6 + ( (v>>1)%2 ) * -10 + ( (v>>37)%2 ) * 2 + ( (v>>49)%2 ) * 2) >= 4 ) << 50 ;
        aux |=(unsigned long long) ( ( ( (v>>49)%2 ) * 2 + ( (v>>20)%2 ) * -2 + ( (v>>1)%2 ) * -2 + ( (v>>0)%2 ) * -2) >= 2 ) << 51 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2 + ( (v>>1)%2 ) * -2 + ( (v>>12)%2 ) * -2) >= 2 ) << 52 ;
        aux |=(unsigned long long) ( ( ( (v>>15)%2 ) * 2 + ( (v>>42)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 4 ) << 53 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 54 ;
        aux |=(unsigned long long) ( (v>>55)%2 )<< 55 ;
        aux |=(unsigned long long) ( (v>>56)%2 )<< 56 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 57 ;
        aux |=(unsigned long long) ( (v>>58)%2 )<< 58 ;
        aux |=(unsigned long long) ( (v>>59)%2 )<< 59 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}


//REDE 9
__global__ void passo_bool_9_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>44)%2 )<< 0 ;
        aux |=(unsigned long long) ( (v>>18)%2 ) | ( (v>>23)%2 ) | ( (v>>16)%2 ) | ( (v>>24)%2 )<< 1 ;
        aux |=(unsigned long long) ( (v>>7)%2 ) | ( (v>>14)%2 )<< 2 ;
        aux |=(unsigned long long) ( (v>>42)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 )<< 4 ;
        aux |=(unsigned long long) ( (v>>17)%2 )<< 5 ;
        aux |=(unsigned long long) ( (v>>43)%2 ) | ( (v>>7)%2 ) | ( (v>>12)%2 )<< 6 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>35)%2 ) | ( (v>>52)%2 )<< 7 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 10 ;
        aux |=(unsigned long long) ( (v>>39)%2 )<< 11 ;
        aux |=(unsigned long long) ( (v>>30)%2 )<< 12 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 13 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 14 ;
        aux |=(unsigned long long) ( (v>>21)%2 ) | ( (v>>50)%2 )<< 15 ;
    
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_9_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 17 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 18 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 20 ;
        aux |=(unsigned long long) ( (v>>11)%2 )<< 21 ;
        aux |=(unsigned long long) ( (v>>34)%2 ) | ( (v>>32)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 23 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 24 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 27 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>12)%2 ) | ( (v>>43)%2 )<< 28 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>43)%2 )<< 29 ;
        aux |=(unsigned long long) ( (v>>31)%2 ) | ( (v>>49)%2 )<< 30 ;
    
        estado[tid] |= aux;
    }
}

__global__ void passo_bool_9_parte3(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>48)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>48)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>42)%2 ) | ( (v>>12)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>3)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 38 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 39 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>38)%2 ) | ( (v>>33)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>8)%2 ) | ( (v>>9)%2 )<< 45 ;
        aux |=(unsigned long long) ( (v>>57)%2 ) | ( (v>>19)%2 )<< 46 ;
        aux |=(unsigned long long) ( (v>>59)%2 )<< 47 ;
        aux |=(unsigned long long) ( (v>>51)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>45)%2 ) | ( (v>>51)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 50 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>12)%2 )<< 52 ;
        aux |=(unsigned long long) ( (v>>57)%2 ) | ( (v>>15)%2 )<< 53 ;
        aux |=(unsigned long long) ( (v>>20)%2 )<< 54 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 55 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 56 ;
        aux |=(unsigned long long) ( (v>>61)%2 )<< 57 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 58 ;
        aux |=(unsigned long long) ( (v>>60)%2 )<< 59 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 60 ;
        aux |=(unsigned long long) ( (v>>61)%2 )<< 61 ;
    
        estado[tid] |= aux;
    }
}


unsigned long long confere_bool_9(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0ULL;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( (v>>44)%2 )<< 0 ;
        aux |=(unsigned long long) ( (v>>18)%2 ) | ( (v>>23)%2 ) | ( (v>>16)%2 ) | ( (v>>24)%2 )<< 1 ;
        aux |=(unsigned long long) ( (v>>7)%2 ) | ( (v>>14)%2 )<< 2 ;
        aux |=(unsigned long long) ( (v>>42)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 3 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 )<< 4 ;
        aux |=(unsigned long long) ( (v>>17)%2 )<< 5 ;
        aux |=(unsigned long long) ( (v>>43)%2 ) | ( (v>>7)%2 ) | ( (v>>12)%2 )<< 6 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>35)%2 ) | ( (v>>52)%2 )<< 7 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>46)%2 )<< 9 ;
        aux |=(unsigned long long) ( (v>>13)%2 )<< 10 ;
        aux |=(unsigned long long) ( (v>>39)%2 )<< 11 ;
        aux |=(unsigned long long) ( (v>>30)%2 )<< 12 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 13 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 14 ;
        aux |=(unsigned long long) ( (v>>21)%2 ) | ( (v>>50)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 16 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 17 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 18 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 19 ;
        aux |=(unsigned long long) ( (v>>53)%2 )<< 20 ;
        aux |=(unsigned long long) ( (v>>11)%2 )<< 21 ;
        aux |=(unsigned long long) ( (v>>34)%2 ) | ( (v>>32)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 23 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 24 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 25 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>0)%2 ) | ( (v>>12)%2 )<< 27 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>12)%2 ) | ( (v>>43)%2 )<< 28 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>43)%2 )<< 29 ;
        aux |=(unsigned long long) ( (v>>31)%2 ) | ( (v>>49)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>48)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>48)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>42)%2 ) | ( (v>>12)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>3)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 37 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 38 ;
        aux |=(unsigned long long) ( (v>>47)%2 )<< 39 ;
        aux |=(unsigned long long) ( (v>>41)%2 )<< 40 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 41 ;
        aux |=(unsigned long long) ( (v>>38)%2 ) | ( (v>>33)%2 )<< 42 ;
        aux |=(unsigned long long) ( (v>>0)%2 ) | ( (v>>22)%2 ) | ( (v>>12)%2 )<< 43 ;
        aux |=(unsigned long long) ( (v>>57)%2 )<< 44 ;
        aux |=(unsigned long long) ( (v>>8)%2 ) | ( (v>>9)%2 )<< 45 ;
        aux |=(unsigned long long) ( (v>>57)%2 ) | ( (v>>19)%2 )<< 46 ;
        aux |=(unsigned long long) ( (v>>59)%2 )<< 47 ;
        aux |=(unsigned long long) ( (v>>51)%2 )<< 48 ;
        aux |=(unsigned long long) ( (v>>45)%2 ) | ( (v>>51)%2 )<< 49 ;
        aux |=(unsigned long long) ( (v>>5)%2 )<< 50 ;
        aux |=(unsigned long long) ( (v>>54)%2 )<< 51 ;
        aux |=(unsigned long long) ( (v>>12)%2 )<< 52 ;
        aux |=(unsigned long long) ( (v>>57)%2 ) | ( (v>>15)%2 )<< 53 ;
        aux |=(unsigned long long) ( (v>>20)%2 )<< 54 ;
        aux |=(unsigned long long) ( (v>>52)%2 )<< 55 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 56 ;
        aux |=(unsigned long long) ( (v>>61)%2 )<< 57 ;
        aux |=(unsigned long long) ( (v>>22)%2 ) | ( (v>>12)%2 )<< 58 ;
        aux |=(unsigned long long) ( (v>>60)%2 )<< 59 ;
        aux |=(unsigned long long) ( (v>>1)%2 )<< 60 ;
        aux |=(unsigned long long) ( (v>>61)%2 )<< 61 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_9_parte1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>44)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>23)%2 ) * 2 + ( (v>>16)%2 ) * 2 + ( (v>>24)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>14)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2 + ( (v>>7)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>35)%2 ) * 2 + ( (v>>52)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 14 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_9_parte2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + ( (v>>50)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>53)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2 + ( (v>>32)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2 + ( (v>>43)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>43)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2 + ( (v>>49)%2 ) * 2) >= 2 ) << 30 ;
        
        estado[tid] |= aux;
    }
}

__global__ void passo_tlf_9_parte3(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2 + ( (v>>33)%2 ) * 2) >= 2 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>9)%2 ) * 2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2 + ( (v>>19)%2 ) * 2) >= 2 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>59)%2 ) * 2) >= 2 ) << 47 ;
        aux |=(unsigned long long) ( ( ( (v>>51)%2 ) * 2) >= 2 ) << 48 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2 + ( (v>>51)%2 ) * 2) >= 2 ) << 49 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 50 ;
        aux |=(unsigned long long) ( ( ( (v>>54)%2 ) * 2) >= 2 ) << 51 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2) >= 2 ) << 52 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2 + ( (v>>15)%2 ) * 2) >= 2 ) << 53 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2) >= 2 ) << 54 ;
        aux |=(unsigned long long) ( ( ( (v>>52)%2 ) * 2) >= 2 ) << 55 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 56 ;
        aux |=(unsigned long long) ( ( ( (v>>61)%2 ) * 2) >= 2 ) << 57 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 58 ;
        aux |=(unsigned long long) ( ( ( (v>>60)%2 ) * 2) >= 2 ) << 59 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 60 ;
        aux |=(unsigned long long) ( (v>>61)%2 )<< 61 ;
        
        estado[tid] |= aux;
    }
}


unsigned long long confere_tlf_9(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>44)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>18)%2 ) * 2 + ( (v>>23)%2 ) * 2 + ( (v>>16)%2 ) * 2 + ( (v>>24)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>7)%2 ) * 2 + ( (v>>14)%2 ) * 2) >= 2 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>17)%2 ) * 2) >= 2 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>43)%2 ) * 2 + ( (v>>7)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>35)%2 ) * 2 + ( (v>>52)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>46)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>30)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2 + ( (v>>50)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2) >= 2 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>53)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2 + ( (v>>32)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>12)%2 ) * 2 + ( (v>>43)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>43)%2 ) * 2) >= 2 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2 + ( (v>>49)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>48)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>42)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>3)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long) ( ( ( (v>>47)%2 ) * 2) >= 2 ) << 39 ;
        aux |=(unsigned long long) ( ( ( (v>>41)%2 ) * 2) >= 2 ) << 40 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 41 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2 + ( (v>>33)%2 ) * 2) >= 2 ) << 42 ;
        aux |=(unsigned long long) ( ( ( (v>>0)%2 ) * 2 + ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 43 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2) >= 2 ) << 44 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>9)%2 ) * 2) >= 2 ) << 45 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2 + ( (v>>19)%2 ) * 2) >= 2 ) << 46 ;
        aux |=(unsigned long long) ( ( ( (v>>59)%2 ) * 2) >= 2 ) << 47 ;
        aux |=(unsigned long long) ( ( ( (v>>51)%2 ) * 2) >= 2 ) << 48 ;
        aux |=(unsigned long long) ( ( ( (v>>45)%2 ) * 2 + ( (v>>51)%2 ) * 2) >= 2 ) << 49 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2) >= 2 ) << 50 ;
        aux |=(unsigned long long) ( ( ( (v>>54)%2 ) * 2) >= 2 ) << 51 ;
        aux |=(unsigned long long) ( ( ( (v>>12)%2 ) * 2) >= 2 ) << 52 ;
        aux |=(unsigned long long) ( ( ( (v>>57)%2 ) * 2 + ( (v>>15)%2 ) * 2) >= 2 ) << 53 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2) >= 2 ) << 54 ;
        aux |=(unsigned long long) ( ( ( (v>>52)%2 ) * 2) >= 2 ) << 55 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 56 ;
        aux |=(unsigned long long) ( ( ( (v>>61)%2 ) * 2) >= 2 ) << 57 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>12)%2 ) * 2) >= 2 ) << 58 ;
        aux |=(unsigned long long) ( ( ( (v>>60)%2 ) * 2) >= 2 ) << 59 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2) >= 2 ) << 60 ;
        aux |=(unsigned long long) ( (v>>61)%2 )<< 61 ;

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

void preenche_init_rand(unsigned long long * init_rand, unsigned long long nSim, unsigned int nEq)
{ 
    for(unsigned long long i = 0; i < nSim; i++)
    {
        
        init_rand[i] = 0;
        unsigned long rand1 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand2 = rand()%((unsigned long)(1<<31)-1);
        for(int j = 0; j < nEq && j < 32; j++)
            init_rand[i] |= ((rand1>>j)%2)<<j;
        for(int j = 32; j < nEq && j < 64 ; j++)
            init_rand[i] |= ((rand2>>(j-32))%2)<<j;
    }
}

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    unsigned int n=0;
    string argv2 ;
    if (argc >= 3) {
        n = atoi(argv[1]);
        argv2 = argv[2];
    } else {
        printf("\nERROR: Number of parameters insufficient\n");
        printf("Example: ./a.out 1 1000\n\n");
        return 1;
    }
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    unsigned long long *h_init_rand, *h_estado;
    h_init_rand = new unsigned long long[MAX_ESTADO];
    h_estado = new unsigned long long[MAX_ESTADO];

    unsigned long long *d_init_rand, *d_estado;
    hipMalloc(&d_init_rand,sizeof(unsigned long long)*MAX_ESTADO);
    hipMalloc(&d_estado,sizeof(unsigned long long)*MAX_ESTADO);
    
    unsigned int nEq = 64;
    srand(MAX_ESTADO);
    preenche_init_rand(h_init_rand,MAX_ESTADO,nEq);

    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i] = 0;

    hipMemcpy(d_init_rand, h_init_rand, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyHostToDevice);
    hipMemcpy(d_estado, h_estado, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyHostToDevice);

    switch (n) {
        case 0: passo_bool_1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 1: passo_tlf_1_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize(); 
                passo_tlf_1_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                
                break;
        case 2: passo_bool_2_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_2_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 3: passo_tlf_2_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_2_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 4: passo_bool_3_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_3_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 5: passo_tlf_3_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_3_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        //case 7: passo_bool_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); break;
        //case 8: passo_tlf_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); break;
        case 6: passo_bool_5_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_5_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 7: passo_tlf_5_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_5_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 8: passo_bool_6_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_6_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_6_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 9: passo_tlf_6_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_6_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                hipDeviceSynchronize();
                passo_tlf_6_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 10: passo_bool_7_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_7_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 11: passo_tlf_7_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_7_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 12: passo_bool_8_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_8_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 13: passo_tlf_8_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_8_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_8_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 14:passo_bool_9_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_9_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_9_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 15: passo_tlf_9_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_9_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_9_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        default: 
                break;
    }
    hipDeviceSynchronize();
    
    hipMemcpy(h_estado, d_estado, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyDeviceToHost);


    unsigned long long i = 0;
    switch (n) {
        case 0:i= confere_bool_1(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 1:i= confere_tlf_1(h_init_rand,h_estado,MAX_ESTADO);
                break;
        case 2: i= confere_bool_2(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 3: i= confere_tlf_2(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 4: i= confere_bool_3(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 5: i= confere_tlf_3(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        //case 7: passo_bool_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        //case 8: passo_tlf_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 6: i= confere_bool_5(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 7: i= confere_tlf_5(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 8: i= confere_bool_6(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 9: i= confere_tlf_6(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 10: i= confere_bool_7(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 11: i= confere_tlf_7(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 12: i= confere_bool_8(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 13: i= confere_tlf_8(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 14: i= confere_bool_9(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 15: i= confere_tlf_9(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        default: i= confere_bool_1(h_init_rand,h_estado,MAX_ESTADO); 
                break;
    }
    
    if(i == MAX_ESTADO)
        cerr << "Resultados da GPU batem com os da CPU\n";
    else
        cerr << "Resultados não batem!\n";
    
    delete [] h_estado;
    delete [] h_init_rand;
    hipFree(d_init_rand);
    hipFree(d_estado);
    hipDeviceReset();
    return 0;
}
