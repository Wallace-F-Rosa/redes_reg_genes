
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <string>
#include <limits>
#include <stdio.h>
#include <stdlib.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;
//REDE 2

__global__ void passo_bool_2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( (v>>25)%2 )<< 0 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 1 ;
        aux |=(unsigned long long) ( ( (v>>13)%2 & ( ( ( (v>>3)%2 & (v>>24)%2 ) ) ) ) & ! ( (v>>17)%2 ) )<< 2 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 3 ;
        aux |=(unsigned long long) ( ( (v>>38)%2 ) & ! ( (v>>1)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) ) | ( ( (v>>12)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>38)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>11)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) ) | ( (v>>10)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 ) & ! ( (v>>17)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 ) & ! ( (v>>37)%2 ) ) | ( ( (v>>15)%2 ) & ! ( (v>>37)%2 ) )<< 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) ) | ( ( ( (v>>7)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( (v>>9)%2 & ( ( ( (v>>8)%2 ) ) ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>39)%2 )<< 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>24)%2 ) ) & ! ( (v>>8)%2 & ( ( ( (v>>10)%2 ) ) ) ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>1)%2 ) & ! ( (v>>0)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>0)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( ( (v>>25)%2 ) & ! ( (v>>18)%2 ) )<< 19 ;
        aux |=(unsigned long long) ( (v>>21)%2 )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>22)%2 ) & ! ( (v>>1)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>1)%2 ) | ( (v>>38)%2 )<< 23 ;
        aux |=(unsigned long long) ( ( (v>>5)%2 ) & ! ( (v>>6)%2 ) )<< 24 ;
        aux |=(unsigned long long) ! ( ( (v>>19)%2 ) )<< 25 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 27 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 28 ;
        aux |=(unsigned long long) ( ( (v>>27)%2 & ( ( ( (v>>28)%2 ) ) ) ) & ! ( (v>>30)%2 ) )<< 29 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 37 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 ) & ! ( (v>>23)%2 ) ) | ( ( (v>>14)%2 ) & ! ( (v>>23)%2 ) )<< 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
        
        printf("%ull\n",aux);
        estado[tid] = aux;
    }
}

unsigned long long confere_bool_2(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( (v>>25)%2 )<< 0 ;
        aux |=(unsigned long long) ( (v>>29)%2 )<< 1 ;
        aux |=(unsigned long long) ( ( (v>>13)%2 & ( ( ( (v>>3)%2 & (v>>24)%2 ) ) ) ) & ! ( (v>>17)%2 ) )<< 2 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 3 ;
        aux |=(unsigned long long) ( ( (v>>38)%2 ) & ! ( (v>>1)%2 ) )<< 4 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) ) | ( ( (v>>12)%2 & ( ( ( (v>>38)%2 ) ) ) ) & ! ( (v>>6)%2 ) )<< 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>4)%2 ) ) & ! ( (v>>38)%2 ) )<< 6 ;
        aux |=(unsigned long long) ( (v>>11)%2 )<< 7 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) ) | ( (v>>10)%2 )<< 8 ;
        aux |=(unsigned long long) ( (v>>8)%2 )<< 9 ;
        aux |=(unsigned long long) ( ( (v>>8)%2 ) & ! ( (v>>17)%2 ) )<< 10 ;
        aux |=(unsigned long long) ( ( (v>>2)%2 ) & ! ( (v>>17)%2 ) ) | ( ( (v>>12)%2 ) & ! ( (v>>17)%2 ) )<< 11 ;
        aux |=(unsigned long long) ( ( (v>>10)%2 ) & ! ( (v>>37)%2 ) ) | ( ( (v>>15)%2 ) & ! ( (v>>37)%2 ) )<< 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) ) | ( ( ( (v>>7)%2 ) & ! ( (v>>1)%2 ) ) & ! ( (v>>17)%2 ) )<< 13 ;
        aux |=(unsigned long long) ( (v>>9)%2 & ( ( ( (v>>8)%2 ) ) ) )<< 14 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 15 ;
        aux |=(unsigned long long) ( (v>>39)%2 )<< 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) & ! ( (v>>24)%2 ) ) & ! ( (v>>8)%2 & ( ( ( (v>>10)%2 ) ) ) ) )<< 17 ;
        aux |=(unsigned long long) ( ( (v>>1)%2 ) & ! ( (v>>0)%2 ) ) | ( ( (v>>26)%2 ) & ! ( (v>>0)%2 ) )<< 18 ;
        aux |=(unsigned long long) ( ( (v>>25)%2 ) & ! ( (v>>18)%2 ) )<< 19 ;
        aux |=(unsigned long long) ( (v>>21)%2 )<< 20 ;
        aux |=(unsigned long long) ( ( (v>>22)%2 ) & ! ( (v>>1)%2 ) )<< 21 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 22 ;
        aux |=(unsigned long long) ( (v>>1)%2 ) | ( (v>>38)%2 )<< 23 ;
        aux |=(unsigned long long) ( ( (v>>5)%2 ) & ! ( (v>>6)%2 ) )<< 24 ;
        aux |=(unsigned long long) ! ( ( (v>>19)%2 ) )<< 25 ;
        aux |=(unsigned long long) ( (v>>36)%2 )<< 26 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 27 ;
        aux |=(unsigned long long) ( (v>>16)%2 )<< 28 ;
        aux |=(unsigned long long) ( ( (v>>27)%2 & ( ( ( (v>>28)%2 ) ) ) ) & ! ( (v>>30)%2 ) )<< 29 ;
        aux |=(unsigned long long) ( (v>>38)%2 )<< 30 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 31 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 32 ;
        aux |=(unsigned long long) ( (v>>40)%2 )<< 33 ;
        aux |=(unsigned long long) ( (v>>32)%2 )<< 34 ;
        aux |=(unsigned long long) ( (v>>34)%2 )<< 35 ;
        aux |=(unsigned long long) ( (v>>31)%2 )<< 36 ;
        aux |=(unsigned long long) ( (v>>35)%2 )<< 37 ;
        aux |=(unsigned long long) ( ( (v>>20)%2 ) & ! ( (v>>23)%2 ) ) | ( ( (v>>14)%2 ) & ! ( (v>>23)%2 ) )<< 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
    
        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_2(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>3)%2 ) * 2 + ( (v>>24)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 6 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>38)%2 ) * 6 + ( (v>>6)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 8 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>4)%2 ) * -2 + ( (v>>38)%2 ) * -2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2 + ( (v>>10)%2 ) * 10) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>37)%2 ) * -6 + ( (v>>15)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>17)%2 ) * -6 + ( (v>>7)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 4 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 6 + ( (v>>24)%2 ) * -6 + ( (v>>8)%2 ) * -2 + ( (v>>10)%2 ) * -2) >= 4 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>0)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>18)%2 ) * -2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>38)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>6)%2 ) * -2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * -2) >= 0 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2 + ( (v>>28)%2 ) * 2 + ( (v>>30)%2 ) * -2) >= 4 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>23)%2 ) * -6 + ( (v>>14)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
        
        estado[tid] = aux;
    }
}

unsigned long long confere_tlf_2(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2) >= 2 ) << 0 ;
        aux |=(unsigned long long) ( ( ( (v>>29)%2 ) * 2) >= 2 ) << 1 ;
        aux |=(unsigned long long) ( ( ( (v>>13)%2 ) * 2 + ( (v>>3)%2 ) * 2 + ( (v>>24)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 6 ) << 2 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 3 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 4 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>38)%2 ) * 6 + ( (v>>6)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 8 ) << 5 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>4)%2 ) * -2 + ( (v>>38)%2 ) * -2) >= 2 ) << 6 ;
        aux |=(unsigned long long) ( ( ( (v>>11)%2 ) * 2) >= 2 ) << 7 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2 + ( (v>>10)%2 ) * 10) >= 2 ) << 8 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2) >= 2 ) << 9 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>17)%2 ) * -2) >= 2 ) << 10 ;
        aux |=(unsigned long long) ( ( ( (v>>2)%2 ) * 2 + ( (v>>17)%2 ) * -6 + ( (v>>12)%2 ) * 2) >= 2 ) << 11 ;
        aux |=(unsigned long long) ( ( ( (v>>10)%2 ) * 2 + ( (v>>37)%2 ) * -6 + ( (v>>15)%2 ) * 2) >= 2 ) << 12 ;
        aux |=(unsigned long long) ( ( ( (v>>8)%2 ) * 2 + ( (v>>1)%2 ) * -6 + ( (v>>17)%2 ) * -6 + ( (v>>7)%2 ) * 2) >= 2 ) << 13 ;
        aux |=(unsigned long long) ( ( ( (v>>9)%2 ) * 2 + ( (v>>8)%2 ) * 2) >= 4 ) << 14 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 15 ;
        aux |=(unsigned long long) ( ( ( (v>>39)%2 ) * 2) >= 2 ) << 16 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 6 + ( (v>>24)%2 ) * -6 + ( (v>>8)%2 ) * -2 + ( (v>>10)%2 ) * -2) >= 4 ) << 17 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>0)%2 ) * -6 + ( (v>>26)%2 ) * 2) >= 2 ) << 18 ;
        aux |=(unsigned long long) ( ( ( (v>>25)%2 ) * 2 + ( (v>>18)%2 ) * -2) >= 2 ) << 19 ;
        aux |=(unsigned long long) ( ( ( (v>>21)%2 ) * 2) >= 2 ) << 20 ;
        aux |=(unsigned long long) ( ( ( (v>>22)%2 ) * 2 + ( (v>>1)%2 ) * -2) >= 2 ) << 21 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 22 ;
        aux |=(unsigned long long) ( ( ( (v>>1)%2 ) * 2 + ( (v>>38)%2 ) * 2) >= 2 ) << 23 ;
        aux |=(unsigned long long) ( ( ( (v>>5)%2 ) * 2 + ( (v>>6)%2 ) * -2) >= 2 ) << 24 ;
        aux |=(unsigned long long) ( ( ( (v>>19)%2 ) * -2) >= 0 ) << 25 ;
        aux |=(unsigned long long) ( ( ( (v>>36)%2 ) * 2) >= 2 ) << 26 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 27 ;
        aux |=(unsigned long long) ( ( ( (v>>16)%2 ) * 2) >= 2 ) << 28 ;
        aux |=(unsigned long long) ( ( ( (v>>27)%2 ) * 2 + ( (v>>28)%2 ) * 2 + ( (v>>30)%2 ) * -2) >= 4 ) << 29 ;
        aux |=(unsigned long long) ( ( ( (v>>38)%2 ) * 2) >= 2 ) << 30 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 31 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 32 ;
        aux |=(unsigned long long) ( ( ( (v>>40)%2 ) * 2) >= 2 ) << 33 ;
        aux |=(unsigned long long) ( ( ( (v>>32)%2 ) * 2) >= 2 ) << 34 ;
        aux |=(unsigned long long) ( ( ( (v>>34)%2 ) * 2) >= 2 ) << 35 ;
        aux |=(unsigned long long) ( ( ( (v>>31)%2 ) * 2) >= 2 ) << 36 ;
        aux |=(unsigned long long) ( ( ( (v>>35)%2 ) * 2) >= 2 ) << 37 ;
        aux |=(unsigned long long) ( ( ( (v>>20)%2 ) * 2 + ( (v>>23)%2 ) * -6 + ( (v>>14)%2 ) * 2) >= 2 ) << 38 ;
        aux |=(unsigned long long)(v>>39%2) << 39;
        aux |=(unsigned long long)(v>>40%2) << 40;
        aux |=(unsigned long long)(v>>41%2) << 41;
        
        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

void preenche_init_rand(unsigned long long * init_rand, unsigned long long nSim, unsigned int nEq)
{
    
    for(unsigned long long i = 0; i < nSim; i++)
    {
        
        init_rand[i] = 0;
        unsigned long rand1 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand2 = rand()%((unsigned long)(1<<31)-1);
        init_rand[i] = rand1;
        for(int j = 0; j < nEq; j++)
            init_rand[i] |= ((rand2>>j)%2)<<j;
    }
}

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[1];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    unsigned long long *h_init_rand, *h_estado;
    h_init_rand = new unsigned long long[MAX_ESTADO];
    h_estado = new unsigned long long[MAX_ESTADO];

    unsigned long long *d_init_rand, *d_estado;
    hipMalloc(&d_init_rand,sizeof(unsigned long long)*MAX_ESTADO);
    hipMalloc(&d_estado,sizeof(unsigned long long)*MAX_ESTADO);
    
    unsigned int nEq = 41;
    srand(MAX_ESTADO);
    preenche_init_rand(h_init_rand,MAX_ESTADO,nEq);

    /* for(unsigned long long i = 0; i < MAX_ESTADO; i++)
    {
        cout << h_init_rand[i] << " ";
    }
    cout << "\n"; */
    hipMemcpy(d_init_rand, h_init_rand, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyHostToDevice);

    passo_bool_2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    hipMemcpy(h_estado, d_estado, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyDeviceToHost);

    unsigned long long i = confere_bool_2(h_init_rand,h_estado,MAX_ESTADO);
    if(i == MAX_ESTADO)
        cerr << "Resultados da GPU batem com os da CPU\n";
    else
        cerr << "Resultados não batem!\n";
    
    hipFree(d_init_rand);
    hipFree(d_estado);
    return 0;
}
