
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <string>
#include <limits>
#include <stdio.h>
#include <stdlib.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;

//REDE 10
__global__ void passo_bool_10_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;

        aux.x |= ( ( (v.x>>41)%2 ) )<< 0 ;
        aux.x |= ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 1 ;
        aux.x |= ( ( (v.x>>17)%2 ) )<< 2 ;
        aux.x |= ! ( ( ( (v.x>>0)%2 ) ) )<< 3 ;
        aux.x |= ( ( ( ( (v.x>>6)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) ) | ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) )<< 4 ;
        aux.x |= ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) )<< 5 ;
        aux.x |= ( ( ( ( (v.x>>49)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) )<< 6 ;
        aux.x |= ( ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 7 ;
        aux.x |= ( ( ( (v.x>>13)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 8 ;
        aux.x |= ( ( ( ( (v.x>>12)%2 ) ) & ! ( ( (v.x>>56)%2 ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 9 ;
        aux.x |= ( ( (v.x>>12)%2 ) )<< 10 ;
        aux.x |= ( ( (v.x>>62)%2 ) )<< 11 ;
        aux.x |= ( ( (v.x>>0)%2 ) ) | ( ( (v.x>>38)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 12 ;
        aux.x |= ( ( (v.x>>38)%2 ) )<< 13 ;
        aux.x |= ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) )<< 14 ;
        aux.x |= ( ( ( (v.x>>23)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.x>>22)%2 ) ) )<< 15 ;
        aux.x |= ( ( ( ( (v.x>>18)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) )<< 16 ;
        aux.x |= ( ( (v.x>>14)%2 ) )<< 17 ;
        aux.x |= ( ( (v.x>>36)%2 ) )<< 18 ;
        aux.x |= ( ( (v.x>>1)%2 ) )<< 19 ;
        aux.x |= ( ( (v.x>>49)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 20 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_10_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;

        aux.x |= ( ( (v.x>>31)%2 ) )<< 21 ;
        aux.x |= ( ( (v.x>>59)%2 ) )<< 22 ;
        aux.x |= ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 23 ;
        aux.x |= ( ( (v.x>>53)%2 ) ) | ( ( (v.x>>23)%2 ) )<< 24 ;
        aux.x |= ( ( (v.x>>60)%2 ) )<< 25 ;
        aux.x |= ( ( (v.x>>47)%2 ) & ( ( ( ( (v.x>>39)%2 ) ) ) ) ) | ( ( (v.x>>0)%2 ) )<< 26 ;
        aux.x |= ( ( (v.x>>59)%2 ) )<< 27 ;
        aux.x |= ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>27)%2 ) ) ) ) )<< 28 ;
        aux.x |= ( ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) & ! ( ( (v.x>>63)%2 ) ) )<< 29 ;
        aux.x |= ( ( ( (v.x>>37)%2 ) ) & ! ( ( (v.x>>43)%2 ) ) )<< 30 ;
        aux.x |= ( ( (v.x>>33)%2 ) )<< 31 ;
        aux.x |= ( ( ( (v.x>>0)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) ) | ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) )<< 32 ;
        aux.x |= ( ( (v.x>>45)%2 ) )<< 33 ;
        aux.x |= ( ( (v.x>>33)%2 ) )<< 34 ;
        aux.x |= ( ( (v.x>>44)%2 ) )<< 35 ;
        aux.x |= ( ( (v.x>>4)%2 ) )<< 36 ;
        aux.x |= ( ( (v.x>>35)%2 ) )<< 37 ;
        aux.x |= ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 38 ;
        aux.x |= ! ( ( ( (v.x>>51)%2 ) ) )<< 39 ;
        aux.x |= ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) )<< 40 ;
        aux.x |= ( ( ( (v.x>>41)%2 ) ) & ! ( ( (v.x>>42)%2 ) ) )<< 41 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_10_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;

        aux.x |= ( ( (v.x>>57)%2 ) )<< 42 ;
        aux.x |= ! ( ( ( (v.y>>0)%2 ) ) )<< 43 ;
        aux.x |= ( ( (v.x>>49)%2 ) )<< 44 ;
        aux.x |= ( ( (v.x>>46)%2 ) )<< 45 ;
        aux.x |= ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) )<< 46 ;
        aux.x |= ( ( (v.y>>0)%2 ) ) | ( ( (v.x>>62)%2 ) & ( ( ( ( (v.y>>1)%2 ) ) ) ) )<< 47 ;
        aux.x |= ! ( ( ( (v.x>>14)%2 ) ) )<< 48 ;
        aux.x |= ( ( ( (v.x>>20)%2 ) ) ) | ! ( ( (v.x>>20)%2 ) | ( (v.x>>43)%2 ) )<< 49 ;
        aux.x |= ( ( (v.x>>36)%2 ) )<< 50 ;
        aux.x |= ! ( ( ( (v.x>>11)%2 ) ) )<< 51 ;
        aux.x |= ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 52 ;
        aux.x |= ( ( (v.x>>30)%2 ) )<< 53 ;
        aux.x |= ! ( ( ( (v.x>>14)%2 ) ) )<< 54 ;
        aux.x |= ( ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>38)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>32)%2 ) ) ) & ! ( ( (v.x>>9)%2 ) ) )<< 55 ;
        aux.x |= ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>9)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>38)%2 ) ) )<< 56 ;
        aux.x |= ( ( ( (v.x>>19)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) ) | ( ( ( (v.x>>30)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) )<< 57 ;
        aux.x |= ( ( (v.x>>26)%2 ) )<< 58 ;
        aux.x |= ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>58)%2 ) ) | ( ( (v.y>>2)%2 ) ) | ( ( (v.x>>43)%2 ) & ( ( ( ( (v.x>>58)%2 ) ) ) ) )<< 59 ;
        aux.x |= ( ( (v.x>>9)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>8)%2 ) & ( ( ( ( (v.x>>10)%2 ) & ( (v.x>>9)%2 ) ) ) ) )<< 60 ;
        aux.x |= ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>28)%2 ) ) & ( ( ( ! ( (v.x>>26)%2 ) ) ) ) ) ) ) & ! ( ( (v.x>>30)%2 ) ) ) | ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>29)%2 ) ) ) ) )<< 61 ;
        aux.x |= ( ( (v.x>>62)%2 ) )<< 62 ;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}


unsigned long long confere_bool_10(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;

        aux.x |= ( ( (v.x>>41)%2 ) )<< 0 ;
        aux.x |= ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 1 ;
        aux.x |= ( ( (v.x>>17)%2 ) )<< 2 ;
        aux.x |= ! ( ( ( (v.x>>0)%2 ) ) )<< 3 ;
        aux.x |= ( ( ( ( (v.x>>6)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) ) | ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) )<< 4 ;
        aux.x |= ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) )<< 5 ;
        aux.x |= ( ( ( ( (v.x>>49)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) )<< 6 ;
        aux.x |= ( ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 7 ;
        aux.x |= ( ( ( (v.x>>13)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 8 ;
        aux.x |= ( ( ( ( (v.x>>12)%2 ) ) & ! ( ( (v.x>>56)%2 ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 9 ;
        aux.x |= ( ( (v.x>>12)%2 ) )<< 10 ;
        aux.x |= ( ( (v.x>>62)%2 ) )<< 11 ;
        aux.x |= ( ( (v.x>>0)%2 ) ) | ( ( (v.x>>38)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 12 ;
        aux.x |= ( ( (v.x>>38)%2 ) )<< 13 ;
        aux.x |= ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) )<< 14 ;
        aux.x |= ( ( ( (v.x>>23)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.x>>22)%2 ) ) )<< 15 ;
        aux.x |= ( ( ( ( (v.x>>18)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) )<< 16 ;
        aux.x |= ( ( (v.x>>14)%2 ) )<< 17 ;
        aux.x |= ( ( (v.x>>36)%2 ) )<< 18 ;
        aux.x |= ( ( (v.x>>1)%2 ) )<< 19 ;
        aux.x |= ( ( (v.x>>49)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 20 ;
        aux.x |= ( ( (v.x>>31)%2 ) )<< 21 ;
        aux.x |= ( ( (v.x>>59)%2 ) )<< 22 ;
        aux.x |= ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 23 ;
        aux.x |= ( ( (v.x>>53)%2 ) ) | ( ( (v.x>>23)%2 ) )<< 24 ;
        aux.x |= ( ( (v.x>>60)%2 ) )<< 25 ;
        aux.x |= ( ( (v.x>>47)%2 ) & ( ( ( ( (v.x>>39)%2 ) ) ) ) ) | ( ( (v.x>>0)%2 ) )<< 26 ;
        aux.x |= ( ( (v.x>>59)%2 ) )<< 27 ;
        aux.x |= ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>27)%2 ) ) ) ) )<< 28 ;
        aux.x |= ( ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) & ! ( ( (v.x>>63)%2 ) ) )<< 29 ;
        aux.x |= ( ( ( (v.x>>37)%2 ) ) & ! ( ( (v.x>>43)%2 ) ) )<< 30 ;
        aux.x |= ( ( (v.x>>33)%2 ) )<< 31 ;
        aux.x |= ( ( ( (v.x>>0)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) ) | ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) )<< 32 ;
        aux.x |= ( ( (v.x>>45)%2 ) )<< 33 ;
        aux.x |= ( ( (v.x>>33)%2 ) )<< 34 ;
        aux.x |= ( ( (v.x>>44)%2 ) )<< 35 ;
        aux.x |= ( ( (v.x>>4)%2 ) )<< 36 ;
        aux.x |= ( ( (v.x>>35)%2 ) )<< 37 ;
        aux.x |= ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 38 ;
        aux.x |= ! ( ( ( (v.x>>51)%2 ) ) )<< 39 ;
        aux.x |= ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) )<< 40 ;
        aux.x |= ( ( ( (v.x>>41)%2 ) ) & ! ( ( (v.x>>42)%2 ) ) )<< 41 ;
        aux.x |= ( ( (v.x>>57)%2 ) )<< 42 ;
        aux.x |= ! ( ( ( (v.y>>0)%2 ) ) )<< 43 ;
        aux.x |= ( ( (v.x>>49)%2 ) )<< 44 ;
        aux.x |= ( ( (v.x>>46)%2 ) )<< 45 ;
        aux.x |= ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) )<< 46 ;
        aux.x |= ( ( (v.y>>0)%2 ) ) | ( ( (v.x>>62)%2 ) & ( ( ( ( (v.y>>1)%2 ) ) ) ) )<< 47 ;
        aux.x |= ! ( ( ( (v.x>>14)%2 ) ) )<< 48 ;
        aux.x |= ( ( ( (v.x>>20)%2 ) ) ) | ! ( ( (v.x>>20)%2 ) | ( (v.x>>43)%2 ) )<< 49 ;
        aux.x |= ( ( (v.x>>36)%2 ) )<< 50 ;
        aux.x |= ! ( ( ( (v.x>>11)%2 ) ) )<< 51 ;
        aux.x |= ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 52 ;
        aux.x |= ( ( (v.x>>30)%2 ) )<< 53 ;
        aux.x |= ! ( ( ( (v.x>>14)%2 ) ) )<< 54 ;
        aux.x |= ( ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>38)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>32)%2 ) ) ) & ! ( ( (v.x>>9)%2 ) ) )<< 55 ;
        aux.x |= ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>9)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>38)%2 ) ) )<< 56 ;
        aux.x |= ( ( ( (v.x>>19)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) ) | ( ( ( (v.x>>30)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) )<< 57 ;
        aux.x |= ( ( (v.x>>26)%2 ) )<< 58 ;
        aux.x |= ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>58)%2 ) ) | ( ( (v.y>>2)%2 ) ) | ( ( (v.x>>43)%2 ) & ( ( ( ( (v.x>>58)%2 ) ) ) ) )<< 59 ;
        aux.x |= ( ( (v.x>>9)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>8)%2 ) & ( ( ( ( (v.x>>10)%2 ) & ( (v.x>>9)%2 ) ) ) ) )<< 60 ;
        aux.x |= ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>28)%2 ) ) & ( ( ( ! ( (v.x>>26)%2 ) ) ) ) ) ) ) & ! ( ( (v.x>>30)%2 ) ) ) | ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>29)%2 ) ) ) ) )<< 61 ;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_10_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;

        aux.x |= ( ( ( (v.x>>41)%2 ) * 2) >= 2 ) <<0;
        aux.x |= ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<1;
        aux.x |= ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<2;
        aux.x |= ( ( ( (v.x>>0)%2 ) * -2) >= 0 ) <<3;
        aux.x |= ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>7)%2 ) * -6 + ( (v.x>>5)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<4;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<5;
        aux.x |= ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>5)%2 ) * -14 + ( (v.x>>7)%2 ) * -14 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<6;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>4)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<7;
        aux.x |= ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<8;
        aux.x |= ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<9;
        aux.x |= ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<10;
        aux.x |= ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<11;
        aux.x |= ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>38)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<12;
        aux.x |= ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<13;
        aux.x |= ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>52)%2 ) * -6 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<14;
        aux.x |= ( ( ( (v.x>>23)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>22)%2 ) * -2) >= 4 ) <<15;
        aux.x |= ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>52)%2 ) * -2 + ( (v.x>>0)%2 ) * -2) >= 2 ) <<16;
        aux.x |= ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<17;
        aux.x |= ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<18;
        aux.x |= ( ( ( (v.x>>1)%2 ) * 2) >= 2 ) <<19;
        aux.x |= ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<20;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_10_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;

        aux.x |= ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<21;
        aux.x |= ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<22;
        aux.x |= ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<23;
        aux.x |= ( ( ( (v.x>>53)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<24;
        aux.x |= ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<25;
        aux.x |= ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>39)%2 ) * 2 + ( (v.x>>0)%2 ) * 6) >= 4 ) <<26;
        aux.x |= ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<27;
        aux.x |= ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>27)%2 ) * 2) >= 4 ) <<28;
        aux.x |= ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * -2) >= 4 ) <<29;
        aux.x |= ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 2 ) <<30;
        aux.x |= ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<31;
        aux.x |= ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>1)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<32;
        aux.x |= ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<33;
        aux.x |= ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<34;
        aux.x |= ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<35;
        aux.x |= ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<36;
        aux.x |= ( ( ( (v.x>>35)%2 ) * 2) >= 2 ) <<37;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<38;
        aux.x |= ( ( ( (v.x>>51)%2 ) * -2) >= 0 ) <<39;
        aux.x |= ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 4 ) <<40;
        aux.x |= ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>42)%2 ) * -2) >= 2 ) <<41;

    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_10_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;

        aux.x |= ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<42;
        aux.x |= ( ( ( (v.y>>0)%2 ) * -2) >= 0 ) <<43;
        aux.x |= ( ( ( (v.x>>49)%2 ) * 2) >= 2 ) <<44;
        aux.x |= ( ( ( (v.x>>46)%2 ) * 2) >= 2 ) <<45;
        aux.x |= ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * 6) >= 4 ) <<46;
        aux.x |= ( ( 0 * 6 + 0 * 2 + ( (v.y>>1)%2 ) * 2) >= 4 ) <<47;
        aux.x |= ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<48;
        aux.x |= ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 0 ) <<49;
        aux.x |= ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<50;
        aux.x |= ( ( ( (v.x>>11)%2 ) * -2) >= 0 ) <<51;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>50)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<52;
        aux.x |= ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<53;
        aux.x |= ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<54;
        aux.x |= ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>38)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>32)%2 ) * -2 + ( (v.x>>9)%2 ) * -2) >= 2 ) <<55;
        aux.x |= ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>9)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>38)%2 ) * -2) >= 2 ) <<56;
        aux.x |= ( ( ( (v.x>>19)%2 ) * 6 + ( (v.x>>32)%2 ) * -6 + ( (v.x>>43)%2 ) * -6 + ( (v.x>>30)%2 ) * 6) >= 0 ) <<57;
        aux.x |= ( ( ( (v.x>>26)%2 ) * 2) >= 2 ) <<58;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 4 + ( (v.x>>58)%2 ) * 4 + 0 * 4 + ( (v.x>>43)%2 ) * 0) >= 4 ) <<59;
        aux.x |= ( ( ( (v.x>>9)%2 ) * 4 + ( (v.x>>10)%2 ) * 4 + ( (v.x>>8)%2 ) * 0) >= 8 ) <<60;
        aux.x |= ( ( ( (v.x>>29)%2 ) * 8 + ( (v.x>>48)%2 ) * 8 + ( (v.x>>28)%2 ) * 8 + ( (v.x>>26)%2 ) * 0 + ( (v.x>>30)%2 ) * 0) >= 24 ) <<61;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_10(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;

        aux.x |= ( ( ( (v.x>>41)%2 ) * 2) >= 2 ) <<0;
        aux.x |= ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<1;
        aux.x |= ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<2;
        aux.x |= ( ( ( (v.x>>0)%2 ) * -2) >= 0 ) <<3;
        aux.x |= ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>7)%2 ) * -6 + ( (v.x>>5)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<4;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<5;
        aux.x |= ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>5)%2 ) * -14 + ( (v.x>>7)%2 ) * -14 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<6;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>4)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<7;
        aux.x |= ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<8;
        aux.x |= ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<9;
        aux.x |= ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<10;
        aux.x |= ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<11;
        aux.x |= ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>38)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<12;
        aux.x |= ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<13;
        aux.x |= ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>52)%2 ) * -6 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<14;
        aux.x |= ( ( ( (v.x>>23)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>22)%2 ) * -2) >= 4 ) <<15;
        aux.x |= ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>52)%2 ) * -2 + ( (v.x>>0)%2 ) * -2) >= 2 ) <<16;
        aux.x |= ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<17;
        aux.x |= ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<18;
        aux.x |= ( ( ( (v.x>>1)%2 ) * 2) >= 2 ) <<19;
        aux.x |= ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<20;
        aux.x |= ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<21;
        aux.x |= ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<22;
        aux.x |= ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<23;
        aux.x |= ( ( ( (v.x>>53)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<24;
        aux.x |= ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<25;
        aux.x |= ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>39)%2 ) * 2 + ( (v.x>>0)%2 ) * 6) >= 4 ) <<26;
        aux.x |= ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<27;
        aux.x |= ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>27)%2 ) * 2) >= 4 ) <<28;
        aux.x |= ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * -2) >= 4 ) <<29;
        aux.x |= ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 2 ) <<30;
        aux.x |= ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<31;
        aux.x |= ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>1)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<32;
        aux.x |= ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<33;
        aux.x |= ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<34;
        aux.x |= ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<35;
        aux.x |= ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<36;
        aux.x |= ( ( ( (v.x>>35)%2 ) * 2) >= 2 ) <<37;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<38;
        aux.x |= ( ( ( (v.x>>51)%2 ) * -2) >= 0 ) <<39;
        aux.x |= ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 4 ) <<40;
        aux.x |= ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>42)%2 ) * -2) >= 2 ) <<41;
        aux.x |= ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<42;
        aux.x |= ( ( ( (v.y>>0)%2 ) * -2) >= 0 ) <<43;
        aux.x |= ( ( ( (v.x>>49)%2 ) * 2) >= 2 ) <<44;
        aux.x |= ( ( ( (v.x>>46)%2 ) * 2) >= 2 ) <<45;
        aux.x |= ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * 6) >= 4 ) <<46;
        aux.x |= ( ( 0 * 6 + 0 * 2 + ( (v.y>>1)%2 ) * 2) >= 4 ) <<47;
        aux.x |= ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<48;
        aux.x |= ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 0 ) <<49;
        aux.x |= ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<50;
        aux.x |= ( ( ( (v.x>>11)%2 ) * -2) >= 0 ) <<51;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>50)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<52;
        aux.x |= ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<53;
        aux.x |= ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<54;
        aux.x |= ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>38)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>32)%2 ) * -2 + ( (v.x>>9)%2 ) * -2) >= 2 ) <<55;
        aux.x |= ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>9)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>38)%2 ) * -2) >= 2 ) <<56;
        aux.x |= ( ( ( (v.x>>19)%2 ) * 6 + ( (v.x>>32)%2 ) * -6 + ( (v.x>>43)%2 ) * -6 + ( (v.x>>30)%2 ) * 6) >= 0 ) <<57;
        aux.x |= ( ( ( (v.x>>26)%2 ) * 2) >= 2 ) <<58;
        aux.x |= ( ( ( (v.x>>61)%2 ) * 4 + ( (v.x>>58)%2 ) * 4 + 0 * 4 + ( (v.x>>43)%2 ) * 0) >= 4 ) <<59;
        aux.x |= ( ( ( (v.x>>9)%2 ) * 4 + ( (v.x>>10)%2 ) * 4 + ( (v.x>>8)%2 ) * 0) >= 8 ) <<60;
        aux.x |= ( ( ( (v.x>>29)%2 ) * 8 + ( (v.x>>48)%2 ) * 8 + ( (v.x>>28)%2 ) * 8 + ( (v.x>>26)%2 ) * 0 + ( (v.x>>30)%2 ) * 0) >= 24 ) <<61;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 11
__global__ void passo_bool_11(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_11(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_11(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_11(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 12
__global__ void passo_bool_12(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_12(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_12(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_12(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 13
__global__ void passo_bool_13(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_13(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_13(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_13(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 14
__global__ void passo_bool_14(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_14(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_14(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_14(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 15
__global__ void passo_bool_15(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_15(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_15(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_15(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 17
__global__ void passo_bool_17(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_17(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_17(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_17(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 18
__global__ void passo_bool_18(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_18(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_18(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = 0;
        aux.y = 0;
        aux.z = 0;
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_18(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}


void preenche_init_rand(ulonglong3 * init_rand, unsigned long long nSim, unsigned int nEq)
{
    
    for(unsigned long long i = 0; i < nSim; i++)
    {
        
        init_rand[i].x = 0;
        init_rand[i].y = 0;
        init_rand[i].z = 0;
        unsigned long rand1 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand2 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand3 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand4 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand5 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand6 = rand()%((unsigned long)(1<<31)-1);
        for(int j = 0; j < nEq && j < 32; j++)
            init_rand[i].x |= ((rand1>>j)%2)<<j;
        for(int j = 32; j < nEq && j < 64; j++)
            init_rand[i].x |= ((rand2>>(j-32))%2)<<j;
        for(int j = 64; j < nEq && j < 96; j++)
            init_rand[i].y |= ((rand3>>(j-64))%2)<<(j-64);
        for(int j = 96; j < nEq && j < 128; j++)
            init_rand[i].y |= ((rand4>>(j-96))%2)<<(j-64);
        for(int j = 128; j < nEq && j < 160; j++)
            init_rand[i].z |= ((rand5>>(j-128))%2)<<(j-128);
        for(int j = 160; j < nEq && j < 192; j++)
            init_rand[i].z |= ((rand6>>(j-160))%2)<<(j-128);
    }
}

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[1];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    ulonglong3 *h_init_rand, *h_estado;
    h_init_rand = new ulonglong3[MAX_ESTADO];
    h_estado = new ulonglong3[MAX_ESTADO];

    ulonglong3 *d_init_rand, *d_estado;
    hipMalloc(&d_init_rand,sizeof(ulonglong3)*MAX_ESTADO);
    hipMalloc(&d_estado,sizeof(ulonglong3)*MAX_ESTADO);
    
    unsigned int nEq = 41;
    srand(MAX_ESTADO);
    preenche_init_rand(h_init_rand,MAX_ESTADO,nEq);

    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i].x = 0;
    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i].y = 0;
    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i].z = 0;

    hipMemcpy(d_init_rand, h_init_rand, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);
    hipMemcpy(d_estado, h_estado, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);

    /* passo_tlf_6_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    cudaDeviceSynchronize();
    passo_tlf_6_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    cudaDeviceSynchronize();
    passo_tlf_6_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    cudaDeviceSynchronize();
    cudaMemcpy(h_estado, d_estado, sizeof(ulonglong3)*MAX_ESTADO, cudaMemcpyDeviceToHost); */

    /* passo_bool_6<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    cudaDeviceSynchronize(); */

    /* unsigned long long i = confere_tlf_6(h_init_rand,h_estado,MAX_ESTADO);
    if(i == MAX_ESTADO)
        cerr << "Resultados da GPU batem com os da CPU\n";
    else
        cerr << "Resultados não batem!\n"; */
    
    delete [] h_estado;
    delete [] h_init_rand;
    hipFree(d_init_rand);
    hipFree(d_estado);
    hipDeviceReset();
    return 0;
}