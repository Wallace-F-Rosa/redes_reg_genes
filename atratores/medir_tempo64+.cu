#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <ctime>
#include <string>
#include <limits>
#include <stdio.h>
#include <stdlib.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;

//REDE 1
__global__ void passo_bool_1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        estado[tid] = aux;
    }
}

unsigned long long confere_bool_1(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];


        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(unsigned long long * init_rand, unsigned long long * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        

    
        estado[tid] = aux;
    }
}

unsigned long long confere_tlf_1(unsigned long long * init_rand, unsigned long long * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}



void preenche_init_rand(unsigned long long * init_rand, unsigned long long nSim, unsigned int nEq)
{
    
    for(unsigned long long i = 0; i < nSim; i++)
    {
        
        init_rand[i] = 0;
        unsigned long rand1 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand2 = rand()%((unsigned long)(1<<31)-1);
        init_rand[i] = rand1;
        for(int j = 0; j < nEq; j++)
            init_rand[i] |= ((rand2>>j)%2)<<j;
    }
}

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[1];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    unsigned long long *h_init_rand, *h_estado;
    h_init_rand = new unsigned long long[MAX_ESTADO];
    h_estado = new unsigned long long[MAX_ESTADO];

    unsigned long long *d_init_rand, *d_estado;
    hipMalloc(&d_init_rand,sizeof(unsigned long long)*MAX_ESTADO);
    hipMalloc(&d_estado,sizeof(unsigned long long)*MAX_ESTADO);
    
    unsigned int nEq = 41;
    srand(MAX_ESTADO);
    preenche_init_rand(h_init_rand,MAX_ESTADO,nEq);

    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i] = 0;

    hipMemcpy(d_init_rand, h_init_rand, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyHostToDevice);
    hipMemcpy(d_estado, h_estado, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyHostToDevice);

    passo_tlf_6_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    passo_tlf_6_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    passo_tlf_6_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    hipMemcpy(h_estado, d_estado, sizeof(unsigned long long)*MAX_ESTADO, hipMemcpyDeviceToHost);

    /* passo_bool_6<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize(); */

    unsigned long long i = confere_tlf_6(h_init_rand,h_estado,MAX_ESTADO);
    if(i == MAX_ESTADO)
        cerr << "Resultados da GPU batem com os da CPU\n";
    else
        cerr << "Resultados não batem!\n";
    
    delete [] h_estado;
    delete [] h_init_rand;
    hipFree(d_init_rand);
    hipFree(d_estado);
    hipDeviceReset();
    return 0;
}