
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <string>
#include <limits>
#include <stdio.h>
#include <stdlib.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;

//REDE 10
__global__ void passo_bool_10_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>0)%2 ) ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>6)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) ) | ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>49)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>12)%2 ) ) & ! ( ( (v.x>>56)%2 ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>0)%2 ) ) | ( ( (v.x>>38)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.x>>22)%2 ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>18)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>1)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 20 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_10_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) ) | ( ( (v.x>>23)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) & ( ( ( ( (v.x>>39)%2 ) ) ) ) ) | ( ( (v.x>>0)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>27)%2 ) ) ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) & ! ( ( (v.x>>63)%2 ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) ) & ! ( ( (v.x>>43)%2 ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) ) | ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>51)%2 ) ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) ) & ! ( ( (v.x>>42)%2 ) ) )<< 41 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_10_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.y>>0)%2 ) ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) ) | ( ( (v.x>>62)%2 ) & ( ( ( ( (v.y>>1)%2 ) ) ) ) )<< 47 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>14)%2 ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) ) ) | ! ( ( (v.x>>20)%2 ) | ( (v.x>>43)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>11)%2 ) ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>14)%2 ) ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>38)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>32)%2 ) ) ) & ! ( ( (v.x>>9)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>9)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>38)%2 ) ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>19)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) ) | ( ( ( (v.x>>30)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>26)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>58)%2 ) ) | ( ( (v.y>>2)%2 ) ) | ( ( (v.x>>43)%2 ) & ( ( ( ( (v.x>>58)%2 ) ) ) ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>8)%2 ) & ( ( ( ( (v.x>>10)%2 ) & ( (v.x>>9)%2 ) ) ) ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>28)%2 ) ) & ( ( ( ! ( (v.x>>26)%2 ) ) ) ) ) ) ) & ! ( ( (v.x>>30)%2 ) ) ) | ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>29)%2 ) ) ) ) )<< 61 ;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}


unsigned long long confere_bool_10(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>0)%2 ) ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>6)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) ) | ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>7)%2 ) ) ) & ! ( ( (v.x>>5)%2 ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>57)%2 ) ) ) & ! ( ( (v.x>>3)%2 ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>49)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) ) | ( ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.x>>5)%2 ) ) ) & ! ( ( (v.x>>7)%2 ) ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>3)%2 ) ) ) & ! ( ( (v.x>>4)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>12)%2 ) ) & ! ( ( (v.x>>56)%2 ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>0)%2 ) ) | ( ( (v.x>>38)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.x>>22)%2 ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>18)%2 ) ) & ! ( ( (v.x>>52)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>1)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) ) | ( ( (v.x>>14)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) ) | ( ( (v.x>>23)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) & ( ( ( ( (v.x>>39)%2 ) ) ) ) ) | ( ( (v.x>>0)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>27)%2 ) ) ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) & ! ( ( (v.x>>63)%2 ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) ) & ! ( ( (v.x>>43)%2 ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) ) | ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>1)%2 ) ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>21)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>51)%2 ) ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) ) & ! ( ( (v.x>>42)%2 ) ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.y>>0)%2 ) ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) ) | ( ( (v.x>>62)%2 ) & ( ( ( ( (v.y>>1)%2 ) ) ) ) )<< 47 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>14)%2 ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) ) ) | ! ( ( (v.x>>20)%2 ) | ( (v.x>>43)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>11)%2 ) ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>61)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) ) | ( ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>50)%2 ) ) ) & ! ( ( (v.x>>57)%2 ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>14)%2 ) ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>38)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>32)%2 ) ) ) & ! ( ( (v.x>>9)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>57)%2 ) ) & ! ( ( (v.x>>9)%2 ) ) ) & ! ( ( (v.x>>0)%2 ) ) ) & ! ( ( (v.x>>38)%2 ) ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>19)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) ) | ( ( ( (v.x>>30)%2 ) ) & ! ( ( (v.x>>32)%2 ) & ( ( ( ( (v.x>>43)%2 ) ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>26)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>58)%2 ) ) | ( ( (v.y>>2)%2 ) ) | ( ( (v.x>>43)%2 ) & ( ( ( ( (v.x>>58)%2 ) ) ) ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>8)%2 ) & ( ( ( ( (v.x>>10)%2 ) & ( (v.x>>9)%2 ) ) ) ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>28)%2 ) ) & ( ( ( ! ( (v.x>>26)%2 ) ) ) ) ) ) ) & ! ( ( (v.x>>30)%2 ) ) ) | ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>48)%2 ) & ( (v.x>>29)%2 ) ) ) ) )<< 61 ;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_10_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * -2) >= 0 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>7)%2 ) * -6 + ( (v.x>>5)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>5)%2 ) * -14 + ( (v.x>>7)%2 ) * -14 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>4)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>38)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>52)%2 ) * -6 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>22)%2 ) * -2) >= 4 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>52)%2 ) * -2 + ( (v.x>>0)%2 ) * -2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<20;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_10_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>39)%2 ) * 2 + ( (v.x>>0)%2 ) * 6) >= 4 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>27)%2 ) * 2) >= 4 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * -2) >= 4 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>1)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>51)%2 ) * -2) >= 0 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 4 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>42)%2 ) * -2) >= 2 ) <<41;

    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_10_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * -2) >= 0 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * 6) >= 4 ) <<46;
        aux.x |=(unsigned long long) ( ( 0 * 6 + 0 * 2 + ( (v.y>>1)%2 ) * 2) >= 4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 0 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * -2) >= 0 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>50)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>38)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>32)%2 ) * -2 + ( (v.x>>9)%2 ) * -2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>9)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>38)%2 ) * -2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>19)%2 ) * 6 + ( (v.x>>32)%2 ) * -6 + ( (v.x>>43)%2 ) * -6 + ( (v.x>>30)%2 ) * 6) >= 0 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 4 + ( (v.x>>58)%2 ) * 4 + 0 * 4 + ( (v.x>>43)%2 ) * 0) >= 4 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 4 + ( (v.x>>10)%2 ) * 4 + ( (v.x>>8)%2 ) * 0) >= 8 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 8 + ( (v.x>>48)%2 ) * 8 + ( (v.x>>28)%2 ) * 8 + ( (v.x>>26)%2 ) * 0 + ( (v.x>>30)%2 ) * 0) >= 24 ) <<61;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_10(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * -2) >= 0 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>7)%2 ) * -6 + ( (v.x>>5)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>5)%2 ) * -14 + ( (v.x>>7)%2 ) * -14 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>3)%2 ) * -6 + ( (v.x>>4)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>38)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>52)%2 ) * -6 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>22)%2 ) * -2) >= 4 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>52)%2 ) * -2 + ( (v.x>>0)%2 ) * -2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 2 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>39)%2 ) * 2 + ( (v.x>>0)%2 ) * 6) >= 4 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>27)%2 ) * 2) >= 4 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * -2) >= 4 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2 + ( (v.x>>1)%2 ) * -6 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>21)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>51)%2 ) * -2) >= 0 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 4 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>42)%2 ) * -2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * -2) >= 0 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>63)%2 ) * 6) >= 4 ) <<46;
        aux.x |=(unsigned long long) ( ( 0 * 6 + 0 * 2 + ( (v.y>>1)%2 ) * 2) >= 4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>43)%2 ) * -2) >= 0 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * -2) >= 0 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>50)%2 ) * -6 + ( (v.x>>57)%2 ) * -6 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * -2) >= 0 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>38)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>32)%2 ) * -2 + ( (v.x>>9)%2 ) * -2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2 + ( (v.x>>9)%2 ) * -2 + ( (v.x>>0)%2 ) * -2 + ( (v.x>>38)%2 ) * -2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>19)%2 ) * 6 + ( (v.x>>32)%2 ) * -6 + ( (v.x>>43)%2 ) * -6 + ( (v.x>>30)%2 ) * 6) >= 0 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 4 + ( (v.x>>58)%2 ) * 4 + 0 * 4 + ( (v.x>>43)%2 ) * 0) >= 4 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 4 + ( (v.x>>10)%2 ) * 4 + ( (v.x>>8)%2 ) * 0) >= 8 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 8 + ( (v.x>>48)%2 ) * 8 + ( (v.x>>28)%2 ) * 8 + ( (v.x>>26)%2 ) * 0 + ( (v.x>>30)%2 ) * 0) >= 24 ) <<61;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 11
__global__ void passo_bool_11_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) ) & ! ( ( (v.x>>62)%2 ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ! ( (v.x>>7)%2 ) ) ) | ( ( ! ( (v.x>>6)%2 ) ) ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>20)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>2)%2 ) & ( ( ( ( (v.x>>16)%2 ) & ( (v.x>>52)%2 ) ) ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>26)%2 ) & ( ( ( ( (v.x>>10)%2 ) & ( (v.x>>4)%2 ) & ( (v.x>>46)%2 ) ) ) ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>1)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>32)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>25)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 20 ;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_11_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>51)%2 ) ) ) ) ) & ! ( ( (v.x>>62)%2 ) ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>55)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) & ( ( ( ( (v.x>>38)%2 ) ) ) ) ) | ( ( (v.x>>41)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>50)%2 ) ) | ( ( (v.x>>34)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>31)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 41 ;
        
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_11_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) & ( ( ( ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.x>>30)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) & ! ( ( (v.y>>1)%2 ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>30)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) | ( ( (v.x>>1)%2 ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) & ( ( ( ( (v.y>>2)%2 ) ) ) ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) & ( ( ( ( (v.x>>42)%2 ) & ( (v.x>>44)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>23)%2 ) ) | ( ( (v.x>>18)%2 ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) & ( ( ( ( (v.x>>39)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( (v.x>>40)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( (v.x>>0)%2 ) )<< 61 ;

    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}


unsigned long long confere_bool_11(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) ) & ! ( ( (v.x>>62)%2 ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ! ( (v.x>>7)%2 ) ) ) | ( ( ! ( (v.x>>6)%2 ) ) ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>20)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>2)%2 ) & ( ( ( ( (v.x>>16)%2 ) & ( (v.x>>52)%2 ) ) ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>26)%2 ) & ( ( ( ( (v.x>>10)%2 ) & ( (v.x>>4)%2 ) & ( (v.x>>46)%2 ) ) ) ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>1)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>32)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>25)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>51)%2 ) ) ) ) ) & ! ( ( (v.x>>62)%2 ) ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>55)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) & ( ( ( ( (v.x>>38)%2 ) ) ) ) ) | ( ( (v.x>>41)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>50)%2 ) ) | ( ( (v.x>>34)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>31)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) & ( ( ( ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.x>>30)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) & ! ( ( (v.y>>1)%2 ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>30)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) | ( ( (v.x>>1)%2 ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) & ( ( ( ( (v.y>>2)%2 ) ) ) ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) & ( ( ( ( (v.x>>42)%2 ) & ( (v.x>>44)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>23)%2 ) ) | ( ( (v.x>>18)%2 ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) & ( ( ( ( (v.x>>39)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( (v.x>>40)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( (v.x>>0)%2 ) )<< 61 ;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_11_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 4 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 4 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2 + ( (v.x>>62)%2 ) * -2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 6 + ( (v.x>>7)%2 ) * -2 + ( (v.x>>6)%2 ) * -2) >= 4 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2 + ( (v.x>>20)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>2)%2 ) * 2 + ( (v.x>>16)%2 ) * 2 + ( (v.x>>52)%2 ) * 2) >= 6 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 2 + ( (v.x>>10)%2 ) * 2 + ( (v.x>>4)%2 ) * 2 + ( (v.x>>46)%2 ) * 2) >= 8 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2 + ( (v.x>>1)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>25)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<20;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_11_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>62)%2 ) * -2) >= 4 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>55)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2 + ( (v.x>>38)%2 ) * 2 + ( (v.x>>41)%2 ) * 6) >= 4 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + ( (v.x>>34)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>31)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<41;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_11_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>23)%2 ) * 2 + ( (v.x>>30)%2 ) * 6) >= 4 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>11)%2 ) * 2 + ( (v.y>>1)%2 ) * -2) >= 4 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>30)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2 + ( (v.x>>1)%2 ) * 6) >= 4 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2 + ( (v.y>>2)%2 ) * 2) >= 4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 4 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2 + ( (v.x>>42)%2 ) * 2 + ( (v.x>>44)%2 ) * 2) >= 6 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2 + ( (v.x>>23)%2 ) * 2 + ( (v.x>>18)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 4 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>39)%2 ) * 2 + 0 * -2 + ( (v.x>>40)%2 ) * 14) >= 4 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2) >= 2 ) <<61;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}


unsigned long long confere_tlf_11(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 4 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 4 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2 + ( (v.x>>62)%2 ) * -2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 6 + ( (v.x>>7)%2 ) * -2 + ( (v.x>>6)%2 ) * -2) >= 4 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2 + ( (v.x>>20)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>2)%2 ) * 2 + ( (v.x>>16)%2 ) * 2 + ( (v.x>>52)%2 ) * 2) >= 6 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 2 + ( (v.x>>10)%2 ) * 2 + ( (v.x>>4)%2 ) * 2 + ( (v.x>>46)%2 ) * 2) >= 8 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2 + ( (v.x>>1)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>25)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>62)%2 ) * -2) >= 4 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>55)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2 + ( (v.x>>38)%2 ) * 2 + ( (v.x>>41)%2 ) * 6) >= 4 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + ( (v.x>>34)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>31)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>23)%2 ) * 2 + ( (v.x>>30)%2 ) * 6) >= 4 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>11)%2 ) * 2 + ( (v.y>>1)%2 ) * -2) >= 4 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>30)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2 + ( (v.x>>1)%2 ) * 6) >= 4 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2 + ( (v.y>>2)%2 ) * 2) >= 4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 4 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2 + ( (v.x>>42)%2 ) * 2 + ( (v.x>>44)%2 ) * 2) >= 6 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2 + ( (v.x>>23)%2 ) * 2 + ( (v.x>>18)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 4 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>39)%2 ) * 2 + 0 * -2 + ( (v.x>>40)%2 ) * 14) >= 4 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2) >= 2 ) <<61;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 12
__global__ void passo_bool_12_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ! ( ( ( (v.x>>61)%2 ) ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>55)%2 ) ) | ( ( (v.x>>2)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) ) | ( ( (v.x>>54)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) ) | ( ( (v.x>>13)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) ) | ( ( (v.x>>17)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) ) | ( ( (v.x>>59)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) ) & ! ( ( (v.x>>18)%2 ) ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>12)%2 ) ) & ! ( ( (v.x>>14)%2 ) ) ) & ! ( ( (v.x>>16)%2 ) ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) ) | ( ( (v.y>>0)%2 ) ) | ( ( (v.y>>1)%2 ) )<< 19 ;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_12_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ! ( ( ( (v.x>>57)%2 ) ) | ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>44)%2 ) ) )<< 20 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>44)%2 ) ) )<< 21 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>44)%2 ) ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) & ( ( ( ( (v.x>>46)%2 ) | ( (v.x>>44)%2 ) ) ) ) )<< 23 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 24 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) ) | ( ( (v.x>>3)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) ) | ( ( (v.x>>3)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) )<< 33 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>43)%2 ) ) | ( ( (v.x>>42)%2 ) ) )<< 34 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 35 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) ) | ( ( (v.x>>3)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) ) | ( ( (v.x>>29)%2 ) )<< 39 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_12_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>38)%2 ) ) & ! ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>47)%2 ) ) ) ) ) ) & ! ( ( (v.x>>58)%2 ) ) ) & ! ( ( (v.x>>48)%2 ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>44)%2 ) | ( (v.x>>57)%2 ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.y>>2)%2 ) ) | ( ( (v.y>>0)%2 ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.y>>3)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>43)%2 ) ) | ( ( (v.x>>42)%2 ) ) )<< 51 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 52 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>42)%2 ) ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>50)%2 ) ) | ( ( (v.x>>55)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>4)%2 ) ) & ! ( ( (v.x>>19)%2 ) ) ) | ( ( ( (v.y>>5)%2 ) ) & ! ( ( (v.x>>19)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>51)%2 ) ) | ( ( (v.y>>0)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>52)%2 ) ) & ! ( ( (v.x>>48)%2 ) ) ) & ! ( ( (v.x>>58)%2 ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) ) | ( ( (v.y>>7)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>40)%2 ) ) | ( ( (v.y>>8)%2 ) )<< 60 ;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_12(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ! ( ( ( (v.x>>61)%2 ) ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>55)%2 ) ) | ( ( (v.x>>2)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) ) | ( ( (v.x>>54)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) ) | ( ( (v.x>>13)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) ) | ( ( (v.x>>17)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) ) | ( ( (v.x>>59)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) ) & ! ( ( (v.x>>18)%2 ) ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>12)%2 ) ) & ! ( ( (v.x>>14)%2 ) ) ) & ! ( ( (v.x>>16)%2 ) ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) ) | ( ( (v.y>>0)%2 ) ) | ( ( (v.y>>1)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>57)%2 ) ) | ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>44)%2 ) ) )<< 20 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>44)%2 ) ) )<< 21 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>44)%2 ) ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) & ( ( ( ( (v.x>>46)%2 ) | ( (v.x>>44)%2 ) ) ) ) )<< 23 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 24 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) ) | ( ( (v.x>>3)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) ) | ( ( (v.x>>3)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) ) & ! ( ( (v.x>>41)%2 ) ) )<< 33 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>43)%2 ) ) | ( ( (v.x>>42)%2 ) ) )<< 34 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 35 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>54)%2 ) ) | ( ( (v.x>>3)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) ) | ( ( (v.x>>29)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>38)%2 ) ) & ! ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>47)%2 ) ) ) ) ) ) & ! ( ( (v.x>>58)%2 ) ) ) & ! ( ( (v.x>>48)%2 ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>44)%2 ) | ( (v.x>>57)%2 ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.y>>2)%2 ) ) | ( ( (v.y>>0)%2 ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.y>>3)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>3)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>43)%2 ) ) | ( ( (v.x>>42)%2 ) ) )<< 51 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) ) )<< 52 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>42)%2 ) ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>50)%2 ) ) | ( ( (v.x>>55)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>4)%2 ) ) & ! ( ( (v.x>>19)%2 ) ) ) | ( ( ( (v.y>>5)%2 ) ) & ! ( ( (v.x>>19)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>51)%2 ) ) | ( ( (v.y>>0)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>52)%2 ) ) & ! ( ( (v.x>>48)%2 ) ) ) & ! ( ( (v.x>>58)%2 ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) ) | ( ( (v.y>>7)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>40)%2 ) ) | ( ( (v.y>>8)%2 ) )<< 60 ;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_12_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * -2) >= 0 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2) >= 0 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>55)%2 ) * 2 + ( (v.x>>2)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>41)%2 ) * -2 + ( (v.x>>13)%2 ) * 6) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>17)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2) >= 0 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2 + ( (v.x>>18)%2 ) * -2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>14)%2 ) * -2 + ( (v.x>>16)%2 ) * -2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 2 + ( (v.y>>1)%2 ) * 2) >= 2 ) <<19;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_12_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * -2 + ( (v.x>>46)%2 ) * -2 + ( (v.x>>44)%2 ) * -2) >= 0 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2 + ( (v.x>>44)%2 ) * -2) >= 0 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2 + ( (v.x>>44)%2 ) * -2) >= 0 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 6 + ( (v.x>>46)%2 ) * 2 + ( (v.x>>44)%2 ) * 2) >= 8 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>3)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2) >= 0 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2 + ( (v.x>>41)%2 ) * -2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2 + ( (v.x>>41)%2 ) * -2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>3)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>41)%2 ) * -2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * -2 + ( (v.x>>42)%2 ) * -2) >= 0 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>3)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2 + ( (v.x>>29)%2 ) * 2) >= 2 ) <<39;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_12_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 6 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>47)%2 ) * -2 + ( (v.x>>58)%2 ) * -6 + ( (v.x>>48)%2 ) * -6) >= 4 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( 0 * 6 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 8 ) <<46;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2) >= 2 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2 + ( (v.x>>43)%2 ) * -2 + ( (v.x>>42)%2 ) * -2) >= 0 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2 + ( (v.x>>42)%2 ) * -2) >= 0 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + ( (v.x>>55)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>19)%2 ) * -6 + ( (v.y>>5)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>51)%2 ) * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2 + ( (v.x>>48)%2 ) * -2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.y>>6)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.y>>7)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>40)%2 ) * 2 + ( (v.y>>8)%2 ) * 2) >= 2 ) <<60;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_12(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * -2) >= 0 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2) >= 0 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>55)%2 ) * 2 + ( (v.x>>2)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>41)%2 ) * -2 + ( (v.x>>13)%2 ) * 6) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>17)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2) >= 0 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2 + ( (v.x>>18)%2 ) * -2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>14)%2 ) * -2 + ( (v.x>>16)%2 ) * -2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 2 + ( (v.y>>1)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * -2 + ( (v.x>>46)%2 ) * -2 + ( (v.x>>44)%2 ) * -2) >= 0 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2 + ( (v.x>>44)%2 ) * -2) >= 0 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2 + ( (v.x>>44)%2 ) * -2) >= 0 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 6 + ( (v.x>>46)%2 ) * 2 + ( (v.x>>44)%2 ) * 2) >= 8 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>3)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2) >= 0 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2 + ( (v.x>>41)%2 ) * -2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2 + ( (v.x>>41)%2 ) * -2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>3)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>41)%2 ) * -2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * -2 + ( (v.x>>42)%2 ) * -2) >= 0 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>3)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2 + ( (v.x>>29)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 6 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>47)%2 ) * -2 + ( (v.x>>58)%2 ) * -6 + ( (v.x>>48)%2 ) * -6) >= 4 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( 0 * 6 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 8 ) <<46;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2) >= 2 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>3)%2 ) * 2) >= 2 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2 + ( (v.x>>43)%2 ) * -2 + ( (v.x>>42)%2 ) * -2) >= 0 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2) >= 0 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * -2 + ( (v.x>>42)%2 ) * -2) >= 0 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + ( (v.x>>55)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>19)%2 ) * -6 + ( (v.y>>5)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>51)%2 ) * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2 + ( (v.x>>48)%2 ) * -2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.y>>6)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.y>>7)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>40)%2 ) * 2 + ( (v.y>>8)%2 ) * 2) >= 2 ) <<60;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 13
__global__ void passo_bool_13_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>0)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>61)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) ) ) | ( ( (v.x>>43)%2 ) ) | ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>4)%2 ) ) ) | ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>37)%2 ) ) ) ) ) | ( ( (v.x>>10)%2 ) ) | ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.x>>37)%2 ) ) ) ) )<< 3 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>33)%2 ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>2)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.y>>2)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>9)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>23)%2 ) ) | ( ( (v.y>>1)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>58)%2 ) ) ) | ! ( ( (v.x>>58)%2 ) | ( (v.x>>55)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>47)%2 ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) ) | ( ( (v.x>>53)%2 ) ) | ( ( (v.x>>1)%2 ) ) | ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>35)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>39)%2 ) & ( ( ( ! ( (v.x>>38)%2 ) & ! ( (v.x>>61)%2 ) ) ) ) ) | ( ( (v.x>>38)%2 ) & ( ( ( ! ( (v.x>>39)%2 ) & ! ( (v.x>>61)%2 ) ) ) ) ) ) | ! ( ( (v.x>>39)%2 ) | ( (v.x>>38)%2 ) | ( (v.x>>61)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) )<< 19 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_13_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.y>>3)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.y>>4)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>27)%2 ) ) | ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>36)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) ) & ! ( ( (v.x>>11)%2 ) ) ) | ( ( (v.x>>29)%2 ) ) | ( ( ( (v.x>>25)%2 ) ) & ! ( ( (v.x>>15)%2 ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>32)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.y>>5)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>42)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>19)%2 ) )<< 39 ;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_13_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) ) | ( ( (v.y>>6)%2 ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>47)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.y>>7)%2 ) ) | ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>12)%2 ) ) | ( ( (v.x>>31)%2 ) ) | ( ( (v.y>>6)%2 ) ) | ( ( ( ( (v.y>>0)%2 ) ) & ! ( ( (v.x>>13)%2 ) ) ) & ! ( ( (v.x>>56)%2 ) ) ) | ( ( ( (v.y>>1)%2 ) ) & ! ( ( (v.x>>56)%2 ) ) ) | ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>49)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) & ( ( ( ! ( (v.x>>50)%2 ) ) ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) ) | ( ( (v.y>>5)%2 ) ) | ( ( (v.x>>38)%2 ) & ( ( ( ! ( (v.y>>5)%2 ) & ! ( (v.x>>46)%2 ) & ! ( (v.x>>61)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.x>>46)%2 ) & ( ( ( ! ( (v.x>>38)%2 ) & ! ( (v.y>>5)%2 ) & ! ( (v.x>>61)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>46)%2 ) ) & ( ( ( ! ( (v.x>>38)%2 ) & ! ( (v.y>>5)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) ) ) ) | ! ( ( (v.x>>38)%2 ) | ( (v.x>>46)%2 ) | ( (v.y>>5)%2 ) | ( (v.x>>61)%2 ) | ( (v.x>>42)%2 ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) ) & ! ( ( (v.x>>40)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.y>>8)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>7)%2 ) ) & ! ( ( (v.x>>51)%2 ) ) ) | ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>26)%2 ) ) | ( ( (v.x>>43)%2 ) ) | ( ( (v.y>>6)%2 ) ) | ( ( (v.x>>8)%2 ) )<< 59 ;


        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_13(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>0)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>61)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) ) ) | ( ( (v.x>>43)%2 ) ) | ( ( ( (v.x>>59)%2 ) ) & ! ( ( (v.x>>4)%2 ) ) ) | ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>37)%2 ) ) ) ) ) | ( ( (v.x>>10)%2 ) ) | ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.x>>37)%2 ) ) ) ) )<< 3 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>33)%2 ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>2)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.y>>2)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>9)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>23)%2 ) ) | ( ( (v.y>>1)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>58)%2 ) ) ) | ! ( ( (v.x>>58)%2 ) | ( (v.x>>55)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>47)%2 ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) ) | ( ( (v.x>>53)%2 ) ) | ( ( (v.x>>1)%2 ) ) | ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>35)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) ) | ( ( (v.x>>39)%2 ) & ( ( ( ! ( (v.x>>38)%2 ) & ! ( (v.x>>61)%2 ) ) ) ) ) | ( ( (v.x>>38)%2 ) & ( ( ( ! ( (v.x>>39)%2 ) & ! ( (v.x>>61)%2 ) ) ) ) ) ) | ! ( ( (v.x>>39)%2 ) | ( (v.x>>38)%2 ) | ( (v.x>>61)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.y>>3)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.y>>4)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>27)%2 ) ) | ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>36)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) ) & ! ( ( (v.x>>11)%2 ) ) ) | ( ( (v.x>>29)%2 ) ) | ( ( ( (v.x>>25)%2 ) ) & ! ( ( (v.x>>15)%2 ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>32)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.y>>5)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>42)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>19)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) ) | ( ( (v.y>>6)%2 ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>47)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.y>>7)%2 ) ) | ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>12)%2 ) ) | ( ( (v.x>>31)%2 ) ) | ( ( (v.y>>6)%2 ) ) | ( ( ( ( (v.y>>0)%2 ) ) & ! ( ( (v.x>>13)%2 ) ) ) & ! ( ( (v.x>>56)%2 ) ) ) | ( ( ( (v.y>>1)%2 ) ) & ! ( ( (v.x>>56)%2 ) ) ) | ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>49)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) & ( ( ( ! ( (v.x>>50)%2 ) ) ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) ) | ( ( (v.y>>5)%2 ) ) | ( ( (v.x>>38)%2 ) & ( ( ( ! ( (v.y>>5)%2 ) & ! ( (v.x>>46)%2 ) & ! ( (v.x>>61)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.x>>46)%2 ) & ( ( ( ! ( (v.x>>38)%2 ) & ! ( (v.y>>5)%2 ) & ! ( (v.x>>61)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>46)%2 ) ) & ( ( ( ! ( (v.x>>38)%2 ) & ! ( (v.y>>5)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) ) ) ) | ! ( ( (v.x>>38)%2 ) | ( (v.x>>46)%2 ) | ( (v.y>>5)%2 ) | ( (v.x>>61)%2 ) | ( (v.x>>42)%2 ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) ) & ! ( ( (v.x>>40)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.y>>8)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>7)%2 ) ) & ! ( ( (v.x>>51)%2 ) ) ) | ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>26)%2 ) ) | ( ( (v.x>>43)%2 ) ) | ( ( (v.y>>6)%2 ) ) | ( ( (v.x>>8)%2 ) )<< 59 ;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_13_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>61)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( 0 * 30 + 0 * 30 + ( (v.x>>43)%2 ) * 90 + ( (v.x>>59)%2 ) * 30 + ( (v.x>>4)%2 ) * -30 + 0 * 18 + ( (v.x>>37)%2 ) * -54 + ( (v.x>>10)%2 ) * 90 + ( (v.y>>1)%2 ) * 18) >= -24 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * -2) >= 0 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>2)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.y>>2)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>9)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.y>>1)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>58)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 0 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * -2) >= 0 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2 + ( (v.x>>53)%2 ) * 2 + ( (v.x>>1)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>35)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>39)%2 ) * -2 + ( (v.x>>38)%2 ) * -2) >= -2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2) >= 2 ) <<19;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_13_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2) >= 2 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>27)%2 ) * 2 + ( (v.x>>30)%2 ) * 2 + ( (v.x>>36)%2 ) * 2) >= 2 ) <<29;
        aux.x |=(unsigned long long) ( ( 0 * 6 + ( (v.x>>11)%2 ) * -6 + ( (v.x>>29)%2 ) * 18 + ( (v.x>>25)%2 ) * 6 + ( (v.x>>15)%2 ) * -6) >= 0 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>19)%2 ) * 2) >= 2 ) <<39;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_13_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2 + ( (v.y>>6)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>47)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( 0 * 22 + ( (v.x>>6)%2 ) * 22 + ( (v.x>>41)%2 ) * 22 + ( (v.x>>12)%2 ) * 22 + ( (v.x>>31)%2 ) * 22 + 0 * 22 + 0 * 2 + ( (v.x>>13)%2 ) * -2 + ( (v.x>>56)%2 ) * -10 + 0 * 6 + ( (v.x>>14)%2 ) * 22 + ( (v.x>>49)%2 ) * 22) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2 + ( (v.x>>50)%2 ) * -2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2) >= 2 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 8 + 0 * 8 + ( (v.x>>38)%2 ) * -4 + ( (v.x>>46)%2 ) * 0 + ( (v.x>>61)%2 ) * -4) >= -4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>40)%2 ) * -2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>51)%2 ) * -2 + ( (v.x>>30)%2 ) * 6 + ( (v.x>>26)%2 ) * 6 + ( (v.x>>43)%2 ) * 6 + 0 * 6 + ( (v.x>>8)%2 ) * 6) >= 2 ) <<59;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_13(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>61)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( 0 * 30 + 0 * 30 + ( (v.x>>43)%2 ) * 90 + ( (v.x>>59)%2 ) * 30 + ( (v.x>>4)%2 ) * -30 + 0 * 18 + ( (v.x>>37)%2 ) * -54 + ( (v.x>>10)%2 ) * 90 + ( (v.y>>1)%2 ) * 18) >= -24 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * -2) >= 0 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>2)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.y>>2)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>9)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.y>>1)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>58)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 0 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * -2) >= 0 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2 + ( (v.x>>53)%2 ) * 2 + ( (v.x>>1)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>35)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>39)%2 ) * -2 + ( (v.x>>38)%2 ) * -2) >= -2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2) >= 2 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>27)%2 ) * 2 + ( (v.x>>30)%2 ) * 2 + ( (v.x>>36)%2 ) * 2) >= 2 ) <<29;
        aux.x |=(unsigned long long) ( ( 0 * 6 + ( (v.x>>11)%2 ) * -6 + ( (v.x>>29)%2 ) * 18 + ( (v.x>>25)%2 ) * 6 + ( (v.x>>15)%2 ) * -6) >= 0 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>19)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2 + ( (v.y>>6)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>47)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( 0 * 22 + ( (v.x>>6)%2 ) * 22 + ( (v.x>>41)%2 ) * 22 + ( (v.x>>12)%2 ) * 22 + ( (v.x>>31)%2 ) * 22 + 0 * 22 + 0 * 2 + ( (v.x>>13)%2 ) * -2 + ( (v.x>>56)%2 ) * -10 + 0 * 6 + ( (v.x>>14)%2 ) * 22 + ( (v.x>>49)%2 ) * 22) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2 + ( (v.x>>50)%2 ) * -2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2) >= 2 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 8 + 0 * 8 + ( (v.x>>38)%2 ) * -4 + ( (v.x>>46)%2 ) * 0 + ( (v.x>>61)%2 ) * -4) >= -4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>54)%2 ) * 2 + ( (v.x>>40)%2 ) * -2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>51)%2 ) * -2 + ( (v.x>>30)%2 ) * 6 + ( (v.x>>26)%2 ) * 6 + ( (v.x>>43)%2 ) * 6 + 0 * 6 + ( (v.x>>8)%2 ) * 6) >= 2 ) <<59;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 14
__global__ void passo_bool_14_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) ) | ( ( (v.x>>46)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) & ( ( ( ! ( (v.x>>5)%2 ) & ! ( (v.x>>39)%2 ) ) ) ) ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) & ( ( ( ! ( (v.x>>46)%2 ) & ! ( (v.x>>4)%2 ) & ! ( (v.x>>1)%2 ) ) ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.x>>11)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>44)%2 ) & ( ( ( ! ( (v.x>>11)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>7)%2 ) & ( ( ( ! ( (v.x>>11)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>50)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>11)%2 ) ) ) ) ) | ( ( (v.x>>44)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>11)%2 ) ) ) ) ) | ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>11)%2 ) ) ) ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) & ( ( ( ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>30)%2 ) & ( ( ( ( (v.x>>50)%2 ) ) ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) ) | ( ( (v.y>>4)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) & ( ( ( ! ( (v.x>>50)%2 ) ) ) ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) & ( ( ( ! ( (v.x>>46)%2 ) ) ) ) ) | ( ( (v.y>>7)%2 ) & ( ( ( ! ( (v.x>>46)%2 ) ) ) ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>55)%2 ) & ! ( (v.y>>0)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>55)%2 ) & ! ( (v.y>>0)%2 ) ) ) ) ) | ( ( (v.x>>52)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>55)%2 ) & ! ( (v.y>>0)%2 ) ) ) ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>49)%2 ) & ! ( (v.x>>18)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>58)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>49)%2 ) & ! ( (v.x>>18)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>15)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>49)%2 ) & ! ( (v.x>>18)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.y>>8)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) ) | ( ( (v.x>>9)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) ) | ( ( (v.x>>37)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) ) | ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) )<< 21 ;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_14_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>23)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.y>>9)%2 ) & ( ( ( ! ( (v.x>>58)%2 ) ) ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.y>>10)%2 ) & ( ( ( ! ( (v.x>>58)%2 ) ) ) ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.y>>11)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.y>>13)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.y>>14)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) & ( ( ( ! ( (v.x>>54)%2 ) ) ) ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>32)%2 ) & ( ( ( ! ( (v.x>>53)%2 ) ) ) ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.y>>15)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) ) ) ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) & ( ( ( ! ( (v.x>>13)%2 ) ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) ) ) | ( ( (v.x>>43)%2 ) & ( ( ( ! ( (v.x>>13)%2 ) ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) & ( ( ( ! ( (v.x>>44)%2 ) ) ) ) ) | ( ( (v.x>>35)%2 ) & ( ( ( ! ( (v.x>>44)%2 ) ) ) ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.y>>3)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.y>>4)%2 ) )<< 43 ;


        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_14_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.y>>16)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>11)%2 ) & ( ( ( ! ( (v.y>>1)%2 ) & ! ( (v.x>>5)%2 ) & ! ( (v.x>>39)%2 ) & ! ( (v.x>>13)%2 ) ) ) ) ) | ( ( (v.x>>57)%2 ) & ( ( ( ! ( (v.y>>1)%2 ) & ! ( (v.x>>5)%2 ) & ! ( (v.x>>39)%2 ) ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) & ( ( ( ! ( (v.x>>15)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>55)%2 ) ) ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>44)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.x>>57)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>7)%2 ) ) ) ) ) | ( ( (v.x>>57)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) & ( ( ( ! ( (v.x>>0)%2 ) ) ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) ) | ( ( (v.x>>54)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>40)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) ) | ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>24)%2 ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>25)%2 ) & ( ( ( ! ( (v.x>>15)%2 ) ) ) ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) & ( ( ( ! ( (v.x>>53)%2 ) ) ) ) ) | ( ( (v.x>>31)%2 ) & ( ( ( ! ( (v.x>>53)%2 ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.y>>1)%2 ) ) | ( ( (v.y>>3)%2 ) ) | ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>48)%2 ) ) | ( ( (v.x>>7)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) | ( ( (v.x>>15)%2 ) ) | ( ( (v.x>>14)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>5)%2 ) )<<2;


        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}


unsigned long long confere_bool_14(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( (v.x>>62)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) ) | ( ( (v.x>>46)%2 ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>12)%2 ) ) | ( ( (v.x>>57)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) & ( ( ( ! ( (v.x>>5)%2 ) & ! ( (v.x>>39)%2 ) ) ) ) ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) & ( ( ( ! ( (v.x>>46)%2 ) & ! ( (v.x>>4)%2 ) & ! ( (v.x>>1)%2 ) ) ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.x>>11)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>44)%2 ) & ( ( ( ! ( (v.x>>11)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>7)%2 ) & ( ( ( ! ( (v.x>>11)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>50)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>11)%2 ) ) ) ) ) | ( ( (v.x>>44)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>11)%2 ) ) ) ) ) | ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>11)%2 ) ) ) ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) & ( ( ( ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>30)%2 ) & ( ( ( ( (v.x>>50)%2 ) ) ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) ) | ( ( (v.y>>4)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) & ( ( ( ! ( (v.x>>50)%2 ) ) ) ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) & ( ( ( ! ( (v.x>>46)%2 ) ) ) ) ) | ( ( (v.y>>7)%2 ) & ( ( ( ! ( (v.x>>46)%2 ) ) ) ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>55)%2 ) & ! ( (v.y>>0)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>55)%2 ) & ! ( (v.y>>0)%2 ) ) ) ) ) | ( ( (v.x>>52)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>55)%2 ) & ! ( (v.y>>0)%2 ) ) ) ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>49)%2 ) & ! ( (v.x>>18)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>58)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>49)%2 ) & ! ( (v.x>>18)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>15)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>49)%2 ) & ! ( (v.x>>18)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.y>>8)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) ) | ( ( (v.x>>9)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) ) | ( ( (v.x>>37)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) ) | ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>55)%2 ) ) ) ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>23)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.y>>9)%2 ) & ( ( ( ! ( (v.x>>58)%2 ) ) ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.y>>10)%2 ) & ( ( ( ! ( (v.x>>58)%2 ) ) ) ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.y>>11)%2 ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.y>>13)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.y>>14)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) & ( ( ( ! ( (v.x>>54)%2 ) ) ) ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>32)%2 ) & ( ( ( ! ( (v.x>>53)%2 ) ) ) ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.y>>15)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) ) ) ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>45)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) & ( ( ( ! ( (v.x>>13)%2 ) ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) ) ) | ( ( (v.x>>43)%2 ) & ( ( ( ! ( (v.x>>13)%2 ) ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) & ( ( ( ! ( (v.x>>44)%2 ) ) ) ) ) | ( ( (v.x>>35)%2 ) & ( ( ( ! ( (v.x>>44)%2 ) ) ) ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.y>>3)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.y>>4)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.y>>16)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>11)%2 ) & ( ( ( ! ( (v.y>>1)%2 ) & ! ( (v.x>>5)%2 ) & ! ( (v.x>>39)%2 ) & ! ( (v.x>>13)%2 ) ) ) ) ) | ( ( (v.x>>57)%2 ) & ( ( ( ! ( (v.y>>1)%2 ) & ! ( (v.x>>5)%2 ) & ! ( (v.x>>39)%2 ) ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) & ( ( ( ! ( (v.x>>15)%2 ) & ! ( (v.y>>0)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>55)%2 ) ) ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>49)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>44)%2 ) & ! ( (v.x>>50)%2 ) ) ) ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.x>>57)%2 ) & ! ( (v.y>>1)%2 ) & ! ( (v.x>>7)%2 ) ) ) ) ) | ( ( (v.x>>57)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) & ( ( ( ! ( (v.x>>0)%2 ) ) ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) ) | ( ( (v.x>>54)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>30)%2 ) ) | ( ( (v.x>>40)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) ) | ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>24)%2 ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>25)%2 ) & ( ( ( ! ( (v.x>>15)%2 ) ) ) ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) & ( ( ( ! ( (v.x>>53)%2 ) ) ) ) ) | ( ( (v.x>>31)%2 ) & ( ( ( ! ( (v.x>>53)%2 ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>44)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( (v.x>>60)%2 ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) & ! ( (v.x>>15)%2 ) & ! ( (v.x>>14)%2 ) ) ) ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.y>>1)%2 ) ) | ( ( (v.y>>3)%2 ) ) | ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>50)%2 ) ) ) ) ) | ( ( (v.x>>48)%2 ) ) | ( ( (v.x>>7)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) | ( ( (v.x>>15)%2 ) ) | ( ( (v.x>>14)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>5)%2 ) )<<2;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_14_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2 + ( (v.x>>46)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 4 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>11)%2 ) * 2 + ( (v.x>>5)%2 ) * -2 + ( (v.x>>39)%2 ) * -2) >= 4 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2 + ( (v.x>>46)%2 ) * -2 + ( (v.x>>4)%2 ) * -2 + ( (v.x>>1)%2 ) * -2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.y>>1)%2 ) * 2 + ( (v.x>>11)%2 ) * -14 + ( (v.x>>50)%2 ) * -14 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>7)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + 0 * -14 + ( (v.y>>1)%2 ) * -14 + ( (v.x>>11)%2 ) * -14 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>8)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>50)%2 ) * 6 + ( (v.x>>30)%2 ) * 2) >= 8 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.y>>4)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * -2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 4 ) <<12;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>46)%2 ) * -6 + ( (v.y>>7)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>48)%2 ) * -14 + ( (v.x>>15)%2 ) * -14 + ( (v.x>>55)%2 ) * -14 + ( (v.y>>0)%2 ) * -14 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>52)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>48)%2 ) * -14 + ( (v.x>>49)%2 ) * -14 + ( (v.x>>18)%2 ) * -14 + ( (v.y>>0)%2 ) * -14 + ( (v.x>>14)%2 ) * -14 + ( (v.x>>58)%2 ) * 2 + ( (v.x>>15)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>55)%2 ) * -30 + ( (v.x>>9)%2 ) * 2 + ( (v.x>>37)%2 ) * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<21;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_14_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) * 2) >= 2 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.y>>13)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>54)%2 ) * -2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * 2 + ( (v.x>>53)%2 ) * -2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.y>>3)%2 ) * -2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2 + ( (v.x>>13)%2 ) * -6 + ( (v.x>>11)%2 ) * 6 + ( (v.x>>43)%2 ) * 2) >= 8 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2 + ( (v.x>>44)%2 ) * -6 + ( (v.x>>35)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 2) >= 2 ) <<43;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_14_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.y>>16)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 6 + ( (v.y>>1)%2 ) * -6 + ( (v.x>>5)%2 ) * -6 + ( (v.x>>39)%2 ) * -6 + ( (v.x>>13)%2 ) * -2 + ( (v.x>>57)%2 ) * 2) >= 6 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 6 + ( (v.x>>15)%2 ) * -6 + ( (v.y>>0)%2 ) * -6 + ( (v.x>>14)%2 ) * -6 + ( (v.x>>51)%2 ) * 30 + ( (v.x>>55)%2 ) * 30) >= 6 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + 0 * -2 + ( (v.y>>1)%2 ) * -2 + ( (v.x>>44)%2 ) * -2 + ( (v.x>>50)%2 ) * -2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 10 + ( (v.x>>57)%2 ) * 6 + ( (v.y>>1)%2 ) * -2 + ( (v.x>>7)%2 ) * -2) >= 10 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>0)%2 ) * -2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>30)%2 ) * 2 + ( (v.x>>40)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>41)%2 ) * 2 + ( (v.x>>29)%2 ) * 2 + ( (v.x>>24)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>25)%2 ) * 2 + ( (v.x>>15)%2 ) * -2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>53)%2 ) * -6 + ( (v.x>>31)%2 ) * 2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<62;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>48)%2 ) * -6 + ( (v.x>>15)%2 ) * -6 + ( (v.x>>14)%2 ) * -6 + ( (v.x>>54)%2 ) * 2) >= 2 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.y>>1)%2 ) * 18 + 0 * 18 + ( (v.y>>0)%2 ) * 6 + ( (v.x>>50)%2 ) * -6 + ( (v.x>>48)%2 ) * 18 + ( (v.x>>7)%2 ) * 6 + ( (v.x>>8)%2 ) * -6 + ( (v.x>>15)%2 ) * 18 + ( (v.x>>14)%2 ) * 18) >= 0 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2) >= 2 ) <<2;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_14(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2 + ( (v.x>>46)%2 ) * 2) >= 2 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 4 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>12)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>11)%2 ) * 2 + ( (v.x>>5)%2 ) * -2 + ( (v.x>>39)%2 ) * -2) >= 4 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2 + ( (v.x>>46)%2 ) * -2 + ( (v.x>>4)%2 ) * -2 + ( (v.x>>1)%2 ) * -2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.y>>1)%2 ) * 2 + ( (v.x>>11)%2 ) * -14 + ( (v.x>>50)%2 ) * -14 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>7)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + 0 * -14 + ( (v.y>>1)%2 ) * -14 + ( (v.x>>11)%2 ) * -14 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>8)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>50)%2 ) * 6 + ( (v.x>>30)%2 ) * 2) >= 8 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.y>>4)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * -2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 4 ) <<12;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>46)%2 ) * -6 + ( (v.y>>7)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>48)%2 ) * -14 + ( (v.x>>15)%2 ) * -14 + ( (v.x>>55)%2 ) * -14 + ( (v.y>>0)%2 ) * -14 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>52)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2 + ( (v.x>>48)%2 ) * -14 + ( (v.x>>49)%2 ) * -14 + ( (v.x>>18)%2 ) * -14 + ( (v.y>>0)%2 ) * -14 + ( (v.x>>14)%2 ) * -14 + ( (v.x>>58)%2 ) * 2 + ( (v.x>>15)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>55)%2 ) * -30 + ( (v.x>>9)%2 ) * 2 + ( (v.x>>37)%2 ) * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) * 2) >= 2 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) * 2) >= 2 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.y>>13)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 2 + ( (v.x>>54)%2 ) * -2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * 2 + ( (v.x>>53)%2 ) * -2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.y>>3)%2 ) * -2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2 + ( (v.x>>13)%2 ) * -6 + ( (v.x>>11)%2 ) * 6 + ( (v.x>>43)%2 ) * 2) >= 8 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2 + ( (v.x>>44)%2 ) * -6 + ( (v.x>>35)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 2) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.y>>16)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2) >= 2 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 6 + ( (v.y>>1)%2 ) * -6 + ( (v.x>>5)%2 ) * -6 + ( (v.x>>39)%2 ) * -6 + ( (v.x>>13)%2 ) * -2 + ( (v.x>>57)%2 ) * 2) >= 6 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2) >= 2 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 6 + ( (v.x>>15)%2 ) * -6 + ( (v.y>>0)%2 ) * -6 + ( (v.x>>14)%2 ) * -6 + ( (v.x>>51)%2 ) * 30 + ( (v.x>>55)%2 ) * 30) >= 6 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>49)%2 ) * 2 + 0 * -2 + ( (v.y>>1)%2 ) * -2 + ( (v.x>>44)%2 ) * -2 + ( (v.x>>50)%2 ) * -2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 10 + ( (v.x>>57)%2 ) * 6 + ( (v.y>>1)%2 ) * -2 + ( (v.x>>7)%2 ) * -2) >= 10 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.x>>0)%2 ) * -2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>54)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>20)%2 ) * 2 + ( (v.x>>30)%2 ) * 2 + ( (v.x>>40)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>41)%2 ) * 2 + ( (v.x>>29)%2 ) * 2 + ( (v.x>>24)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>25)%2 ) * 2 + ( (v.x>>15)%2 ) * -2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>53)%2 ) * -6 + ( (v.x>>31)%2 ) * 2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 2) >= 2 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<62;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>48)%2 ) * -6 + ( (v.x>>15)%2 ) * -6 + ( (v.x>>14)%2 ) * -6 + ( (v.x>>54)%2 ) * 2) >= 2 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.y>>1)%2 ) * 18 + 0 * 18 + ( (v.y>>0)%2 ) * 6 + ( (v.x>>50)%2 ) * -6 + ( (v.x>>48)%2 ) * 18 + ( (v.x>>7)%2 ) * 6 + ( (v.x>>8)%2 ) * -6 + ( (v.x>>15)%2 ) * 18 + ( (v.x>>14)%2 ) * 18) >= 0 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2) >= 2 ) <<2;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 15
__global__ void passo_bool_15_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) ) & !( ( (v.y>>7)%2 ) ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>8)%2 ) & ( ( ( ( (v.x>>42)%2 ) & ( (v.x>>46)%2 ) ) ) ) ) & !( ( (v.y>>9)%2 ) ) ) & !( ( (v.y>>10)%2 ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>63)%2 ) ) & !( ( (v.x>>27)%2 ) ) ) & !( ( (v.x>>7)%2 ) ) ) & !( ( (v.x>>3)%2 ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>43)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) ) ) & !( ( (v.y>>7)%2 ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) ) | ( ( (v.x>>9)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) ) | ( ( (v.y>>11)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>25)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>26)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) ) | ( ( (v.x>>41)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) ) | ( ( (v.x>>6)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>26)%2 ) ) | ( ( (v.x>>19)%2 ) & ( ( ( ( (v.x>>46)%2 ) ) ) ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<< 22 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>40)%2 ) ) ) ) | !( ( (v.x>>40)%2 ) | ( (v.x>>6)%2 ) )<< 23 ;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_15_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) ) & !( ( (v.x>>23)%2 ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>56)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.y>>13)%2 ) & ( ( ( ( (v.x>>15)%2 ) ) ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.x>>57)%2 ) ) ) & !( ( (v.x>>56)%2 ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>58)%2 ) ) ) ) | !( ( (v.x>>58)%2 ) | ( (v.x>>59)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) ) | ( ( (v.x>>19)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>32)%2 ) ) | ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) ) ) ) | !( ( (v.x>>32)%2 ) | ( (v.x>>56)%2 ) | ( (v.x>>57)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>58)%2 ) ) | ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) ) | ( ( (v.x>>59)%2 ) ) ) ) | !( ( (v.x>>56)%2 ) | ( (v.x>>58)%2 ) | ( (v.x>>57)%2 ) | ( (v.x>>59)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>51)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.y>>14)%2 ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.y>>14)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>1)%2 ) ) | ( ( (v.x>>19)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>15)%2 ) ) ) ) ) | ( ( (v.x>>53)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) ) & !( ( (v.y>>16)%2 ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>37)%2 ) )<< 47 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_15_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) ) | ( ( (v.x>>8)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>44)%2 ) ) | ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>19)%2 ) ) ) | !( ( (v.x>>8)%2 ) | ( (v.x>>63)%2 ) | ( (v.x>>44)%2 ) | ( (v.x>>20)%2 ) | ( (v.x>>19)%2 ) | ( (v.x>>6)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) ) & !( ( (v.x>>54)%2 ) ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) ) & !( ( (v.x>>57)%2 ) ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) & ( ( ( ( (v.y>>7)%2 ) & ( (v.x>>63)%2 ) ) ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.y>>17)%2 ) ) ) & !( ( (v.x>>59)%2 ) ) ) & !( ( (v.y>>18)%2 ) ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>22)%2 ) & ( ( ( ( (v.x>>56)%2 ) ) ) ) ) & !( ( (v.y>>18)%2 ) ) ) & !( ( (v.y>>17)%2 ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) ) | ( ( (v.y>>7)%2 ) ) | ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>19)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.x>>56)%2 ) ) ) & !( ( (v.y>>19)%2 ) ) ) & !( ( (v.x>>57)%2 ) ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>18)%2 ) & ( ( ( ( (v.x>>60)%2 ) ) ) ) ) & !( ( (v.y>>20)%2 ) ) ) & !( ( (v.y>>21)%2 ) ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.x>>56)%2 ) ) ) & !( ( (v.y>>19)%2 ) ) ) & !( ( (v.x>>57)%2 ) ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>17)%2 ) & ( ( ( ( (v.x>>62)%2 ) ) ) ) ) & !( ( (v.y>>20)%2 ) ) ) & !( ( (v.y>>22)%2 ) ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) )<<2;
        aux.y |=(unsigned long long) ( ( (v.x>>58)%2 ) )<<3;
        aux.y |=(unsigned long long) ( ( (v.x>>59)%2 ) )<<4;
        aux.y |=(unsigned long long) ( ( (v.x>>8)%2 ) ) | ( ( (v.x>>9)%2 ) )<<5;
        aux.y |=(unsigned long long) ( ( ( (v.x>>16)%2 ) ) & !( ( (v.x>>58)%2 ) ) )<<6;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_15(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) ) & !( ( (v.y>>7)%2 ) ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>8)%2 ) & ( ( ( ( (v.x>>42)%2 ) & ( (v.x>>46)%2 ) ) ) ) ) & !( ( (v.y>>9)%2 ) ) ) & !( ( (v.y>>10)%2 ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>63)%2 ) ) & !( ( (v.x>>27)%2 ) ) ) & !( ( (v.x>>7)%2 ) ) ) & !( ( (v.x>>3)%2 ) ) )<< 2 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>5)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>43)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) ) ) & !( ( (v.y>>7)%2 ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) ) | ( ( (v.x>>9)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( (v.x>>34)%2 ) ) | ( ( (v.y>>11)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>25)%2 ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>26)%2 ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) ) | ( ( (v.x>>41)%2 ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) ) | ( ( (v.x>>6)%2 ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.x>>36)%2 ) )<< 19 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>26)%2 ) ) | ( ( (v.x>>19)%2 ) & ( ( ( ( (v.x>>46)%2 ) ) ) ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>21)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<< 22 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>40)%2 ) ) ) ) | !( ( (v.x>>40)%2 ) | ( (v.x>>6)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) ) & !( ( (v.x>>23)%2 ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>56)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>1)%2 ) ) ) ) | !( ( (v.x>>1)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.y>>13)%2 ) & ( ( ( ( (v.x>>15)%2 ) ) ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>28)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.x>>57)%2 ) ) ) & !( ( (v.x>>56)%2 ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>59)%2 ) ) | ( ( (v.x>>58)%2 ) ) ) ) | !( ( (v.x>>58)%2 ) | ( (v.x>>59)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>47)%2 ) ) | ( ( (v.x>>19)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>32)%2 ) ) | ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) ) ) ) | !( ( (v.x>>32)%2 ) | ( (v.x>>56)%2 ) | ( (v.x>>57)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( !( ( ( (v.x>>58)%2 ) ) | ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) ) | ( ( (v.x>>59)%2 ) ) ) ) | !( ( (v.x>>56)%2 ) | ( (v.x>>58)%2 ) | ( (v.x>>57)%2 ) | ( (v.x>>59)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>51)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.y>>14)%2 ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.y>>14)%2 ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>41)%2 ) ) | ( ( (v.x>>1)%2 ) ) | ( ( (v.x>>19)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>15)%2 ) ) ) ) ) | ( ( (v.x>>53)%2 ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) ) & !( ( (v.y>>16)%2 ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>37)%2 ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.x>>9)%2 ) ) | ( ( (v.x>>8)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>44)%2 ) ) | ( ( (v.x>>6)%2 ) ) | ( ( (v.x>>19)%2 ) ) ) | !( ( (v.x>>8)%2 ) | ( (v.x>>63)%2 ) | ( (v.x>>44)%2 ) | ( (v.x>>20)%2 ) | ( (v.x>>19)%2 ) | ( (v.x>>6)%2 ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) ) & !( ( (v.x>>54)%2 ) ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) ) & !( ( (v.x>>57)%2 ) ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) & ( ( ( ( (v.y>>7)%2 ) & ( (v.x>>63)%2 ) ) ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.y>>17)%2 ) ) ) & !( ( (v.x>>59)%2 ) ) ) & !( ( (v.y>>18)%2 ) ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>22)%2 ) & ( ( ( ( (v.x>>56)%2 ) ) ) ) ) & !( ( (v.y>>18)%2 ) ) ) & !( ( (v.y>>17)%2 ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.x>>63)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) ) | ( ( (v.y>>7)%2 ) ) | ( ( (v.x>>63)%2 ) ) | ( ( (v.x>>19)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.x>>56)%2 ) ) ) & !( ( (v.y>>19)%2 ) ) ) & !( ( (v.x>>57)%2 ) ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>18)%2 ) & ( ( ( ( (v.x>>60)%2 ) ) ) ) ) & !( ( (v.y>>20)%2 ) ) ) & !( ( (v.y>>21)%2 ) ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>34)%2 ) ) ) ) ) & !( ( (v.x>>56)%2 ) ) ) & !( ( (v.y>>19)%2 ) ) ) & !( ( (v.x>>57)%2 ) ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>17)%2 ) & ( ( ( ( (v.x>>62)%2 ) ) ) ) ) & !( ( (v.y>>20)%2 ) ) ) & !( ( (v.y>>22)%2 ) ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) ) | ( ( (v.x>>57)%2 ) )<<2;
        aux.y |=(unsigned long long) ( ( (v.x>>58)%2 ) )<<3;
        aux.y |=(unsigned long long) ( ( (v.x>>59)%2 ) )<<4;
        aux.y |=(unsigned long long) ( ( (v.x>>8)%2 ) ) | ( ( (v.x>>9)%2 ) )<<5;
        aux.y |=(unsigned long long) ( ( ( (v.x>>16)%2 ) ) & !( ( (v.x>>58)%2 ) ) )<<6;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_15_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
        
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2 + ( (v.y>>7)%2 ) * -2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>42)%2 ) * 2 + ( (v.x>>46)%2 ) * 2 + 0 * -2 + ( (v.y>>10)%2 ) * -2) >= 6 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2 + ( (v.x>>27)%2 ) * -2 + ( (v.x>>7)%2 ) * -2 + ( (v.x>>3)%2 ) * -2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2 + ( (v.x>>43)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>10)%2 ) * 2 + ( (v.x>>63)%2 ) * 2 + ( (v.y>>7)%2 ) * -2) >= 4 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 2 + ( (v.x>>9)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2 + ( (v.y>>11)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>25)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>41)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>6)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 6 + ( (v.x>>19)%2 ) * 2 + ( (v.x>>46)%2 ) * 2) >= 4 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 4 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * -2 + ( (v.x>>40)%2 ) * -2) >= 0 ) <<23;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_15_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
        
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>23)%2 ) * -2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>56)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<27;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>15)%2 ) * 2) >= 4 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>57)%2 ) * -2 + ( (v.x>>56)%2 ) * -2) >= 4 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * -2 + ( (v.x>>58)%2 ) * -2) >= 0 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>19)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * -2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>57)%2 ) * -2) >= 0 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>58)%2 ) * -2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>57)%2 ) * -2 + ( (v.x>>59)%2 ) * -2) >= 0 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>51)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>1)%2 ) * 2 + ( (v.x>>19)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + 0 * 2 + ( (v.x>>53)%2 ) * 6) >= 4 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2 + ( (v.y>>16)%2 ) * -2) >= 2 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2) >= 2 ) <<47;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_15_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
        
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2 + ( (v.x>>8)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 6 + ( (v.x>>44)%2 ) * 6 + ( (v.x>>6)%2 ) * 6 + ( (v.x>>19)%2 ) * 6 + ( (v.x>>8)%2 ) * -2 + ( (v.x>>20)%2 ) * -2) >= 0 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.y>>6)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 2 + ( (v.x>>54)%2 ) * -2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.x>>57)%2 ) * -2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + 0 * 2 + ( (v.x>>63)%2 ) * 2) >= 6 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + 0 * -2 + ( (v.x>>59)%2 ) * -2 + ( (v.y>>18)%2 ) * -2) >= 4 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.x>>56)%2 ) * 2 + 0 * -2 + ( (v.y>>17)%2 ) * -2) >= 4 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2 + 0 * 2 + ( (v.x>>63)%2 ) * 2 + ( (v.x>>19)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + 0 * -2 + ( (v.x>>57)%2 ) * -2) >= 4 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>60)%2 ) * 2 + 0 * -2 + ( (v.y>>21)%2 ) * -2) >= 4 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + 0 * -2 + ( (v.x>>57)%2 ) * -2) >= 4 ) <<62;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>62)%2 ) * 2 + 0 * -2 + ( (v.y>>22)%2 ) * -2) >= 4 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<2;
        aux.y |=(unsigned long long) ( ( ( (v.x>>58)%2 ) * 2) >= 2 ) <<3;
        aux.y |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<4;
        aux.y |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 2 + ( (v.x>>9)%2 ) * 2) >= 2 ) <<5;
        aux.y |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<6;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_15(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2 + ( (v.y>>7)%2 ) * -2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>42)%2 ) * 2 + ( (v.x>>46)%2 ) * 2 + 0 * -2 + ( (v.y>>10)%2 ) * -2) >= 6 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2 + ( (v.x>>27)%2 ) * -2 + ( (v.x>>7)%2 ) * -2 + ( (v.x>>3)%2 ) * -2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>5)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2 + ( (v.x>>43)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>10)%2 ) * 2 + ( (v.x>>63)%2 ) * 2 + ( (v.y>>7)%2 ) * -2) >= 4 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 2 + ( (v.x>>9)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.x>>34)%2 ) * 2 + ( (v.y>>11)%2 ) * 2) >= 2 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>25)%2 ) * 2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 2 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>41)%2 ) * 2) >= 2 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>6)%2 ) * 2) >= 2 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.x>>36)%2 ) * 2) >= 2 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 6 + ( (v.x>>19)%2 ) * 2 + ( (v.x>>46)%2 ) * 2) >= 4 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>21)%2 ) * 2 + ( (v.x>>14)%2 ) * 2) >= 4 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * -2 + ( (v.x>>40)%2 ) * -2) >= 0 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>23)%2 ) * -2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>56)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<27;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>15)%2 ) * 2) >= 4 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>57)%2 ) * -2 + ( (v.x>>56)%2 ) * -2) >= 4 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * -2 + ( (v.x>>58)%2 ) * -2) >= 0 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>47)%2 ) * 2 + ( (v.x>>19)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>32)%2 ) * -2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>57)%2 ) * -2) >= 0 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>58)%2 ) * -2 + ( (v.x>>56)%2 ) * -2 + ( (v.x>>57)%2 ) * -2 + ( (v.x>>59)%2 ) * -2) >= 0 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>51)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.x>>1)%2 ) * 2 + ( (v.x>>19)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + 0 * 2 + ( (v.x>>53)%2 ) * 6) >= 4 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 2 + ( (v.y>>16)%2 ) * -2) >= 2 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2) >= 2 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2) >= 2 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.x>>9)%2 ) * 2 + ( (v.x>>8)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 6 + ( (v.x>>44)%2 ) * 6 + ( (v.x>>6)%2 ) * 6 + ( (v.x>>19)%2 ) * 6 + ( (v.x>>8)%2 ) * -2 + ( (v.x>>20)%2 ) * -2) >= 0 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.y>>6)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>26)%2 ) * 2 + ( (v.x>>54)%2 ) * -2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.x>>57)%2 ) * -2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + 0 * 2 + ( (v.x>>63)%2 ) * 2) >= 6 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + 0 * -2 + ( (v.x>>59)%2 ) * -2 + ( (v.y>>18)%2 ) * -2) >= 4 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.x>>56)%2 ) * 2 + 0 * -2 + ( (v.y>>17)%2 ) * -2) >= 4 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>63)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2 + 0 * 2 + ( (v.x>>63)%2 ) * 2 + ( (v.x>>19)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + 0 * -2 + ( (v.x>>57)%2 ) * -2) >= 4 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.x>>18)%2 ) * 2 + ( (v.x>>60)%2 ) * 2 + 0 * -2 + ( (v.y>>21)%2 ) * -2) >= 4 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>56)%2 ) * -2 + 0 * -2 + ( (v.x>>57)%2 ) * -2) >= 4 ) <<62;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>62)%2 ) * 2 + 0 * -2 + ( (v.y>>22)%2 ) * -2) >= 4 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 2 ) <<2;
        aux.y |=(unsigned long long) ( ( ( (v.x>>58)%2 ) * 2) >= 2 ) <<3;
        aux.y |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<4;
        aux.y |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 2 + ( (v.x>>9)%2 ) * 2) >= 2 ) <<5;
        aux.y |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + ( (v.x>>58)%2 ) * -2) >= 2 ) <<6;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 17
__global__ void passo_bool_17_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>51)%2 ) & ( (v.x>>40)%2 ) ) ) ) ) & ! ( ( (v.y>>13)%2 ) ) ) | ( ( ( (v.x>>49)%2 ) & ( ( ( ( (v.y>>12)%2 ) & ( (v.x>>40)%2 ) ) ) ) ) & ! ( ( (v.y>>13)%2 ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>1)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) )<< 3 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>1)%2 ) ) ) ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) ) ) & ! ( ( (v.y>>13)%2 ) ) ) | ( ( ( (v.x>>28)%2 ) ) & ! ( ( (v.y>>13)%2 ) ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) ) & ! ( ( (v.x>>25)%2 ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>33)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>6)%2 ) & ( ( ( ( (v.x>>56)%2 ) ) ) ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>18)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>21)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) ) | ! ( ( (v.y>>17)%2 ) | ( (v.y>>22)%2 ) | ( (v.y>>20)%2 ) | ( (v.y>>21)%2 ) | ( (v.y>>15)%2 ) | ( (v.y>>3)%2 ) | ( (v.y>>18)%2 ) | ( (v.y>>16)%2 ) | ( (v.y>>19)%2 ) | ( (v.x>>13)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>21)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>18)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>30)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) )<< 18 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_17_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>30)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>30)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>31)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>21)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>30)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>31)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>51)%2 ) ) | ( ( (v.x>>24)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>19)%2 ) ) | ( ( (v.x>>16)%2 ) ) | ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>15)%2 ) ) | ( ( (v.x>>17)%2 ) ) | ( ( (v.y>>0)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) & ( ( ( ( (v.x>>1)%2 ) ) ) ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>37)%2 ) & ( ( ( ( (v.x>>35)%2 ) ) ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) ) | ( ( (v.x>>58)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) ) | ( ( (v.x>>31)%2 ) ) | ( ( (v.x>>32)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 37 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_17_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) & ( ( ( ( (v.y>>32)%2 ) ) ) ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.y>>32)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>17)%2 ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) ) & ! ( ( (v.y>>33)%2 ) ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>36)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>44)%2 ) & ( (v.x>>39)%2 ) ) ) ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>12)%2 ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) | ( ( (v.x>>41)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>1)%2 ) & ( ( ( ( (v.x>>50)%2 ) ) ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>16)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) ) | ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) ) | ( ( (v.x>>20)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) ) | ( ( (v.y>>34)%2 ) & ( ( ( ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.x>>59)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>50)%2 ) ) & ! ( ( (v.y>>1)%2 ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>5)%2 ) & ( (v.x>>11)%2 ) ) ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.y>>35)%2 ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) ) & ! ( ( (v.x>>63)%2 ) ) )<< 56 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_17_parte4(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) ) | ( ( (v.y>>11)%2 ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) ) & ! ( ( (v.x>>2)%2 ) ) ) | ( ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) ) & ! ( ( (v.x>>2)%2 ) ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) ) & ! ( ( (v.x>>60)%2 ) ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.y>>4)%2 ) ) )<< 60 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.y>>10)%2 ) ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>19)%2 ) ) | ( ( (v.x>>16)%2 ) ) | ( ( (v.x>>15)%2 ) ) | ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>17)%2 ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.y>>38)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) )<<2;
        aux.y |=(unsigned long long) ( ( (v.y>>2)%2 ) )<<3;
        aux.y |=(unsigned long long) ( ( (v.x>>23)%2 ) )<<4;
        aux.y |=(unsigned long long) ( ( ( (v.y>>39)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.x>>47)%2 ) ) )<<5;
        aux.y |=(unsigned long long) ( ( (v.y>>39)%2 ) & ( ( ( ( (v.x>>51)%2 ) & ( (v.y>>37)%2 ) & ( (v.y>>34)%2 ) ) ) ) )<<6;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) )<<7;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) )<<8;
        aux.y |=(unsigned long long) ( ( (v.x>>19)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) ) | ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) )<<9;
        aux.y |= ! ( ( ( (v.x>>1)%2 ) ) )<<10;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.x>>51)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<<11;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}


unsigned long long confere_bool_17(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( (v.x>>29)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>51)%2 ) & ( (v.x>>40)%2 ) ) ) ) ) & ! ( ( (v.y>>13)%2 ) ) ) | ( ( ( (v.x>>49)%2 ) & ( ( ( ( (v.y>>12)%2 ) & ( (v.x>>40)%2 ) ) ) ) ) & ! ( ( (v.y>>13)%2 ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>1)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) )<< 3 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>1)%2 ) ) ) ) ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) ) ) & ! ( ( (v.y>>13)%2 ) ) ) | ( ( ( (v.x>>28)%2 ) ) & ! ( ( (v.y>>13)%2 ) ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) ) & ! ( ( (v.x>>25)%2 ) ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) ) | ( ( (v.x>>33)%2 ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>6)%2 ) & ( ( ( ( (v.x>>56)%2 ) ) ) ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>18)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>21)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) ) | ! ( ( (v.y>>17)%2 ) | ( (v.y>>22)%2 ) | ( (v.y>>20)%2 ) | ( (v.y>>21)%2 ) | ( (v.y>>15)%2 ) | ( (v.y>>3)%2 ) | ( (v.y>>18)%2 ) | ( (v.y>>16)%2 ) | ( (v.y>>19)%2 ) | ( (v.x>>13)%2 ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>21)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>18)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) & ! ( ( (v.y>>23)%2 ) ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>30)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) ) | ( ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) & ! ( ( (v.y>>3)%2 ) ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>23)%2 ) ) ) ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>30)%2 ) & ( (v.y>>23)%2 ) ) ) ) ) | ( ( (v.y>>30)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>31)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>20)%2 ) & ( ( ( ( (v.y>>16)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>22)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>21)%2 ) & ( ( ( ( (v.y>>23)%2 ) & ( (v.y>>29)%2 ) ) ) ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) ) | ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) ) ) ) ) & ! ( ( (v.y>>23)%2 ) ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.y>>24)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>17)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>30)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>19)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>31)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>29)%2 ) ) ) ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.x>>30)%2 ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>51)%2 ) ) | ( ( (v.x>>24)%2 ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>19)%2 ) ) | ( ( (v.x>>16)%2 ) ) | ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>15)%2 ) ) | ( ( (v.x>>17)%2 ) ) | ( ( (v.y>>0)%2 ) )<< 24 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.x>>46)%2 ) & ( ( ( ( (v.x>>1)%2 ) ) ) ) ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>53)%2 ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( (v.x>>37)%2 ) & ( ( ( ( (v.x>>35)%2 ) ) ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>48)%2 ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) ) | ( ( (v.x>>58)%2 ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.x>>43)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) ) | ( ( (v.x>>31)%2 ) ) | ( ( (v.x>>32)%2 ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.x>>57)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.x>>61)%2 ) & ( ( ( ( (v.y>>32)%2 ) ) ) ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.y>>32)%2 ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>17)%2 ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) ) & ! ( ( (v.y>>33)%2 ) ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>35)%2 ) ) | ( ( (v.x>>34)%2 ) ) | ( ( (v.x>>36)%2 ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>28)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>44)%2 ) & ( (v.x>>39)%2 ) ) ) ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>12)%2 ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( (v.x>>46)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) | ( ( (v.x>>41)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>1)%2 ) & ( ( ( ( (v.x>>50)%2 ) ) ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>16)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) ) | ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) ) | ( ( (v.x>>20)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) ) | ( ( (v.y>>34)%2 ) & ( ( ( ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.x>>59)%2 ) & ( ( ( ( (v.y>>34)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>50)%2 ) ) & ! ( ( (v.y>>1)%2 ) ) ) & ! ( ( (v.x>>55)%2 ) ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>5)%2 ) & ( (v.x>>11)%2 ) ) ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.x>>4)%2 ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( (v.y>>35)%2 ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) ) & ! ( ( (v.x>>63)%2 ) ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( (v.y>>6)%2 ) ) | ( ( (v.y>>11)%2 ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) ) & ! ( ( (v.x>>2)%2 ) ) ) | ( ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) ) ) ) & ! ( ( (v.x>>2)%2 ) ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) ) & ! ( ( (v.x>>60)%2 ) ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.y>>4)%2 ) ) )<< 60 ;
        aux.x |=(unsigned long long) ! ( ( ( (v.y>>10)%2 ) ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( (v.x>>59)%2 ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>21)%2 ) ) | ( ( (v.x>>19)%2 ) ) | ( ( (v.x>>16)%2 ) ) | ( ( (v.x>>15)%2 ) ) | ( ( (v.x>>18)%2 ) ) | ( ( (v.x>>20)%2 ) ) | ( ( (v.x>>17)%2 ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.y>>38)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) )<<2;
        aux.y |=(unsigned long long) ( ( (v.y>>2)%2 ) )<<3;
        aux.y |=(unsigned long long) ( ( (v.x>>23)%2 ) )<<4;
        aux.y |=(unsigned long long) ( ( ( (v.y>>39)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.x>>47)%2 ) ) )<<5;
        aux.y |=(unsigned long long) ( ( (v.y>>39)%2 ) & ( ( ( ( (v.x>>51)%2 ) & ( (v.y>>37)%2 ) & ( (v.y>>34)%2 ) ) ) ) )<<6;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) )<<7;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) )<<8;
        aux.y |=(unsigned long long) ( ( (v.x>>19)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) ) | ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>36)%2 ) ) ) ) )<<9;
        aux.y |= ! ( ( ( (v.x>>1)%2 ) ) )<<10;
        aux.y |=(unsigned long long) ( ( (v.x>>14)%2 ) & ( ( ( ( (v.x>>51)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) )<<11;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_17_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( 0 * 6 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>40)%2 ) * 6 + 0 * -6 + ( (v.x>>49)%2 ) * 2) >= 14 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2 + ( (v.x>>8)%2 ) * 2) >= 4 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * -2 + ( (v.x>>1)%2 ) * -2) >= -2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2 + ( (v.x>>22)%2 ) * 2 + 0 * -10 + ( (v.x>>28)%2 ) * 6) >= 4 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + ( (v.x>>25)%2 ) * -2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2 + ( (v.x>>33)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + ( (v.y>>14)%2 ) * -2) >= 2 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>56)%2 ) * 2) >= 4 ) <<13;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 506 + ( (v.y>>3)%2 ) * -510 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.x>>13)%2 ) * -2) >= 0 ) <<14;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 126 + 0 * 126 + ( (v.y>>3)%2 ) * -126 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>22)%2 ) * 2) >= 254 ) <<15;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 766 + 0 * 766 + ( (v.y>>3)%2 ) * -766 + 0 * -254 + 0 * 2 + 0 * 2 + 0 * 258 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 1534 ) <<16;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 510 + 0 * 510 + 0 * -510 + ( (v.y>>3)%2 ) * -510 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 1022 ) <<17;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 126 + 0 * 126 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 254 ) <<18;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_17_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( 0 * 12 + 0 * 16372 + 0 * 16372 + 0 * 12 + 0 * 12 + 0 * 4096 + 0 * 4108 + 0 * 12 + 0 * 4100 + 0 * 4100 + 0 * 12 + 0 * 12 + 0 * 12 + 0 * 12 + ( (v.y>>21)%2 ) * 12) >= 24572 ) <<19;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 30 + 0 * 30 + 0 * -30 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 62 ) <<20;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 254 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 256 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>24)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>21)%2 ) * 2 + ( (v.x>>19)%2 ) * 2 + ( (v.x>>16)%2 ) * 2 + ( (v.x>>18)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>17)%2 ) * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2 + ( (v.x>>1)%2 ) * -2) >= -2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>35)%2 ) * 2) >= 4 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2 + ( (v.x>>58)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2 + ( (v.x>>31)%2 ) * 2 + ( (v.x>>32)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<37;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_17_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
        
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.y>>32)%2 ) * 2) >= 4 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.y>>32)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>21)%2 ) * 2 + ( (v.x>>17)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.y>>33)%2 ) * -2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>36)%2 ) * 2) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>39)%2 ) * 2) >= 6 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.y>>12)%2 ) * 2) >= 4 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>57)%2 ) * 6 + ( (v.x>>41)%2 ) * 2) >= 8 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.y>>1)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 4 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + 0 * 62 + ( (v.x>>18)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>23)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 64 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + ( (v.y>>1)%2 ) * -2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>5)%2 ) * 2 + ( (v.x>>11)%2 ) * 2) >= 6 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.y>>35)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2 + ( (v.x>>63)%2 ) * -2) >= 2 ) <<56;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_17_parte4(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
        
        aux.x |=(unsigned long long) ( ( ( (v.y>>6)%2 ) * 2 + ( (v.y>>11)%2 ) * 2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 6 + 0 * 2 + ( (v.x>>2)%2 ) * -6 + ( (v.x>>48)%2 ) * 2) >= 8 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2 + ( (v.x>>60)%2 ) * -2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.y>>4)%2 ) * -2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.y>>10)%2 ) * -2) >= 0 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<62;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>14)%2 ) * 2) >= 4 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>21)%2 ) * 2 + ( (v.x>>19)%2 ) * 2 + ( (v.x>>16)%2 ) * 2 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>18)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>17)%2 ) * 2) >= 2 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.y>>38)%2 ) * 2) >= 2 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<2;
        aux.y |=(unsigned long long) ( ( ( (v.y>>2)%2 ) * 2) >= 2 ) <<3;
        aux.y |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2) >= 2 ) <<4;
        aux.y |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>47)%2 ) * -2) >= 4 ) <<5;
        aux.y |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>51)%2 ) * 2 + 0 * 2 + ( (v.y>>34)%2 ) * 2) >= 8 ) <<6;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.y>>36)%2 ) * 2) >= 4 ) <<7;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.y>>36)%2 ) * 2) >= 4 ) <<8;
        aux.y |=(unsigned long long) ( ( ( (v.x>>19)%2 ) * 2 + 0 * 6 + ( (v.x>>14)%2 ) * 2) >= 8 ) <<9;
        aux.y |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<10;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 6 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>49)%2 ) * 2) >= 8 ) <<11;
    
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}


unsigned long long confere_tlf_17(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>29)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( 0 * 6 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>40)%2 ) * 6 + 0 * -6 + ( (v.x>>49)%2 ) * 2) >= 14 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2 + ( (v.x>>8)%2 ) * 2) >= 4 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * -2 + ( (v.x>>1)%2 ) * -2) >= -2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2) >= 2 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2 + ( (v.x>>22)%2 ) * 2 + 0 * -10 + ( (v.x>>28)%2 ) * 6) >= 4 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + ( (v.x>>25)%2 ) * -2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2 + ( (v.x>>33)%2 ) * 2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + ( (v.y>>14)%2 ) * -2) >= 2 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>6)%2 ) * 2 + ( (v.x>>56)%2 ) * 2) >= 4 ) <<13;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 506 + ( (v.y>>3)%2 ) * -510 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.x>>13)%2 ) * -2) >= 0 ) <<14;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 126 + 0 * 126 + ( (v.y>>3)%2 ) * -126 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>22)%2 ) * 2) >= 254 ) <<15;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 766 + 0 * 766 + ( (v.y>>3)%2 ) * -766 + 0 * -254 + 0 * 2 + 0 * 2 + 0 * 258 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 1534 ) <<16;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 510 + 0 * 510 + 0 * -510 + ( (v.y>>3)%2 ) * -510 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 1022 ) <<17;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 126 + 0 * 126 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 254 ) <<18;
        aux.x |=(unsigned long long) ( ( 0 * 12 + 0 * 16372 + 0 * 16372 + 0 * 12 + 0 * 12 + 0 * 4096 + 0 * 4108 + 0 * 12 + 0 * 4100 + 0 * 4100 + 0 * 12 + 0 * 12 + 0 * 12 + 0 * 12 + ( (v.y>>21)%2 ) * 12) >= 24572 ) <<19;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 30 + 0 * 30 + 0 * -30 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 62 ) <<20;
        aux.x |=(unsigned long long) ( ( 0 * 2 + 0 * 254 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + 0 * 2 + ( (v.y>>27)%2 ) * 2) >= 256 ) <<21;
        aux.x |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 2) >= 2 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>24)%2 ) * 2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>21)%2 ) * 2 + ( (v.x>>19)%2 ) * 2 + ( (v.x>>16)%2 ) * 2 + ( (v.x>>18)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>17)%2 ) * 2 + ( (v.y>>0)%2 ) * 2) >= 2 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * -2 + ( (v.x>>1)%2 ) * -2) >= -2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 2) >= 2 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>53)%2 ) * 2) >= 2 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 2 + ( (v.x>>35)%2 ) * 2) >= 4 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>48)%2 ) * 2) >= 2 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2 + ( (v.x>>58)%2 ) * 2) >= 2 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.x>>43)%2 ) * 2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2 + ( (v.x>>31)%2 ) * 2 + ( (v.x>>32)%2 ) * 2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 2) >= 2 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.x>>57)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 2 + ( (v.y>>32)%2 ) * 2) >= 4 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.y>>32)%2 ) * 2) >= 2 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>21)%2 ) * 2 + ( (v.x>>17)%2 ) * 2) >= 2 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.y>>33)%2 ) * -2) >= 2 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>35)%2 ) * 2 + ( (v.x>>34)%2 ) * 2 + ( (v.x>>36)%2 ) * 2) >= 2 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<44;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>44)%2 ) * 2 + ( (v.x>>39)%2 ) * 2) >= 6 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 2 + ( (v.y>>12)%2 ) * 2) >= 4 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>46)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 4 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.x>>57)%2 ) * 6 + ( (v.x>>41)%2 ) * 2) >= 8 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.y>>1)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>55)%2 ) * -2) >= 4 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + 0 * 62 + ( (v.x>>18)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>23)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 64 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>50)%2 ) * 2 + ( (v.y>>1)%2 ) * -2 + ( (v.x>>55)%2 ) * -2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>5)%2 ) * 2 + ( (v.x>>11)%2 ) * 2) >= 6 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.x>>4)%2 ) * 2) >= 2 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.y>>35)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>62)%2 ) * 2 + ( (v.x>>63)%2 ) * -2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.y>>6)%2 ) * 2 + ( (v.y>>11)%2 ) * 2) >= 2 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 6 + 0 * 2 + ( (v.x>>2)%2 ) * -6 + ( (v.x>>48)%2 ) * 2) >= 8 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2 + ( (v.x>>60)%2 ) * -2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.y>>4)%2 ) * -2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.y>>10)%2 ) * -2) >= 0 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>59)%2 ) * 2) >= 2 ) <<62;
        aux.x |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>14)%2 ) * 2) >= 4 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>21)%2 ) * 2 + ( (v.x>>19)%2 ) * 2 + ( (v.x>>16)%2 ) * 2 + ( (v.x>>15)%2 ) * 2 + ( (v.x>>18)%2 ) * 2 + ( (v.x>>20)%2 ) * 2 + ( (v.x>>17)%2 ) * 2) >= 2 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.y>>38)%2 ) * 2) >= 2 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<2;
        aux.y |=(unsigned long long) ( ( ( (v.y>>2)%2 ) * 2) >= 2 ) <<3;
        aux.y |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2) >= 2 ) <<4;
        aux.y |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>47)%2 ) * -2) >= 4 ) <<5;
        aux.y |=(unsigned long long) ( ( 0 * 2 + ( (v.x>>51)%2 ) * 2 + 0 * 2 + ( (v.y>>34)%2 ) * 2) >= 8 ) <<6;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.y>>36)%2 ) * 2) >= 4 ) <<7;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.y>>36)%2 ) * 2) >= 4 ) <<8;
        aux.y |=(unsigned long long) ( ( ( (v.x>>19)%2 ) * 2 + 0 * 6 + ( (v.x>>14)%2 ) * 2) >= 8 ) <<9;
        aux.y |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * -2) >= 0 ) <<10;
        aux.y |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 6 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>49)%2 ) * 2) >= 8 ) <<11;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 18
__global__ void passo_bool_18_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( (v.y>>10)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>42)%2 ) & ( ( ( ( (v.z>>2)%2 ) ) & ( ( ( ( (v.x>>33)%2 ) ) ) & ( ( ( (v.x>>31)%2 ) ) ) ) ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) ) | ( ( (v.y>>4)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.y>>53)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) & ( ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) & ( ( ( (v.x>>39)%2 ) & ( (v.y>>45)%2 ) ) ) & ( ( ! ( (v.x>>6)%2 ) ) ) ) ) | ( ( (v.x>>6)%2 ) & ( ( ( ! ( (v.y>>14)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) & ( ( ( ( (v.x>>39)%2 ) & ( (v.y>>45)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) & ( ( ! ( (v.x>>6)%2 ) ) ) ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.y>>55)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.y>>20)%2 ) ) | ( ( (v.y>>21)%2 ) ) | ( ( (v.y>>19)%2 ) ) | ( ( (v.y>>22)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) ) & ! ( ( (v.z>>3)%2 ) ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>10)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>11)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) ) & ! ( ( (v.y>>41)%2 ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) & ( ( ( ! ( (v.y>>38)%2 ) ) ) & ( ( ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.y>>23)%2 ) & ( ( ( ! ( (v.y>>26)%2 ) & ! ( (v.x>>16)%2 ) ) ) & ( ( ( (v.x>>33)%2 ) & ( (v.x>>61)%2 ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) & ( ( ( ( (v.x>>24)%2 ) | ( (v.y>>45)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) & ( ( ( ( (v.x>>33)%2 ) | ( (v.y>>7)%2 ) | ( (v.x>>32)%2 ) | ( (v.x>>34)%2 ) ) ) | ( ( ! ( (v.y>>41)%2 ) & ! ( (v.x>>33)%2 ) & ! ( (v.y>>7)%2 ) & ! ( (v.x>>32)%2 ) & ! ( (v.x>>34)%2 ) ) ) ) ) | ( ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) & ( (v.x>>15)%2 ) ) ) ) ) & ! ( ( (v.y>>41)%2 ) ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.x>>20)%2 ) & ( ( ( ( (v.x>>19)%2 ) ) ) ) ) ) | ( ( ( (v.y>>12)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.x>>20)%2 ) & ( ( ( ( (v.x>>19)%2 ) ) ) ) ) ) | ( ( (v.x>>19)%2 ) & ( ( ( ! ( (v.x>>20)%2 ) ) ) ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.y>>8)%2 ) & ( ( ( ( (v.x>>19)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>60)%2 ) & ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>22)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.z>>4)%2 ) & ( ( ( ! ( (v.y>>8)%2 ) ) ) ) ) | ( ( (v.y>>58)%2 ) & ( ( ( ( (v.y>>8)%2 ) & ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.y>>57)%2 ) & ( ( ( ( (v.y>>8)%2 ) & ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.y>>56)%2 ) & ( ( ( ( (v.y>>8)%2 ) & ( (v.x>>10)%2 ) ) ) ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) ) | ( ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) ) | ( ( ( (v.x>>42)%2 ) & ( ( ( ( (v.y>>49)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.y>>52)%2 ) )<< 25 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_18_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>33)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>34)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>32)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>35)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ! ( (v.y>>61)%2 ) ) ) & ( ( ( (v.x>>32)%2 ) ) ) ) ) | ( ( (v.y>>56)%2 ) & ( ( ( ! ( (v.x>>32)%2 ) & ! ( (v.x>>27)%2 ) ) ) ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) & ( ( ( ( (v.y>>5)%2 ) & ( (v.x>>22)%2 ) ) ) ) ) & ! ( ( (v.y>>41)%2 ) ) ) | ( ( ( (v.x>>37)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) & ( ( ( ! ( (v.x>>28)%2 ) ) ) ) ) ) ) & ! ( ( (v.y>>41)%2 ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) & ( ( ( ! ( (v.y>>24)%2 ) ) ) & ( ( ( (v.x>>29)%2 ) ) ) ) ) | ( ( (v.y>>57)%2 ) & ( ( ( ! ( (v.x>>33)%2 ) & ! ( (v.x>>29)%2 ) ) ) ) ) | ( ( (v.y>>7)%2 ) & ( ( ( ! ( (v.x>>29)%2 ) ) ) & ( ( ! ( (v.y>>59)%2 ) ) ) & ( ( ( (v.z>>5)%2 ) ) ) & ( ( ! ( (v.x>>33)%2 ) ) ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.y>>58)%2 ) & ( ( ( ! ( (v.x>>30)%2 ) ) & ( ( ( ! ( (v.x>>34)%2 ) ) ) ) ) ) ) | ( ( (v.x>>30)%2 ) & ( ( ( ! ( (v.y>>24)%2 ) & ! ( (v.y>>11)%2 ) ) ) & ( ( ( (v.x>>34)%2 ) ) ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.y>>59)%2 ) & ( ( ( ! ( (v.x>>35)%2 ) ) ) & ( ( ! ( (v.y>>7)%2 ) ) ) & ( ( ! ( (v.x>>31)%2 ) ) ) ) ) | ( ( (v.x>>35)%2 ) & ( ( ( ! ( (v.y>>24)%2 ) ) ) & ( ( ( (v.x>>31)%2 ) ) ) ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) ) | ( ( (v.y>>56)%2 ) & ( ( ( ! ( (v.x>>27)%2 ) ) ) & ( ( ! ( (v.x>>32)%2 ) ) ) ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.y>>57)%2 ) & ( ( ( ! ( (v.x>>29)%2 ) ) ) & ( ( ! ( (v.x>>33)%2 ) ) ) ) ) | ( ( (v.x>>29)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.y>>58)%2 ) & ( ( ( ! ( (v.x>>30)%2 ) ) ) & ( ( ! ( (v.x>>34)%2 ) ) ) ) ) | ( ( (v.x>>30)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) ) | ( ( (v.y>>59)%2 ) & ( ( ( ! ( (v.x>>35)%2 ) ) ) & ( ( ! ( (v.x>>31)%2 ) ) ) ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) | ( ( (v.x>>22)%2 ) ) | ( ( (v.y>>43)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.z>>6)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.y>>11)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) | ( ( (v.y>>12)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>33)%2 ) ) & ! ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) ) ) ) ) ) & ! ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ( (v.x>>10)%2 ) ) ) ) ) ) & ! ( ( (v.x>>41)%2 ) & ( ( ( ( (v.x>>11)%2 ) & ( (v.x>>33)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) & ! ( (v.y>>14)%2 ) & ! ( (v.y>>7)%2 ) & ! ( (v.x>>10)%2 ) ) ) ) ) ) | ( ( ( ( (v.x>>40)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) & ! ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) ) ) ) ) ) & ! ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ( (v.x>>10)%2 ) ) ) ) ) ) | ( ( ( ( ( (v.y>>7)%2 ) ) & ! ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) ) ) ) ) ) & ! ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ( (v.x>>10)%2 ) ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) ) ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.y>>49)%2 ) & ( ( ( ( (v.z>>2)%2 ) ) ) & ( ( ! ( (v.x>>39)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ! ( (v.x>>39)%2 ) & ! ( (v.z>>2)%2 ) & ! ( (v.y>>49)%2 ) & ! ( (v.y>>14)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.y>>14)%2 ) & ( ( ( ! ( (v.x>>42)%2 ) ) ) & ( ( ( (v.x>>39)%2 ) & ( (v.z>>2)%2 ) & ( (v.y>>49)%2 ) ) ) ) ) | ( ( (v.x>>42)%2 ) & ( ( ( ! ( (v.x>>39)%2 ) & ! ( (v.y>>45)%2 ) ) ) ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>50)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>46)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>55)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>47)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>46)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>47)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>50)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>53)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>55)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.z>>0)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) | ( ( (v.x>>23)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) | ( ( (v.y>>40)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>11)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) & ( ( ! ( (v.y>>7)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) & ! ( ( (v.x>>61)%2 ) ) ) | ( ( ( ( (v.x>>23)%2 ) & ( ( ( ! ( (v.y>>7)%2 ) ) ) & ( ( ! ( (v.x>>61)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) & ! ( ( (v.x>>61)%2 ) ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( ( ( ( ( ( (v.y>>7)%2 ) & ( ( ( ( (v.y>>39)%2 ) ) ) ) ) & ! ( ( (v.x>>39)%2 ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.x>>61)%2 ) ) ) & ! ( ( (v.y>>27)%2 ) ) ) ) | ! ( ( (v.x>>39)%2 ) | ( (v.y>>8)%2 ) | ( (v.y>>39)%2 ) | ( (v.y>>7)%2 ) | ( (v.y>>27)%2 ) | ( (v.x>>61)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.y>>26)%2 ) ) | ( ( (v.x>>16)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>16)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) | ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) ) | ( ( (v.y>>26)%2 ) ) | ( ( (v.x>>16)%2 ) )<< 51 ;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_18_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) & ( ( ( ( (v.y>>51)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.y>>51)%2 ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.x>>53)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.x>>54)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>51)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.x>>55)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ( (v.y>>43)%2 ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) | ( ( (v.x>>58)%2 ) & ( ( ( ( (v.y>>43)%2 ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) | ( ( (v.x>>37)%2 ) & ( ( ( ( (v.y>>43)%2 ) ) ) ) ) | ( ( (v.y>>52)%2 ) ) | ( ( (v.y>>26)%2 ) ) | ( ( (v.y>>35)%2 ) ) | ( ( (v.x>>25)%2 ) ) | ( ( (v.x>>16)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>54)%2 ) ) ) & ( ( ( (v.x>>22)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) & ( ( ! ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) & ( ( ! ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.x>>37)%2 ) & ( ( ( ! ( (v.x>>54)%2 ) ) ) & ( ( ( (v.x>>22)%2 ) ) ) ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>52)%2 ) ) & ! ( ( (v.x>>28)%2 ) ) ) | ( ( ( (v.y>>26)%2 ) ) & ! ( ( (v.x>>28)%2 ) ) ) | ( ( ( (v.x>>38)%2 ) ) & ! ( ( (v.x>>28)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>16)%2 ) ) | ( ( (v.y>>26)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) & ( ( ( ! ( (v.x>>57)%2 ) ) ) | ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>47)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) ) & ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>61)%2 ) & ( ( ( ! ( (v.x>>57)%2 ) ) ) | ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.y>>39)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) ) ) | ( ( ! ( (v.x>>57)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>47)%2 ) ) ) & ( ( ! ( (v.x>>57)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>57)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.y>>46)%2 ) ) | ( ( (v.x>>59)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.y>>13)%2 ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) & ( ( ( ! ( (v.x>>6)%2 ) ) ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) | ( ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>61)%2 ) ) & ( ( ( ( (v.y>>26)%2 ) | ( (v.x>>16)%2 ) ) ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) & ! ( ( (v.y>>16)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) & ( ( ( ! ( (v.x>>6)%2 ) ) ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.y>>7)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( (v.z>>1)%2 ) ) | ( ( (v.y>>45)%2 ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.x>>33)%2 ) ) | ( ( (v.x>>28)%2 ) ) | ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>17)%2 ) ) | ( ( (v.y>>35)%2 ) ) | ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) ) | ( ( (v.y>>8)%2 ) ) | ( ( (v.x>>3)%2 ) ) | ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>30)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>60)%2 ) ) | ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>49)%2 ) ) ) ) ) | ( ( (v.y>>39)%2 ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ! ( (v.y>>2)%2 ) ) ) & ( ( ! ( (v.y>>49)%2 ) ) ) & ( ( ( (v.x>>24)%2 ) ) ) ) ) | ( ( (v.x>>3)%2 ) )<<2;
        aux.y |=(unsigned long long) ( ( (v.y>>3)%2 ) & ( ( ( ! ( (v.y>>15)%2 ) ) ) & ( ( ! ( (v.y>>2)%2 ) ) ) ) ) | ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) ) ) & ( ( ( (v.y>>0)%2 ) ) ) ) )<<3;
        aux.y |=(unsigned long long) ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) | ( ( (v.y>>1)%2 ) & ( ( ( ( (v.y>>2)%2 ) ) ) ) ) | ( ( (v.y>>4)%2 ) )<<4;
        aux.y |=(unsigned long long) ( ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>3)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) )<<5;
        aux.y |=(unsigned long long) ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>3)%2 ) ) ) & ( ( ! ( (v.y>>6)%2 ) ) ) ) ) & ! ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) & ! ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>6)%2 ) & ( ( ( ! ( (v.y>>0)%2 ) ) ) & ( ( ! ( (v.y>>2)%2 ) ) ) ) ) & ! ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) & ! ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.y>>6)%2 ) ) ) ) ) & ! ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) & ! ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) )<<6;
        aux.y |=(unsigned long long) ( ( ( (v.y>>7)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>7)%2 ) ) ) ) ) ) | ( ( ( (v.x>>63)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>7)%2 ) ) ) ) ) )<<7;
        aux.y |=(unsigned long long) ( ( ( (v.x>>0)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) & ( ( ( (v.y>>9)%2 ) ) ) ) ) & ! ( ( (v.y>>53)%2 ) & ( ( ( ( (v.y>>8)%2 ) ) ) ) ) ) | ( ( ( (v.y>>8)%2 ) & ( ( ( ! ( (v.y>>14)%2 ) ) ) & ( ( ! ( (v.y>>53)%2 ) ) ) ) ) & ! ( ( (v.y>>53)%2 ) & ( ( ( ( (v.y>>8)%2 ) ) ) ) ) ) | ( ( ( (v.x>>19)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) & ( ( ( (v.y>>9)%2 ) ) ) ) ) & ! ( ( (v.y>>53)%2 ) & ( ( ( ( (v.y>>8)%2 ) ) ) ) ) )<<8;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) & ( ( ( ! ( (v.y>>9)%2 ) ) ) & ( ( ( (v.x>>63)%2 ) ) ) ) ) | ( ( (v.y>>9)%2 ) & ( ( ( ! ( (v.y>>8)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) & ( ( ( ! ( (v.y>>8)%2 ) ) ) ) )<<9;
        aux.y |=(unsigned long long) ( ( (v.y>>4)%2 ) & ( ( ( ( (v.x>>23)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) | ( ( (v.y>>5)%2 ) & ( ( ( ( (v.x>>12)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) ) ) ) ) | ( ( (v.x>>12)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>23)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) )<<10;
        aux.y |=(unsigned long long) ( ( (v.x>>30)%2 ) ) | ( ( ( (v.x>>33)%2 ) & ( ( ( ( (v.y>>11)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) & ( ( ( ! ( (v.x>>30)%2 ) ) ) ) ) )<<11;
        aux.y |=(unsigned long long) ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) & ( (v.y>>5)%2 ) ) ) ) ) | ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) & ( ( ! ( (v.y>>5)%2 ) & ! ( (v.x>>0)%2 ) & ! ( (v.x>>22)%2 ) & ! ( (v.x>>60)%2 ) ) ) ) ) | ( ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) & ! ( ( (v.x>>60)%2 ) & ( ( ( ! ( (v.x>>24)%2 ) & ! ( (v.y>>45)%2 ) ) ) & ( ( ( (v.x>>0)%2 ) ) ) ) ) )<<12;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>4)%2 ) ) ) | ( ( ( (v.y>>5)%2 ) ) ) ) ) ) ) | ( ( (v.y>>8)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>4)%2 ) ) ) | ( ( ( (v.y>>5)%2 ) ) ) ) ) ) ) | ( ( (v.x>>3)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) | ( ( ( (v.y>>4)%2 ) ) ) ) ) | ( ( (v.y>>26)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>5)%2 ) ) ) | ( ( ( (v.y>>4)%2 ) ) ) ) ) ) ) | ( ( (v.x>>16)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>5)%2 ) ) ) | ( ( ( (v.y>>4)%2 ) ) ) ) ) ) )<<13;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_18_parte4(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.y |=(unsigned long long) ( ! ( ( ( (v.x>>22)%2 ) ) ) ) | ! ( ( (v.x>>22)%2 ) )<<14;
        aux.y |=(unsigned long long) ( ( ( (v.z>>7)%2 ) ) & ! ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>15)%2 ) ) ) ) ) ) | ( ( ( (v.y>>23)%2 ) & ( ( ( ( (v.y>>0)%2 ) ) ) & ( ( ( (v.y>>37)%2 ) ) ) & ( ( ( (v.x>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>15)%2 ) ) ) ) ) )<<15;
        aux.y |=(unsigned long long) ( ! ( ( ( (v.z>>7)%2 ) ) | ( ( (v.x>>22)%2 ) & ( ( ( ( (v.z>>4)%2 ) ) ) ) ) ) ) | ! ( ( (v.x>>22)%2 ) | ( (v.z>>7)%2 ) | ( (v.z>>4)%2 ) )<<16;
        aux.y |=(unsigned long long) ( ( ( ( (v.x>>42)%2 ) & ( ( ( ( (v.z>>2)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>7)%2 ) ) )<<17;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) )<<18;
        aux.y |=(unsigned long long) ( ( (v.y>>56)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<19;
        aux.y |=(unsigned long long) ( ( (v.y>>57)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<20;
        aux.y |=(unsigned long long) ( ( (v.y>>58)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<21;
        aux.y |=(unsigned long long) ( ( (v.y>>59)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<22;
        aux.y |=(unsigned long long) ( ( (v.y>>3)%2 ) & ( ( ( ( (v.x>>9)%2 ) ) ) ) ) | ( ( (v.y>>5)%2 ) & ( ( ( ( (v.x>>9)%2 ) ) ) ) )<<23;
        aux.y |=(unsigned long long) ( ( (v.x>>11)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) )<<24;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) )<<25;
        aux.y |=(unsigned long long) ( ( ( ( ( (v.y>>36)%2 ) & ( ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) & ! ( ( (v.y>>32)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>63)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) ) | ( ( ( (v.y>>23)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) & ( ( ( ! ( (v.y>>38)%2 ) ) ) & ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) & ( ( ! ( (v.y>>26)%2 ) ) ) & ( ( ! ( (v.x>>21)%2 ) & ! ( (v.y>>36)%2 ) & ! ( (v.y>>50)%2 ) ) ) & ( ( ( (v.x>>16)%2 ) ) ) ) ) | ( ( ! ( (v.x>>33)%2 ) ) & ( ( ( ( (v.x>>16)%2 ) ) ) & ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) & ( ( ! ( (v.y>>26)%2 ) ) ) ) ) | ( ( ( (v.x>>33)%2 ) & ( (v.x>>61)%2 ) ) & ( ( ( ! ( (v.y>>26)%2 ) & ! ( (v.x>>16)%2 ) ) ) & ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) ) | ( ( ( ( ( (v.y>>50)%2 ) & ( ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) & ! ( ( (v.y>>32)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>63)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) ) | ( ( ( ( ( (v.x>>21)%2 ) & ( ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) & ! ( ( (v.y>>32)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>63)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) )<<26;
        aux.y |=(unsigned long long) ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) & ( (v.x>>61)%2 ) ) ) & ( ( ! ( (v.y>>27)%2 ) ) ) ) ) | ( ( (v.y>>35)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ! ( (v.y>>7)%2 ) & ! ( (v.x>>6)%2 ) & ! ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ! ( (v.y>>35)%2 ) & ! ( (v.x>>6)%2 ) & ! ( (v.x>>23)%2 ) ) ) & ( ( ( (v.y>>27)%2 ) ) ) ) )<<27;
        aux.y |=(unsigned long long) ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>30)%2 ) ) ) & ( ( ! ( (v.y>>28)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ! ( (v.y>>29)%2 ) ) ) ) )<<28;
        aux.y |=(unsigned long long) ( ( (v.y>>29)%2 ) & ( ( ( ! ( (v.y>>27)%2 ) ) ) ) ) | ( ( (v.y>>35)%2 ) & ( ( ( ! ( (v.y>>29)%2 ) ) ) & ( ( ( (v.y>>28)%2 ) ) ) ) )<<29;
        aux.y |=(unsigned long long) ( ( ( (v.y>>30)%2 ) & ( ( ( ! ( (v.y>>28)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ! ( (v.y>>27)%2 ) & ! ( (v.y>>30)%2 ) ) ) ) ) ) | ! ( ( (v.y>>27)%2 ) | ( (v.y>>28)%2 ) | ( (v.y>>30)%2 ) )<<30;
        aux.y |=(unsigned long long) ( ( (v.x>>11)%2 ) ) | ( ( (v.y>>33)%2 ) ) | ( ( (v.x>>2)%2 ) )<<31;
        aux.y |=(unsigned long long) ( ( (v.y>>31)%2 ) )<<32;
        aux.y |=(unsigned long long) ( ( ( ( (v.y>>59)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>57)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>58)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>56)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) )<<33;
        aux.y |=(unsigned long long) ( ( (v.y>>7)%2 ) & ( ( ( ( (v.y>>45)%2 ) & ( (v.y>>60)%2 ) ) ) & ( ( ! ( (v.y>>34)%2 ) | ! ( (v.x>>29)%2 ) ) ) ) ) | ( ( (v.x>>12)%2 ) & ( ( ( ( (v.y>>45)%2 ) & ( (v.y>>60)%2 ) ) ) & ( ( ! ( (v.y>>34)%2 ) | ! ( (v.x>>29)%2 ) ) ) ) )<<34;
        aux.y |=(unsigned long long) ( ( (v.y>>36)%2 ) ) | ( ( (v.y>>41)%2 ) ) | ( ( (v.y>>44)%2 ) )<<35;
        aux.y |=(unsigned long long) ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>16)%2 ) ) ) ) ) | ( ( (v.x>>19)%2 ) & ( ( ( ( (v.x>>16)%2 ) ) ) ) )<<36;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ! ( (v.x>>36)%2 ) & ! ( (v.y>>63)%2 ) & ! ( (v.y>>7)%2 ) ) ) ) ) | ( ( (v.y>>61)%2 ) & ( ( ( ! ( (v.y>>38)%2 ) & ! ( (v.y>>37)%2 ) ) ) ) )<<37;
        aux.y |=(unsigned long long) ( ! ( ( ( (v.x>>0)%2 ) ) | ( ( (v.y>>8)%2 ) ) | ( ( (v.y>>4)%2 ) ) ) ) | ! ( ( (v.y>>4)%2 ) | ( (v.x>>0)%2 ) | ( (v.y>>8)%2 ) )<<38;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) )<<39;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_bool_18_parte5(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;
    
        aux.y |=(unsigned long long) ( ( ( ( (v.x>>45)%2 ) ) & ! ( ( (v.x>>46)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>42)%2 ) ) & ! ( ( (v.x>>46)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) )<<40;
        aux.y |=(unsigned long long) ( ( (v.x>>28)%2 ) )<<41;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>50)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>46)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>53)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>55)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<<42;
        aux.y |=(unsigned long long) ( ( ( (v.x>>22)%2 ) & ( ( ( ( (v.x>>24)%2 ) & ( (v.y>>45)%2 ) ) ) ) ) & ! ( ( (v.y>>43)%2 ) & ( ( ( ( (v.x>>24)%2 ) & ( (v.x>>22)%2 ) & ( (v.y>>45)%2 ) & ( (v.y>>15)%2 ) ) ) ) ) )<<43;
        aux.y |=(unsigned long long) ( ( ( (v.x>>37)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( (v.x>>59)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) & ( ( ( (v.x>>17)%2 ) ) ) ) )<<44;
        aux.y |=(unsigned long long) ( ( ( (v.x>>31)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.y>>18)%2 ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.y>>59)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>22)%2 ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) )<<45;
        aux.y |=(unsigned long long) ( ( (v.y>>48)%2 ) )<<46;
        aux.y |=(unsigned long long) ( ( (v.z>>7)%2 ) )<<47;
        aux.y |=(unsigned long long) ( ( ( (v.y>>52)%2 ) ) & ! ( ( (v.z>>0)%2 ) ) )<<48;
        aux.y |=(unsigned long long) ( ( (v.y>>4)%2 ) & ( ( ( ! ( (v.y>>49)%2 ) ) ) ) ) | ( ( (v.y>>49)%2 ) & ( ( ( ! ( (v.y>>45)%2 ) ) ) ) )<<49;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) & ( ( ( ( (v.y>>4)%2 ) | ( (v.y>>35)%2 ) | ( (v.y>>34)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>4)%2 ) | ( (v.y>>35)%2 ) | ( (v.y>>34)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) ) ) | ( ( (v.x>>12)%2 ) & ( ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) | ( (v.y>>35)%2 ) | ( (v.y>>34)%2 ) ) ) ) )<<50;
        aux.y |=(unsigned long long) ( ( (v.y>>52)%2 ) )<<51;
        aux.y |=(unsigned long long) ( ( (v.x>>38)%2 ) )<<52;
        aux.y |=(unsigned long long) ( ( (v.z>>7)%2 ) ) | ( ( (v.y>>52)%2 ) )<<53;
        aux.y |=(unsigned long long) ( ( (v.y>>49)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) | ( ( (v.x>>5)%2 ) & ( ( ( ( (v.y>>49)%2 ) & ( (v.y>>54)%2 ) ) ) & ( ( ! ( (v.y>>4)%2 ) ) ) ) )<<54;
        aux.y |=(unsigned long long) ( ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>4)%2 ) | ( (v.x>>59)%2 ) | ( (v.x>>37)%2 ) ) ) & ( ( ( (v.x>>17)%2 ) & ( (v.x>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) ) | ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>17)%2 ) & ( (v.x>>16)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) | ( (v.x>>59)%2 ) | ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) & ( ( ( ! ( (v.y>>17)%2 ) & ! ( (v.x>>17)%2 ) ) ) & ( ( ( (v.x>>24)%2 ) | ( (v.y>>45)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) | ( (v.x>>59)%2 ) | ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) )<<55;
        aux.y |=(unsigned long long) ( ( ( (v.z>>8)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>19)%2 ) & ! ( (v.z>>8)%2 ) & ! ( (v.y>>56)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) ) ) ) ) ) | ( ( ( (v.y>>19)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>19)%2 ) & ! ( (v.z>>8)%2 ) & ! ( (v.y>>56)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) ) ) ) ) ) | ( ( ( (v.y>>56)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>19)%2 ) & ! ( (v.z>>8)%2 ) & ! ( (v.y>>56)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) ) ) ) ) )<<56;
        aux.y |=(unsigned long long) ( ( ( (v.z>>9)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>9)%2 ) & ! ( (v.y>>57)%2 ) & ! ( (v.y>>20)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>9)%2 ) & ! ( (v.y>>57)%2 ) & ! ( (v.y>>20)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>57)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>9)%2 ) & ! ( (v.y>>57)%2 ) & ! ( (v.y>>20)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) ) ) ) ) )<<57;
        aux.y |=(unsigned long long) ( ( ( (v.z>>10)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>20)%2 ) ) ) | ( ( ! ( (v.z>>10)%2 ) & ! ( (v.y>>58)%2 ) & ! ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>20)%2 ) ) ) | ( ( ! ( (v.z>>10)%2 ) & ! ( (v.y>>58)%2 ) & ! ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>58)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>20)%2 ) ) ) | ( ( ! ( (v.z>>10)%2 ) & ! ( (v.y>>58)%2 ) & ! ( (v.y>>20)%2 ) ) ) ) ) )<<58;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>5)%2 ) & ! ( (v.y>>59)%2 ) & ! ( (v.y>>22)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) ) ) ) ) ) | ( ( ( (v.z>>5)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>5)%2 ) & ! ( (v.y>>59)%2 ) & ! ( (v.y>>22)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) ) ) ) ) ) | ( ( ( (v.y>>22)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>5)%2 ) & ! ( (v.y>>59)%2 ) & ! ( (v.y>>22)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) ) ) ) ) )<<59;
        aux.y |=(unsigned long long) ( ( ( (v.x>>1)%2 ) ) & ! ( ( (v.x>>62)%2 ) ) ) | ( ( ( (v.y>>60)%2 ) ) & ! ( ( (v.x>>62)%2 ) ) )<<60;
        aux.y |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) )<<61;
        aux.y |=(unsigned long long) ( ( ( ( (v.y>>5)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( (v.y>>3)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( (v.y>>4)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( (v.x>>10)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( ( (v.x>>22)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>41)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) )<<62;
        aux.y |=(unsigned long long) ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) | ( ( ! ( (v.y>>63)%2 ) ) ) | ( ( ! ( (v.y>>62)%2 ) ) ) ) )<<63;
        aux.z |=(unsigned long long) ( ( ( ( (v.x>>44)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) ) | ( ( ( ( (v.x>>43)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) ) | ( ( ( ( (v.y>>42)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) )<<0;
        aux.z |=(unsigned long long) ( ( (v.x>>23)%2 ) & ( ( ( ! ( (v.z>>1)%2 ) ) ) & ( ( ( (v.x>>63)%2 ) ) ) ) )<<1;

        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_bool_18(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( (v.y>>10)%2 ) )<< 0 ;
        aux.x |=(unsigned long long) ( ( (v.x>>42)%2 ) & ( ( ( ( (v.z>>2)%2 ) ) & ( ( ( ( (v.x>>33)%2 ) ) ) & ( ( ( (v.x>>31)%2 ) ) ) ) ) ) )<< 1 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) )<< 2 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) ) | ( ( (v.y>>4)%2 ) )<< 3 ;
        aux.x |=(unsigned long long) ( ( (v.y>>53)%2 ) )<< 4 ;
        aux.x |=(unsigned long long) ( ( (v.x>>7)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) )<< 5 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) & ( ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) & ( ( ( (v.x>>39)%2 ) & ( (v.y>>45)%2 ) ) ) & ( ( ! ( (v.x>>6)%2 ) ) ) ) ) | ( ( (v.x>>6)%2 ) & ( ( ( ! ( (v.y>>14)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) & ( ( ( ( (v.x>>39)%2 ) & ( (v.y>>45)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) & ( ( ! ( (v.x>>6)%2 ) ) ) ) )<< 6 ;
        aux.x |=(unsigned long long) ( ( (v.y>>55)%2 ) )<< 7 ;
        aux.x |=(unsigned long long) ( ( (v.y>>20)%2 ) ) | ( ( (v.y>>21)%2 ) ) | ( ( (v.y>>19)%2 ) ) | ( ( (v.y>>22)%2 ) )<< 8 ;
        aux.x |=(unsigned long long) ( ( (v.x>>39)%2 ) )<< 9 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) ) & ! ( ( (v.z>>3)%2 ) ) )<< 10 ;
        aux.x |=(unsigned long long) ( ( (v.x>>10)%2 ) )<< 11 ;
        aux.x |=(unsigned long long) ( ( (v.x>>13)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) ) ) )<< 12 ;
        aux.x |=(unsigned long long) ( ( (v.x>>11)%2 ) )<< 13 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) ) ) )<< 14 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) ) & ! ( ( (v.y>>41)%2 ) ) )<< 15 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) & ( ( ( ! ( (v.y>>38)%2 ) ) ) & ( ( ( (v.y>>23)%2 ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.y>>23)%2 ) & ( ( ( ! ( (v.y>>26)%2 ) & ! ( (v.x>>16)%2 ) ) ) & ( ( ( (v.x>>33)%2 ) & ( (v.x>>61)%2 ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) )<< 16 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) & ( ( ( ( (v.x>>24)%2 ) | ( (v.y>>45)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) )<< 17 ;
        aux.x |=(unsigned long long) ( ( (v.x>>15)%2 ) & ( ( ( ( (v.x>>33)%2 ) | ( (v.y>>7)%2 ) | ( (v.x>>32)%2 ) | ( (v.x>>34)%2 ) ) ) | ( ( ! ( (v.y>>41)%2 ) & ! ( (v.x>>33)%2 ) & ! ( (v.y>>7)%2 ) & ! ( (v.x>>32)%2 ) & ! ( (v.x>>34)%2 ) ) ) ) ) | ( ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) & ( (v.x>>15)%2 ) ) ) ) ) & ! ( ( (v.y>>41)%2 ) ) )<< 18 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.x>>20)%2 ) & ( ( ( ( (v.x>>19)%2 ) ) ) ) ) ) | ( ( ( (v.y>>12)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.x>>20)%2 ) & ( ( ( ( (v.x>>19)%2 ) ) ) ) ) ) | ( ( (v.x>>19)%2 ) & ( ( ( ! ( (v.x>>20)%2 ) ) ) ) )<< 19 ;
        aux.x |=(unsigned long long) ( ( (v.y>>8)%2 ) & ( ( ( ( (v.x>>19)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>60)%2 ) & ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>22)%2 ) )<< 20 ;
        aux.x |=(unsigned long long) ( ( (v.x>>17)%2 ) & ( ( ( ( (v.x>>14)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) )<< 21 ;
        aux.x |=(unsigned long long) ( ( (v.z>>4)%2 ) & ( ( ( ! ( (v.y>>8)%2 ) ) ) ) ) | ( ( (v.y>>58)%2 ) & ( ( ( ( (v.y>>8)%2 ) & ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.y>>57)%2 ) & ( ( ( ( (v.y>>8)%2 ) & ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.y>>56)%2 ) & ( ( ( ( (v.y>>8)%2 ) & ( (v.x>>10)%2 ) ) ) ) )<< 22 ;
        aux.x |=(unsigned long long) ( ( (v.x>>52)%2 ) ) | ( ( ( ( (v.x>>23)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) )<< 23 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) ) | ( ( ( (v.x>>42)%2 ) & ( ( ( ( (v.y>>49)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) )<< 24 ;
        aux.x |=(unsigned long long) ( ( (v.y>>52)%2 ) )<< 25 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>33)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>34)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>32)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( ( ( (v.x>>35)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>25)%2 ) ) ) & ! ( ( (v.x>>23)%2 ) ) )<< 26 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ! ( (v.y>>61)%2 ) ) ) & ( ( ( (v.x>>32)%2 ) ) ) ) ) | ( ( (v.y>>56)%2 ) & ( ( ( ! ( (v.x>>32)%2 ) & ! ( (v.x>>27)%2 ) ) ) ) )<< 27 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) & ( ( ( ( (v.y>>5)%2 ) & ( (v.x>>22)%2 ) ) ) ) ) & ! ( ( (v.y>>41)%2 ) ) ) | ( ( ( (v.x>>37)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) & ( ( ( ! ( (v.x>>28)%2 ) ) ) ) ) ) ) & ! ( ( (v.y>>41)%2 ) ) )<< 28 ;
        aux.x |=(unsigned long long) ( ( (v.x>>33)%2 ) & ( ( ( ! ( (v.y>>24)%2 ) ) ) & ( ( ( (v.x>>29)%2 ) ) ) ) ) | ( ( (v.y>>57)%2 ) & ( ( ( ! ( (v.x>>33)%2 ) & ! ( (v.x>>29)%2 ) ) ) ) ) | ( ( (v.y>>7)%2 ) & ( ( ( ! ( (v.x>>29)%2 ) ) ) & ( ( ! ( (v.y>>59)%2 ) ) ) & ( ( ( (v.z>>5)%2 ) ) ) & ( ( ! ( (v.x>>33)%2 ) ) ) ) )<< 29 ;
        aux.x |=(unsigned long long) ( ( (v.y>>58)%2 ) & ( ( ( ! ( (v.x>>30)%2 ) ) & ( ( ( ! ( (v.x>>34)%2 ) ) ) ) ) ) ) | ( ( (v.x>>30)%2 ) & ( ( ( ! ( (v.y>>24)%2 ) & ! ( (v.y>>11)%2 ) ) ) & ( ( ( (v.x>>34)%2 ) ) ) ) )<< 30 ;
        aux.x |=(unsigned long long) ( ( (v.y>>59)%2 ) & ( ( ( ! ( (v.x>>35)%2 ) ) ) & ( ( ! ( (v.y>>7)%2 ) ) ) & ( ( ! ( (v.x>>31)%2 ) ) ) ) ) | ( ( (v.x>>35)%2 ) & ( ( ( ! ( (v.y>>24)%2 ) ) ) & ( ( ( (v.x>>31)%2 ) ) ) ) )<< 31 ;
        aux.x |=(unsigned long long) ( ( (v.x>>27)%2 ) ) | ( ( (v.y>>56)%2 ) & ( ( ( ! ( (v.x>>27)%2 ) ) ) & ( ( ! ( (v.x>>32)%2 ) ) ) ) )<< 32 ;
        aux.x |=(unsigned long long) ( ( (v.y>>57)%2 ) & ( ( ( ! ( (v.x>>29)%2 ) ) ) & ( ( ! ( (v.x>>33)%2 ) ) ) ) ) | ( ( (v.x>>29)%2 ) )<< 33 ;
        aux.x |=(unsigned long long) ( ( (v.y>>58)%2 ) & ( ( ( ! ( (v.x>>30)%2 ) ) ) & ( ( ! ( (v.x>>34)%2 ) ) ) ) ) | ( ( (v.x>>30)%2 ) )<< 34 ;
        aux.x |=(unsigned long long) ( ( (v.x>>31)%2 ) ) | ( ( (v.y>>59)%2 ) & ( ( ( ! ( (v.x>>35)%2 ) ) ) & ( ( ! ( (v.x>>31)%2 ) ) ) ) )<< 35 ;
        aux.x |=(unsigned long long) ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) )<< 36 ;
        aux.x |=(unsigned long long) ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) ) ) | ( ( (v.x>>22)%2 ) ) | ( ( (v.y>>43)%2 ) )<< 37 ;
        aux.x |=(unsigned long long) ( ( (v.z>>6)%2 ) )<< 38 ;
        aux.x |=(unsigned long long) ( ( (v.y>>5)%2 ) )<< 39 ;
        aux.x |=(unsigned long long) ( ( (v.y>>11)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) | ( ( (v.y>>12)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) )<< 40 ;
        aux.x |=(unsigned long long) ( ( ( ( ( (v.x>>33)%2 ) ) & ! ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) ) ) ) ) ) & ! ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ( (v.x>>10)%2 ) ) ) ) ) ) & ! ( ( (v.x>>41)%2 ) & ( ( ( ( (v.x>>11)%2 ) & ( (v.x>>33)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) & ! ( (v.y>>14)%2 ) & ! ( (v.y>>7)%2 ) & ! ( (v.x>>10)%2 ) ) ) ) ) ) | ( ( ( ( (v.x>>40)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) & ! ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) ) ) ) ) ) & ! ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ( (v.x>>10)%2 ) ) ) ) ) ) | ( ( ( ( ( (v.y>>7)%2 ) ) & ! ( ( (v.x>>10)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ! ( (v.x>>40)%2 ) ) ) ) ) ) & ! ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) & ( ( ( (v.x>>10)%2 ) ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>41)%2 ) ) ) ) ) )<< 41 ;
        aux.x |=(unsigned long long) ( ( (v.y>>49)%2 ) & ( ( ( ( (v.z>>2)%2 ) ) ) & ( ( ! ( (v.x>>39)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ! ( (v.x>>39)%2 ) & ! ( (v.z>>2)%2 ) & ! ( (v.y>>49)%2 ) & ! ( (v.y>>14)%2 ) & ! ( (v.x>>42)%2 ) ) ) ) ) | ( ( (v.y>>14)%2 ) & ( ( ( ! ( (v.x>>42)%2 ) ) ) & ( ( ( (v.x>>39)%2 ) & ( (v.z>>2)%2 ) & ( (v.y>>49)%2 ) ) ) ) ) | ( ( (v.x>>42)%2 ) & ( ( ( ! ( (v.x>>39)%2 ) & ! ( (v.y>>45)%2 ) ) ) ) )<< 42 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>50)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>46)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>55)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>47)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<< 43 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>46)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>47)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<< 44 ;
        aux.x |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>50)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>53)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>55)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<< 45 ;
        aux.x |=(unsigned long long) ( ( (v.z>>0)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) | ( ( (v.x>>23)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) | ( ( (v.y>>40)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) )<< 46 ;
        aux.x |=(unsigned long long) ( ( ( ( (v.x>>11)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) & ( ( ! ( (v.y>>7)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) & ! ( ( (v.x>>61)%2 ) ) ) | ( ( ( ( (v.x>>23)%2 ) & ( ( ( ! ( (v.y>>7)%2 ) ) ) & ( ( ! ( (v.x>>61)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) & ! ( ( (v.x>>61)%2 ) ) )<< 47 ;
        aux.x |=(unsigned long long) ( ( ( ( ( ( ( (v.y>>7)%2 ) & ( ( ( ( (v.y>>39)%2 ) ) ) ) ) & ! ( ( (v.x>>39)%2 ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.x>>61)%2 ) ) ) & ! ( ( (v.y>>27)%2 ) ) ) ) | ! ( ( (v.x>>39)%2 ) | ( (v.y>>8)%2 ) | ( (v.y>>39)%2 ) | ( (v.y>>7)%2 ) | ( (v.y>>27)%2 ) | ( (v.x>>61)%2 ) )<< 48 ;
        aux.x |=(unsigned long long) ( ( (v.y>>26)%2 ) ) | ( ( (v.x>>16)%2 ) )<< 49 ;
        aux.x |=(unsigned long long) ( ( (v.x>>16)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) | ( ( (v.y>>26)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) )<< 50 ;
        aux.x |=(unsigned long long) ( ( (v.x>>38)%2 ) ) | ( ( (v.y>>26)%2 ) ) | ( ( (v.x>>16)%2 ) )<< 51 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) & ( ( ( ( (v.y>>51)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.y>>51)%2 ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.x>>53)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.x>>54)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.y>>27)%2 ) & ( ( ( ( (v.y>>51)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) ) | ( ( ( (v.x>>55)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.x>>52)%2 ) ) ) ) ) )<< 52 ;
        aux.x |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ( (v.y>>43)%2 ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) | ( ( (v.x>>58)%2 ) & ( ( ( ( (v.y>>43)%2 ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) | ( ( (v.x>>37)%2 ) & ( ( ( ( (v.y>>43)%2 ) ) ) ) ) | ( ( (v.y>>52)%2 ) ) | ( ( (v.y>>26)%2 ) ) | ( ( (v.y>>35)%2 ) ) | ( ( (v.x>>25)%2 ) ) | ( ( (v.x>>16)%2 ) )<< 53 ;
        aux.x |=(unsigned long long) ( ( (v.y>>0)%2 ) & ( ( ( ! ( (v.x>>54)%2 ) ) ) & ( ( ( (v.x>>22)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) & ( ( ! ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.y>>12)%2 ) & ( ( ( ( (v.x>>22)%2 ) ) ) & ( ( ! ( (v.x>>54)%2 ) ) ) ) ) | ( ( (v.x>>37)%2 ) & ( ( ( ! ( (v.x>>54)%2 ) ) ) & ( ( ( (v.x>>22)%2 ) ) ) ) )<< 54 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>52)%2 ) ) & ! ( ( (v.x>>28)%2 ) ) ) | ( ( ( (v.y>>26)%2 ) ) & ! ( ( (v.x>>28)%2 ) ) ) | ( ( ( (v.x>>38)%2 ) ) & ! ( ( (v.x>>28)%2 ) ) )<< 55 ;
        aux.x |=(unsigned long long) ( ( (v.x>>16)%2 ) ) | ( ( (v.y>>26)%2 ) )<< 56 ;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) & ( ( ( ! ( (v.x>>57)%2 ) ) ) | ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>47)%2 ) & ( ( ( ( (v.x>>11)%2 ) ) ) & ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>61)%2 ) & ( ( ( ! ( (v.x>>57)%2 ) ) ) | ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.y>>39)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) ) ) | ( ( ! ( (v.x>>57)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>47)%2 ) ) ) & ( ( ! ( (v.x>>57)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) ) | ( ( ( (v.x>>57)%2 ) & ( ( ( ! ( (v.x>>48)%2 ) ) ) ) ) & ! ( ( (v.x>>48)%2 ) & ( ( ( ( (v.x>>57)%2 ) ) ) ) ) )<< 57 ;
        aux.x |=(unsigned long long) ( ( (v.y>>46)%2 ) ) | ( ( (v.x>>59)%2 ) )<< 58 ;
        aux.x |=(unsigned long long) ( ( (v.x>>14)%2 ) ) | ( ( (v.x>>22)%2 ) )<< 59 ;
        aux.x |=(unsigned long long) ( ( (v.y>>13)%2 ) )<< 60 ;
        aux.x |=(unsigned long long) ( ( ( (v.y>>26)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) & ( ( ( ! ( (v.x>>6)%2 ) ) ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) | ( ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>61)%2 ) ) & ( ( ( ( (v.y>>26)%2 ) | ( (v.x>>16)%2 ) ) ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) ) & ! ( ( (v.y>>16)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) & ( ( ( ( (v.x>>59)%2 ) ) & ( ( ( ! ( (v.x>>6)%2 ) ) ) ) ) | ( ( ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) ) )<< 61 ;
        aux.x |=(unsigned long long) ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.y>>7)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) )<< 62 ;
        aux.x |=(unsigned long long) ( ( (v.z>>1)%2 ) ) | ( ( (v.y>>45)%2 ) )<< 63 ;
        aux.y |=(unsigned long long) ( ( (v.x>>33)%2 ) ) | ( ( (v.x>>28)%2 ) ) | ( ( (v.x>>24)%2 ) ) | ( ( (v.x>>17)%2 ) ) | ( ( (v.y>>35)%2 ) ) | ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) )<<0;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) ) | ( ( (v.y>>8)%2 ) ) | ( ( (v.x>>3)%2 ) ) | ( ( (v.x>>29)%2 ) ) | ( ( (v.x>>30)%2 ) )<<1;
        aux.y |=(unsigned long long) ( ( (v.x>>60)%2 ) ) | ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>49)%2 ) ) ) ) ) | ( ( (v.y>>39)%2 ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ! ( (v.y>>2)%2 ) ) ) & ( ( ! ( (v.y>>49)%2 ) ) ) & ( ( ( (v.x>>24)%2 ) ) ) ) ) | ( ( (v.x>>3)%2 ) )<<2;
        aux.y |=(unsigned long long) ( ( (v.y>>3)%2 ) & ( ( ( ! ( (v.y>>15)%2 ) ) ) & ( ( ! ( (v.y>>2)%2 ) ) ) ) ) | ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.y>>3)%2 ) ) ) & ( ( ( (v.y>>0)%2 ) ) ) ) )<<3;
        aux.y |=(unsigned long long) ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) | ( ( (v.y>>1)%2 ) & ( ( ( ( (v.y>>2)%2 ) ) ) ) ) | ( ( (v.y>>4)%2 ) )<<4;
        aux.y |=(unsigned long long) ( ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>4)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>3)%2 ) ) ) ) ) & ! ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) )<<5;
        aux.y |=(unsigned long long) ( ( ( ( (v.y>>15)%2 ) & ( ( ( ( (v.y>>3)%2 ) ) ) & ( ( ! ( (v.y>>6)%2 ) ) ) ) ) & ! ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) & ! ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>6)%2 ) & ( ( ( ! ( (v.y>>0)%2 ) ) ) & ( ( ! ( (v.y>>2)%2 ) ) ) ) ) & ! ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) & ! ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>1)%2 ) & ( ( ( ! ( (v.y>>6)%2 ) ) ) ) ) & ! ( ( (v.y>>0)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) ) & ! ( ( (v.y>>2)%2 ) & ( ( ( ( (v.y>>6)%2 ) ) ) ) ) )<<6;
        aux.y |=(unsigned long long) ( ( ( (v.y>>7)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>7)%2 ) ) ) ) ) ) | ( ( ( (v.x>>63)%2 ) & ( ( ( ( (v.y>>60)%2 ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>7)%2 ) ) ) ) ) )<<7;
        aux.y |=(unsigned long long) ( ( ( (v.x>>0)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) & ( ( ( (v.y>>9)%2 ) ) ) ) ) & ! ( ( (v.y>>53)%2 ) & ( ( ( ( (v.y>>8)%2 ) ) ) ) ) ) | ( ( ( (v.y>>8)%2 ) & ( ( ( ! ( (v.y>>14)%2 ) ) ) & ( ( ! ( (v.y>>53)%2 ) ) ) ) ) & ! ( ( (v.y>>53)%2 ) & ( ( ( ( (v.y>>8)%2 ) ) ) ) ) ) | ( ( ( (v.x>>19)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) & ( ( ( (v.y>>9)%2 ) ) ) ) ) & ! ( ( (v.y>>53)%2 ) & ( ( ( ( (v.y>>8)%2 ) ) ) ) ) )<<8;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) & ( ( ( ! ( (v.y>>9)%2 ) ) ) & ( ( ( (v.x>>63)%2 ) ) ) ) ) | ( ( (v.y>>9)%2 ) & ( ( ( ! ( (v.y>>8)%2 ) ) ) ) ) | ( ( (v.x>>63)%2 ) & ( ( ( ! ( (v.y>>8)%2 ) ) ) ) )<<9;
        aux.y |=(unsigned long long) ( ( (v.y>>4)%2 ) & ( ( ( ( (v.x>>23)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) | ( ( (v.y>>5)%2 ) & ( ( ( ( (v.x>>12)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) ) ) ) ) | ( ( (v.x>>12)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) ) | ( ( (v.x>>23)%2 ) & ( ( ( ( (v.x>>10)%2 ) ) ) ) )<<10;
        aux.y |=(unsigned long long) ( ( (v.x>>30)%2 ) ) | ( ( ( (v.x>>33)%2 ) & ( ( ( ( (v.y>>11)%2 ) ) ) ) ) & ! ( ( (v.y>>7)%2 ) & ( ( ( ! ( (v.x>>30)%2 ) ) ) ) ) )<<11;
        aux.y |=(unsigned long long) ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) & ( (v.y>>5)%2 ) ) ) ) ) | ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) & ( ( ! ( (v.y>>5)%2 ) & ! ( (v.x>>0)%2 ) & ! ( (v.x>>22)%2 ) & ! ( (v.x>>60)%2 ) ) ) ) ) | ( ( ( (v.x>>22)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) & ! ( ( (v.x>>60)%2 ) & ( ( ( ! ( (v.x>>24)%2 ) & ! ( (v.y>>45)%2 ) ) ) & ( ( ( (v.x>>0)%2 ) ) ) ) ) )<<12;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>4)%2 ) ) ) | ( ( ( (v.y>>5)%2 ) ) ) ) ) ) ) | ( ( (v.y>>8)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>4)%2 ) ) ) | ( ( ( (v.y>>5)%2 ) ) ) ) ) ) ) | ( ( (v.x>>3)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) | ( ( ( (v.y>>4)%2 ) ) ) ) ) | ( ( (v.y>>26)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>5)%2 ) ) ) | ( ( ( (v.y>>4)%2 ) ) ) ) ) ) ) | ( ( (v.x>>16)%2 ) & ( ( ( ! ( (v.x>>3)%2 ) ) ) & ( ( ( (v.x>>5)%2 ) ) & ( ( ( ( (v.y>>5)%2 ) ) ) | ( ( ( (v.y>>4)%2 ) ) ) ) ) ) )<<13;
        aux.y |=(unsigned long long) ( ! ( ( ( (v.x>>22)%2 ) ) ) ) | ! ( ( (v.x>>22)%2 ) )<<14;
        aux.y |=(unsigned long long) ( ( ( (v.z>>7)%2 ) ) & ! ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>15)%2 ) ) ) ) ) ) | ( ( ( (v.y>>23)%2 ) & ( ( ( ( (v.y>>0)%2 ) ) ) & ( ( ( (v.y>>37)%2 ) ) ) & ( ( ( (v.x>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>15)%2 ) ) ) ) ) )<<15;
        aux.y |=(unsigned long long) ( ! ( ( ( (v.z>>7)%2 ) ) | ( ( (v.x>>22)%2 ) & ( ( ( ( (v.z>>4)%2 ) ) ) ) ) ) ) | ! ( ( (v.x>>22)%2 ) | ( (v.z>>7)%2 ) | ( (v.z>>4)%2 ) )<<16;
        aux.y |=(unsigned long long) ( ( ( ( (v.x>>42)%2 ) & ( ( ( ( (v.z>>2)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>7)%2 ) ) )<<17;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) )<<18;
        aux.y |=(unsigned long long) ( ( (v.y>>56)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<19;
        aux.y |=(unsigned long long) ( ( (v.y>>57)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<20;
        aux.y |=(unsigned long long) ( ( (v.y>>58)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<21;
        aux.y |=(unsigned long long) ( ( (v.y>>59)%2 ) & ( ( ( ( (v.x>>26)%2 ) ) ) ) )<<22;
        aux.y |=(unsigned long long) ( ( (v.y>>3)%2 ) & ( ( ( ( (v.x>>9)%2 ) ) ) ) ) | ( ( (v.y>>5)%2 ) & ( ( ( ( (v.x>>9)%2 ) ) ) ) )<<23;
        aux.y |=(unsigned long long) ( ( (v.x>>11)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) )<<24;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) )<<25;
        aux.y |=(unsigned long long) ( ( ( ( ( (v.y>>36)%2 ) & ( ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) & ! ( ( (v.y>>32)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>63)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) ) | ( ( ( (v.y>>23)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) & ( ( ( ! ( (v.y>>38)%2 ) ) ) & ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) & ( ( ! ( (v.y>>26)%2 ) ) ) & ( ( ! ( (v.x>>21)%2 ) & ! ( (v.y>>36)%2 ) & ! ( (v.y>>50)%2 ) ) ) & ( ( ( (v.x>>16)%2 ) ) ) ) ) | ( ( ! ( (v.x>>33)%2 ) ) & ( ( ( ( (v.x>>16)%2 ) ) ) & ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) & ( ( ! ( (v.y>>26)%2 ) ) ) ) ) | ( ( ( (v.x>>33)%2 ) & ( (v.x>>61)%2 ) ) & ( ( ( ! ( (v.y>>26)%2 ) & ! ( (v.x>>16)%2 ) ) ) & ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) ) | ( ( ( ( ( (v.y>>50)%2 ) & ( ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) & ! ( ( (v.y>>32)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>63)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) ) | ( ( ( ( ( (v.x>>21)%2 ) & ( ( ( ( (v.z>>2)%2 ) & ( (v.x>>42)%2 ) ) ) ) ) & ! ( ( (v.y>>32)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>63)%2 ) & ( ( ( ( (v.y>>26)%2 ) ) ) ) ) ) & ! ( ( (v.y>>38)%2 ) & ( ( ( ! ( (v.x>>61)%2 ) ) ) ) ) )<<26;
        aux.y |=(unsigned long long) ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>25)%2 ) & ( (v.y>>29)%2 ) & ( (v.x>>61)%2 ) ) ) & ( ( ! ( (v.y>>27)%2 ) ) ) ) ) | ( ( (v.y>>35)%2 ) & ( ( ( ( (v.y>>27)%2 ) ) ) ) ) | ( ( (v.y>>27)%2 ) & ( ( ( ! ( (v.y>>7)%2 ) & ! ( (v.x>>6)%2 ) & ! ( (v.x>>23)%2 ) ) ) ) ) | ( ( (v.x>>61)%2 ) & ( ( ( ! ( (v.y>>35)%2 ) & ! ( (v.x>>6)%2 ) & ! ( (v.x>>23)%2 ) ) ) & ( ( ( (v.y>>27)%2 ) ) ) ) )<<27;
        aux.y |=(unsigned long long) ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>30)%2 ) ) ) & ( ( ! ( (v.y>>28)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ! ( (v.y>>29)%2 ) ) ) ) )<<28;
        aux.y |=(unsigned long long) ( ( (v.y>>29)%2 ) & ( ( ( ! ( (v.y>>27)%2 ) ) ) ) ) | ( ( (v.y>>35)%2 ) & ( ( ( ! ( (v.y>>29)%2 ) ) ) & ( ( ( (v.y>>28)%2 ) ) ) ) )<<29;
        aux.y |=(unsigned long long) ( ( ( (v.y>>30)%2 ) & ( ( ( ! ( (v.y>>28)%2 ) ) ) ) ) | ( ( (v.y>>28)%2 ) & ( ( ( ! ( (v.y>>27)%2 ) & ! ( (v.y>>30)%2 ) ) ) ) ) ) | ! ( ( (v.y>>27)%2 ) | ( (v.y>>28)%2 ) | ( (v.y>>30)%2 ) )<<30;
        aux.y |=(unsigned long long) ( ( (v.x>>11)%2 ) ) | ( ( (v.y>>33)%2 ) ) | ( ( (v.x>>2)%2 ) )<<31;
        aux.y |=(unsigned long long) ( ( (v.y>>31)%2 ) )<<32;
        aux.y |=(unsigned long long) ( ( ( ( (v.y>>59)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>57)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>58)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>56)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.y>>8)%2 ) ) ) & ! ( ( (v.y>>35)%2 ) & ( ( ( ( (v.x>>63)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) ) ) ) ) )<<33;
        aux.y |=(unsigned long long) ( ( (v.y>>7)%2 ) & ( ( ( ( (v.y>>45)%2 ) & ( (v.y>>60)%2 ) ) ) & ( ( ! ( (v.y>>34)%2 ) | ! ( (v.x>>29)%2 ) ) ) ) ) | ( ( (v.x>>12)%2 ) & ( ( ( ( (v.y>>45)%2 ) & ( (v.y>>60)%2 ) ) ) & ( ( ! ( (v.y>>34)%2 ) | ! ( (v.x>>29)%2 ) ) ) ) )<<34;
        aux.y |=(unsigned long long) ( ( (v.y>>36)%2 ) ) | ( ( (v.y>>41)%2 ) ) | ( ( (v.y>>44)%2 ) )<<35;
        aux.y |=(unsigned long long) ( ( (v.x>>11)%2 ) & ( ( ( ( (v.x>>16)%2 ) ) ) ) ) | ( ( (v.x>>19)%2 ) & ( ( ( ( (v.x>>16)%2 ) ) ) ) )<<36;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) & ( ( ( ! ( (v.x>>36)%2 ) & ! ( (v.y>>63)%2 ) & ! ( (v.y>>7)%2 ) ) ) ) ) | ( ( (v.y>>61)%2 ) & ( ( ( ! ( (v.y>>38)%2 ) & ! ( (v.y>>37)%2 ) ) ) ) )<<37;
        aux.y |=(unsigned long long) ( ! ( ( ( (v.x>>0)%2 ) ) | ( ( (v.y>>8)%2 ) ) | ( ( (v.y>>4)%2 ) ) ) ) | ! ( ( (v.y>>4)%2 ) | ( (v.x>>0)%2 ) | ( (v.y>>8)%2 ) )<<38;
        aux.y |=(unsigned long long) ( ( (v.y>>37)%2 ) )<<39;
        aux.y |=(unsigned long long) ( ( ( ( (v.x>>45)%2 ) ) & ! ( ( (v.x>>46)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) ) | ( ( ( ( (v.y>>42)%2 ) ) & ! ( ( (v.x>>46)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) ) & ! ( ( (v.y>>14)%2 ) & ( ( ( ( (v.y>>40)%2 ) ) ) ) ) )<<40;
        aux.y |=(unsigned long long) ( ( (v.x>>28)%2 ) )<<41;
        aux.y |=(unsigned long long) ( ( (v.x>>56)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>49)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>50)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>46)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>53)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.y>>51)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>54)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) ) | ( ( (v.x>>55)%2 ) & ( ( ( ( (v.x>>4)%2 ) ) ) ) )<<42;
        aux.y |=(unsigned long long) ( ( ( (v.x>>22)%2 ) & ( ( ( ( (v.x>>24)%2 ) & ( (v.y>>45)%2 ) ) ) ) ) & ! ( ( (v.y>>43)%2 ) & ( ( ( ( (v.x>>24)%2 ) & ( (v.x>>22)%2 ) & ( (v.y>>45)%2 ) & ( (v.y>>15)%2 ) ) ) ) ) )<<43;
        aux.y |=(unsigned long long) ( ( ( (v.x>>37)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) ) & ! ( ( (v.x>>23)%2 ) ) ) | ( ( (v.x>>59)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) & ( ( ( (v.x>>17)%2 ) ) ) ) )<<44;
        aux.y |=(unsigned long long) ( ( ( (v.x>>31)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.y>>18)%2 ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.y>>59)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>14)%2 ) & ( ( ( ( (v.y>>16)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>22)%2 ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) ) | ( ( ( (v.x>>29)%2 ) & ( ( ( ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>18)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) )<<45;
        aux.y |=(unsigned long long) ( ( (v.y>>48)%2 ) )<<46;
        aux.y |=(unsigned long long) ( ( (v.z>>7)%2 ) )<<47;
        aux.y |=(unsigned long long) ( ( ( (v.y>>52)%2 ) ) & ! ( ( (v.z>>0)%2 ) ) )<<48;
        aux.y |=(unsigned long long) ( ( (v.y>>4)%2 ) & ( ( ( ! ( (v.y>>49)%2 ) ) ) ) ) | ( ( (v.y>>49)%2 ) & ( ( ( ! ( (v.y>>45)%2 ) ) ) ) )<<49;
        aux.y |=(unsigned long long) ( ( (v.y>>8)%2 ) & ( ( ( ( (v.y>>4)%2 ) | ( (v.y>>35)%2 ) | ( (v.y>>34)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.y>>4)%2 ) | ( (v.y>>35)%2 ) | ( (v.y>>34)%2 ) ) ) & ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) ) ) | ( ( (v.x>>12)%2 ) & ( ( ( ( (v.y>>5)%2 ) | ( (v.y>>3)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) | ( (v.y>>35)%2 ) | ( (v.y>>34)%2 ) ) ) ) )<<50;
        aux.y |=(unsigned long long) ( ( (v.y>>52)%2 ) )<<51;
        aux.y |=(unsigned long long) ( ( (v.x>>38)%2 ) )<<52;
        aux.y |=(unsigned long long) ( ( (v.z>>7)%2 ) ) | ( ( (v.y>>52)%2 ) )<<53;
        aux.y |=(unsigned long long) ( ( (v.y>>49)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) | ( ( (v.x>>5)%2 ) & ( ( ( ( (v.y>>49)%2 ) & ( (v.y>>54)%2 ) ) ) & ( ( ! ( (v.y>>4)%2 ) ) ) ) )<<54;
        aux.y |=(unsigned long long) ( ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>4)%2 ) | ( (v.x>>59)%2 ) | ( (v.x>>37)%2 ) ) ) & ( ( ( (v.x>>17)%2 ) & ( (v.x>>16)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) ) | ( ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>17)%2 ) & ( (v.x>>16)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) | ( (v.x>>59)%2 ) | ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) ) | ( ( ( (v.x>>16)%2 ) & ( ( ( ! ( (v.y>>17)%2 ) & ! ( (v.x>>17)%2 ) ) ) & ( ( ( (v.x>>24)%2 ) | ( (v.y>>45)%2 ) ) ) & ( ( ( (v.y>>4)%2 ) | ( (v.x>>59)%2 ) | ( (v.x>>37)%2 ) ) ) ) ) & ! ( ( (v.y>>17)%2 ) ) )<<55;
        aux.y |=(unsigned long long) ( ( ( (v.z>>8)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>19)%2 ) & ! ( (v.z>>8)%2 ) & ! ( (v.y>>56)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) ) ) ) ) ) | ( ( ( (v.y>>19)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>19)%2 ) & ! ( (v.z>>8)%2 ) & ! ( (v.y>>56)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) ) ) ) ) ) | ( ( ( (v.y>>56)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.y>>19)%2 ) & ! ( (v.z>>8)%2 ) & ! ( (v.y>>56)%2 ) ) ) | ( ( ( (v.y>>19)%2 ) ) ) ) ) )<<56;
        aux.y |=(unsigned long long) ( ( ( (v.z>>9)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>9)%2 ) & ! ( (v.y>>57)%2 ) & ! ( (v.y>>20)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>9)%2 ) & ! ( (v.y>>57)%2 ) & ! ( (v.y>>20)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>57)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>9)%2 ) & ! ( (v.y>>57)%2 ) & ! ( (v.y>>20)%2 ) ) ) | ( ( ( (v.y>>20)%2 ) ) ) ) ) )<<57;
        aux.y |=(unsigned long long) ( ( ( (v.z>>10)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>20)%2 ) ) ) | ( ( ! ( (v.z>>10)%2 ) & ! ( (v.y>>58)%2 ) & ! ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>20)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>20)%2 ) ) ) | ( ( ! ( (v.z>>10)%2 ) & ! ( (v.y>>58)%2 ) & ! ( (v.y>>20)%2 ) ) ) ) ) ) | ( ( ( (v.y>>58)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ( (v.y>>20)%2 ) ) ) | ( ( ! ( (v.z>>10)%2 ) & ! ( (v.y>>58)%2 ) & ! ( (v.y>>20)%2 ) ) ) ) ) )<<58;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>5)%2 ) & ! ( (v.y>>59)%2 ) & ! ( (v.y>>22)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) ) ) ) ) ) | ( ( ( (v.z>>5)%2 ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>5)%2 ) & ! ( (v.y>>59)%2 ) & ! ( (v.y>>22)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) ) ) ) ) ) | ( ( ( (v.y>>22)%2 ) & ( ( ( ! ( (v.x>>8)%2 ) ) ) ) ) & ! ( ( (v.x>>8)%2 ) & ( ( ( ! ( (v.z>>5)%2 ) & ! ( (v.y>>59)%2 ) & ! ( (v.y>>22)%2 ) ) ) | ( ( ( (v.y>>22)%2 ) ) ) ) ) )<<59;
        aux.y |=(unsigned long long) ( ( ( (v.x>>1)%2 ) ) & ! ( ( (v.x>>62)%2 ) ) ) | ( ( ( (v.y>>60)%2 ) ) & ! ( ( (v.x>>62)%2 ) ) )<<60;
        aux.y |=(unsigned long long) ( ( (v.x>>27)%2 ) & ( ( ( ( (v.y>>5)%2 ) ) ) ) )<<61;
        aux.y |=(unsigned long long) ( ( ( ( (v.y>>5)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( (v.y>>3)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( (v.y>>4)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( (v.x>>10)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) ) | ( ( ( ( ( (v.x>>22)%2 ) ) & ! ( ( (v.x>>24)%2 ) ) ) & ! ( ( (v.y>>41)%2 ) ) ) & ! ( ( (v.y>>45)%2 ) ) )<<62;
        aux.y |=(unsigned long long) ( ( (v.x>>24)%2 ) & ( ( ( ( (v.y>>45)%2 ) ) ) ) ) | ( ( (v.y>>45)%2 ) & ( ( ( ( (v.x>>24)%2 ) ) ) | ( ( ! ( (v.y>>63)%2 ) ) ) | ( ( ! ( (v.y>>62)%2 ) ) ) ) )<<63;
        aux.z |=(unsigned long long) ( ( ( ( (v.x>>44)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) ) | ( ( ( ( (v.x>>43)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) ) | ( ( ( ( (v.y>>42)%2 ) ) & ! ( ( (v.y>>14)%2 ) ) ) & ! ( ( (v.x>>46)%2 ) ) )<<0;
        aux.z |=(unsigned long long) ( ( (v.x>>23)%2 ) & ( ( ( ! ( (v.z>>1)%2 ) ) ) & ( ( ( (v.x>>63)%2 ) ) ) ) )<<1;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_18_parte1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.y>>10)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + 0 * 2 + ( (v.x>>33)%2 ) * 2 + ( (v.x>>31)%2 ) * 2) >= 8 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2 + ( (v.y>>4)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.y>>53)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 4 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 12 + ( (v.y>>5)%2 ) * 12 + ( (v.y>>3)%2 ) * 12 + ( (v.x>>39)%2 ) * 36 + ( (v.y>>45)%2 ) * 36 + ( (v.x>>6)%2 ) * 92 + ( (v.y>>14)%2 ) * -128 + ( (v.x>>63)%2 ) * 12) >= -32 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.y>>55)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.y>>20)%2 ) * 2 + ( (v.y>>21)%2 ) * 2 + ( (v.y>>19)%2 ) * 2 + ( (v.y>>22)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.z>>3)%2 ) * -2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>10)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>11)%2 ) * 2) >= 4 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 6 + ( (v.x>>24)%2 ) * 6 + ( (v.y>>17)%2 ) * -2 + ( (v.x>>14)%2 ) * -2) >= 10 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 2 + ( (v.y>>41)%2 ) * -2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 26 + ( (v.y>>38)%2 ) * -34 + ( (v.y>>23)%2 ) * 38 + ( (v.y>>45)%2 ) * -2 + ( (v.y>>26)%2 ) * -6 + ( (v.x>>33)%2 ) * 6 + ( (v.x>>61)%2 ) * 6) >= 16 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 6 + ( (v.x>>24)%2 ) * 2 + ( (v.y>>45)%2 ) * 2 + ( (v.y>>17)%2 ) * -6) >= 8 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 248 + ( (v.x>>33)%2 ) * 8 + ( (v.y>>7)%2 ) * 8 + ( (v.x>>32)%2 ) * 8 + ( (v.x>>34)%2 ) * 8 + ( (v.y>>41)%2 ) * -8 + ( (v.x>>24)%2 ) * 0 + ( (v.y>>45)%2 ) * 0) >= 248 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) * 4 + ( (v.y>>4)%2 ) * 12 + ( (v.x>>20)%2 ) * -16 + ( (v.x>>19)%2 ) * 4 + ( (v.y>>12)%2 ) * 4) >= 0 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 14 + ( (v.x>>19)%2 ) * 14 + ( (v.y>>45)%2 ) * 6 + ( (v.x>>60)%2 ) * 6 + ( (v.x>>10)%2 ) * 6 + ( (v.x>>22)%2 ) * 42) >= 18 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>14)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 6 ) <<21;
        aux.x |=(unsigned long long) ( ( 0 * 32 + ( (v.y>>8)%2 ) * -4 + ( (v.y>>58)%2 ) * 4 + ( (v.x>>10)%2 ) * 28 + ( (v.y>>57)%2 ) * 4 + ( (v.y>>56)%2 ) * 4) >= 28 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 14 + ( (v.x>>23)%2 ) * 2 + ( (v.y>>14)%2 ) * -2 + ( (v.x>>46)%2 ) * -2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 6 + ( (v.x>>24)%2 ) * 2 + ( (v.y>>15)%2 ) * -10 + ( (v.x>>42)%2 ) * 10 + ( (v.y>>49)%2 ) * 10) >= 8 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2) >= 2 ) <<25;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_18_parte2(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 6 + ( (v.y>>4)%2 ) * 90 + ( (v.y>>25)%2 ) * -154 + ( (v.x>>23)%2 ) * -154 + ( (v.x>>34)%2 ) * 6 + ( (v.x>>8)%2 ) * 34 + ( (v.y>>45)%2 ) * 34 + ( (v.x>>32)%2 ) * 6 + ( (v.x>>35)%2 ) * 6) >= 68 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 0 + ( (v.y>>61)%2 ) * -4 + ( (v.x>>32)%2 ) * 0 + ( (v.y>>56)%2 ) * 4) >= 0 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 0 + ( (v.y>>5)%2 ) * 4 + ( (v.x>>22)%2 ) * 8 + ( (v.y>>41)%2 ) * -8 + ( (v.x>>37)%2 ) * 4) >= 12 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * -4 + ( (v.y>>24)%2 ) * -32 + ( (v.x>>29)%2 ) * -4 + ( (v.y>>57)%2 ) * 28 + ( (v.y>>7)%2 ) * 4 + ( (v.y>>59)%2 ) * -4 + ( (v.z>>5)%2 ) * 4) >= -24 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.y>>58)%2 ) * 8 + ( (v.x>>30)%2 ) * -4 + ( (v.x>>34)%2 ) * -4 + ( (v.y>>24)%2 ) * -4 + ( (v.y>>11)%2 ) * -4) >= -8 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 4 + ( (v.x>>35)%2 ) * 4 + ( (v.y>>7)%2 ) * -4 + ( (v.x>>31)%2 ) * 4 + ( (v.y>>24)%2 ) * -8) >= -4 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 6 + ( (v.y>>56)%2 ) * 2 + ( (v.x>>32)%2 ) * -2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.y>>57)%2 ) * 2 + ( (v.x>>29)%2 ) * 6 + ( (v.x>>33)%2 ) * -2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.y>>58)%2 ) * 2 + ( (v.x>>30)%2 ) * 6 + ( (v.x>>34)%2 ) * -2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 6 + ( (v.y>>59)%2 ) * 2 + ( (v.x>>35)%2 ) * -2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.y>>45)%2 ) * 2) >= 4 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>22)%2 ) * 6 + ( (v.y>>43)%2 ) * 6) >= 4 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.z>>6)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) * 2 + ( (v.y>>4)%2 ) * 6 + ( (v.y>>12)%2 ) * 2) >= 8 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 46 + ( (v.x>>10)%2 ) * -14 + ( (v.x>>41)%2 ) * -50 + ( (v.x>>40)%2 ) * 26 + ( (v.x>>11)%2 ) * -18 + ( (v.y>>14)%2 ) * -6 + ( (v.y>>7)%2 ) * 34) >= -44 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.y>>49)%2 ) * 10 + 0 * 10 + ( (v.x>>39)%2 ) * -22 + ( (v.x>>42)%2 ) * 2 + ( (v.y>>45)%2 ) * -14 + ( (v.y>>14)%2 ) * 2) >= -14 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 2046 + ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.y>>46)%2 ) * 2 + ( (v.y>>51)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>55)%2 ) * 2 + ( (v.x>>61)%2 ) * 2 + ( (v.y>>47)%2 ) * 2) >= 2048 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 126 + ( (v.x>>61)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.y>>46)%2 ) * 2 + ( (v.y>>51)%2 ) * 2 + ( (v.y>>47)%2 ) * 2) >= 128 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 254 + ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>53)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>55)%2 ) * 2) >= 256 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.z>>0)%2 ) * 2 + ( (v.y>>60)%2 ) * 14 + ( (v.x>>23)%2 ) * 2 + ( (v.y>>40)%2 ) * 2) >= 16 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.x>>61)%2 ) * -6 + ( (v.y>>7)%2 ) * -6 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.y>>7)%2 ) * 0 + ( (v.y>>39)%2 ) * 0 + ( (v.x>>39)%2 ) * -4 + ( (v.y>>8)%2 ) * -4 + ( (v.x>>61)%2 ) * -4 + ( (v.y>>27)%2 ) * -4) >= 0 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.y>>26)%2 ) * 2 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + ( (v.y>>40)%2 ) * 6 + ( (v.y>>26)%2 ) * 2) >= 8 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2 + ( (v.y>>26)%2 ) * 2 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<51;

        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_18_parte3(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 0 + ( (v.y>>51)%2 ) * 108 + ( (v.y>>14)%2 ) * -92 + ( (v.x>>52)%2 ) * -92 + ( (v.x>>53)%2 ) * 12 + ( (v.y>>27)%2 ) * 84 + ( (v.x>>54)%2 ) * 12 + ( (v.x>>55)%2 ) * 12) >= 4 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 4 + ( (v.y>>43)%2 ) * 8 + ( (v.x>>37)%2 ) * 8 + ( (v.x>>58)%2 ) * 4 + ( (v.y>>52)%2 ) * 12 + ( (v.y>>26)%2 ) * 12 + ( (v.y>>35)%2 ) * 12 + ( (v.x>>25)%2 ) * 12 + ( (v.x>>16)%2 ) * 12) >= 12 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>54)%2 ) * -30 + ( (v.x>>22)%2 ) * 30 + ( (v.y>>45)%2 ) * 2 + ( (v.y>>12)%2 ) * 2 + ( (v.x>>37)%2 ) * 2) >= 32 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2 + ( (v.x>>28)%2 ) * -14 + ( (v.y>>26)%2 ) * 2 + ( (v.x>>38)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + ( (v.y>>26)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 12 + ( (v.x>>57)%2 ) * -52 + ( (v.x>>48)%2 ) * -64 + ( (v.x>>47)%2 ) * 4 + ( (v.x>>11)%2 ) * 4 + ( (v.x>>61)%2 ) * 12 + ( (v.y>>39)%2 ) * 12) >= -56 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.y>>46)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.y>>13)%2 ) * 2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.y>>26)%2 ) * 86 + ( (v.x>>59)%2 ) * 42 + ( (v.x>>6)%2 ) * -42 + ( (v.x>>37)%2 ) * 126 + ( (v.y>>7)%2 ) * -258 + ( (v.y>>45)%2 ) * 18 + ( (v.x>>61)%2 ) * 18 + ( (v.x>>16)%2 ) * 86 + ( (v.y>>16)%2 ) * -18) >= 80 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 6 + ( (v.x>>23)%2 ) * -2 + ( (v.y>>7)%2 ) * 2) >= 6 ) <<62;
        aux.x |=(unsigned long long) ( ( ( (v.z>>1)%2 ) * 2 + ( (v.y>>45)%2 ) * 2) >= 2 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 6 + ( (v.x>>28)%2 ) * 6 + ( (v.x>>24)%2 ) * 6 + ( (v.x>>17)%2 ) * 6 + ( (v.y>>35)%2 ) * 6 + ( (v.x>>22)%2 ) * 2 + ( (v.y>>45)%2 ) * 2) >= 4 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 2 + ( (v.y>>8)%2 ) * 2 + ( (v.x>>3)%2 ) * 2 + ( (v.x>>29)%2 ) * 2 + ( (v.x>>30)%2 ) * 2) >= 2 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 22 + ( (v.y>>2)%2 ) * 6 + ( (v.y>>49)%2 ) * 6 + ( (v.y>>39)%2 ) * 22 + ( (v.y>>45)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>3)%2 ) * 22) >= 4 ) <<2;
        aux.y |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 0 + ( (v.y>>15)%2 ) * -8 + ( (v.y>>2)%2 ) * -8 + ( (v.y>>1)%2 ) * 8 + ( (v.y>>0)%2 ) * 8) >= 0 ) <<3;
        aux.y |=(unsigned long long) ( ( ( (v.y>>15)%2 ) * 6 + ( (v.y>>5)%2 ) * 6 + ( (v.y>>1)%2 ) * 6 + ( (v.y>>2)%2 ) * 6 + ( (v.y>>4)%2 ) * 18) >= 12 ) <<4;
        aux.y |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 18 + ( (v.y>>4)%2 ) * 18 + ( (v.y>>15)%2 ) * -14 + ( (v.y>>5)%2 ) * -14 + ( (v.y>>2)%2 ) * 18 + ( (v.y>>3)%2 ) * 18) >= 22 ) <<5;
        aux.y |=(unsigned long long) ( ( ( (v.y>>15)%2 ) * 8 + ( (v.y>>3)%2 ) * 8 + ( (v.y>>6)%2 ) * -24 + ( (v.y>>0)%2 ) * -16 + ( (v.y>>2)%2 ) * -16 + ( (v.y>>1)%2 ) * 24) >= -24 ) <<6;
        aux.y |=(unsigned long long) ( ( ( (v.y>>7)%2 ) * 0 + ( (v.y>>60)%2 ) * 8 + ( (v.y>>14)%2 ) * -4 + ( (v.x>>63)%2 ) * 4) >= 8 ) <<7;
        aux.y |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 10 + ( (v.x>>10)%2 ) * 30 + ( (v.y>>9)%2 ) * 30 + ( (v.y>>53)%2 ) * -38 + ( (v.y>>8)%2 ) * 14 + ( (v.y>>14)%2 ) * -26 + ( (v.x>>19)%2 ) * 10) >= 6 ) <<8;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * -4 + ( (v.y>>9)%2 ) * 0 + ( (v.x>>63)%2 ) * 4) >= 0 ) <<9;
        aux.y |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 6 + ( (v.x>>23)%2 ) * 10 + ( (v.y>>5)%2 ) * 6 + ( (v.x>>12)%2 ) * 10 + ( (v.x>>10)%2 ) * 18) >= 22 ) <<10;
        aux.y |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 14 + ( (v.x>>33)%2 ) * 2 + ( (v.y>>11)%2 ) * 2 + ( (v.y>>7)%2 ) * -2) >= 4 ) <<11;
        aux.y |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 12 + ( (v.x>>24)%2 ) * 12 + ( (v.y>>5)%2 ) * 36 + ( (v.x>>0)%2 ) * -4 + ( (v.x>>22)%2 ) * 20 + ( (v.x>>60)%2 ) * -4) >= 24 ) <<12;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 6 + ( (v.x>>3)%2 ) * 102 + ( (v.x>>5)%2 ) * 90 + ( (v.y>>4)%2 ) * 94 + ( (v.y>>5)%2 ) * 94 + ( (v.y>>8)%2 ) * 6 + ( (v.y>>26)%2 ) * 6 + ( (v.x>>16)%2 ) * 6) >= 190 ) <<13;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_18_parte4(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.y |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * -2) >= 0 ) <<14;
        aux.y |=(unsigned long long) ( ( 0 * 90 + ( (v.y>>45)%2 ) * -34 + ( (v.y>>15)%2 ) * -34 + ( (v.y>>23)%2 ) * 6 + ( (v.y>>0)%2 ) * 6 + ( (v.y>>37)%2 ) * 6 + ( (v.x>>16)%2 ) * 6) >= -10 ) <<15;
        aux.y |=(unsigned long long) ( ( 0 * -6 + ( (v.x>>22)%2 ) * -2 + ( (v.z>>4)%2 ) * -2) >= -2 ) <<16;
        aux.y |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + 0 * 2 + ( (v.y>>8)%2 ) * -2 + ( (v.y>>7)%2 ) * -2) >= 4 ) <<17;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<18;
        aux.y |=(unsigned long long) ( ( ( (v.y>>56)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<19;
        aux.y |=(unsigned long long) ( ( ( (v.y>>57)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<20;
        aux.y |=(unsigned long long) ( ( ( (v.y>>58)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<21;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<22;
        aux.y |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2 + ( (v.x>>9)%2 ) * 6 + ( (v.y>>5)%2 ) * 2) >= 8 ) <<23;
        aux.y |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 4 ) <<24;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<25;
        aux.y |=(unsigned long long) ( ( ( (v.y>>36)%2 ) * 192 + 0 * 1728 + ( (v.x>>42)%2 ) * 1728 + ( (v.y>>32)%2 ) * -336 + ( (v.y>>26)%2 ) * -1056 + ( (v.y>>63)%2 ) * -336 + ( (v.y>>38)%2 ) * -576 + ( (v.x>>61)%2 ) * 576 + ( (v.y>>23)%2 ) * 48 + ( (v.x>>21)%2 ) * 192 + ( (v.y>>50)%2 ) * 192 + ( (v.x>>16)%2 ) * 16 + ( (v.x>>33)%2 ) * 0) >= 2592 ) <<26;
        aux.y |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 32 + ( (v.y>>25)%2 ) * 32 + ( (v.y>>29)%2 ) * 32 + ( (v.x>>61)%2 ) * 48 + ( (v.y>>27)%2 ) * 272 + ( (v.y>>35)%2 ) * 208 + ( (v.y>>7)%2 ) * -16 + ( (v.x>>6)%2 ) * -48 + ( (v.x>>23)%2 ) * -48) >= 32 ) <<27;
        aux.y |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 4 + ( (v.y>>30)%2 ) * 4 + ( (v.y>>28)%2 ) * 4 + ( (v.y>>29)%2 ) * -8) >= 0 ) <<28;
        aux.y |=(unsigned long long) ( ( ( (v.y>>29)%2 ) * 4 + ( (v.y>>27)%2 ) * -8 + ( (v.y>>35)%2 ) * 4 + ( (v.y>>28)%2 ) * 4) >= 0 ) <<29;
        aux.y |=(unsigned long long) ( ( ( (v.y>>30)%2 ) * 0 + ( (v.y>>28)%2 ) * -4 + ( (v.y>>27)%2 ) * -4) >= -4 ) <<30;
        aux.y |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.y>>33)%2 ) * 2 + ( (v.x>>2)%2 ) * 2) >= 2 ) <<31;
        aux.y |=(unsigned long long) ( ( ( (v.y>>31)%2 ) * 2) >= 2 ) <<32;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 14 + ( (v.x>>8)%2 ) * 210 + ( (v.y>>8)%2 ) * -210 + ( (v.y>>35)%2 ) * -30 + ( (v.x>>63)%2 ) * -30 + ( (v.y>>5)%2 ) * -30 + ( (v.y>>57)%2 ) * 14 + ( (v.y>>58)%2 ) * 14 + ( (v.y>>56)%2 ) * 14) >= 164 ) <<33;
        aux.y |=(unsigned long long) ( ( ( (v.y>>7)%2 ) * 6 + ( (v.y>>45)%2 ) * 18 + ( (v.y>>60)%2 ) * 18 + ( (v.y>>34)%2 ) * -6 + ( (v.x>>29)%2 ) * -6 + ( (v.x>>12)%2 ) * 6) >= 36 ) <<34;
        aux.y |=(unsigned long long) ( ( ( (v.y>>36)%2 ) * 2 + ( (v.y>>41)%2 ) * 2 + ( (v.y>>44)%2 ) * 2) >= 2 ) <<35;
        aux.y |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.x>>16)%2 ) * 6 + ( (v.x>>19)%2 ) * 2) >= 8 ) <<36;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * -8 + ( (v.x>>36)%2 ) * -8 + ( (v.y>>63)%2 ) * -8 + ( (v.y>>7)%2 ) * -8 + ( (v.y>>61)%2 ) * 16 + ( (v.y>>38)%2 ) * -16) >= -24 ) <<37;
        aux.y |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * -2 + ( (v.y>>8)%2 ) * -2 + ( (v.y>>4)%2 ) * -2) >= 0 ) <<38;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 2) >= 2 ) <<39;
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

__global__ void passo_tlf_18_parte5(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        aux.x = v.x = init_rand[tid].x;
        aux.y = v.y = init_rand[tid].y;
        aux.z = v.z = init_rand[tid].z;

        aux.y |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 10 + ( (v.x>>46)%2 ) * -6 + ( (v.y>>40)%2 ) * -18 + ( (v.y>>14)%2 ) * -6 + ( (v.y>>42)%2 ) * 10) >= -8 ) <<40;
        aux.y |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2) >= 2 ) <<41;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 1022 + ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.y>>46)%2 ) * 2 + ( (v.x>>53)%2 ) * 2 + ( (v.y>>51)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>55)%2 ) * 2) >= 1024 ) <<42;
        aux.y |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 6 + ( (v.x>>24)%2 ) * 6 + ( (v.y>>45)%2 ) * 6 + ( (v.y>>43)%2 ) * -2 + ( (v.y>>15)%2 ) * -2) >= 16 ) <<43;
        aux.y |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 6 + ( (v.y>>5)%2 ) * 14 + ( (v.x>>23)%2 ) * -6 + ( (v.x>>59)%2 ) * 6 + ( (v.x>>17)%2 ) * 6) >= 20 ) <<44;
        aux.y |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 30 + ( (v.x>>8)%2 ) * 210 + ( (v.x>>18)%2 ) * -934 + ( (v.y>>45)%2 ) * -934 + ( (v.y>>18)%2 ) * 270 + ( (v.y>>59)%2 ) * 30 + ( (v.x>>24)%2 ) * 54 + ( (v.y>>16)%2 ) * 162 + ( (v.x>>14)%2 ) * 54 + ( (v.x>>22)%2 ) * 270 + ( (v.x>>29)%2 ) * 30) >= -718 ) <<45;
        aux.y |=(unsigned long long) ( ( ( (v.y>>48)%2 ) * 2) >= 2 ) <<46;
        aux.y |=(unsigned long long) ( ( ( (v.z>>7)%2 ) * 2) >= 2 ) <<47;
        aux.y |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2 + ( (v.z>>0)%2 ) * -2) >= 2 ) <<48;
        aux.y |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 4 + ( (v.y>>49)%2 ) * 0 + ( (v.y>>45)%2 ) * -4) >= 0 ) <<49;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 42 + ( (v.y>>4)%2 ) * 42 + ( (v.y>>35)%2 ) * 42 + ( (v.y>>34)%2 ) * 42 + ( (v.y>>5)%2 ) * 98 + ( (v.y>>3)%2 ) * 98 + ( (v.y>>45)%2 ) * 42 + ( (v.x>>12)%2 ) * 42) >= 182 ) <<50;
        aux.y |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2) >= 2 ) <<51;
        aux.y |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<52;
        aux.y |=(unsigned long long) ( ( 0 * 2 + ( (v.y>>52)%2 ) * 2) >= 2 ) <<53;
        aux.y |=(unsigned long long) ( ( ( (v.y>>49)%2 ) * 18 + ( (v.y>>45)%2 ) * 14 + ( (v.x>>5)%2 ) * 2 + ( (v.y>>54)%2 ) * 2 + ( (v.y>>4)%2 ) * -2) >= 22 ) <<54;
        aux.y |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 28 + ( (v.y>>4)%2 ) * 12 + ( (v.x>>59)%2 ) * 12 + ( (v.x>>37)%2 ) * 12 + ( (v.x>>17)%2 ) * 0 + ( (v.x>>16)%2 ) * 84 + ( (v.y>>17)%2 ) * -84 + ( (v.y>>45)%2 ) * 28) >= 124 ) <<55;
        aux.y |=(unsigned long long) ( ( 0 * 4 + ( (v.x>>8)%2 ) * -8 + ( (v.y>>19)%2 ) * -4 + ( (v.y>>56)%2 ) * 4) >= -4 ) <<56;
        aux.y |=(unsigned long long) ( ( 0 * 4 + ( (v.x>>8)%2 ) * -8 + ( (v.y>>57)%2 ) * 4 + ( (v.y>>20)%2 ) * -4) >= -4 ) <<57;
        aux.y |=(unsigned long long) ( ( 0 * 4 + ( (v.x>>8)%2 ) * -8 + ( (v.y>>20)%2 ) * -4 + ( (v.y>>58)%2 ) * 4) >= -4 ) <<58;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 4 + ( (v.x>>8)%2 ) * -8 + 0 * 4 + ( (v.y>>22)%2 ) * -4) >= -4 ) <<59;
        aux.y |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * 2 + ( (v.x>>62)%2 ) * -6 + ( (v.y>>60)%2 ) * 2) >= 2 ) <<60;
        aux.y |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 4 ) <<61;
        aux.y |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 6 + ( (v.x>>24)%2 ) * -122 + ( (v.y>>45)%2 ) * -122 + ( (v.y>>3)%2 ) * 6 + ( (v.y>>4)%2 ) * 6 + ( (v.x>>10)%2 ) * 6 + ( (v.x>>22)%2 ) * 2 + ( (v.y>>41)%2 ) * -2) >= 2 ) <<62;
        aux.y |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.y>>45)%2 ) * 14 + ( (v.y>>63)%2 ) * -2 + ( (v.y>>62)%2 ) * -2) >= 12 ) <<63;
        aux.z |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2 + ( (v.y>>14)%2 ) * -14 + ( (v.x>>46)%2 ) * -14 + ( (v.x>>43)%2 ) * 2 + ( (v.y>>42)%2 ) * 2) >= 2 ) <<0;
        aux.z |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.z>>1)%2 ) * -2 + ( (v.x>>63)%2 ) * 2) >= 4 ) <<1;
    
        
        estado[tid].x |= aux.x;
        estado[tid].y |= aux.y;
        estado[tid].z |= aux.z;
    }
}

unsigned long long confere_tlf_18(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = init_rand[i].x;
        aux.y = v.y = init_rand[i].y;
        aux.z = v.z = init_rand[i].z;

        aux.x |=(unsigned long long) ( ( ( (v.y>>10)%2 ) * 2) >= 2 ) <<0;
        aux.x |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + 0 * 2 + ( (v.x>>33)%2 ) * 2 + ( (v.x>>31)%2 ) * 2) >= 8 ) <<1;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2) >= 2 ) <<2;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2 + ( (v.y>>4)%2 ) * 2) >= 2 ) <<3;
        aux.x |=(unsigned long long) ( ( ( (v.y>>53)%2 ) * 2) >= 2 ) <<4;
        aux.x |=(unsigned long long) ( ( ( (v.x>>7)%2 ) * 2 + ( (v.x>>57)%2 ) * 2) >= 4 ) <<5;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 12 + ( (v.y>>5)%2 ) * 12 + ( (v.y>>3)%2 ) * 12 + ( (v.x>>39)%2 ) * 36 + ( (v.y>>45)%2 ) * 36 + ( (v.x>>6)%2 ) * 92 + ( (v.y>>14)%2 ) * -128 + ( (v.x>>63)%2 ) * 12) >= -32 ) <<6;
        aux.x |=(unsigned long long) ( ( ( (v.y>>55)%2 ) * 2) >= 2 ) <<7;
        aux.x |=(unsigned long long) ( ( ( (v.y>>20)%2 ) * 2 + ( (v.y>>21)%2 ) * 2 + ( (v.y>>19)%2 ) * 2 + ( (v.y>>22)%2 ) * 2) >= 2 ) <<8;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 2) >= 2 ) <<9;
        aux.x |=(unsigned long long) ( ( ( (v.x>>41)%2 ) * 2 + ( (v.z>>3)%2 ) * -2) >= 2 ) <<10;
        aux.x |=(unsigned long long) ( ( ( (v.x>>10)%2 ) * 2) >= 2 ) <<11;
        aux.x |=(unsigned long long) ( ( ( (v.x>>13)%2 ) * 2 + ( (v.x>>11)%2 ) * 2) >= 4 ) <<12;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2) >= 2 ) <<13;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 6 + ( (v.x>>24)%2 ) * 6 + ( (v.y>>17)%2 ) * -2 + ( (v.x>>14)%2 ) * -2) >= 10 ) <<14;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 2 + ( (v.y>>41)%2 ) * -2) >= 2 ) <<15;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 26 + ( (v.y>>38)%2 ) * -34 + ( (v.y>>23)%2 ) * 38 + ( (v.y>>45)%2 ) * -2 + ( (v.y>>26)%2 ) * -6 + ( (v.x>>33)%2 ) * 6 + ( (v.x>>61)%2 ) * 6) >= 16 ) <<16;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 6 + ( (v.x>>24)%2 ) * 2 + ( (v.y>>45)%2 ) * 2 + ( (v.y>>17)%2 ) * -6) >= 8 ) <<17;
        aux.x |=(unsigned long long) ( ( ( (v.x>>15)%2 ) * 248 + ( (v.x>>33)%2 ) * 8 + ( (v.y>>7)%2 ) * 8 + ( (v.x>>32)%2 ) * 8 + ( (v.x>>34)%2 ) * 8 + ( (v.y>>41)%2 ) * -8 + ( (v.x>>24)%2 ) * 0 + ( (v.y>>45)%2 ) * 0) >= 248 ) <<18;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) * 4 + ( (v.y>>4)%2 ) * 12 + ( (v.x>>20)%2 ) * -16 + ( (v.x>>19)%2 ) * 4 + ( (v.y>>12)%2 ) * 4) >= 0 ) <<19;
        aux.x |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 14 + ( (v.x>>19)%2 ) * 14 + ( (v.y>>45)%2 ) * 6 + ( (v.x>>60)%2 ) * 6 + ( (v.x>>10)%2 ) * 6 + ( (v.x>>22)%2 ) * 42) >= 18 ) <<20;
        aux.x |=(unsigned long long) ( ( ( (v.x>>17)%2 ) * 2 + ( (v.x>>14)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 6 ) <<21;
        aux.x |=(unsigned long long) ( ( 0 * 32 + ( (v.y>>8)%2 ) * -4 + ( (v.y>>58)%2 ) * 4 + ( (v.x>>10)%2 ) * 28 + ( (v.y>>57)%2 ) * 4 + ( (v.y>>56)%2 ) * 4) >= 28 ) <<22;
        aux.x |=(unsigned long long) ( ( ( (v.x>>52)%2 ) * 14 + ( (v.x>>23)%2 ) * 2 + ( (v.y>>14)%2 ) * -2 + ( (v.x>>46)%2 ) * -2) >= 2 ) <<23;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 6 + ( (v.x>>24)%2 ) * 2 + ( (v.y>>15)%2 ) * -10 + ( (v.x>>42)%2 ) * 10 + ( (v.y>>49)%2 ) * 10) >= 8 ) <<24;
        aux.x |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2) >= 2 ) <<25;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 6 + ( (v.y>>4)%2 ) * 90 + ( (v.y>>25)%2 ) * -154 + ( (v.x>>23)%2 ) * -154 + ( (v.x>>34)%2 ) * 6 + ( (v.x>>8)%2 ) * 34 + ( (v.y>>45)%2 ) * 34 + ( (v.x>>32)%2 ) * 6 + ( (v.x>>35)%2 ) * 6) >= 68 ) <<26;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 0 + ( (v.y>>61)%2 ) * -4 + ( (v.x>>32)%2 ) * 0 + ( (v.y>>56)%2 ) * 4) >= 0 ) <<27;
        aux.x |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 0 + ( (v.y>>5)%2 ) * 4 + ( (v.x>>22)%2 ) * 8 + ( (v.y>>41)%2 ) * -8 + ( (v.x>>37)%2 ) * 4) >= 12 ) <<28;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * -4 + ( (v.y>>24)%2 ) * -32 + ( (v.x>>29)%2 ) * -4 + ( (v.y>>57)%2 ) * 28 + ( (v.y>>7)%2 ) * 4 + ( (v.y>>59)%2 ) * -4 + ( (v.z>>5)%2 ) * 4) >= -24 ) <<29;
        aux.x |=(unsigned long long) ( ( ( (v.y>>58)%2 ) * 8 + ( (v.x>>30)%2 ) * -4 + ( (v.x>>34)%2 ) * -4 + ( (v.y>>24)%2 ) * -4 + ( (v.y>>11)%2 ) * -4) >= -8 ) <<30;
        aux.x |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 4 + ( (v.x>>35)%2 ) * 4 + ( (v.y>>7)%2 ) * -4 + ( (v.x>>31)%2 ) * 4 + ( (v.y>>24)%2 ) * -8) >= -4 ) <<31;
        aux.x |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 6 + ( (v.y>>56)%2 ) * 2 + ( (v.x>>32)%2 ) * -2) >= 2 ) <<32;
        aux.x |=(unsigned long long) ( ( ( (v.y>>57)%2 ) * 2 + ( (v.x>>29)%2 ) * 6 + ( (v.x>>33)%2 ) * -2) >= 2 ) <<33;
        aux.x |=(unsigned long long) ( ( ( (v.y>>58)%2 ) * 2 + ( (v.x>>30)%2 ) * 6 + ( (v.x>>34)%2 ) * -2) >= 2 ) <<34;
        aux.x |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 6 + ( (v.y>>59)%2 ) * 2 + ( (v.x>>35)%2 ) * -2) >= 2 ) <<35;
        aux.x |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.y>>45)%2 ) * 2) >= 4 ) <<36;
        aux.x |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>22)%2 ) * 6 + ( (v.y>>43)%2 ) * 6) >= 4 ) <<37;
        aux.x |=(unsigned long long) ( ( ( (v.z>>6)%2 ) * 2) >= 2 ) <<38;
        aux.x |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 2) >= 2 ) <<39;
        aux.x |=(unsigned long long) ( ( ( (v.y>>11)%2 ) * 2 + ( (v.y>>4)%2 ) * 6 + ( (v.y>>12)%2 ) * 2) >= 8 ) <<40;
        aux.x |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 46 + ( (v.x>>10)%2 ) * -14 + ( (v.x>>41)%2 ) * -50 + ( (v.x>>40)%2 ) * 26 + ( (v.x>>11)%2 ) * -18 + ( (v.y>>14)%2 ) * -6 + ( (v.y>>7)%2 ) * 34) >= -44 ) <<41;
        aux.x |=(unsigned long long) ( ( ( (v.y>>49)%2 ) * 10 + 0 * 10 + ( (v.x>>39)%2 ) * -22 + ( (v.x>>42)%2 ) * 2 + ( (v.y>>45)%2 ) * -14 + ( (v.y>>14)%2 ) * 2) >= -14 ) <<42;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 2046 + ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.y>>46)%2 ) * 2 + ( (v.y>>51)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>55)%2 ) * 2 + ( (v.x>>61)%2 ) * 2 + ( (v.y>>47)%2 ) * 2) >= 2048 ) <<43;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 126 + ( (v.x>>61)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.y>>46)%2 ) * 2 + ( (v.y>>51)%2 ) * 2 + ( (v.y>>47)%2 ) * 2) >= 128 ) <<44;
        aux.x |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 254 + ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.x>>53)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>55)%2 ) * 2) >= 256 ) <<45;
        aux.x |=(unsigned long long) ( ( ( (v.z>>0)%2 ) * 2 + ( (v.y>>60)%2 ) * 14 + ( (v.x>>23)%2 ) * 2 + ( (v.y>>40)%2 ) * 2) >= 16 ) <<46;
        aux.x |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.x>>61)%2 ) * -6 + ( (v.y>>7)%2 ) * -6 + ( (v.x>>23)%2 ) * 2) >= 2 ) <<47;
        aux.x |=(unsigned long long) ( ( ( (v.y>>7)%2 ) * 0 + ( (v.y>>39)%2 ) * 0 + ( (v.x>>39)%2 ) * -4 + ( (v.y>>8)%2 ) * -4 + ( (v.x>>61)%2 ) * -4 + ( (v.y>>27)%2 ) * -4) >= 0 ) <<48;
        aux.x |=(unsigned long long) ( ( ( (v.y>>26)%2 ) * 2 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<49;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + ( (v.y>>40)%2 ) * 6 + ( (v.y>>26)%2 ) * 2) >= 8 ) <<50;
        aux.x |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2 + ( (v.y>>26)%2 ) * 2 + ( (v.x>>16)%2 ) * 2) >= 2 ) <<51;
        aux.x |=(unsigned long long) ( ( ( (v.x>>61)%2 ) * 0 + ( (v.y>>51)%2 ) * 108 + ( (v.y>>14)%2 ) * -92 + ( (v.x>>52)%2 ) * -92 + ( (v.x>>53)%2 ) * 12 + ( (v.y>>27)%2 ) * 84 + ( (v.x>>54)%2 ) * 12 + ( (v.x>>55)%2 ) * 12) >= 4 ) <<52;
        aux.x |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 4 + ( (v.y>>43)%2 ) * 8 + ( (v.x>>37)%2 ) * 8 + ( (v.x>>58)%2 ) * 4 + ( (v.y>>52)%2 ) * 12 + ( (v.y>>26)%2 ) * 12 + ( (v.y>>35)%2 ) * 12 + ( (v.x>>25)%2 ) * 12 + ( (v.x>>16)%2 ) * 12) >= 12 ) <<53;
        aux.x |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 2 + ( (v.x>>54)%2 ) * -30 + ( (v.x>>22)%2 ) * 30 + ( (v.y>>45)%2 ) * 2 + ( (v.y>>12)%2 ) * 2 + ( (v.x>>37)%2 ) * 2) >= 32 ) <<54;
        aux.x |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2 + ( (v.x>>28)%2 ) * -14 + ( (v.y>>26)%2 ) * 2 + ( (v.x>>38)%2 ) * 2) >= 2 ) <<55;
        aux.x |=(unsigned long long) ( ( ( (v.x>>16)%2 ) * 2 + ( (v.y>>26)%2 ) * 2) >= 2 ) <<56;
        aux.x |=(unsigned long long) ( ( ( (v.x>>39)%2 ) * 12 + ( (v.x>>57)%2 ) * -52 + ( (v.x>>48)%2 ) * -64 + ( (v.x>>47)%2 ) * 4 + ( (v.x>>11)%2 ) * 4 + ( (v.x>>61)%2 ) * 12 + ( (v.y>>39)%2 ) * 12) >= -56 ) <<57;
        aux.x |=(unsigned long long) ( ( ( (v.y>>46)%2 ) * 2 + ( (v.x>>59)%2 ) * 2) >= 2 ) <<58;
        aux.x |=(unsigned long long) ( ( ( (v.x>>14)%2 ) * 2 + ( (v.x>>22)%2 ) * 2) >= 2 ) <<59;
        aux.x |=(unsigned long long) ( ( ( (v.y>>13)%2 ) * 2) >= 2 ) <<60;
        aux.x |=(unsigned long long) ( ( ( (v.y>>26)%2 ) * 86 + ( (v.x>>59)%2 ) * 42 + ( (v.x>>6)%2 ) * -42 + ( (v.x>>37)%2 ) * 126 + ( (v.y>>7)%2 ) * -258 + ( (v.y>>45)%2 ) * 18 + ( (v.x>>61)%2 ) * 18 + ( (v.x>>16)%2 ) * 86 + ( (v.y>>16)%2 ) * -18) >= 80 ) <<61;
        aux.x |=(unsigned long long) ( ( ( (v.x>>8)%2 ) * 6 + ( (v.x>>23)%2 ) * -2 + ( (v.y>>7)%2 ) * 2) >= 6 ) <<62;
        aux.x |=(unsigned long long) ( ( ( (v.z>>1)%2 ) * 2 + ( (v.y>>45)%2 ) * 2) >= 2 ) <<63;
        aux.y |=(unsigned long long) ( ( ( (v.x>>33)%2 ) * 6 + ( (v.x>>28)%2 ) * 6 + ( (v.x>>24)%2 ) * 6 + ( (v.x>>17)%2 ) * 6 + ( (v.y>>35)%2 ) * 6 + ( (v.x>>22)%2 ) * 2 + ( (v.y>>45)%2 ) * 2) >= 4 ) <<0;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 2 + ( (v.y>>8)%2 ) * 2 + ( (v.x>>3)%2 ) * 2 + ( (v.x>>29)%2 ) * 2 + ( (v.x>>30)%2 ) * 2) >= 2 ) <<1;
        aux.y |=(unsigned long long) ( ( ( (v.x>>60)%2 ) * 22 + ( (v.y>>2)%2 ) * 6 + ( (v.y>>49)%2 ) * 6 + ( (v.y>>39)%2 ) * 22 + ( (v.y>>45)%2 ) * 2 + ( (v.x>>24)%2 ) * 2 + ( (v.x>>3)%2 ) * 22) >= 4 ) <<2;
        aux.y |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 0 + ( (v.y>>15)%2 ) * -8 + ( (v.y>>2)%2 ) * -8 + ( (v.y>>1)%2 ) * 8 + ( (v.y>>0)%2 ) * 8) >= 0 ) <<3;
        aux.y |=(unsigned long long) ( ( ( (v.y>>15)%2 ) * 6 + ( (v.y>>5)%2 ) * 6 + ( (v.y>>1)%2 ) * 6 + ( (v.y>>2)%2 ) * 6 + ( (v.y>>4)%2 ) * 18) >= 12 ) <<4;
        aux.y |=(unsigned long long) ( ( ( (v.y>>0)%2 ) * 18 + ( (v.y>>4)%2 ) * 18 + ( (v.y>>15)%2 ) * -14 + ( (v.y>>5)%2 ) * -14 + ( (v.y>>2)%2 ) * 18 + ( (v.y>>3)%2 ) * 18) >= 22 ) <<5;
        aux.y |=(unsigned long long) ( ( ( (v.y>>15)%2 ) * 8 + ( (v.y>>3)%2 ) * 8 + ( (v.y>>6)%2 ) * -24 + ( (v.y>>0)%2 ) * -16 + ( (v.y>>2)%2 ) * -16 + ( (v.y>>1)%2 ) * 24) >= -24 ) <<6;
        aux.y |=(unsigned long long) ( ( ( (v.y>>7)%2 ) * 0 + ( (v.y>>60)%2 ) * 8 + ( (v.y>>14)%2 ) * -4 + ( (v.x>>63)%2 ) * 4) >= 8 ) <<7;
        aux.y |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * 10 + ( (v.x>>10)%2 ) * 30 + ( (v.y>>9)%2 ) * 30 + ( (v.y>>53)%2 ) * -38 + ( (v.y>>8)%2 ) * 14 + ( (v.y>>14)%2 ) * -26 + ( (v.x>>19)%2 ) * 10) >= 6 ) <<8;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * -4 + ( (v.y>>9)%2 ) * 0 + ( (v.x>>63)%2 ) * 4) >= 0 ) <<9;
        aux.y |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 6 + ( (v.x>>23)%2 ) * 10 + ( (v.y>>5)%2 ) * 6 + ( (v.x>>12)%2 ) * 10 + ( (v.x>>10)%2 ) * 18) >= 22 ) <<10;
        aux.y |=(unsigned long long) ( ( ( (v.x>>30)%2 ) * 14 + ( (v.x>>33)%2 ) * 2 + ( (v.y>>11)%2 ) * 2 + ( (v.y>>7)%2 ) * -2) >= 4 ) <<11;
        aux.y |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 12 + ( (v.x>>24)%2 ) * 12 + ( (v.y>>5)%2 ) * 36 + ( (v.x>>0)%2 ) * -4 + ( (v.x>>22)%2 ) * 20 + ( (v.x>>60)%2 ) * -4) >= 24 ) <<12;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 6 + ( (v.x>>3)%2 ) * 102 + ( (v.x>>5)%2 ) * 90 + ( (v.y>>4)%2 ) * 94 + ( (v.y>>5)%2 ) * 94 + ( (v.y>>8)%2 ) * 6 + ( (v.y>>26)%2 ) * 6 + ( (v.x>>16)%2 ) * 6) >= 190 ) <<13;
        aux.y |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * -2) >= 0 ) <<14;
        aux.y |=(unsigned long long) ( ( 0 * 90 + ( (v.y>>45)%2 ) * -34 + ( (v.y>>15)%2 ) * -34 + ( (v.y>>23)%2 ) * 6 + ( (v.y>>0)%2 ) * 6 + ( (v.y>>37)%2 ) * 6 + ( (v.x>>16)%2 ) * 6) >= -10 ) <<15;
        aux.y |=(unsigned long long) ( ( 0 * -6 + ( (v.x>>22)%2 ) * -2 + ( (v.z>>4)%2 ) * -2) >= -2 ) <<16;
        aux.y |=(unsigned long long) ( ( ( (v.x>>42)%2 ) * 2 + 0 * 2 + ( (v.y>>8)%2 ) * -2 + ( (v.y>>7)%2 ) * -2) >= 4 ) <<17;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<18;
        aux.y |=(unsigned long long) ( ( ( (v.y>>56)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<19;
        aux.y |=(unsigned long long) ( ( ( (v.y>>57)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<20;
        aux.y |=(unsigned long long) ( ( ( (v.y>>58)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<21;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 2 + ( (v.x>>26)%2 ) * 2) >= 4 ) <<22;
        aux.y |=(unsigned long long) ( ( ( (v.y>>3)%2 ) * 2 + ( (v.x>>9)%2 ) * 6 + ( (v.y>>5)%2 ) * 2) >= 8 ) <<23;
        aux.y |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 4 ) <<24;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 2) >= 2 ) <<25;
        aux.y |=(unsigned long long) ( ( ( (v.y>>36)%2 ) * 192 + 0 * 1728 + ( (v.x>>42)%2 ) * 1728 + ( (v.y>>32)%2 ) * -336 + ( (v.y>>26)%2 ) * -1056 + ( (v.y>>63)%2 ) * -336 + ( (v.y>>38)%2 ) * -576 + ( (v.x>>61)%2 ) * 576 + ( (v.y>>23)%2 ) * 48 + ( (v.x>>21)%2 ) * 192 + ( (v.y>>50)%2 ) * 192 + ( (v.x>>16)%2 ) * 16 + ( (v.x>>33)%2 ) * 0) >= 2592 ) <<26;
        aux.y |=(unsigned long long) ( ( ( (v.y>>45)%2 ) * 32 + ( (v.y>>25)%2 ) * 32 + ( (v.y>>29)%2 ) * 32 + ( (v.x>>61)%2 ) * 48 + ( (v.y>>27)%2 ) * 272 + ( (v.y>>35)%2 ) * 208 + ( (v.y>>7)%2 ) * -16 + ( (v.x>>6)%2 ) * -48 + ( (v.x>>23)%2 ) * -48) >= 32 ) <<27;
        aux.y |=(unsigned long long) ( ( ( (v.y>>14)%2 ) * 4 + ( (v.y>>30)%2 ) * 4 + ( (v.y>>28)%2 ) * 4 + ( (v.y>>29)%2 ) * -8) >= 0 ) <<28;
        aux.y |=(unsigned long long) ( ( ( (v.y>>29)%2 ) * 4 + ( (v.y>>27)%2 ) * -8 + ( (v.y>>35)%2 ) * 4 + ( (v.y>>28)%2 ) * 4) >= 0 ) <<29;
        aux.y |=(unsigned long long) ( ( ( (v.y>>30)%2 ) * 0 + ( (v.y>>28)%2 ) * -4 + ( (v.y>>27)%2 ) * -4) >= -4 ) <<30;
        aux.y |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.y>>33)%2 ) * 2 + ( (v.x>>2)%2 ) * 2) >= 2 ) <<31;
        aux.y |=(unsigned long long) ( ( ( (v.y>>31)%2 ) * 2) >= 2 ) <<32;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 14 + ( (v.x>>8)%2 ) * 210 + ( (v.y>>8)%2 ) * -210 + ( (v.y>>35)%2 ) * -30 + ( (v.x>>63)%2 ) * -30 + ( (v.y>>5)%2 ) * -30 + ( (v.y>>57)%2 ) * 14 + ( (v.y>>58)%2 ) * 14 + ( (v.y>>56)%2 ) * 14) >= 164 ) <<33;
        aux.y |=(unsigned long long) ( ( ( (v.y>>7)%2 ) * 6 + ( (v.y>>45)%2 ) * 18 + ( (v.y>>60)%2 ) * 18 + ( (v.y>>34)%2 ) * -6 + ( (v.x>>29)%2 ) * -6 + ( (v.x>>12)%2 ) * 6) >= 36 ) <<34;
        aux.y |=(unsigned long long) ( ( ( (v.y>>36)%2 ) * 2 + ( (v.y>>41)%2 ) * 2 + ( (v.y>>44)%2 ) * 2) >= 2 ) <<35;
        aux.y |=(unsigned long long) ( ( ( (v.x>>11)%2 ) * 2 + ( (v.x>>16)%2 ) * 6 + ( (v.x>>19)%2 ) * 2) >= 8 ) <<36;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * -8 + ( (v.x>>36)%2 ) * -8 + ( (v.y>>63)%2 ) * -8 + ( (v.y>>7)%2 ) * -8 + ( (v.y>>61)%2 ) * 16 + ( (v.y>>38)%2 ) * -16) >= -24 ) <<37;
        aux.y |=(unsigned long long) ( ( ( (v.x>>0)%2 ) * -2 + ( (v.y>>8)%2 ) * -2 + ( (v.y>>4)%2 ) * -2) >= 0 ) <<38;
        aux.y |=(unsigned long long) ( ( ( (v.y>>37)%2 ) * 2) >= 2 ) <<39;
        aux.y |=(unsigned long long) ( ( ( (v.x>>45)%2 ) * 10 + ( (v.x>>46)%2 ) * -6 + ( (v.y>>40)%2 ) * -18 + ( (v.y>>14)%2 ) * -6 + ( (v.y>>42)%2 ) * 10) >= -8 ) <<40;
        aux.y |=(unsigned long long) ( ( ( (v.x>>28)%2 ) * 2) >= 2 ) <<41;
        aux.y |=(unsigned long long) ( ( ( (v.x>>56)%2 ) * 2 + ( (v.x>>4)%2 ) * 1022 + ( (v.x>>49)%2 ) * 2 + ( (v.x>>50)%2 ) * 2 + ( (v.x>>51)%2 ) * 2 + ( (v.y>>46)%2 ) * 2 + ( (v.x>>53)%2 ) * 2 + ( (v.y>>51)%2 ) * 2 + ( (v.x>>54)%2 ) * 2 + ( (v.x>>55)%2 ) * 2) >= 1024 ) <<42;
        aux.y |=(unsigned long long) ( ( ( (v.x>>22)%2 ) * 6 + ( (v.x>>24)%2 ) * 6 + ( (v.y>>45)%2 ) * 6 + ( (v.y>>43)%2 ) * -2 + ( (v.y>>15)%2 ) * -2) >= 16 ) <<43;
        aux.y |=(unsigned long long) ( ( ( (v.x>>37)%2 ) * 6 + ( (v.y>>5)%2 ) * 14 + ( (v.x>>23)%2 ) * -6 + ( (v.x>>59)%2 ) * 6 + ( (v.x>>17)%2 ) * 6) >= 20 ) <<44;
        aux.y |=(unsigned long long) ( ( ( (v.x>>31)%2 ) * 30 + ( (v.x>>8)%2 ) * 210 + ( (v.x>>18)%2 ) * -934 + ( (v.y>>45)%2 ) * -934 + ( (v.y>>18)%2 ) * 270 + ( (v.y>>59)%2 ) * 30 + ( (v.x>>24)%2 ) * 54 + ( (v.y>>16)%2 ) * 162 + ( (v.x>>14)%2 ) * 54 + ( (v.x>>22)%2 ) * 270 + ( (v.x>>29)%2 ) * 30) >= -718 ) <<45;
        aux.y |=(unsigned long long) ( ( ( (v.y>>48)%2 ) * 2) >= 2 ) <<46;
        aux.y |=(unsigned long long) ( ( ( (v.z>>7)%2 ) * 2) >= 2 ) <<47;
        aux.y |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2 + ( (v.z>>0)%2 ) * -2) >= 2 ) <<48;
        aux.y |=(unsigned long long) ( ( ( (v.y>>4)%2 ) * 4 + ( (v.y>>49)%2 ) * 0 + ( (v.y>>45)%2 ) * -4) >= 0 ) <<49;
        aux.y |=(unsigned long long) ( ( ( (v.y>>8)%2 ) * 42 + ( (v.y>>4)%2 ) * 42 + ( (v.y>>35)%2 ) * 42 + ( (v.y>>34)%2 ) * 42 + ( (v.y>>5)%2 ) * 98 + ( (v.y>>3)%2 ) * 98 + ( (v.y>>45)%2 ) * 42 + ( (v.x>>12)%2 ) * 42) >= 182 ) <<50;
        aux.y |=(unsigned long long) ( ( ( (v.y>>52)%2 ) * 2) >= 2 ) <<51;
        aux.y |=(unsigned long long) ( ( ( (v.x>>38)%2 ) * 2) >= 2 ) <<52;
        aux.y |=(unsigned long long) ( ( 0 * 2 + ( (v.y>>52)%2 ) * 2) >= 2 ) <<53;
        aux.y |=(unsigned long long) ( ( ( (v.y>>49)%2 ) * 18 + ( (v.y>>45)%2 ) * 14 + ( (v.x>>5)%2 ) * 2 + ( (v.y>>54)%2 ) * 2 + ( (v.y>>4)%2 ) * -2) >= 22 ) <<54;
        aux.y |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 28 + ( (v.y>>4)%2 ) * 12 + ( (v.x>>59)%2 ) * 12 + ( (v.x>>37)%2 ) * 12 + ( (v.x>>17)%2 ) * 0 + ( (v.x>>16)%2 ) * 84 + ( (v.y>>17)%2 ) * -84 + ( (v.y>>45)%2 ) * 28) >= 124 ) <<55;
        aux.y |=(unsigned long long) ( ( 0 * 4 + ( (v.x>>8)%2 ) * -8 + ( (v.y>>19)%2 ) * -4 + ( (v.y>>56)%2 ) * 4) >= -4 ) <<56;
        aux.y |=(unsigned long long) ( ( 0 * 4 + ( (v.x>>8)%2 ) * -8 + ( (v.y>>57)%2 ) * 4 + ( (v.y>>20)%2 ) * -4) >= -4 ) <<57;
        aux.y |=(unsigned long long) ( ( 0 * 4 + ( (v.x>>8)%2 ) * -8 + ( (v.y>>20)%2 ) * -4 + ( (v.y>>58)%2 ) * 4) >= -4 ) <<58;
        aux.y |=(unsigned long long) ( ( ( (v.y>>59)%2 ) * 4 + ( (v.x>>8)%2 ) * -8 + 0 * 4 + ( (v.y>>22)%2 ) * -4) >= -4 ) <<59;
        aux.y |=(unsigned long long) ( ( ( (v.x>>1)%2 ) * 2 + ( (v.x>>62)%2 ) * -6 + ( (v.y>>60)%2 ) * 2) >= 2 ) <<60;
        aux.y |=(unsigned long long) ( ( ( (v.x>>27)%2 ) * 2 + ( (v.y>>5)%2 ) * 2) >= 4 ) <<61;
        aux.y |=(unsigned long long) ( ( ( (v.y>>5)%2 ) * 6 + ( (v.x>>24)%2 ) * -122 + ( (v.y>>45)%2 ) * -122 + ( (v.y>>3)%2 ) * 6 + ( (v.y>>4)%2 ) * 6 + ( (v.x>>10)%2 ) * 6 + ( (v.x>>22)%2 ) * 2 + ( (v.y>>41)%2 ) * -2) >= 2 ) <<62;
        aux.y |=(unsigned long long) ( ( ( (v.x>>24)%2 ) * 2 + ( (v.y>>45)%2 ) * 14 + ( (v.y>>63)%2 ) * -2 + ( (v.y>>62)%2 ) * -2) >= 12 ) <<63;
        aux.z |=(unsigned long long) ( ( ( (v.x>>44)%2 ) * 2 + ( (v.y>>14)%2 ) * -14 + ( (v.x>>46)%2 ) * -14 + ( (v.x>>43)%2 ) * 2 + ( (v.y>>42)%2 ) * 2) >= 2 ) <<0;
        aux.z |=(unsigned long long) ( ( ( (v.x>>23)%2 ) * 2 + ( (v.z>>1)%2 ) * -2 + ( (v.x>>63)%2 ) * 2) >= 4 ) <<1;

        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}


void preenche_init_rand(ulonglong3 * init_rand, unsigned long long nSim, unsigned int nEq)
{
    
    for(unsigned long long i = 0; i < nSim; i++)
    {
        
        init_rand[i].x = 0;
        init_rand[i].y = 0;
        init_rand[i].z = 0;
        unsigned long rand1 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand2 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand3 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand4 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand5 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand6 = rand()%((unsigned long)(1<<31)-1);
        for(int j = 0; j < nEq && j < 32; j++)
            init_rand[i].x |=(unsigned long long) ((rand1>>j)%2)<<j;
        for(int j = 32; j < nEq && j < 64; j++)
            init_rand[i].x |=(unsigned long long) ((rand2>>(j-32))%2)<<j;
        for(int j = 64; j < nEq && j < 96; j++)
            init_rand[i].y |=(unsigned long long) ((rand3>>(j-64))%2)<<(j-64);
        for(int j = 96; j < nEq && j < 128; j++)
            init_rand[i].y |=(unsigned long long) ((rand4>>(j-96))%2)<<(j-64);
        for(int j = 128; j < nEq && j < 160; j++)
            init_rand[i].z |=(unsigned long long) ((rand5>>(j-128))%2)<<(j-128);
        for(int j = 160; j < nEq && j < 192; j++)
            init_rand[i].z |=(unsigned long long) ((rand6>>(j-160))%2)<<(j-128);
    }
}

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    unsigned int n=0;
    string argv2 ;
    if (argc >= 3) {
        n = atoi(argv[1]);
        argv2 = argv[2];
    } else {
        printf("\nERROR: Number of parameters insufficient\n");
        printf("Example: ./a.out 1 1000\n\n");
        return 1;
    }
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    ulonglong3 *h_init_rand, *h_estado;
    h_init_rand = new ulonglong3[MAX_ESTADO];
    h_estado = new ulonglong3[MAX_ESTADO];

    ulonglong3 *d_init_rand, *d_estado;
    hipMalloc(&d_init_rand,sizeof(ulonglong3)*MAX_ESTADO);
    hipMalloc(&d_estado,sizeof(ulonglong3)*MAX_ESTADO);
    
    unsigned int nEq = 192;
    srand(MAX_ESTADO);
    preenche_init_rand(h_init_rand,MAX_ESTADO,nEq);

    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
    {
        h_estado[i].x = 0;
        h_estado[i].y = 0;
        h_estado[i].z = 0;
    }
        

    hipMemcpy(d_init_rand, h_init_rand, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);
    hipMemcpy(d_estado, h_estado, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);

    switch (n) {
        case 0: passo_bool_10_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_10_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_10_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 1: passo_tlf_10_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_10_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_10_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 2: passo_bool_11_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_11_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_11_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 3: passo_tlf_11_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_11_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_11_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 4: passo_bool_12_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_12_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_12_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 5: passo_tlf_12_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_12_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_12_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        //case 7: passo_bool_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); break;
        //case 8: passo_tlf_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); break;
        case 6: passo_bool_13_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_13_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_13_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 7: passo_tlf_13_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_13_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_13_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 8: passo_bool_14_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_14_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_14_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 9: passo_tlf_14_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_14_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_14_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 10: passo_bool_15_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_15_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_15_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 11:passo_tlf_15_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_15_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_15_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 12:passo_bool_17_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_17_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_17_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 13: passo_tlf_17_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_17_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_17_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_17_parte4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        case 14:passo_bool_18_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_18_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_18_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_18_parte4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_bool_18_parte5<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 15: passo_tlf_18_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_18_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                hipDeviceSynchronize();
                passo_tlf_18_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
                break;
        default: 
                break;
    }
    hipDeviceSynchronize();
    
    hipMemcpy(h_estado, d_estado, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyDeviceToHost);


    unsigned long long i = 0;
    switch (n) {
        case 0:i= confere_bool_10(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 1:i= confere_tlf_10(h_init_rand,h_estado,MAX_ESTADO);
                break;
        case 2: i= confere_bool_11(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 3: i= confere_tlf_11(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 4: i= confere_bool_12(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 5: i= confere_tlf_12(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        //case 7: passo_bool_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        //case 8: passo_tlf_4<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO); 
                break;
        case 6: i= confere_bool_13(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 7: i= confere_tlf_13(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 8: i= confere_bool_14(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 9: i= confere_tlf_14(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 10: i= confere_bool_15(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 11: i= confere_tlf_15(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 12: i= confere_bool_17(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 13: i= confere_tlf_17(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 14: i= confere_bool_18(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        case 15: i= confere_tlf_18(h_init_rand,h_estado,MAX_ESTADO); 
                break;
        default: 
                break;
    }
    
    if(i == MAX_ESTADO)
        cerr << "Resultados da GPU batem com os da CPU\n";
    else
        cerr << "Resultados não batem!\n";
    
    delete [] h_estado;
    delete [] h_init_rand;
    hipFree(d_init_rand);
    hipFree(d_estado);
    hipDeviceReset();
    return 0;
}