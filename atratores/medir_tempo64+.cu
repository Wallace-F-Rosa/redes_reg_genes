#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <ctime>
#include <string>
#include <limits>
#include <stdio.h>
#include <stdlib.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;

//REDE 10
__global__ void passo_bool_10(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_10(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_10(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_10(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 11
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 12
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 13
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 14
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 15
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 17
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

//REDE 18
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long  tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long tid = threadIdx.x + blockIdx.x* blockDim.x;
    ulonglong3 v,aux;
    if(tid < MAX_ESTADO)
    {
        v.x = init_rand[tid].x;
        v.y = init_rand[tid].y;
        v.z = init_rand[tid].z;
    
        estado[tid].x = aux.x;
        estado[tid].y = aux.y;
        estado[tid].z = aux.z;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    ulonglong3 v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux.x = v.x = 0;
        aux.y = v.y = 0;
        aux.z = v.z = 0;
        
        v.x = init_rand[i].x;
        v.y = init_rand[i].y;
        v.z = init_rand[i].z;


        if(aux.x != estado_gpu[i].x || aux.y != estado_gpu[i].y || aux.z != estado_gpu[i].z ){
            cerr << "Estado : " << init_rand[i].x << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i].x << "\n" << "CPU : " << aux.x << "\n";
            return i;
        } 
    }
    
    return nSim;
}


void preenche_init_rand(ulonglong3 * init_rand, unsigned long long nSim, unsigned int nEq)
{
    
    for(unsigned long long i = 0; i < nSim; i++)
    {
        
        init_rand[i].x = 0;
        init_rand[i].y = 0;
        init_rand[i].z = 0;
        unsigned long rand1 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand2 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand3 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand4 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand5 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand6 = rand()%((unsigned long)(1<<31)-1);
        for(int j = 0; j < nEq && j < 32; j++)
            init_rand[i].x |= ((rand1>>j)%2)<<j;
        for(int j = 32; j < nEq && j < 64; j++)
            init_rand[i].x |= ((rand2>>(j-32))%2)<<j;
        for(int j = 64; j < nEq && j < 96; j++)
            init_rand[i].y |= ((rand3>>(j-64))%2)<<(j-64);
        for(int j = 96; j < nEq && j < 128; j++)
            init_rand[i].y |= ((rand4>>(j-96))%2)<<(j-64);
        for(int j = 128; j < nEq && j < 160; j++)
            init_rand[i].z |= ((rand5>>(j-128))%2)<<(j-128);
        for(int j = 160; j < nEq && j < 192; j++)
            init_rand[i].z |= ((rand6>>(j-160))%2)<<(j-128);
    }
}

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[1];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    ulonglong3 *h_init_rand, *h_estado;
    h_init_rand = new ulonglong3[MAX_ESTADO];
    h_estado = new ulonglong3[MAX_ESTADO];

    ulonglong3 *d_init_rand, *d_estado;
    hipMalloc(&d_init_rand,sizeof(ulonglong3)*MAX_ESTADO);
    hipMalloc(&d_estado,sizeof(ulonglong3)*MAX_ESTADO);
    
    unsigned int nEq = 41;
    srand(MAX_ESTADO);
    preenche_init_rand(h_init_rand,MAX_ESTADO,nEq);

    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i].x = 0;
    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i].y = 0;
    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i].z = 0;

    hipMemcpy(d_init_rand, h_init_rand, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);
    hipMemcpy(d_estado, h_estado, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);

    /* passo_tlf_6_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    passo_tlf_6_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    passo_tlf_6_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    hipMemcpy(h_estado, d_estado, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyDeviceToHost); */

    /* passo_bool_6<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize(); */

    /* unsigned long long i = confere_tlf_6(h_init_rand,h_estado,MAX_ESTADO);
    if(i == MAX_ESTADO)
        cerr << "Resultados da GPU batem com os da CPU\n";
    else
        cerr << "Resultados não batem!\n"; */
    
    delete [] h_estado;
    delete [] h_init_rand;
    hipFree(d_init_rand);
    hipFree(d_estado);
    hipDeviceReset();
    return 0;
}