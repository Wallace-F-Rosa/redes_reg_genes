#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <ctime>
#include <string>
#include <limits>
#include <stdio.h>
#include <stdlib.h>

#define CLOCKS_PER_SEC_CPU 1100000000
using namespace std;

//REDE 1
__global__ void passo_bool_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{   
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        estado[tid] = aux;
    }
}

unsigned long long confere_bool_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];


        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}

__global__ void passo_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado, unsigned long long MAX_ESTADO)
{
    unsigned long long v=0,aux=0, tid = threadIdx.x + blockIdx.x* blockDim.x;
    if(tid < MAX_ESTADO)
    {
        v = init_rand[tid];
    
        

    
        estado[tid] = aux;
    }
}

unsigned long long confere_tlf_1(ulonglong3 * init_rand, ulonglong3 * estado_gpu, unsigned long long nSim)
{  
    unsigned long long v,aux;
    for(unsigned long long i = 0; i < nSim; i++)
    {   
        aux = v = 0;
        
        v = init_rand[i];

        

        if(aux != estado_gpu[i]){
            cerr << "Estado : " << init_rand[i] << " Posição :"<<i<<"\n";
            cerr << "GPU : " << estado_gpu[i] << "\n" << "CPU : " << aux << "\n";
            return i;
        } 
    }
    
    return nSim;
}



void preenche_init_rand(ulonglong3 * init_rand, unsigned long long nSim, unsigned int nEq)
{
    
    for(unsigned long long i = 0; i < nSim; i++)
    {
        
        init_rand[i] = 0;
        unsigned long rand1 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand2 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand3 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand4 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand5 = rand()%((unsigned long)(1<<31)-1);
        unsigned long rand6 = rand()%((unsigned long)(1<<31)-1);
        for(int j = 0; j < nEq && j < 32; j++)
            init_rand[i].x |= ((rand1>>j)%2)<<j;
        for(int j = 32; j < nEq && j < 64; j++)
            init_rand[i].x |= ((rand2>>(j-32))%2)<<j;
        for(int j = 64; j < nEq && j < 96; j++)
            init_rand[i].y |= ((rand3>>(j-64))%2)<<(j-64);
        for(int j = 96; j < nEq && j < 128; j++)
            init_rand[i].y |= ((rand4>>(j-96))%2)<<(j-64);
        for(int j = 128; j < nEq && j < 160; j++)
            init_rand[i].z |= ((rand5>>(j-128))%2)<<(j-128);
        for(int j = 160; j < nEq && j < 192; j++)
            init_rand[i].z |= ((rand6>>(j-160))%2)<<(j-128);
    }
}

int main(int argc, char **argv)
{
    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[1];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);
    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);
    dim3 grid((MAX_ESTADO + block.x -1)/block.x);
    ulonglong3 *h_init_rand, *h_estado;
    h_init_rand = new ulonglong3[MAX_ESTADO];
    h_estado = new ulonglong3[MAX_ESTADO];

    ulonglong3 *d_init_rand, *d_estado;
    hipMalloc(&d_init_rand,sizeof(ulonglong3)*MAX_ESTADO);
    hipMalloc(&d_estado,sizeof(ulonglong3)*MAX_ESTADO);
    
    unsigned int nEq = 41;
    srand(MAX_ESTADO);
    preenche_init_rand(h_init_rand,MAX_ESTADO,nEq);

    for(unsigned long long i = 0; i < MAX_ESTADO; i++)
        h_estado[i] = 0;

    hipMemcpy(d_init_rand, h_init_rand, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);
    hipMemcpy(d_estado, h_estado, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyHostToDevice);

    /* passo_tlf_6_parte1<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    passo_tlf_6_parte2<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    passo_tlf_6_parte3<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize();
    hipMemcpy(h_estado, d_estado, sizeof(ulonglong3)*MAX_ESTADO, hipMemcpyDeviceToHost); */

    /* passo_bool_6<<<grid,block>>>(d_init_rand,d_estado,MAX_ESTADO);
    hipDeviceSynchronize(); */

    unsigned long long i = confere_tlf_6(h_init_rand,h_estado,MAX_ESTADO);
    if(i == MAX_ESTADO)
        cerr << "Resultados da GPU batem com os da CPU\n";
    else
        cerr << "Resultados não batem!\n";
    
    delete [] h_estado;
    delete [] h_init_rand;
    hipFree(d_init_rand);
    hipFree(d_estado);
    hipDeviceReset();
    return 0;
}