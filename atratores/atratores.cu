#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <fstream>
#include <string>
#include <vector>

using namespace std;

#define TABLE_SIZE 1024
#define BUCKET_SIZE 200
#define TAM_REDE CONSTANTE_REDE
#define TAM_PESOS CONSTANTE_PESOS
#define TAM_ESTADO (TAM_REDE/32 + (TAM_REDE%32 != 0))

//#define TAM_ESTADO TAM_REDE/32 + (TAM_REDE%32 != 0) //tamanho máximo de cada estado na rede (em bits)


typedef struct
{
    int *peso, *pesoIni, *eqSize, *T, nEq; //Grafo em forma de tabela
} Grafo;

//imprime o grafo lido
ostream & operator << (ostream & out, const Grafo & g)
{
    out << g.nEq <<"\n";
    for(int i = 0; i < g.nEq; i++)
        out << g.eqSize[i] << " ";
    out << "\n";

    int posPeso;
    for(int i = 0; i < g.nEq; i++)
    {
        posPeso = g.pesoIni[i];
        for(int j = 0; j < g.eqSize[i]; j++,posPeso+=2)
        {
            out << g.peso[posPeso] << " " << g.peso[posPeso+1]<<" ";
        }
        out << g.T[i] <<"\n";
    }
    return out;
}

typedef struct
{
    unsigned int *atr; //estados que o atrator contém. no caso de um atrator grande, a cada TAM_ESTADO bits teremos um novo atrator 
    unsigned long long cont; //conta quantos estados caem neste atrator
    int periodo; //período do atrator

} Atrator;

void atrator_tabela_sincrono_cpu(const Grafo &g, Atrator * Tabela, unsigned long long MAX_ESTADO)
{
    bool aleatorio = false;
    if(g.nEq > 40){ aleatorio = true; srand(MAX_ESTADO);}  //rede grande, estados aleatórios   
    unsigned int * s0, *s1;
    s0 = (unsigned int *)malloc(TAM_ESTADO*sizeof(unsigned int));
    s1 = (unsigned int *)malloc(TAM_ESTADO*sizeof(unsigned int));
    for(unsigned long long estado = 0; estado < MAX_ESTADO; estado++)
    {
        //variáveis necessárias para realizar o passo
        
        for(int i = 0; i < TAM_ESTADO; i++)
            s0[i] = s1[i] = 0; 

        
        //inicializando estado inicial(o bit mais alto representa o vértice (nEq-1) e o bit mais baixo representa o vértice 0)
        if(aleatorio)
        {
            for(int i = 0; i < TAM_ESTADO; i++)
                s0[i] = (unsigned int)rand() % UINT_MAX; //preenche o estado com numeros aleatórios
        }
        else
        {
            for(int i = 0; i < g.nEq; i++)
            {
                int var = g.nEq -1 - i; //variável desejada
                int posAtr = TAM_ESTADO - var/32 - (var%32!=0); //posição de atr onde se encontra o bit da variável desejada
                s0[posAtr] |= ((estado >> i)%2) << (var%32);
            }
        }
        

        for(int i = 0; i < TAM_ESTADO; i++)
            s1[i] = s0[i];

         //variaveis auxiliares
         unsigned int * newEstado = (unsigned int *)malloc(sizeof(unsigned int)*TAM_ESTADO);
        
         bool diferente = false;
        
        //procurando o atrator onde os estados caem
        do
        {
            //s0 anda um passo

            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < g.nEq; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s0[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-i; //variavel que será atualizada no novo estado
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar - 32*posAtr);
            }
            //atualiza s0
            for(int i = 0; i < TAM_ESTADO; i++) s0[i] = newEstado[i];

            //s1 anda dois passos
            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < g.nEq; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar - 32*posAtr);
            }
            //atualiza s1
            for(int i = 0; i < TAM_ESTADO; i++) s1[i] = newEstado[i];

            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < g.nEq; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar - 32*posAtr);
            }
            //atualiza s1
            for(int i = 0; i < TAM_ESTADO; i++) s1[i] = newEstado[i];

            //testando se s0 != s1
            diferente = false;
            for(int i = 0; i < TAM_ESTADO; i++) if(s0[i] != s1[i]){ diferente = true; break;}
        }while(diferente);

        int upperBit = -1, lowerBit = -1, hash = 0;
        #pragma unroll
        for(int i = 0; i < g.nEq; i++)
        {
            int bitVar = g.nEq -1 - i;
            int posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
            bool bit = (s1[posAtr] >>(bitVar%32))% 2;
            if(lowerBit == -1 && (bit == 1))
                lowerBit = i+1;
            
            if(bit == 1)
            {
                upperBit = i + 1;
                hash += upperBit;
            }
        }
        if(upperBit == -1) hash = 0;
        else hash = (hash)/1024 + upperBit;

        //insere o estado na tabela hash :
        //assert(hash >= TABLE_SIZE || hash < 0);
        if(hash >= TABLE_SIZE || hash < 0){
            printf("Estado : ");
            for(int i = 0; i < TAM_ESTADO; i++) printf("%d",s1[i]);
            printf(" , hash : %d\n",hash);
            return;
        }

        //salvar o atrator na tabela
        bool igual = true;
        for(int i = 0; i < TAM_ESTADO; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;}
        if(igual)
        {
            Tabela[hash].cont+=1;//se dois estados caem no mesmo balde, soma mais um no estado
        }
        else
        {
            //procura um balde vazio desde que o estado encontrado nao seja igual ao dos baldes encontrados no caminho
            while(Tabela[hash].cont != 0 && (!igual))
            {   
                hash++;
                igual = true;
                for(int i = 0; i < TAM_ESTADO; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;} 
                
            } 
            if(!igual)
                for(int i = 0; i < TAM_ESTADO; i++) Tabela[hash].atr[i]=s1[i];
            Tabela[hash].cont+=1;//se dois estados caem no mesmo balde, soma mais um no estado
            Tabela[hash].periodo=1;
        }
        
    }

    free(s0);
    free(s1);
    return;
}

/* __device__ void passo(unsigned int * estado, unsigned int TAM_ESTADO, const Grafo &g)
{
    unsigned int * newEstado;
    newEstado = (unsigned int *)malloc(sizeof(unsigned int)*TAM_ESTADO);

    //passo
    for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

    //calcula novo estado
    for(int i = 0; i < g.nEq; i++)
    {
        int bitVar = 0; //bit no qual a variavel i é representada
        int posAtr = 0; //posição do bit no vetor do estado
        int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
        //aplicando a tlf
        for(int j = 0; j < eqsize;  j++, pos += 2){
            bitVar = (g.nEq-1)-g.peso[pos];
            posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
            sum_prod += ((estado[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
        }
        bitVar = (g.nEq-1)-i; //variavel que será atualizada no novo estado
        newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
    }
    //atualiza s0
    for(int i = 0; i < TAM_ESTADO; i++) estado[i] = newEstado[i];

    free(newEstado);

} */


__global__ void atrator_tabela_sincrono(hiprandState * curstate, const Grafo g, Atrator *Tabela, unsigned long long MAX_THREAD_ID)
{
    //o id da tread é calculado para evitar que as threads excedentes sejam utilizadas
    unsigned long long idx = blockDim.x*blockIdx.x + threadIdx.x;
    //calculo do estado : estado = blockDim.x*blockIdx.x + threadIdx.x 
    if(idx < MAX_THREAD_ID)
    {
        //copia do grafo na memória shared
        __shared__ int eqSize[TAM_REDE];
        __shared__ int pesoIni[TAM_REDE];
        __shared__ int T[TAM_REDE];
        __shared__ int peso[TAM_PESOS*2];

        //inicializa a cópía do grafo na memória shared
        if(threadIdx.x<TAM_REDE)
        {
            eqSize[threadIdx.x] = g.eqSize[threadIdx.x];
            pesoIni[threadIdx.x] = g.pesoIni[threadIdx.x];
            T[threadIdx.x] = g.T[threadIdx.x];
        }
        if(TAM_PESOS <= 1024)
        {
            if(threadIdx.x<TAM_PESOS)
            {
                peso[threadIdx.x*2] = peso[threadIdx.x*2];
                peso[threadIdx.x*2+1] = peso[threadIdx.x*2+1]; 
            }
        }
        else
        {
            int total = TAM_PESOS;
            while(total >= blockDim.x)
            {
                peso[threadIdx.x*2] = peso[threadIdx.x*2];
                peso[threadIdx.x*2+1] = peso[threadIdx.x*2+1];
                total = total - blockDim.x;
            }
            if(threadIdx.x<total)
            {
                peso[threadIdx.x*2] = peso[threadIdx.x*2];
                peso[threadIdx.x*2+1] = peso[threadIdx.x*2+1]; 
            }
        }
        __syncthreads();

        //rede grande, estado aleatório
        if(TAM_REDE > 25)
            hiprand_init(idx, idx, 0, curstate + idx);//inicia a seed


        //variáveis necessárias para realizar o passo
        unsigned int s0[TAM_ESTADO], s1[TAM_ESTADO];
        
        //testa se a memoria foi alocada corretamente
        /* assert(s0 != NULL);
        assert(s1 != NULL); */

        for(int i = 0; i < TAM_ESTADO; i++)
            s0[i] = s1[i] = 0; 
        
        if(TAM_REDE > 25)
        {
            for(int i = 0; i < TAM_ESTADO; i++)
               s1[i] = s0[i] = hiprand(curstate + idx);
        }
        else
        {
            //inicializando estado inicial(o bit mais alto representa o vértice (nEq-1) e o bit mais baixo representa o vértice 0)
            for(int i = 0; i < TAM_REDE; i++)
            {
                int var = TAM_REDE -1 - i; //variável desejada
                int posAtr = TAM_ESTADO - var/32 - (var%32!=0) ; //posição de atr onde se encontra o bit da variável desejada
                s0[posAtr] |= ((idx >> i)%2) << (var%32);
                s1[posAtr] |= ((idx >> i)%2) << (var%32);
            }
        }
        
        
        //variaveis auxiliares
        unsigned int newEstado[TAM_ESTADO];
        //assert(newEstado != NULL); //testa se a nenoria foi alocada corretamente

        //testando se s0 != s1
        bool diferente = false;
        
        //procurando o atrator onde os estados caem
        do
        {
            //s0 anda um passo
            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < TAM_REDE; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = pesoIni[i] , eqsize = eqSize[i], Teq = T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (TAM_REDE-1)-peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s0[posAtr] >>(bitVar%32))% 2)*peso[pos+1];
                }
                bitVar = (TAM_REDE-1)-i; //variavel que será atualizada no novo estado
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza s0
            for(int i = 0; i < TAM_ESTADO; i++) s0[i] = newEstado[i];

            //passo(s0,TAM_ESTADO,sh_g);

            //s1 anda dois passos
            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < TAM_REDE; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = pesoIni[i] , eqsize = eqSize[i], Teq = T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (TAM_REDE-1)-peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*peso[pos+1];
                }
                bitVar = (TAM_REDE-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza s1
            for(int i = 0; i < TAM_ESTADO; i++) s1[i] = newEstado[i];

            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < TAM_REDE; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = pesoIni[i] , eqsize = eqSize[i], Teq = T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (TAM_REDE-1)-peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*peso[pos+1];
                }
                bitVar = (TAM_REDE-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza s1
            for(int i = 0; i < TAM_ESTADO; i++) s1[i] = newEstado[i];
            /* passo(s1,TAM_ESTADO,sh_g);
            passo(s1,TAM_ESTADO,sh_g); */

            //testando se s0 != s1
            diferente = false;
            for(int i = 0; i < TAM_ESTADO; i++) if(s0[i] != s1[i]){ diferente = true; break;}
        }while(diferente);
        __syncthreads();
        //Neste ponto s0 == s1



        //salva na memória global sequencialmente

	
        int upperBit = -1, lowerBit = -1, hash = 0;
        #pragma unroll
        for(int i = 0; i < TAM_REDE; i++)
        {
            int bitVar = TAM_REDE -1 - i;
            int posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
            bool bit = (s1[posAtr] >> (bitVar%32))% 2;
            if(lowerBit == -1 && (bit == 1))
                lowerBit = i+1;
            
            if(bit == 1)
            {
                upperBit = i + 1;
                hash += upperBit;
            }
        }
        if(upperBit == -1) hash = 0;
        else hash = (hash)/1024 + upperBit;
        
        assert(hash < TABLE_SIZE);
        //insere o estado na tabela hash :
        if(hash >= TABLE_SIZE || hash < 0){
            printf("Erro ao calcular hash\n");
            printf("Estado : ");
            for(int i = 0; i < TAM_ESTADO; i++) printf("%X",s1[i]);
            printf(" , hash : %d\n",hash);
            return;
        }

        //confere se o balde já está cheio e acha um balde vazio
        bool igual = true;
        for(int i = 0; i < TAM_ESTADO; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;}
        if(igual)
        {
            atomicAdd((unsigned long long *)&(Tabela[hash].cont), (unsigned long long)1);//se dois estados caem no mesmo balde, soma mais um no estado
        }
        else
        {
            //procura um balde vazio desde que o estado encontrado nao seja igual ao dos baldes encontrados no caminho
            while(Tabela[hash].cont != 0 && (!igual))
            {   
                hash++;
                igual = true;
                for(int i = 0; i < TAM_ESTADO; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;}
            }
            __syncthreads();
            
            for(int i = 0; i < TAM_ESTADO; i++){ Tabela[hash].atr[i]=s1[i];} 
            atomicAdd((unsigned long long *)&(Tabela[hash].cont), (unsigned long long)1);
            Tabela[hash].periodo=1;
        }    		
        __syncthreads();		
     
        
        /* if(threadIdx.x == 0)
        {
            for(int i = 0; i < TAM_ESTADO; i++) printf("%X",s1[i]);
            printf("\n");
        } */
        __syncthreads();
         
    }
}

__global__ void gen_rand(hiprandState * curstate, const Grafo g, const unsigned long long MAX_TREAD_ID)
{
    unsigned long long idx = blockDim.x*blockIdx.x + threadIdx.x;

    
    if(idx < MAX_TREAD_ID)
    {
        //inicia a seed
        hiprand_init(idx, idx, 0, curstate + idx);
        unsigned int * estado;
        estado = (unsigned int *)malloc(sizeof(unsigned int)*TAM_ESTADO);
        for(int i = 0; i < TAM_ESTADO; i++) estado[i] = 0;  
        
       
        
        for(int i = 0; i < TAM_ESTADO; i++)
            estado[i] = hiprand(curstate + idx);

        /* for(int i = 0; i < TAM_ESTADO; i++)
        {
            float randf = hiprand_uniform(curstate + idx);
            unsigned int max = (2<<31)-1;
            randf *= ( max + 0.999999);
            randf += 0;
            unsigned int num = (unsigned int)truncf(randf);
            estado[i] = num;
        } */
        
        /* for(int i = 0; i < g.nEq; i++)
        {
            int var = g.nEq -1 - i; //variável desejada
            int posAtr = var/32 + (var%32!=0) - 1; //posição de atr onde se encontra o bit da variável desejada

            float randf = hiprand_uniform(curstate + idx);
            randf *= (1 - 0 + 0.999999);
            randf += 0;
            unsigned int bit = (unsigned int)truncf(randf);
            estado[posAtr] |= bit << (var - 32*posAtr);
        } */
        
        
        for(unsigned long long id = 0; id < MAX_TREAD_ID; id++)
        {
            __syncthreads();
            if(id == idx)
            {
                for(int i = 0; i < TAM_ESTADO; i++) printf("%u",estado[i]);
                printf("\n");
                __syncthreads();
            }
            __syncthreads();
        }
        __syncthreads();
        
        /*
        free(estado); */

    }

    return;

}


Atrator * junta_atrator(Atrator * Tabela, const Grafo &g, const string tec)
{
    //ajeita tabela da GPU
    /* if(tec == "GPU")
    {
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            if(Tabela[i].cont != 0)
            {
                bool igual;
                int j;
                for(j = i+1; j < TABLE_SIZE; j++)
                {
                    igual = false;
                    if(Tabela[j].cont != 0)
                    {
                        igual = true;
                        for(int z = 0; z < TAM_ESTADO; z++) if(Tabela[i].atr[z] != Tabela[j].atr[z]){ igual = false; break;}
                    }
                    if(igual)
                    {
                        Tabela[i].cont += Tabela[j].cont;
                        Tabela[j].cont = 0;
                    }
                }
                
            }
        }
    } */


    

    Atrator * resultado = new Atrator[TABLE_SIZE];

    //inicializa tabela
    for(int i = 0; i < TABLE_SIZE; i++) 
    {
        resultado[i].cont = 0;
        resultado[i].periodo = 0;
    }

    //variaveis auxiliares
    unsigned int * newEstado = new unsigned int[TAM_ESTADO];
    unsigned int * aux = new unsigned int[TAM_ESTADO]; //auxiliar para saber os próximos estados do atratror

    
    //junta os atratores da Tabela no resultado
    for(int i = 0; i < TABLE_SIZE; i++)
    {
        if(Tabela[i].cont != 0)
        {
            resultado[i].periodo = Tabela[i].periodo;
            resultado[i].cont = Tabela[i].cont;
            vector<unsigned int> atr(TAM_ESTADO); // atrator completo que será armazenado em resultado[i].atr

            //zera a posição visitada na Tabela
            Tabela[i].cont = 0;

            for(int j = 0; j < TAM_ESTADO; j++) aux[j] = atr[j] = Tabela[i].atr[j];//inicialização de aux e atr

            //aplica um passo em atr para encontrar o próximo estado do atrator
           
            //passo
            for(int j = 0; j < TAM_ESTADO; j++) newEstado[j]=0; //zera o newEstado

            //calcula novo estado
            for(int j = 0; j < g.nEq; j++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[j] , eqsize = g.eqSize[j], Teq = g.T[j];
                //aplicando a tlf
                for(int z = 0; z < eqsize;  z++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((aux[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-j;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza aux
            for(int j = 0; j < TAM_ESTADO; j++) aux[j] = newEstado[j];
            
            //testar se aux != Tabela[i].atr[j]
            bool diferente = false;
            for(int j = 0; j < TAM_ESTADO; j++) if(aux[j] != Tabela[i].atr[j]){ diferente = true; break;}
            while(diferente)
            {

               /*  for(int j = 0; j < TAM_ESTADO; j++) printf("%X",aux[j]);
                cout << endl; */

                //calcula o hash do estado atual
                int upperBit = -1, lowerBit = -1, hash = 0;
                #pragma unroll
                for(int j = 0; j < g.nEq; j++)
                {
                    int bitVar = g.nEq -1 - j;
                    int posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    bool bit = (aux[posAtr] >>(bitVar%32))% 2;
                    if(lowerBit == -1 && (bit == 1))
                        lowerBit = j+1;
                    
                    if(bit == 1)
                    {
                        upperBit = j + 1;
                        hash += upperBit;
                    }
                }
                if(upperBit == -1) hash = 0;
                else hash = (hash)/1024 + upperBit;
                    

                bool igual = true;
                assert(hash < TABLE_SIZE);

                //procurando se o estado encontrado no ciclo esta na tabela
                for(int j = 0; j < TAM_ESTADO; j++) if(Tabela[hash].atr[j] != aux[j]) {igual = false; break;}
                //procurando se o estado encontrado no ciclo esta na tabela
                if(hash < TABLE_SIZE) //se o estado estiver na tabela extraimos seus dados
                {
                    while(hash < TABLE_SIZE && !igual)
                    {
                        hash++;
                        igual = true;
                        if(hash < TABLE_SIZE) for(int j = 0; j < TAM_ESTADO; j++) if(Tabela[hash].atr[j] != aux[j]) {igual = false; break;}
                    } 
                }
                

                for(int j = 0; j < TAM_ESTADO; j++) atr.push_back(aux[j]);
                resultado[i].periodo++;
                
                if(hash < TABLE_SIZE) //se o estado estiver na tabela extraimos seus dados
                {
                    resultado[i].cont += Tabela[hash].cont;
                    Tabela[hash].cont = 0; // já contamos esse estado em algum atrator 
                    assert(Tabela[hash].cont == 0);
                }

                //da um passo em aux
                for(int j = 0; j < TAM_ESTADO; j++) newEstado[j]=0; //zera o newEstado

                //calcula novo estado
                for(int j = 0; j < g.nEq; j++)
                {
                    int bitVar = 0; //bit no qual a variavel i é representada
                    int posAtr = 0; //posição do bit no vetor do estado
                    int sum_prod =0, pos = g.pesoIni[j] , eqsize = g.eqSize[j], Teq = g.T[j];
                    //aplicando a tlf
                    for(int z = 0; z < eqsize;  z++, pos += 2){
                        bitVar = (g.nEq-1)-g.peso[pos];
                        posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                        sum_prod += ((aux[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                    }
                    bitVar = (g.nEq-1)-j;
                    newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
                }
                //atualiza aux
                for(int j = 0; j < TAM_ESTADO; j++) aux[j] = newEstado[j];


                //aux != Tabela[i].atr ?
                diferente = false;
                for(int j = 0; j < TAM_ESTADO; j++) if(aux[j] != Tabela[i].atr[j]){ diferente = true; break;}
            }


            //copia o atrator completo para a tabela de resultado
            resultado[i].atr = new unsigned int[atr.size()];
            for(int j = 0; j < atr.size(); j++) resultado[i].atr[j] = atr[j];


        }

    }
    
    delete [] aux;
    delete [] newEstado;

    return resultado;
}


__global__ void testekernel()
{
    if(threadIdx.x == 0)
        printf("Bloco %d Thread %d\n",blockIdx.x,threadIdx.x);
}


int main(int argc, char **argv)
{

    int nEq; // numero de equações

    //ler o grafo pelo arquivo de entrada
    filebuf fb;
    if(!fb.open(argv[1],ios::in)) 
    {
        cerr << "Erro ao abrir arquivo de entrada " << argv[1] <<endl;
        exit(0);
    }

    istream is(&fb);
    is >> nEq; //numero de equações, tamanho da rede
    
    Grafo g, d_g; //grafo da cpu e da gpu 
    g.eqSize = new int[nEq];
    g.pesoIni = new int[nEq];
    g.T = new int[nEq];
    g.nEq = nEq;

    hipMalloc((int **)&(d_g.eqSize),sizeof(int)*nEq);
    hipMalloc((int **)&(d_g.pesoIni),sizeof(int)*nEq);
    hipMalloc((int **)&(d_g.T),sizeof(int)*nEq);

    //ler os tamanhos das equações a seguir
    int nPesos = 0; // numero de pesos

    for(int i = 0; i < TAM_REDE; i++)
    {
        is >> g.eqSize[i];
        nPesos+= g.eqSize[i];
    }

    nPesos *= 2;
    g.peso = new int [nPesos];
    hipMalloc((int **)&(d_g.peso),nPesos*sizeof(int));

    int posPeso = 0; //posição dos pesos
    for(int i = 0; i < TAM_REDE; i++)
    {
        int var=0, peso=0, T=0; // variavel da equação, peso e threshold
        g.pesoIni[i] = posPeso;
        for(int j = 0; j < g.eqSize[i]; j++,posPeso+=2)
        {
            is >> var >> peso;
            g.peso[posPeso] = var;
            g.peso[posPeso + 1] = peso;
        }
        is >> T;
        g.T[i] = T;
    }
    fb.close(); // fim da leitura

    
    //inicializando objeto que vai para a gpu
    hipMemcpy(d_g.eqSize, g.eqSize, sizeof(int)*nEq, hipMemcpyHostToDevice);
    hipMemcpy(d_g.pesoIni, g.pesoIni, sizeof(int)*nEq, hipMemcpyHostToDevice);
    hipMemcpy(d_g.T, g.T, sizeof(int)*nEq, hipMemcpyHostToDevice);
    hipMemcpy(d_g.peso, g.peso, sizeof(int)*nPesos, hipMemcpyHostToDevice);
    d_g.nEq = g.nEq;
    //hipMemcpy(&(d_g.nEq), &(g.nEq), sizeof(int), hipMemcpyHostToDevice);


    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);

    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[2];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);

    dim3 grid((MAX_ESTADO + block.x -1)/block.x);

    //Alocando tabela de atratores para cpu e para gpu
    Atrator *Tabela,*d_Tabela;
    Tabela = new Atrator[TABLE_SIZE];
    hipMalloc((Atrator**)&d_Tabela, TABLE_SIZE*sizeof(Atrator));
    unsigned int * d_atr[TABLE_SIZE];
    
    for(int i = 0; i < TABLE_SIZE; i++)
    {
        Tabela[i].atr = new unsigned int[TAM_ESTADO];
        Tabela[i].cont = 0;
        Tabela[i].periodo = 0;

        for(int j = 0; j < TAM_ESTADO; j++) Tabela[i].atr[j] = 0;
        
        hipMalloc((void **)&(d_atr[i]), sizeof(unsigned int)*TAM_ESTADO);
        hipMemcpy(d_atr[i], Tabela[i].atr, sizeof(unsigned int)*TAM_ESTADO, hipMemcpyHostToDevice);

        hipMemcpy(&(d_Tabela[i]), &(Tabela[i]), sizeof(Atrator), hipMemcpyHostToDevice);
        hipMemcpy(&(d_Tabela[i].atr), &(d_atr[i]), sizeof(unsigned int*), hipMemcpyHostToDevice);
    }
    

    double MB_por_Estado = (9.0/40000.0) * MAX_ESTADO;
    size_t HeapSize = MB_por_Estado * 1024 * 1024;

    string tec = argv[3];
    if(tec == "GPU")
    {
        hiprandState * d_state;
        hipMalloc((void **)&d_state, sizeof(hiprandState) * MAX_ESTADO);
        //hipDeviceSetLimit(hipLimitMallocHeapSize, HeapSize);
    
        atrator_tabela_sincrono<<<grid,block>>>(d_state,d_g,d_Tabela,MAX_ESTADO);
        hipDeviceSynchronize();
        //traz o resultado da GPU
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            hipMemcpy(Tabela[i].atr, d_atr[i], TAM_ESTADO*sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&(Tabela[i].cont), &(d_Tabela[i].cont), sizeof(unsigned long long), hipMemcpyDeviceToHost);
            hipMemcpy(&(Tabela[i].periodo), &(d_Tabela[i].periodo), sizeof(unsigned long long), hipMemcpyDeviceToHost);
        }
    }
    else if(tec == "CPU") atrator_tabela_sincrono_cpu(g,Tabela,MAX_ESTADO);
    else
    {
        cerr << "Argumento "<<tec<<" inválido : escolha entre GPU ou CPU!"<<endl;
        //desalocando memória
        for(int i = 0; i < TABLE_SIZE; i++){ delete [] Tabela[i].atr; hipFree(d_atr[i]);}  
        delete [] Tabela;
        hipFree(d_Tabela);

        delete [] g.peso;
        delete [] g.pesoIni;
        delete [] g.eqSize;
        delete [] g.T;
        hipFree(d_g.peso);
        hipFree(d_g.pesoIni);
        hipFree(d_g.eqSize);
        hipFree(d_g.T);
        return 0;
    }

    //junta os atratores
    Atrator * resultado ;//= Tabela;
    resultado = junta_atrator(Tabela,g,tec);
    //return 0;
    //imprimindo resultado
    for(int i = 0; i < TABLE_SIZE; i++)
    {
        if(resultado[i].periodo != 0)
        {
            printf("%u ",resultado[i].periodo);
            for(int j = 0; j < resultado[i].periodo; j++)
            {
                for(int z = 0; z < TAM_ESTADO; z++) printf("%X",resultado[i].atr[j*TAM_ESTADO + z]);
                printf(" ");
            }
                
            //binario
              for(int j = 0; j < g.nEq; j++)
            {
                int var = g.nEq -1 - j; //variável desejada
                int posAtr = var/32 + (var%32!=0) - 1; //posição de atr onde se encontra o bit da variável desejada
                bool valor = (Tabela[i].atr[posAtr]>>(var-32*posAtr))%2;
                printf("%d",valor);
            } 
            printf("%llu\n", resultado[i].cont); 
        }   
    }
    
    //desalocando memória
    for(int i = 0; i < TABLE_SIZE; i++){ hipFree(d_atr[i]); free(Tabela[i].atr);}  
    free(Tabela);
    free(resultado);
    hipFree(d_Tabela);

    delete [] g.peso;
    delete [] g.pesoIni;
    delete [] g.eqSize;
    delete [] g.T;
    hipFree(d_g.peso);
    hipFree(d_g.pesoIni);
    hipFree(d_g.eqSize);
    hipFree(d_g.T);

    //reinicia o device
    hipDeviceReset();
    return 0;
}
