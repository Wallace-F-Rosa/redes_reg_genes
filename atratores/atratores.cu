#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <fstream>
#include <string>
#include <vector>

using namespace std;

#define TABLE_SIZE 2048
#define BUCKET_SIZE 200
#define TAM_REDE CONSTANTE_REDE
#define TAM_PESOS CONSTANTE_PESOS
#define TAM_ESTADO (TAM_REDE/32 + (TAM_REDE%32 != 0))
#define CLOCK_PER_SEC_CPU 1007.780*1000000
#define CLOCK_PER_SEC_GPU 1506*1000000

//#define TAM_ESTADO TAM_REDE/32 + (TAM_REDE%32 != 0) //tamanho máximo de cada estado na rede (em bits)


typedef struct
{
    int *peso, *pesoIni, *eqSize, *T, nEq; //Grafo em forma de tabela
} Grafo;

//imprime o grafo lido
ostream & operator << (ostream & out, const Grafo & g)
{
    out << g.nEq <<"\n";
    for(int i = 0; i < g.nEq; i++)
        out << g.eqSize[i] << " ";
    out << "\n";

    int posPeso;
    for(int i = 0; i < g.nEq; i++)
    {
        posPeso = g.pesoIni[i];
        for(int j = 0; j < g.eqSize[i]; j++,posPeso+=2)
        {
            out << g.peso[posPeso] << " " << g.peso[posPeso+1]<<" ";
        }
        out << g.T[i] <<"\n";
    }
    return out;
}

typedef struct
{
    unsigned int *atr; //estados que o atrator contém. no caso de um atrator grande, a cada TAM_ESTADO bits teremos um novo atrator 
    unsigned long long cont; //conta quantos estados caem neste atrator
    int periodo; //período do atrator

} Atrator;

void atrator_tabela_sincrono_cpu(const Grafo &g, Atrator * Tabela, unsigned long long MAX_ESTADO)
{
    bool aleatorio = false;
       
    int tamEstado = g.nEq/32 + (g.nEq%32 != 0);
    unsigned int * s0, *s1;
    s0 = (unsigned int *)malloc(tamEstado*sizeof(unsigned int));
    s1 = (unsigned int *)malloc(tamEstado*sizeof(unsigned int));
    for(unsigned long long estado = 0; estado < MAX_ESTADO; estado++)
    {
        //variáveis necessárias para realizar o passo
        if(g.nEq > 25){ aleatorio = true; srand(estado);}  //rede grande, estados aleatórios
        for(int i = 0; i < tamEstado; i++)
            s0[i] = s1[i] = 0; 

        
        //inicializando estado inicial(o bit mais alto representa o vértice (nEq-1) e o bit mais baixo representa o vértice 0)
        if(aleatorio)
        {
            for(int i = 0; i < tamEstado; i++)
                s0[i] = (unsigned int)rand() % UINT_MAX; //preenche o estado com numeros aleatórios

            if(TAM_REDE < 32*TAM_ESTADO)
            {
                for(int bitVar = TAM_REDE; bitVar < 32*TAM_ESTADO; bitVar++)
                {
                    int posAtr = (bitVar)/32;
                    s0[posAtr] =(s0[posAtr] & ~(1 << (bitVar%32)));
                }
            }
        }
        else
        {
            for(int i = 0; i < g.nEq; i++)
            {
                int var = g.nEq -1 - i; //variável desejada
                int posAtr = tamEstado - var/32 - (var%32!=0); //posição de atr onde se encontra o bit da variável desejada
                s0[posAtr] |= ((estado >> i)%2) << (var%32);
            }
        }
        

        for(int i = 0; i < tamEstado; i++)
            s1[i] = s0[i];

         //variaveis auxiliares
         unsigned int * newEstado = (unsigned int *)malloc(sizeof(unsigned int)*tamEstado);
        
         bool diferente = false;
        
        //procurando o atrator onde os estados caem
        do
        {
            //s0 anda um passo

            //passo
            for(int i = 0; i < tamEstado; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < g.nEq; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s0[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-i; //variavel que será atualizada no novo estado
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar - 32*posAtr);
            }
            //atualiza s0
            for(int i = 0; i < tamEstado; i++) s0[i] = newEstado[i];

            //s1 anda dois passos
            //passo
            for(int i = 0; i < tamEstado; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < g.nEq; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar - 32*posAtr);
            }
            //atualiza s1
            for(int i = 0; i < tamEstado; i++) s1[i] = newEstado[i];

            //passo
            for(int i = 0; i < tamEstado; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < g.nEq; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar - 32*posAtr);
            }
            //atualiza s1
            for(int i = 0; i < tamEstado; i++) s1[i] = newEstado[i];

            //testando se s0 != s1
            diferente = false;
            for(int i = 0; i < tamEstado; i++) if(s0[i] != s1[i]){ diferente = true; break;}
        }while(diferente);

        int upperBit = -1, lowerBit = -1, hash = 0;
        #pragma unroll
        for(int i = 0; i < g.nEq; i++)
        {
            int bitVar = g.nEq -1 - i;
            int posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
            bool bit = (s1[posAtr] >>(bitVar%32))% 2;
            if(lowerBit == -1 && (bit == 1))
                lowerBit = i+1;
            
            if(bit == 1)
            {
                upperBit = i + 1;
                hash += upperBit;
            }
        }
        if(upperBit == -1) hash = 0;
        else hash = (hash)/1024 + upperBit;

        //insere o estado na tabela hash :
        //assert(hash >= TABLE_SIZE || hash < 0);
        if(hash >= TABLE_SIZE || hash < 0){
            printf("Estado : ");
            for(int i = 0; i < tamEstado; i++) printf("%d",s1[i]);
            printf(" , hash : %d\n",hash);
            return;
        }

        //salvar o atrator na tabela
        bool igual = true;
        for(int i = 0; i < tamEstado; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;}
        if(igual)
        {
            Tabela[hash].cont+=1;//se dois estados caem no mesmo balde, soma mais um no estado
        }
        else
        {
            //procura um balde vazio desde que o estado encontrado nao seja igual ao dos baldes encontrados no caminho
            while(Tabela[hash].cont != 0 && (!igual))
            {   
                hash++;
                igual = true;
                for(int i = 0; i < tamEstado; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;} 
                
            } 
            if(!igual)
                for(int i = 0; i < tamEstado; i++) Tabela[hash].atr[i]=s1[i];
            Tabela[hash].cont+=1;//se dois estados caem no mesmo balde, soma mais um no estado
            Tabela[hash].periodo=1;
        }
        
    }

    free(s0);
    free(s1);
    return;
}

__device__ void passo(unsigned int * estado, unsigned int tamEstado, const Grafo &g)
{
    unsigned int * newEstado;
    newEstado = (unsigned int *)malloc(sizeof(unsigned int)*tamEstado);

    //passo
    for(int i = 0; i < tamEstado; i++) newEstado[i]=0; //zera o newEstado

    //calcula novo estado
    for(int i = 0; i < g.nEq; i++)
    {
        int bitVar = 0; //bit no qual a variavel i é representada
        int posAtr = 0; //posição do bit no vetor do estado
        int sum_prod =0, pos = g.pesoIni[i] , eqsize = g.eqSize[i], Teq = g.T[i];
        //aplicando a tlf
        for(int j = 0; j < eqsize;  j++, pos += 2){
            bitVar = (g.nEq-1)-g.peso[pos];
            posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
            sum_prod += ((estado[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
        }
        bitVar = (g.nEq-1)-i; //variavel que será atualizada no novo estado
        newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
    }
    //atualiza s0
    for(int i = 0; i < tamEstado; i++) estado[i] = newEstado[i];

    free(newEstado);

}


__global__ void atrator_tabela_sincrono(hiprandState * curstate, const Grafo g, Atrator *Tabela, unsigned long long MAX_TREAD_ID, unsigned int * C)
{
    //o id da tread é calculado para evitar que as threads excedentes sejam utilizadas
    unsigned long long idx = blockDim.x*blockIdx.x + threadIdx.x;

    //copia do grafo na memória shared
    __shared__ int eqSize[TAM_REDE];
    __shared__ int pesoIni[TAM_REDE];
    __shared__ int T[TAM_REDE];
    __shared__ int peso[TAM_PESOS*2];

    //inicializa a cópía do grafo na memória shared
    if(threadIdx.x<TAM_REDE)
    {   
        eqSize[threadIdx.x] = g.eqSize[threadIdx.x];
        pesoIni[threadIdx.x] = g.pesoIni[threadIdx.x];
        T[threadIdx.x] = g.T[threadIdx.x];
    }
    if(threadIdx.x < TAM_PESOS)
    {
        peso[threadIdx.x*2] = g.peso[threadIdx.x*2];
        peso[threadIdx.x*2+1] = g.peso[threadIdx.x*2+1];
    }
    __syncthreads();

    
    //calculo do estado : estado = blockDim.x*blockIdx.x + threadIdx.x 
    if(idx < MAX_TREAD_ID)
    {
        //rede grande, estado aleatório
        if(TAM_REDE > 25)
            hiprand_init(idx, idx, 0, curstate + idx);//inicia a seed

        //variáveis necessárias para realizar o passo
        unsigned int s0[TAM_REDE], s1[TAM_REDE];

        for(int i = 0; i < TAM_ESTADO; i++)
            s0[i] = s1[i] = 0; 
        
        if(TAM_REDE > 25)
        {
            for(int i = 0; i < TAM_ESTADO; i++)
               s1[i] = s0[i] = hiprand(curstate + idx);

            //desligando bits que sobram
            if(TAM_REDE < 32*TAM_ESTADO)
            {
                for(int bitVar = TAM_REDE; bitVar < 32*TAM_ESTADO; bitVar++)
                {
                    int posAtr = (bitVar)/32;
                    s1[posAtr] = s0[posAtr] =(s0[posAtr] & ~(1 << (bitVar%32)));
                }
            }
        }
        else
        {
            //inicializando estado inicial(o bit mais alto representa o vértice (nEq-1) e o bit mais baixo representa o vértice 0)
            for(int i = 0; i < TAM_REDE; i++)
            {
                int var = TAM_REDE -1 - i; //variável desejada
                int posAtr = TAM_ESTADO - var/32 - (var%32!=0) ; //posição de atr onde se encontra o bit da variável desejada
                s0[posAtr] |= ((idx >> i)%2) << (var%32);
                s1[posAtr] |= ((idx >> i)%2) << (var%32);
            }
        }
        
        
        //variaveis auxiliares
        unsigned int newEstado[TAM_REDE];
        
        
        //assert(newEstado != NULL); //testa se a nenoria foi alocada corretamente

        //testando se s0 != s1
        bool diferente = false;
        unsigned int c1=0,c2=0;
        //procurando o atrator onde os estados caem
        do
        {
            //s0 anda um passo
            
            
            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < TAM_REDE; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = pesoIni[i] , eqsize = eqSize[i], Teq = T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (TAM_REDE-1)-peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s0[posAtr] >>(bitVar%32))% 2)*peso[pos+1];
                }
                bitVar = (TAM_REDE-1)-i; //variavel que será atualizada no novo estado
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza s0
            for(int i = 0; i < TAM_ESTADO; i++) s0[i] = newEstado[i];
            
            //passo(s0,tamEstado,sh_g);

            //s1 anda dois passos
            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < TAM_REDE; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = pesoIni[i] , eqsize = eqSize[i], Teq = T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (TAM_REDE-1)-peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*peso[pos+1];
                }
                bitVar = (TAM_REDE-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza s1
            for(int i = 0; i < TAM_ESTADO; i++) s1[i] = newEstado[i];

            asm("mov.u32 %0,%%clock;":"=r"(c1));
            //passo
            for(int i = 0; i < TAM_ESTADO; i++) newEstado[i]=0; //zera o newEstado

            //calcula novo estado
            for(int i = 0; i < TAM_REDE; i++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = pesoIni[i] , eqsize = eqSize[i], Teq = T[i];
                //aplicando a tlf
                for(int j = 0; j < eqsize;  j++, pos += 2){
                    bitVar = (TAM_REDE-1)-peso[pos];
                    posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((s1[posAtr] >>(bitVar%32))% 2)*peso[pos+1];
                }
                bitVar = (TAM_REDE-1)-i;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza s1
            for(int i = 0; i < TAM_REDE; i++) s1[i] = newEstado[i];
            asm("mov.u32 %0,%%clock;":"=r"(c2));
            /* passo(s1,tamEstado,sh_g);
            passo(s1,tamEstado,sh_g); */

            //testando se s0 != s1
            diferente = false;
            for(int i = 0; i < TAM_ESTADO; i++) if(s0[i] != s1[i]){ diferente = true; break;}
        }while(diferente);
        __syncthreads();
        //Neste ponto s0 == s1

    //tempo do ultimo passo
    C[idx] = c2 - c1;
        //salva na memória global sequencialmente

	for(unsigned int block = 0; block < gridDim.x; block++)
	{
	    if(blockIdx.x == block)
	    {
            for(unsigned int thread = 0; thread < blockDim.x; thread++)
            {
                if(threadIdx.x == thread)
                {
                    int upperBit = -1, lowerBit = -1, hash = 0;
                    #pragma unroll
                    for(int i = 0; i < TAM_REDE; i++)
                    {
                        int bitVar = TAM_REDE -1 - i;
                        int posAtr = TAM_ESTADO - bitVar/32 - (bitVar%32!=0) ;
                        bool bit = (s1[posAtr] >> (bitVar%32))% 2;
                        if(lowerBit == -1 && (bit == 1))
                            lowerBit = i+1;
                        
                        if(bit == 1)
                        {
                            upperBit = i + 1;
                            hash += upperBit;
                        }
                    }
                    if(upperBit == -1) hash = 0;
                    else hash = (hash)/1024 + upperBit;
                    
                    assert(hash < TABLE_SIZE);
                    //insere o estado na tabela hash :
                    if(hash >= TABLE_SIZE || hash < 0){
                        printf("Estado : ");
                        for(int i = 0; i < TAM_ESTADO; i++) printf("%X",s1[i]);
                        printf(" , hash : %d\n",hash);
                        return;
                    }

                    //confere se o balde já está cheio e acha um balde vazio
                    bool igual = true;
                    for(int i = 0; i < TAM_ESTADO; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;}
                    if(igual)
                    {
                        atomicAdd((unsigned long long *)&(Tabela[hash].cont), (unsigned long long)1);//se dois estados caem no mesmo balde, soma mais um no estado
                    }
                    else
                    {
                        //procura um balde vazio desde que o estado encontrado nao seja igual ao dos baldes encontrados no caminho
                        while(Tabela[hash].cont != 0 && (!igual))
                        {   
                            hash++;
                            igual = true;
                            for(int i = 0; i < TAM_ESTADO; i++) if(Tabela[hash].atr[i] != s1[i]) {igual = false; break;}
                        }
                        
                        
                        for(int i = 0; i < TAM_ESTADO; i++){ Tabela[hash].atr[i]=s1[i];} 
                        atomicAdd((unsigned long long *)&(Tabela[hash].cont), (unsigned long long)1);
                        Tabela[hash].periodo=1;
                    }    		
                } 
                __syncthreads();		
            }
            __syncthreads();
	    }
	    __syncthreads();
	}
    __syncthreads();
        
       
    }
}

__global__ void gen_rand(hiprandState * curstate, const Grafo g, const unsigned long long MAX_TREAD_ID)
{
    unsigned long long idx = blockDim.x*blockIdx.x + threadIdx.x;

    
    if(idx < MAX_TREAD_ID)
    {
        //inicia a seed
        hiprand_init(idx, idx, 0, curstate + idx);
        
        unsigned int tamEstado = g.nEq/32 + (g.nEq%32!=0);
        unsigned int * estado;
        estado = (unsigned int *)malloc(sizeof(unsigned int)*tamEstado);
        for(int i = 0; i < tamEstado; i++) estado[i] = 0;  
        
       
        
        for(int i = 0; i < tamEstado; i++)
            estado[i] = hiprand(curstate + idx);
        
        
        for(unsigned long long id = 0; id < MAX_TREAD_ID; id++)
        {
            __syncthreads();
            if(id == idx)
            {
                for(int i = 0; i < tamEstado; i++) printf("%u",estado[i]);
                printf("\n");
                __syncthreads();
            }
            __syncthreads();
        }
        __syncthreads();
        
        /*
        free(estado); */

    }

    return;

}


Atrator * junta_atrator(Atrator * Tabela, const Grafo &g, const string tec)
{
    int tamEstado = g.nEq/32 + (g.nEq%32 != 0);
    //ajeita tabela da GPU
    if(tec == "bananas")
    {
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            if(Tabela[i].cont != 0)
            {
                bool igual;
                int j;
                for(j = i+1; j < TABLE_SIZE; j++)
                {
                    igual = false;
                    if(Tabela[j].cont != 0)
                    {
                        igual = true;
                        for(int z = 0; z < tamEstado; z++) if(Tabela[i].atr[z] != Tabela[j].atr[z]){ igual = false; break;}
                    }
                    if(igual)
                    {
                        Tabela[i].cont += Tabela[j].cont;
                        Tabela[j].cont = 0;
                    }
                }
                
            }
        }
    }


    Atrator * resultado = new Atrator[TABLE_SIZE];

    //inicializa tabela
    for(int i = 0; i < TABLE_SIZE; i++) 
    {
        resultado[i].cont = 0;
        resultado[i].periodo = 0;
    }

    //variaveis auxiliares
    unsigned int * newEstado = new unsigned int[tamEstado];
    unsigned int * aux = new unsigned int[tamEstado]; //auxiliar para saber os próximos estados do atratror

    
    //junta os atratores da Tabela no resultado
    for(int i = 0; i < TABLE_SIZE; i++)
    {
        if(Tabela[i].cont != 0)
        {
            resultado[i].periodo = Tabela[i].periodo;
            resultado[i].cont = Tabela[i].cont;
            vector<unsigned int> atr(tamEstado); // atrator completo que será armazenado em resultado[i].atr

            //zera a posição visitada na Tabela
            Tabela[i].cont = 0;

            for(int j = 0; j < tamEstado; j++) aux[j] = atr[j] = Tabela[i].atr[j];//inicialização de aux e atr

            //aplica um passo em atr para encontrar o próximo estado do atrator
           
            //passo
            for(int j = 0; j < tamEstado; j++) newEstado[j]=0; //zera o newEstado

            //calcula novo estado
            for(int j = 0; j < g.nEq; j++)
            {
                int bitVar = 0; //bit no qual a variavel i é representada
                int posAtr = 0; //posição do bit no vetor do estado
                int sum_prod =0, pos = g.pesoIni[j] , eqsize = g.eqSize[j], Teq = g.T[j];
                //aplicando a tlf
                for(int z = 0; z < eqsize;  z++, pos += 2){
                    bitVar = (g.nEq-1)-g.peso[pos];
                    posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
                    sum_prod += ((aux[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                }
                bitVar = (g.nEq-1)-j;
                newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
            }
            //atualiza aux
            for(int j = 0; j < tamEstado; j++) aux[j] = newEstado[j];
            
            //testar se aux != Tabela[i].atr[j]
            bool diferente = false;
            for(int j = 0; j < tamEstado; j++) if(aux[j] != Tabela[i].atr[j]){ diferente = true; break;}
            while(diferente)
            {

               /*  for(int j = 0; j < tamEstado; j++) printf("%X",aux[j]);
                cout << endl; */

                //calcula o hash do estado atual
                int upperBit = -1, lowerBit = -1, hash = 0;
                #pragma unroll
                for(int j = 0; j < g.nEq; j++)
                {
                    int bitVar = g.nEq -1 - j;
                    int posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
                    bool bit = (aux[posAtr] >>(bitVar%32))% 2;
                    if(lowerBit == -1 && (bit == 1))
                        lowerBit = j+1;
                    
                    if(bit == 1)
                    {
                        upperBit = j + 1;
                        hash += upperBit;
                    }
                }
                if(upperBit == -1) hash = 0;
                else hash = (hash)/1024 + upperBit;
                    

                bool igual = true;
                assert(hash < TABLE_SIZE);

                //procurando se o estado encontrado no ciclo esta na tabela
                for(int j = 0; j < tamEstado; j++) if(Tabela[hash].atr[j] != aux[j]) {igual = false; break;}
                //procurando se o estado encontrado no ciclo esta na tabela
                if(hash < TABLE_SIZE) //se o estado estiver na tabela extraimos seus dados
                {
                    while(hash < TABLE_SIZE && !igual)
                    {
                        hash++;
                        igual = true;
                        if(hash < TABLE_SIZE) for(int j = 0; j < tamEstado; j++) if(Tabela[hash].atr[j] != aux[j]) {igual = false; break;}
                    } 
                }
                

                for(int j = 0; j < tamEstado; j++) atr.push_back(aux[j]);
                resultado[i].periodo++;
                
                if(hash < TABLE_SIZE) //se o estado estiver na tabela extraimos seus dados
                {
                    resultado[i].cont += Tabela[hash].cont;
                    Tabela[hash].cont = 0; // já contamos esse estado em algum atrator 
                    assert(Tabela[hash].cont == 0);
                }

                //da um passo em aux
                for(int j = 0; j < tamEstado; j++) newEstado[j]=0; //zera o newEstado

                //calcula novo estado
                for(int j = 0; j < g.nEq; j++)
                {
                    int bitVar = 0; //bit no qual a variavel i é representada
                    int posAtr = 0; //posição do bit no vetor do estado
                    int sum_prod =0, pos = g.pesoIni[j] , eqsize = g.eqSize[j], Teq = g.T[j];
                    //aplicando a tlf
                    for(int z = 0; z < eqsize;  z++, pos += 2){
                        bitVar = (g.nEq-1)-g.peso[pos];
                        posAtr = tamEstado - bitVar/32 - (bitVar%32!=0) ;
                        sum_prod += ((aux[posAtr] >>(bitVar%32))% 2)*g.peso[pos+1];
                    }
                    bitVar = (g.nEq-1)-j;
                    newEstado[posAtr] |= (sum_prod >= Teq) << (bitVar%32);
                }
                //atualiza aux
                for(int j = 0; j < tamEstado; j++) aux[j] = newEstado[j];


                //aux != Tabela[i].atr ?
                diferente = false;
                for(int j = 0; j < tamEstado; j++) if(aux[j] != Tabela[i].atr[j]){ diferente = true; break;}
            }


            //copia o atrator completo para a tabela de resultado
            resultado[i].atr = new unsigned int[atr.size()];
            for(int j = 0; j < atr.size(); j++) resultado[i].atr[j] = atr[j];


        }

    }
    
    delete [] aux;
    delete [] newEstado;

    return resultado;
}



int main(int argc, char **argv)
{

    int nEq; // numero de equações

    //ler o grafo pelo arquivo de entrada
    filebuf fb;
    if(!fb.open(argv[1],ios::in)) 
    {
        cerr << "Erro ao abrir arquivo de entrada " << argv[1] <<endl;
        exit(0);
    }

    istream is(&fb);
    is >> nEq; //numero de equações, tamanho da rede
    
    Grafo g, d_g; //grafo da cpu e da gpu 
    g.eqSize = new int[nEq];
    g.pesoIni = new int[nEq];
    g.T = new int[nEq];
    g.nEq = nEq;

    hipMalloc((int **)&(d_g.eqSize),sizeof(int)*nEq);
    hipMalloc((int **)&(d_g.pesoIni),sizeof(int)*nEq);
    hipMalloc((int **)&(d_g.T),sizeof(int)*nEq);

    //ler os tamanhos das equações a seguir
    int nPesos = 0; // numero de pesos

    for(int i = 0; i < nEq; i++)
    {
        is >> g.eqSize[i];
        nPesos+= g.eqSize[i];
    }

    nPesos *= 2;
    g.peso = new int [nPesos];
    hipMalloc((int **)&(d_g.peso),nPesos*sizeof(int));

    int posPeso = 0; //posição dos pesos
    for(int i = 0; i < nEq; i++)
    {
        int var=0, peso=0, T=0; // variavel da equação, peso e threshold
        g.pesoIni[i] = posPeso;
        for(int j = 0; j < g.eqSize[i]; j++,posPeso+=2)
        {
            is >> var >> peso;
            g.peso[posPeso] = var;
            g.peso[posPeso + 1] = peso;
        }
        is >> T;
        g.T[i] = T;
    }
    fb.close(); // fim da leitura

    
    //inicializando objeto que vai para a gpu
    hipMemcpy(d_g.eqSize, g.eqSize, sizeof(int)*nEq, hipMemcpyHostToDevice);
    hipMemcpy(d_g.pesoIni, g.pesoIni, sizeof(int)*nEq, hipMemcpyHostToDevice);
    hipMemcpy(d_g.T, g.T, sizeof(int)*nEq, hipMemcpyHostToDevice);
    hipMemcpy(d_g.peso, g.peso, sizeof(int)*nPesos, hipMemcpyHostToDevice);
    d_g.nEq = g.nEq;
    //hipMemcpy(&(d_g.nEq), &(g.nEq), sizeof(int), hipMemcpyHostToDevice);


    //definindo tamanho da grid e dos blocos
    int threads = 1024;
    dim3 block(threads);

    unsigned long long MAX_ESTADO = 0;
    string argv2 = argv[2];
    for(int i = 0; i < argv2.size() ; i++)
        MAX_ESTADO += ((unsigned long int)(argv2[i] - '0'))*pow(10,argv2.size()-i-1);

    dim3 grid((MAX_ESTADO + block.x -1)/block.x);

    //Alocando tabela de atratores para cpu e para gpu
    Atrator *Tabela,*d_Tabela;
    Tabela = new Atrator[TABLE_SIZE];
    hipMalloc((Atrator**)&d_Tabela, TABLE_SIZE*sizeof(Atrator));
    unsigned int * d_atr[TABLE_SIZE];

    int tamEstado = nEq/32 + (nEq%32 != 0);

    
    for(int i = 0; i < TABLE_SIZE; i++)
    {
        Tabela[i].atr = new unsigned int[tamEstado];
        Tabela[i].cont = 0;
        Tabela[i].periodo = 0;

        for(int j = 0; j < tamEstado; j++) Tabela[i].atr[j] = 0;
        
        hipMalloc((void **)&(d_atr[i]), sizeof(unsigned int)*tamEstado);
        hipMemcpy(d_atr[i], Tabela[i].atr, sizeof(unsigned int)*tamEstado, hipMemcpyHostToDevice);

        hipMemcpy(&(d_Tabela[i]), &(Tabela[i]), sizeof(Atrator), hipMemcpyHostToDevice);
        hipMemcpy(&(d_Tabela[i].atr), &(d_atr[i]), sizeof(unsigned int*), hipMemcpyHostToDevice);
    }
    

    double MB_por_Estado = (9.0/40000.0) * MAX_ESTADO;
    size_t HeapSize = MB_por_Estado * 1024 * 1024;

    string tec = argv[3];
    
    unsigned int * h_C = new unsigned int[MAX_ESTADO];
    if(tec == "GPU")
    {
        //estado para gerar números aleatórios
        hiprandState * d_state;
        hipMalloc((void **)&d_state, sizeof(hiprandState) * MAX_ESTADO);
        //hipDeviceSetLimit(hipLimitMallocHeapSize, HeapSize);
        
        unsigned int * d_C;
        hipMalloc(&d_C,sizeof(unsigned int)*MAX_ESTADO);

        //gen_rand<<<1,1024>>>(d_state,d_g,MAX_ESTADO);
        atrator_tabela_sincrono<<<grid,block>>>(d_state,d_g,d_Tabela,MAX_ESTADO,d_C);
        hipDeviceSynchronize();
        hipMemcpy(h_C, d_C, sizeof(unsigned int)*MAX_ESTADO, hipMemcpyDeviceToHost);
        //traz o resultado da GPU
        for(int i = 0; i < TABLE_SIZE; i++)
        {
            hipMemcpy(Tabela[i].atr, d_atr[i], tamEstado*sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&(Tabela[i].cont), &(d_Tabela[i].cont), sizeof(unsigned long long), hipMemcpyDeviceToHost);
            hipMemcpy(&(Tabela[i].periodo), &(d_Tabela[i].periodo), sizeof(unsigned long long), hipMemcpyDeviceToHost);
        }

    }
    else if(tec == "CPU") atrator_tabela_sincrono_cpu(g,Tabela,MAX_ESTADO);
    else
    {
        cerr << "Argumento "<<tec<<" inválido : escolha entre GPU ou CPU!"<<endl;
        //desalocando memória
        for(int i = 0; i < TABLE_SIZE; i++){ delete [] Tabela[i].atr; hipFree(d_atr[i]);}  
        delete [] Tabela;
        hipFree(d_Tabela);

        delete [] g.peso;
        delete [] g.pesoIni;
        delete [] g.eqSize;
        delete [] g.T;
        hipFree(d_g.peso);
        hipFree(d_g.pesoIni);
        hipFree(d_g.eqSize);
        hipFree(d_g.T);
        return 0;
    }
    //junta os atratores
    Atrator * resultado ;//= Tabela;
    resultado = junta_atrator(Tabela,g,tec);
    //return 0;
    //imprimindo resultado
    for(int i = 0; i < TABLE_SIZE; i++)
    {
        if(resultado[i].cont)
        {
            printf("%d ",resultado[i].periodo);
            for(int j = 0; j < resultado[i].periodo; j++)
            {
                for(int z = 0; z < tamEstado; z++) printf("%X",resultado[i].atr[j*tamEstado + z]);
                printf(" ");
            }
            printf("%llu\n", resultado[i].cont);
        }
    }

    double avgT = 0;
    for(int i = 0; i < MAX_ESTADO; i++)
        avgT += h_C[i];
    avgT /= MAX_ESTADO;
    if(tec == "CPU") avgT /= CLOCK_PER_SEC_CPU;
    else avgT /= CLOCK_PER_SEC_GPU;

    cerr << "Tempo de 1 passo : "<<h_C[1] << "s\n";

    //desalocando memória
    for(int i = 0; i < TABLE_SIZE; i++){ hipFree(d_atr[i]); free(Tabela[i].atr);}  
    free(Tabela);
    free(resultado);
    hipFree(d_Tabela);

    delete [] g.peso;
    delete [] g.pesoIni;
    delete [] g.eqSize;
    delete [] g.T;
    hipFree(d_g.peso);
    hipFree(d_g.pesoIni);
    hipFree(d_g.eqSize);
    hipFree(d_g.T);

    //reinicia o device
    hipDeviceReset();
    return 0;
}
